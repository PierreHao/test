#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2014 by Contributors
 * \file tensor_gpu-inl.cuh
 * \brief implementation of GPU code using CUDA
 * \author Bing Xu, Tianqi Chen
 */
#ifndef MSHADOW_CUDA_TENSOR_GPU_INL_CUH_
#define MSHADOW_CUDA_TENSOR_GPU_INL_CUH_
#include "../tensor.h"
#include "./reduce.cuh"

namespace mshadow {
namespace cuda {
/* load unit for memory access, if CUDAARCH not defined, this is advanced nvcc */
#if MSHADOW_OLD_CUDA
const int kMemUnitBits = 4;
const int kMaxThreadsPerBlock = 512;
#else
const int kMemUnitBits = 5;
const int kMaxThreadsPerBlock = 1024;
#endif
/*! \brief number of units that can do synchronized update, half warp size */
const int kMemUnit = 1 << kMemUnitBits;
/*! \brief mask that could be helpful sometime */
const int kMemUnitMask = kMemUnit - 1;
/*! \brief suggested thread number(logscale) for mapping kernel */
const int kBaseThreadBits = 8;
/*! \brief suggested thread number for mapping kernel */
const int kBaseThreadNum  = 1 << kBaseThreadBits;
/*! \brief maximum value of grid */
const int kMaxGridNum = 65535;
/*! \brief suggested grid number for mapping kernel */
const int kBaseGridNum = 1024;
/*! \brief get align stride for given size in x dimension */
inline index_t GetAlignStride(index_t xsize) {
  if (xsize >= MSHADOW_MIN_PAD_RATIO * 32) {
    return ((xsize  + kMemUnit - 1) >> kMemUnitBits) << kMemUnitBits;
  } else {
    // if originally space is not aligned, no necessary to to alligned thread allocation
    return xsize;
  }
}
inline void CheckLaunchParam(dim3 dimGrid, dim3 dimBlock, const char *estr = "") {
  if (dimBlock.x * dimBlock.y * dimBlock.z > static_cast<unsigned>(kMaxThreadsPerBlock) ||
      dimGrid.x > 65535 || dimGrid.y > 65535) {
    LOG(FATAL) << "too large launch parameter: "
      << estr << "["
      << dimBlock.x << ","
      << dimBlock.y << ","
      << dimBlock.z << "]";
  }
}
template<typename Saver, typename DstPlan,
         typename Plan, int block_dim_bits>
__device__ void MapPlanProc(DstPlan dst, index_t xstride,
                            Shape<2> dshape, const Plan exp, int block_idx) {
  const index_t tid = (block_idx << block_dim_bits) + threadIdx.x;
  const int y = tid / xstride;
  const int x = tid % xstride;
  if (y < dshape[0] && x < dshape[1]) {
    Saver::Save(dst.REval(y, x), exp.Eval(y,x));
  }
}
template<typename Saver,int block_dim_bits,
         typename DstPlan, typename Plan>
__global__ void MapPlanKernel(DstPlan dst, index_t xstride,
                              Shape<2> dshape, const Plan exp) {
  MapPlanProc<Saver, DstPlan, Plan, block_dim_bits>
      (dst, xstride, dshape, exp, blockIdx.x);
}
template<typename Saver, int block_dim_bits, int grid_size,
         typename DstPlan, typename Plan>
__global__ void MapPlanLargeKernel(DstPlan dst, index_t xstride,
                                   Shape<2> dshape, const Plan exp, int repeat) {
  for (int i = 0; i < repeat; ++i) {
  MapPlanProc<Saver, DstPlan, Plan, block_dim_bits>
      (dst, xstride, dshape, exp, blockIdx.x + i * grid_size);
  }
}

template<typename Saver, typename DstExp, typename E, typename DType>
inline void MapPlan(expr::Plan<DstExp, DType> dst,
                    const expr::Plan<E, DType> &plan,
                    Shape<2> dshape,
                    hipStream_t stream) {
  const index_t xstride = GetAlignStride(dshape[1]);
  const int num_block = (dshape[0] * xstride + kBaseThreadNum-1) / kBaseThreadNum;
  dim3 dimBlock(kBaseThreadNum, 1, 1);

  if (num_block < kMaxGridNum) {
    dim3 dimGrid(num_block, 1, 1);
    MapPlanKernel<Saver, kBaseThreadBits,
                  expr::Plan<DstExp, DType>,
                  expr::Plan<E, DType> >
        <<<dimGrid, dimBlock, 0, stream>>>(dst, xstride, dshape, plan);
  } else {
    int repeat = (num_block + kBaseGridNum-1) / kBaseGridNum;
    dim3 dimGrid(kBaseGridNum, 1 , 1);
    MapPlanLargeKernel<Saver, kBaseThreadBits, kBaseGridNum,
                       expr::Plan<DstExp, DType>,
                       expr::Plan<E, DType> >
        <<<dimGrid, dimBlock, 0, stream>>>(dst, xstride, dshape, plan, repeat);
  }
}

template<typename Saver,typename Reducer, int warp_bits,
         typename DType, typename DstPlan, typename Plan>
__global__ void MapRedKeepLowestKernel(DstPlan dst, Plan plan,
                                       DType scale, Shape<2> eshape) {
  const unsigned warp_size = 1 << warp_bits;
  const unsigned x = (blockIdx.x << warp_bits) + threadIdx.x;
  // to avoid bank conflict
  __shared__ DType s_res[warp_size][warp_size + 1];
  // note: reverse store [y][x], so that we can reduce over threadIdx.x, use warp optimization
  if (threadIdx.y < eshape[0] && x < eshape[1]) {
    s_res[threadIdx.x][threadIdx.y] = plan.Eval(threadIdx.y, x);
  }
  for (unsigned y = warp_size; y < eshape[0]; y += warp_size) {
    if (threadIdx.y + y < eshape[0] && x < eshape[1]) {
      Reducer::Reduce(s_res[threadIdx.x][threadIdx.y], plan.Eval(threadIdx.y + y, x));
    }
  }
  __syncthreads();
  if (eshape[0] >= warp_size) {
    Reduce1D<Reducer, warp_bits>(s_res[threadIdx.y]);
  } else {
    Reduce1DNotAlign<Reducer, warp_bits>(s_res[threadIdx.y], eshape[0]);
  }
  __syncthreads();

  if (threadIdx.y == 0 && x < eshape[1]) {
    Saver::Save(dst.REval(0, x),  s_res[threadIdx.x][0] * scale);
  }
}

template<typename Saver, typename Reducer,
         typename DstExp, typename E, typename DType>
inline void MapReduceKeepLowest(expr::Plan<DstExp, DType> dst,
                                const expr::Plan<E, DType> &plan,
                                DType scale, Shape<2> eshape,
                                hipStream_t stream) {
  dim3 dimBlock(kMemUnit, kMemUnit);
  dim3 dimGrid((eshape[1] + kMemUnit - 1) >> kMemUnitBits);
  CheckLaunchParam(dimGrid, dimBlock, "MapRedKeepLowestKernel");
  MapRedKeepLowestKernel<Saver, Reducer, kMemUnitBits, DType,
                         expr::Plan<DstExp, DType>,
                         expr::Plan<E, DType> >
      <<<dimGrid, dimBlock, 0, stream>>>(dst, plan, scale, eshape);
}

template<typename Saver, typename Reducer, int block_dim_bits,
         typename DType, typename DstPlan, typename Plan>
__global__ void MapReduceKeepDim1Kernel(DstPlan dst, Plan plan, DType scale, Shape<4> pshape) {
  const int block_size = 1 << block_dim_bits;
  __shared__ DType s_rec[block_size];
  const int c = blockIdx.x;
  const index_t tot = pshape[3] * pshape[2] * pshape[0];

  DType res; Reducer::SetInitValue(res);
  for (index_t i_offset = 0; i_offset < tot; i_offset += block_size) {
    index_t i = i_offset + threadIdx.x;
    if (i< tot) {
      const index_t x = i % pshape[3];
      i /= pshape[3];
      const index_t y = i % pshape[2];
      const index_t n = i / pshape[2];
      Reducer::Reduce(res, plan.Eval((n * pshape[1] + c) * pshape[2] + y, x));
    }
  }
  s_rec[threadIdx.x] = res;
  __syncthreads();
  Reduce1D<Reducer, block_dim_bits>(s_rec);
  if (threadIdx.x == 0) {
    Saver::Save(dst.REval(0, c), s_rec[0] * scale);
  }
}

template<typename Saver, typename Reducer, typename DstExp, typename E, typename DType>
inline void MapReduceKeepDim1(expr::Plan<DstExp, DType> dst,
                              const expr::Plan<E, DType> &plan,
                              DType scale, Shape<4> pshape,
                              hipStream_t stream) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid (pshape[1]);
  CheckLaunchParam(dimGrid, dimBlock, "MapReduceKeepDim1");
  MapReduceKeepDim1Kernel<Saver,Reducer,kBaseThreadBits, DType,
                          expr::Plan<DstExp, DType>,
                          expr::Plan<E, DType> >
      <<<dimGrid, dimBlock, 0, stream>>>(dst, plan, scale, pshape);
}

template<int x_bits, typename DType, typename DstPlan, typename SrcPlan1, typename SrcPlan2>
__global__ void SoftmaxGradKernel(DstPlan dst, SrcPlan1 src, SrcPlan2 label, index_t xmax) {
  const unsigned x_size = 1 << x_bits;
  const int y = blockIdx.x;
  const int k = static_cast<int>(label.Eval(0, y));

  // calculate normalizer, with writeback
  for (unsigned x = 0; x < xmax; x += x_size) {
    const unsigned xindex = x + threadIdx.x;
    if (xindex < xmax) {
      if (xindex == k) {
        dst.REval(y, xindex) = src.Eval(y, xindex) - 1.0f;
      } else {
        dst.REval(y, xindex) = src.Eval(y, xindex);
      }
    }
  }
}

template<int x_bits, typename DType,  typename DstPlan, typename SrcPlan>
__global__ void SoftmaxKernel(DstPlan dst, SrcPlan src, index_t xmax) {
  const unsigned x_size = 1 << x_bits;
  const int y = blockIdx.x;
  __shared__ DType s_rec[x_size];
  // step 1: get max
  if (threadIdx.x < xmax) {
    s_rec[threadIdx.x] = src.Eval(y, threadIdx.x);
  }
  for (unsigned x = x_size; x < xmax; x += x_size) {
    if (x + threadIdx.x < xmax) {
      DType a = src.Eval(y, x + threadIdx.x);
      s_rec[threadIdx.x] = max(a, s_rec[threadIdx.x]);
    }
  }
  __syncthreads();
  if (threadIdx.x >= xmax) {
    s_rec[threadIdx.x] = s_rec[0];
  }
  __syncthreads();
  Reduce1D<red::maximum, x_bits>(s_rec);
  __syncthreads();
  DType smax = s_rec[0];
  __syncthreads();
  s_rec[threadIdx.x] = 0.0f;
  __syncthreads();

  // calculate normalizer, with writeback
  for (unsigned x = 0; x < xmax; x += x_size) {
    if (x + threadIdx.x < xmax) {
      DType p = expf(src.Eval(y, x + threadIdx.x) - smax);
      s_rec[threadIdx.x] += p;
      // write back first, will fetch later
      dst.REval(y, x + threadIdx.x) = p;
    }
  }
  // calculate normalizer
  __syncthreads();
  Reduce1D<red::sum, x_bits>(s_rec);
  __syncthreads();
  DType ssum = s_rec[0];

  for (unsigned x = 0; x < xmax; x += x_size) {
    if (x + threadIdx.x < xmax) {
      dst.REval(y, x + threadIdx.x) /= ssum;
    }
  }
}

template<typename DType>
inline void Softmax(Tensor<gpu, 2, DType> &dst,
                    const Tensor<gpu, 2, DType> &src) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  CHECK_EQ(dst.shape_, src.shape_) << "Softmax: shape mismatch";
  CheckLaunchParam(dimGrid, dimBlock, "Softmax");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  SoftmaxKernel<kBaseThreadBits, DType>
      <<<dimGrid, dimBlock, 0, stream>>>
      (expr::MakePlan(dst),
       expr::MakePlan(src),
       dst.size(1));
}

template<typename DType>
inline void SoftmaxGrad(Tensor<gpu, 2, DType> &dst,
                        const Tensor<gpu, 2, DType> &src,
                        const Tensor<gpu, 1, DType> &label) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  CHECK_EQ(dst.shape_, src.shape_) << "SoftmaxGrad: shape mismatch";
  CHECK_EQ(dst.size(0), label.size(0)) << "SoftmaxGrad: label shape mismatch";
  CheckLaunchParam(dimGrid, dimBlock, "SoftmaxGrad");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  SoftmaxGradKernel<kBaseThreadBits, DType>
      <<<dimGrid, dimBlock, 0, stream>>>
      (expr::MakePlan(dst),
       expr::MakePlan(src),
       expr::MakePlan(label),
       dst.size(1));
}

template<int n_bits, typename DType>
__global__ void Softmax3DGradKernel(Tensor<gpu, 3, DType> dst,
                                    const Tensor<gpu, 3, DType> src,
                                    const Tensor<gpu, 2, DType> label) {
  const index_t xmax = dst.size(1);
  const index_t nmax = dst.size(2);
  const unsigned n_size = 1 << n_bits;
  const int y = blockIdx.x;
  const int n = threadIdx.x;

  for (index_t n_index = n; n_index < nmax; n_index += n_size) {
    const int k = static_cast<int>(label[y][n_index]);
    for (index_t i = 0; i < xmax; ++i) {
      if (i == k) {
        dst[y][i][n_index] = src[y][i][n_index] - 1.0f;
      } else {
        dst[y][i][n_index] = src[y][i][n_index];
      }
    }
  }
}

template<int n_bits, typename DType>
__global__ void Softmax3DGradKernel(Tensor<gpu, 3, DType> dst,
                                    const Tensor<gpu, 3, DType> src,
                                    const Tensor<gpu, 2, DType> label,
                                    DType ignore_label) {
  const index_t xmax = dst.size(1);
  const index_t nmax = dst.size(2);
  const unsigned n_size = 1 << n_bits;
  const int y = blockIdx.x;
  const int n = threadIdx.x;
  for (index_t n_index = n; n_index < nmax; n_index += n_size) {
    int k = static_cast<int>(label[y][n_index]);
    if (k == static_cast<int>(ignore_label)) {
      for (index_t i = 0; i < xmax; ++i) {
        dst[y][i][n_index] = 0.0f;
      }
    } else {
      for (index_t i = 0; i < xmax; ++i) {
        if (i == k) {
          dst[y][i][n_index] = src[y][i][n_index] - 1.0f;
        } else {
          dst[y][i][n_index] = src[y][i][n_index];
        }
      }
    }
  }
}

template<int n_bits, typename DType>
__global__ void Softmax3DKernel(Tensor<gpu, 3, DType> dst,
                    const Tensor<gpu, 3, DType> src) {
  const index_t xmax = dst.size(1);
  const index_t nmax = dst.size(2);
  const unsigned n_size = 1 << n_bits;
  const int y = blockIdx.x;
  const int n = threadIdx.x;

  for (index_t n_index = n; n_index < nmax; n_index += n_size) {
    DType smax = src[y][0][n_index];
    for (index_t i = 1; i < xmax; ++i) {
      smax = max(smax, src[y][i][n_index]);
    }
    DType ssum = 0.0f;
    for (index_t i = 0; i < xmax; ++i) {
      DType p = expf(src[y][i][n_index] - smax);
      ssum += p;
      dst[y][i][n_index] = p;
    }
    for (index_t i = 0; i < xmax; ++i) {
      dst[y][i][n_index] /= ssum;
    }
  }
}

template<typename DType>
inline void Softmax(Tensor<gpu, 3, DType> &dst,
                    const Tensor<gpu, 3, DType> &src) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  CHECK_EQ(dst.shape_, src.shape_) << "Softmax: shape mismatch";
  CheckLaunchParam(dimGrid, dimBlock, "Softmax");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  Softmax3DKernel<kBaseThreadBits, DType><<<dimGrid, dimBlock, 0, stream>>>(dst, src);
}

template<typename DType>
inline void SoftmaxGrad(Tensor<gpu, 3, DType> &dst,
                        const Tensor<gpu, 3, DType> &src,
                        const Tensor<gpu, 2, DType> &label) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  CHECK_EQ(dst.shape_, src.shape_) << "SoftmaxGrad: shape mismatch";
  CHECK_EQ(dst.size(0), label.size(0)) << "SoftmaxGrad: label shape mismatch";
  CHECK_EQ(dst.size(2), label.size(1)) << "SoftmaxGrad: label shape mismatch";
  CheckLaunchParam(dimGrid, dimBlock, "SoftmaxGrad");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  Softmax3DGradKernel<kBaseThreadBits, DType><<<dimGrid, dimBlock, 0, stream>>>(dst, src, label);
}

template<typename DType>
inline void SoftmaxGrad(Tensor<gpu, 3, DType> &dst,
                        const Tensor<gpu, 3, DType> &src,
                        const Tensor<gpu, 2, DType> &label,
                        const DType &ignore_label) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(0));
  CHECK_EQ(dst.shape_, src.shape_) << "SoftmaxGrad: shape mismatch";
  CHECK_EQ(dst.size(0), label.size(0)) << "SoftmaxGrad: label shape mismatch";
  CHECK_EQ(dst.size(2), label.size(1)) << "SoftmaxGrad: label shape mismatch";
  CheckLaunchParam(dimGrid, dimBlock, "SoftmaxGrad");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  Softmax3DGradKernel<kBaseThreadBits, DType><<<dimGrid, dimBlock, 0, stream>>>(dst, src, label, ignore_label);
}

#include <cmath>
#include <cfloat>

template <typename Dtype>
void ROIPoolingLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->cpu_data();
  const Dtype* bottom_rois = bottom[1]->cpu_data();
  // Number of ROIs
  int num_rois = bottom[1]->num();
  int batch_size = bottom[0]->num();
  int top_count = top[0]->count();
  Dtype* top_data = top[0]->mutable_cpu_data();
  caffe_set(top_count, Dtype(-FLT_MAX), top_data);
  int* argmax_data = max_idx_.mutable_cpu_data();
  caffe_set(top_count, -1, argmax_data);

  // For each ROI R = [batch_index x1 y1 x2 y2]: max pool over R
  for (int n = 0; n < num_rois; ++n) {
    int roi_batch_ind = bottom_rois[0];
    int roi_start_w = round(bottom_rois[1] * spatial_scale_);
    int roi_start_h = round(bottom_rois[2] * spatial_scale_);
    int roi_end_w = round(bottom_rois[3] * spatial_scale_);
    int roi_end_h = round(bottom_rois[4] * spatial_scale_);
    CHECK_GE(roi_batch_ind, 0);
    CHECK_LT(roi_batch_ind, batch_size);

    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    const Dtype bin_size_h = static_cast<Dtype>(roi_height)
                             / static_cast<Dtype>(pooled_height_);
    const Dtype bin_size_w = static_cast<Dtype>(roi_width)
                             / static_cast<Dtype>(pooled_width_);

    const Dtype* batch_data = bottom_data + bottom[0]->offset(roi_batch_ind);

    for (int c = 0; c < channels_; ++c) {
      for (int ph = 0; ph < pooled_height_; ++ph) {
        for (int pw = 0; pw < pooled_width_; ++pw) {
          // Compute pooling region for this output unit:
          //  start (included) = floor(ph * roi_height / pooled_height_)
          //  end (excluded) = ceil((ph + 1) * roi_height / pooled_height_)
          int hstart = static_cast<int>(floor(static_cast<Dtype>(ph)
                                              * bin_size_h));
          int wstart = static_cast<int>(floor(static_cast<Dtype>(pw)
                                              * bin_size_w));
          int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1)
                                           * bin_size_h));
          int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1)
                                           * bin_size_w));

          hstart = min(max(hstart + roi_start_h, 0), height_);
          hend = min(max(hend + roi_start_h, 0), height_);
          wstart = min(max(wstart + roi_start_w, 0), width_);
          wend = min(max(wend + roi_start_w, 0), width_);

          bool is_empty = (hend <= hstart) || (wend <= wstart);

          const int pool_index = ph * pooled_width_ + pw;
          if (is_empty) {
            top_data[pool_index] = 0;
            argmax_data[pool_index] = -1;
          }

          for (int h = hstart; h < hend; ++h) {
            for (int w = wstart; w < wend; ++w) {
              const int index = h * width_ + w;
              if (batch_data[index] > top_data[pool_index]) {
                top_data[pool_index] = batch_data[index];
                argmax_data[pool_index] = index;
              }
            }
          }
        }
      }
      // Increment all data pointers by one channel
      batch_data += bottom[0]->offset(0, 1);
      top_data += top[0]->offset(0, 1);
      argmax_data += max_idx_.offset(0, 1);
    }
    // Increment ROI data pointer
    bottom_rois += bottom[1]->offset(1);
  }
}

template<typename Dtype>
__global__ void ROIPoolForward(const int nthreads, const Dtype* bottom_data,
                               const Dtype spatial_scale, const int channels, const int height,
                               const int width, const int pooled_height, const int pooled_width,
                               const Dtype* bottom_rois, Dtype* top_data, int* argmax_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads; index += blockDim.x * gridDim.x) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];
    int roi_start_w = std::round(bottom_rois[1] * spatial_scale);
    int roi_start_h = std::round(bottom_rois[2] * spatial_scale);
    int roi_end_w = std::round(bottom_rois[3] * spatial_scale);
    int roi_end_h = std::round(bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = std::max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = std::max(roi_end_h - roi_start_h + 1, 1);
    Dtype bin_size_h = static_cast<Dtype>(roi_height)
                       / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w = static_cast<Dtype>(roi_width)
                       / static_cast<Dtype>(pooled_width);

    int hstart = static_cast<int>(std::floor(static_cast<Dtype>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(std::floor(static_cast<Dtype>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(std::ceil(static_cast<Dtype>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(std::ceil(static_cast<Dtype>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = std::min(std::max(hstart + roi_start_h, 0), height);
    hend = std::min(std::max(hend + roi_start_h, 0), height);
    wstart = std::min(std::max(wstart + roi_start_w, 0), width);
    wend = std::min(std::max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (bottom_data[bottom_index] > maxval) {
          maxval = bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename Dtype>
__global__ void ROIPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* argmax_data, const int num_rois, const Dtype spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
    const Dtype* bottom_rois) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads; index += blockDim.x * gridDim.x) {
    // (n, c, h, w) coords in bottom data
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    Dtype gradient = 0;
    // Accumulate gradient over all ROIs that pooled this element
    for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
      const Dtype* offset_bottom_rois = bottom_rois + roi_n * 5;
      int roi_batch_ind = offset_bottom_rois[0];
      // Skip if ROI's batch index doesn't match n
      if (n != roi_batch_ind) {
        continue;
      }

      int roi_start_w = std::round(offset_bottom_rois[1] * spatial_scale);
      int roi_start_h = std::round(offset_bottom_rois[2] * spatial_scale);
      int roi_end_w = std::round(offset_bottom_rois[3] * spatial_scale);
      int roi_end_h = std::round(offset_bottom_rois[4] * spatial_scale);

      // Skip if ROI doesn't include (h, w)
      const bool in_roi = (w >= roi_start_w && w <= roi_end_w &&
                           h >= roi_start_h && h <= roi_end_h);
      if (!in_roi) {
        continue;
      }

      int offset = (roi_n * channels + c) * pooled_height * pooled_width;
      const Dtype* offset_top_diff = top_diff + offset;
      const int* offset_argmax_data = argmax_data + offset;

      // Compute feasible set of pooled units that could have pooled
      // this bottom unit

      // Force malformed ROIs to be 1x1
      int roi_width = std::max(roi_end_w - roi_start_w + 1, 1);
      int roi_height = std::max(roi_end_h - roi_start_h + 1, 1);

      Dtype bin_size_h = static_cast<Dtype>(roi_height)
                         / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = static_cast<Dtype>(roi_width)
                         / static_cast<Dtype>(pooled_width);

      int phstart = std::floor(static_cast<Dtype>(h - roi_start_h) / bin_size_h);
      int phend = std::ceil(static_cast<Dtype>(h - roi_start_h + 1) / bin_size_h);
      int pwstart = std::floor(static_cast<Dtype>(w - roi_start_w) / bin_size_w);
      int pwend = std::ceil(static_cast<Dtype>(w - roi_start_w + 1) / bin_size_w);

      phstart = std::min(std::max(phstart, 0), pooled_height);
      phend = std::min(std::max(phend, 0), pooled_height);
      pwstart = std::min(std::max(pwstart, 0), pooled_width);
      pwend = std::min(std::max(pwend, 0), pooled_width);

      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (offset_argmax_data[ph * pooled_width + pw] == (h * width + w)) {
            gradient += offset_top_diff[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

#include <vector>
#include <iostream>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _nms(int* keep_out, int* num_out, const float* boxes_host, int boxes_num,
          int boxes_dim, float nms_overlap_thresh) {
  float* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));
}

}  // namespace cuda
}  // namespace mshadow
#endif  // MSHADOW_CUDA_TENSOR_GPU_INL_CUH_
