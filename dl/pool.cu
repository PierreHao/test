#include "hip/hip_runtime.h"
#include "layer.h"

// --------------------------------------------------------------------------
// kernel code
//   max_pool_{gpu, cpu}
// --------------------------------------------------------------------------

// max-pooling bottom3d (C x H x W) -> top3d (C x H' x W')
//   given (c, h', w'),
//     top3d[c][h'][w'] = max_{h, w} bottom3d[c][h][w]
//     argmax3d[c][h'][w'] = argmax_{h, w} bottom3d[c][h][w]
//   for
//     h = (-pad_h + stride_h * h') + { 0, 1, ..., kernel_h - 1 }
//     w = (-pad_w + stride_w * w') + { 0, 1, ..., kernel_w - 1 }
#ifdef GPU
__global__
void max_pool_gpu(const real* const bottom3d,
                  real* const top3d,
                  int* const argmax3d,
                  const int C, const int bottom_H, const int bottom_W,
                  const int top_H, const int top_W,
                  const int kernel_h, const int kernel_w,
                  const int pad_h, const int pad_w,
                  const int stride_h, const int stride_w)
{
  // thread index: (c, h', w') = c*H'*W' + h'*W' + w'
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < C * top_H * top_W;
       index += blockDim.x) {
    // parse thread index -> (c, h', w')
    const int c = index / top_H / top_W;
    const int ht = (index / top_W) % top_H;
    const int wt = index % top_W; 

    // pooling range in bottom
    //   h = (-pad_h + stride_h * h') + { 0, 1, ..., kernel_h - 1}
    //   w = (-pad_w + stride_w * w') + { 0, 1, ..., kernel_w - 1}
    const int h_start = MAX(0, -pad_h + stride_h * ht);
    const int w_start = MAX(0, -pad_w + stride_w * wt);
    const int h_end = MIN(bottom_H, -pad_h + stride_h * ht + kernel_h);
    const int w_end = MIN(bottom_W, -pad_w + stride_w * wt + kernel_w);

    // if pooling range is invalid, assign 0
    if (h_start >= h_end || w_start >= w_end) {
      top3d[index] = 0;
      argmax3d[index] = -1;
      continue;
    }

    // otherwise,
    //   top3d[c][h'][w'] = max_{h,w} bottom3d[c][h][w]
    //   argmax3d[c][h'][w'] = argmax_{h,w} bottom3d[c][h][w]
    {
      const real* const p_bottom3d = bottom3d + c * bottom_H * bottom_W;
      int maxidx = h_start * bottom_W + w_start;
      real maxval = p_bottom3d[maxidx];
      for (int h = h_start; h < h_end; ++h) {
        for (int w = w_start; w < w_end; ++w) {
          if (p_bottom3d[h * bottom_W + w] > maxval) {
            maxidx = h * bottom_W + w;
            maxval = p_bottom3d[maxidx];
          }
        }
      }
      top3d[index] = maxval;
      argmax3d[index] = maxidx;
    }
  }
}
#else
void max_pool_cpu(const real* const bottom3d,
                  real* const top3d,
                  int* const argmax3d,
                  const int C, const int bottom_H, const int bottom_W,
                  const int top_H, const int top_W,
                  const int kernel_h, const int kernel_w,
                  const int pad_h, const int pad_w,
                  const int stride_h, const int stride_w)
{
  // thread index: (c, h', w') = c*H'*W' + h'*W' + w'
  for (int index = 0; index < C * top_H * top_W; ++index) {
    // parse thread index -> (c, h', w')
    const int c = index / top_H / top_W;
    const int ht = (index / top_W) % top_H;
    const int wt = index % top_W; 

    // pooling range in bottom
    //   h = (-pad_h + stride_h * h') + { 0, 1, ..., kernel_h - 1}
    //   w = (-pad_w + stride_w * w') + { 0, 1, ..., kernel_w - 1}
    const int h_start = MAX(0, -pad_h + stride_h * ht);
    const int w_start = MAX(0, -pad_w + stride_w * wt);
    const int h_end = MIN(bottom_H, -pad_h + stride_h * ht + kernel_h);
    const int w_end = MIN(bottom_W, -pad_w + stride_w * wt + kernel_w);

    // if pooling range is invalid, assign 0
    if (h_start >= h_end || w_start >= w_end) {
      top3d[index] = 0;
      argmax3d[index] = -1;
      continue;
    }

    // otherwise,
    //   top3d[c][h'][w'] = max_{h,w} bottom3d[c][h][w]
    //   argmax3d[c][h'][w'] = argmax_{h,w} bottom3d[c][h][w]
    {
      const real* const p_bottom3d = bottom3d + c * bottom_H * bottom_W;
      int maxidx = h_start * bottom_W + w_start;
      real maxval = p_bottom3d[maxidx];
      for (int h = h_start; h < h_end; ++h) {
        for (int w = w_start; w < w_end; ++w) {
          if (p_bottom3d[h * bottom_W + w] > maxval) {
            maxidx = h * bottom_W + w;
            maxval = p_bottom3d[maxidx];
          }
        }
      }
      top3d[index] = maxval;
      argmax3d[index] = maxidx;
    }
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   pool_forward
// --------------------------------------------------------------------------

// max-pooling: bottom -> top
//   bottom: C x H x W
//   top: C x H' x W'
//   argmax: C x H' x W' array
void pool_forward(const Tensor* const bottom3d,
                  Tensor* const top3d,
                  int* const argmax_data,
                  const PoolOption* const option)
{
  // kernel size, padding size & stride size
  const int kernel_h = option->kernel_h;
  const int kernel_w = option->kernel_w;
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom3d->data;
  real* p_top_item = top3d->data;
  int* p_argmax_item = argmax_data;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: C x H x W
    const int C = bottom3d->shape[n][0];  // C
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // set top shape: C x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H
        = 1 + DIV_THEN_CEIL(bottom_H + 2 * pad_h - kernel_h, stride_h);
    const int top_W
        = 1 + DIV_THEN_CEIL(bottom_W + 2 * pad_w - kernel_w, stride_w);
    top3d->shape[n][0] = C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // max-pooling
    //   bottom3d (C x H x W) -> top3d (C x H' x W')
    #ifdef GPU
    {
      const int num_threads = C * top_H * top_W;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads, threads_per_block);
      max_pool_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item, p_top_item, p_argmax_item,
          C, bottom_H, bottom_W, top_H, top_W,
          kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w);
    }
    #else
    {
      max_pool_cpu(
          p_bottom_item, p_top_item, p_argmax_item,
          C, bottom_H, bottom_W, top_H, top_W,
          kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w);
    }
    #endif

    // locate next item
    {
      const int bottom_size = C * bottom_H * bottom_W;
      const int top_size = C * top_H * top_W;
      p_bottom_item += bottom_size;
      p_top_item += top_size;
      p_argmax_item += top_size;
    }
  } // endfor batch

  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;
}



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST
#include <stdio.h>
#include <stdlib.h>

#define DATA_SIZE 128*72*92

int main(int argc, char *argv[])
{
  // variable declaration & memory allocation
  Tensor X, Y;
  real* const X_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* const Y_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* const Y_true_data = (real*)malloc(DATA_SIZE * sizeof(real));
  int* p_argmax_data;
  PoolOption option;

  // set option
  {
    option.kernel_h = 3;
    option.kernel_w = 3;
    option.pad_h = 0;
    option.pad_w = 0;
    option.stride_h = 2;
    option.stride_w = 2;
  }

  // set data shapes
  {
    X.ndim = 3;
    X.num_items = 1;
    for (int i = 0; i < X.num_items; ++i) {
      X.shape[i][0] = 128;
      X.shape[i][1] = 72;
      X.shape[i][2] = 92;
    }

    Y.ndim = 3;
    Y.num_items = 1;
    for (int i = 0; i < Y.num_items; ++i) {
      Y.shape[i][0] = X.shape[i][0];
      Y.shape[i][1] = 1 + DIV_THEN_CEIL(
          X.shape[i][1] + 2 * option.pad_h - option.kernel_h,
          option.stride_h);
      Y.shape[i][2] = 1 + DIV_THEN_CEIL(
          X.shape[i][2] + 2 * option.pad_w - option.kernel_w,
          option.stride_w);
    }
  }
 
  // load data
  {
    FILE* fp;
    const int X_size = flatten_size(&X);
    const int Y_size = flatten_size(&Y);

    printf("data loading\n");

    fp = fopen("../data/temp/pool_bottom0.bin", "rb");
    if ((int)fread(X_data, sizeof(real), X_size, fp) != X_size) {
      printf("Error while reading pool_bottom0\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/pool_top0.bin", "rb");
    if ((int)fread(Y_true_data, sizeof(real), Y_size, fp) != Y_size) {
      printf("Error while reading pool_top0\n");
    }
    fclose(fp);
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    CUDA_CHECK(hipSetDevice(0));
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const int X_size = flatten_size(&X);
    const int Y_size = flatten_size(&Y);

    printf("gpu malloc\n");
    CUDA_CHECK(hipMalloc(&X.data, X_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&Y.data, Y_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&p_argmax_data, Y_size * sizeof(int)));

    printf("memcpy: cpu -> gpu\n");
    CUDA_CHECK(hipMemcpy(X.data, X_data, X_size * sizeof(real),
                          hipMemcpyHostToDevice));
  }
  #else
  {
    const int Y_size = flatten_size(&Y);

    X.data = X_data;
    Y.data = Y_data;
    p_argmax_data = (int*)malloc(Y_size * sizeof(int));
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    pool_forward(&X, &Y, p_argmax_data, &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const int Y_size = flatten_size(&Y);

    printf("memcpy: cpu <- gpu\n");
    CUDA_CHECK(hipMemcpy(Y_data, Y.data, Y_size * sizeof(real),
                          hipMemcpyDeviceToHost));
  }
  #endif

  // verify results
  {
    int i = 0;

    printf("verification\n");

    for (int n = 0; n < Y.num_items; ++n) {
      for (int c = 0; c < Y.shape[n][0]; ++c) {
        for (int h = 0; h < Y.shape[n][1]; ++h) {
          for (int w = 0; w < Y.shape[n][2]; ++w) {
            if (Y_data[i] != Y_true_data[i]) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
              ++i;
            }
          } // endfor w
        } // endfor h
      } // endfor c
    } // endfor n
  }

  // memory deallocation
  {
    printf("free\n");
    free(X_data);
    free(Y_data);
    free(Y_true_data);
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    CUDA_CHECK(hipFree(X.data));
    CUDA_CHECK(hipFree(Y.data));
    CUDA_CHECK(hipFree(p_argmax_data));

  }
  #else
  {
    free(p_argmax_data);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
