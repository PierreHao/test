#include "hip/hip_runtime.h"
#include "layer.h"
#include <string.h>

// --------------------------------------------------------------------------
// kernel code
//   max_pool_{gpu, cpu}
// --------------------------------------------------------------------------

// max-pooling bottom3d (C x H x W) -> top3d (C x H' x W')
//   given (c, h', w'),
//     top3d[c][h'][w'] = max_{h, w} bottom3d[c][h][w]
//   for
//     h = (-pad_h + stride_h * h') + { 0, 1, ..., kernel_h - 1 }
//     w = (-pad_w + stride_w * w') + { 0, 1, ..., kernel_w - 1 }
#ifdef GPU
__global__
static
void max_pool_gpu(const real bottom3d[],
                  real top3d[],
                  const int C, const int bottom_H, const int bottom_W,
                  const int top_H, const int top_W,
                  const int kernel_h, const int kernel_w,
                  const int pad_h, const int pad_w,
                  const int stride_h, const int stride_w)
{
  // thread index: (c, h', w') = c*H'*W' + h'*W' + w'
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < C * top_H * top_W) {
    // parse thread index -> (c, h', w')
    const int c = index / top_H / top_W;
    const int ht = (index / top_W) % top_H;
    const int wt = index % top_W; 

    // pooling range in bottom
    //   h = (-pad_h + stride_h * h') + { 0, 1, ..., kernel_h - 1}
    //   w = (-pad_w + stride_w * w') + { 0, 1, ..., kernel_w - 1}
    const int h_start = MAX(0,  -pad_h + stride_h * ht);
    const int w_start = MAX(0,  -pad_w + stride_w * wt);
    const int h_end = MIN(-pad_h + stride_h * ht + kernel_h,  bottom_H);
    const int w_end = MIN(-pad_w + stride_w * wt + kernel_w,  bottom_W);

    // if pooling region is not empty,
    //   top3d[c][h'][w'] = "max in the region"
    // otherwise, assign 0
    if (h_start < h_end && w_start < w_end) {
      // find maximum in the pooling region
      const real* const p_bottom3d = bottom3d + c * bottom_H * bottom_W;
      int maxidx = h_start * bottom_W + w_start;
      real maxval = p_bottom3d[maxidx];
      for (int h = h_start; h < h_end; ++h) {
        for (int w = w_start; w < w_end; ++w) {
          if (p_bottom3d[h * bottom_W + w] > maxval) {
            maxidx = h * bottom_W + w;
            maxval = p_bottom3d[maxidx];
          }
        }
      }
      top3d[index] = maxval;
    }
    else {
      top3d[index] = 0;
    }
  }
}
#else
static
void max_pool_cpu(const real bottom3d[],
                  real top3d[],
                  const int C, const int bottom_H, const int bottom_W,
                  const int top_H, const int top_W,
                  const int kernel_h, const int kernel_w,
                  const int pad_h, const int pad_w,
                  const int stride_h, const int stride_w)
{
  for (int c = 0; c < C; ++c) {
  for (int ht = 0; ht < top_H; ++ht) {
    const int h_start = MAX(0,  -pad_h + stride_h * ht);
    const int h_end = MIN(-pad_h + stride_h * ht + kernel_h,  bottom_H);
    if (h_start >= h_end) {
      real* const p_top3d = top3d + (c * top_H + ht) * top_W;
      memset(p_top3d, 0, top_W * sizeof(real));
      continue;
    }

    for (int wt = 0; wt < top_W; ++wt) {
      const int w_start = MAX(0,  -pad_w + stride_w * wt);
      const int w_end = MIN(-pad_w + stride_w * wt + kernel_w,  bottom_W);
      real* const p_top3d = top3d + (c * top_H + ht) * top_W + wt;
      if (w_start >= w_end) {
        *p_top3d = 0;
      }

      // find maximum in the pooling region
      else {
        const real* const p_bottom3d = bottom3d + c * bottom_H * bottom_W;
        int maxidx = h_start * bottom_W + w_start;
        real maxval = p_bottom3d[maxidx];
        for (int h = h_start; h < h_end; ++h) {
          for (int w = w_start; w < w_end; ++w) {
            if (p_bottom3d[h * bottom_W + w] > maxval) {
              maxidx = h * bottom_W + w;
              maxval = p_bottom3d[maxidx];
            }
          }
        }
        *p_top3d = maxval;
      }
    } // endfor wt
  }} // endfor ht, c
}
#endif



// --------------------------------------------------------------------------
// layer operator code
// --------------------------------------------------------------------------

// max-pooling: bottom -> top
//   bottom: C x H x W
//   top: C x H' x W'
static
void pool_forward(const Tensor* const bottom3d,
                  Tensor* const top3d,
                  const LayerOption* const option)
{
  // kernel size, padding size & stride size
  const int kernel_h = option->kernel_h;
  const int kernel_w = option->kernel_w;
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom3d->data;
  real* p_top_item = top3d->data;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: C x H x W
    const int C = bottom3d->shape[n][0];  // C
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // set top shape: C x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H
        = 1 + DIV_THEN_CEIL(bottom_H + 2 * pad_h - kernel_h,  stride_h);
    const int top_W
        = 1 + DIV_THEN_CEIL(bottom_W + 2 * pad_w - kernel_w,  stride_w);
    top3d->shape[n][0] = C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // max-pooling
    //   bottom3d (C x H x W) -> top3d (C x H' x W')
    #ifdef GPU
    {
      const int num_threads = C * top_H * top_W;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);
      max_pool_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item,  p_top_item,
          C,  bottom_H,  bottom_W,  top_H,  top_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    }
    #else
    {
      max_pool_cpu(
          p_bottom_item,  p_top_item,
          C,  bottom_H,  bottom_W,  top_H,  top_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    }
    #endif

    // locate next item
    {
      const int bottom_size = C * bottom_H * bottom_W;
      const int top_size = C * top_H * top_W;
      p_bottom_item += bottom_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------
static
void pool_shape(const Tensor* const bottom3d,
                Tensor* const top3d,
                const LayerOption* const option)
{
  const int kernel_h = option->kernel_h;
  const int kernel_w = option->kernel_w;
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // calculate shape for each item in the batch
  int total_size = 0;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: C x H x W
    const int C = bottom3d->shape[n][0];  // C
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // top shape: C x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H
        = 1 + DIV_THEN_CEIL(bottom_H + 2 * pad_h - kernel_h,  stride_h);
    const int top_W
        = 1 + DIV_THEN_CEIL(bottom_W + 2 * pad_w - kernel_w,  stride_w);
    top3d->shape[n][0] = C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // start position for n-th item in top3d->data
    top3d->start[n] = total_size;
    total_size += C * top_H * top_W;
  }
  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void forward_max_pool_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;

  pool_forward(get_bottom(layer, 0), get_top(layer, 0), &layer->option);
}

void shape_pool_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;

  pool_shape(get_bottom(layer, 0), get_top(layer, 0), &layer->option);
}
