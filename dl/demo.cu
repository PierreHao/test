#include "hip/hip_runtime.h"
#include "layer.h"
#include <string.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include <time.h>

static
const char* gs_class_names[] = {
  "__unknown__",
  "bicycle",
  "bird",
  "bus",
  "car",
  "cat",
  "dog",
  "horse",
  "motorbike",
  "person",
  "train",
  "aeroplane",
  "boat",
  "bottle",
  "chair",
  "cow",
  "diningtable",
  "pottedplant",
  "sheep",
  "sofa",
  "tvmonitor",
  "cake",
  "vase"
};

static
void draw_boxes(cv::Mat* const image,
                const real* const out_data,
                const int num_boxes,
                const float time)
{
  char label[128];
  for (int r = 0; r < num_boxes; ++r) {
    const real* const p_box = out_data + r * 6;
    const char* const class_name = gs_class_names[(int)p_box[0]];
    const real score = p_box[5];
    const int x1 = (int)ROUND(p_box[1]);
    const int y1 = (int)ROUND(p_box[2]);
    const int x2 = (int)ROUND(p_box[3]);
    const int y2 = (int)ROUND(p_box[4]);
    const int w = x2 - x1 + 1;
    const int h = y2 - y1 + 1;
    sprintf(label, "%s(%.2f)", class_name, score);

    if (score >= 0.8) {
      cv::rectangle(*image, cv::Rect(x1, y1, w, h),
                    cv::Scalar(0, 0, 255), 2);
    }
    else {
      cv::rectangle(*image, cv::Rect(x1, y1, w, h),
                    cv::Scalar(255, 0, 0), 1);
    }
    cv::putText(*image, label, cv::Point(x1, y1 + 15),
            2, 0.5, cv::Scalar(0, 0, 0), 2);
    cv::putText(*image, label, cv::Point(x1, y1 + 15),
            2, 0.5, cv::Scalar(255, 255, 255), 1);
  }
  if (time > 0) {
    sprintf(label, "%.3f sec", time);
    cv::putText(*image, label, cv::Point(10, 10),
                2, 0.5, cv::Scalar(0, 0, 0), 2);
    cv::putText(*image, label, cv::Point(10, 10),
                2, 0.5, cv::Scalar(255, 255, 255), 1);
  }
}

static
void detect_frame(Net* const net,
                  cv::Mat* const image)
{
  if (image && image->data) {
    const clock_t tick0 = clock();
    real time = 0;

    process_pvanet(net, image->data, image->rows, image->cols, NULL);

    {
      clock_t tick1 = clock();
      if (time == 0) {
        time = (real)(tick1 - tick0) / CLOCKS_PER_SEC;
      }
      else {
        time = time * 0.9f + (real)(tick1 - tick0) / CLOCKS_PER_SEC * 0.1f;
      }
    }

    draw_boxes(image, net->output_cpu_data, net->num_output_boxes, time);

    cv::imshow("faster-rcnn", *image);
  }
}

static
void test_stream(Net* const net, cv::VideoCapture& vc)
{
  cv::Mat image;

  while (1) {
    vc >> image;
    if (image.empty()) break;

    detect_frame(net, &image);

    if (cv::waitKey(1) == 27) break; //ESC
  }
}

static
void test_image(Net* const net, const char* const filename)
{
  cv::Mat image = cv::imread(filename);
  if (!image.data) {
    printf("Cannot open image: %s\n", filename);
    return;
  }

  detect_frame(net, &image);

  cv::waitKey(0);
}

static
void test_database(Net* const net,
                   const char* const db_filename,
                   const char* const out_filename)
{
  #if BATCH_SIZE == 4
  static const int batch_size = 4;
  #else
  static const int batch_size = 1;
  #endif

  char buf[10240];
  char* line[20];
  int total_count = 0, count = 0, buf_count = 0;
  FILE* fp_list = fopen(db_filename, "r");

  #ifndef DEMO
  FILE* fp_out = fopen(out_filename, "wb");
  #else
  FILE* fp_out = NULL;
  #endif

  clock_t tick0, tick1;
  float a_time[2] = { 0, };

  if (!fp_list) {
    printf("File not found: %s\n", db_filename);
  }

  #ifndef DEMO
  if (!fp_out) {
    printf("File write error: %s\n", out_filename);
  }
  #endif

  tick0 = clock();

  while (fgets(&buf[buf_count], 1024, fp_list))
  {
    {
      const int len = strlen(&buf[buf_count]);

      buf[buf_count + len - 1] = 0;
      line[count] = &buf[buf_count];
      ++count;
      buf_count += len;
    }

    if (count == batch_size)
    {
    #if BATCH_SIZE == 4
      cv::Mat images[] = {
        cv::imread(line[0]), cv::imread(line[1]),
        cv::imread(line[2]), cv::imread(line[3])
      };
      const unsigned char* const images_data[] = {
        images[0].data, images[1].data, images[2].data, images[3].data
      };
      const int heights[] = {
        images[0].rows, images[1].rows, images[2].rows, images[3].rows
      };
      const int widths[] = {
        images[0].cols, images[1].cols, images[2].cols, images[3].cols
      };
    #else
      cv::Mat images[] = { cv::imread(line[0]) };
      const unsigned char* const images_data[] = { images[0].data };
      const int heights[] = { images[0].rows };
      const int widths[] = { images[0].cols };
    #endif

      process_batch_pvanet(net, images_data, heights, widths, batch_size,
                           fp_out);

      tick1 = clock();
      a_time[0] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;
      a_time[1] += (float)(tick1 - tick0) / CLOCKS_PER_SEC;
      tick0 = tick1;
      printf("Running time: %.2f (current), %.2f (average)\n",
             a_time[0] * 1000 / count,
             a_time[1] * 1000 / (total_count + count));

      total_count += count;
      count = 0;
      buf_count = 0;
    }
  }

  if (count > 0) {
    for (int n = 0; n < count; ++n) {
      cv::Mat image = cv::imread(line[n]);
      process_pvanet(net, image.data, image.rows, image.cols, fp_out);
    }

    tick1 = clock();
    a_time[0] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;
    a_time[1] += (float)(tick1 - tick0) / CLOCKS_PER_SEC;
    tick0 = tick1;
    printf("Running time: %.2f (current), %.2f (average)\n",
           a_time[0] * 1000 / count,
           a_time[1] * 1000 / (total_count + count));
  }

  if (fp_list) {
    fclose(fp_list);
  }
  if (fp_out) {
    fclose(fp_out);
  }
}

static
void print_usage(void)
{
  printf("[Usage] ./demo_gpu.bin <command> <model path> <arg1> <arg2> ...\n");
  printf("  1. [Live demo using WebCam] ./demo_gpu.bin live <model path> <camera id> <width> <height>\n");
  printf("  2. [Image file] ./demo_gpu.bin snapshot <model path> <image filename>\n");
  printf("  3. [Video file] ./demo_gpu.bin video <model path> <video filename>\n");
  printf("  4. [List of images] ./demo_gpu.bin database <model path> <DB filename> <output filename>\n");
}

static
int test(const char* const args[], const int num_args)
{
  Net pvanet;
  const char* const command = args[0];
  const char* const model_path = args[1];

  #ifdef GPU
  hipSetDevice(0);
  #endif

  construct_pvanet(&pvanet, model_path);

  if (strcmp(command, "live") == 0) {
    if (num_args >= 5) {
      const int camera_id = atoi(args[2]);
      const int frame_width = atoi(args[3]);
      const int frame_height = atoi(args[4]);

      cv::imshow("faster-rcnn", 0);
      cv::VideoCapture vc(camera_id);
      if (!vc.isOpened()) {
        printf("Cannot open camera(%d)\n", camera_id);
        cv::destroyAllWindows();
        return -1;
      }
      vc.set(CV_CAP_PROP_FRAME_WIDTH, frame_width);
      vc.set(CV_CAP_PROP_FRAME_HEIGHT, frame_height);
      test_stream(&pvanet, vc);
      cv::destroyAllWindows();
    }
    else {
      print_usage();
      return -1;
    }
  }

  else if (strcmp(command, "snapshot") == 0) {
    if (num_args > 2) {
      const char* const filename = args[2];

      cv::imshow("faster-rcnn", 0);
      test_image(&pvanet, filename);
      cv::destroyAllWindows();
    }
    else {
      print_usage();
      return -1;
    }
  }

  else if (strcmp(command, "video") == 0) {
    if (num_args > 2) {
      const char* const filename = args[2];

      cv::imshow("faster-rcnn", 0);
      cv::VideoCapture vc(filename);
      if (!vc.isOpened()) {
        printf("Cannot open video: %s\n", filename);
        cv::destroyAllWindows();
        return -1;
      }
      test_stream(&pvanet, vc);
      cv::destroyAllWindows();
    }
    else {
      print_usage();
      return -1;
    }
  }

  else if (strcmp(command, "database") == 0) {
    if (num_args > 3) {
      const char* const db_filename = args[2];
      const char* const out_filename = args[3];

      test_database(&pvanet, db_filename, out_filename);
    }
    else {
      print_usage();
      return -1;
    }
  }

  else {
    print_usage();
    return -1;
  }

  return 0;
}

#ifdef TEST
int main(int argc, char* argv[])
{
  if (argc >= 3) {
    test(argv + 1, argc - 1);
  }
  else {
    print_usage();
  }

  return 0;
}
#endif
