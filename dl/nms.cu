#include "hip/hip_runtime.h"
#include "layer.h"

// --------------------------------------------------------------------------
// kernel code
//   iou: compute overlap between two boxes
//   nms_mask: given a set of boxes, compute overlap between all box pairs
// --------------------------------------------------------------------------

// "IoU = intersection area / union area" of two boxes A, B
//   A, B: 4-dim array (x1, y1, x2, y2)
#ifdef GPU
__device__
#endif
static
real iou(const real A[], const real B[])
{
  #ifndef GPU
  if (A[0] > B[2] || A[1] > B[3] || A[2] < B[0] || A[3] < B[1]) {
    return 0;
  }
  else {
  #endif

  // overlapped region (= box)
  const real x1 = MAX(A[0],  B[0]);
  const real y1 = MAX(A[1],  B[1]);
  const real x2 = MIN(A[2],  B[2]);
  const real y2 = MIN(A[3],  B[3]);

  // intersection area
  const real width = MAX(0.0f,  x2 - x1 + 1.0f);
  const real height = MAX(0.0f,  y2 - y1 + 1.0f);
  const real area = width * height;

  // area of A, B
  const real A_area = (A[2] - A[0] + 1.0f) * (A[3] - A[1] + 1.0f);
  const real B_area = (B[2] - B[0] + 1.0f) * (B[3] - B[1] + 1.0f);

  // IoU
  return area / (A_area + B_area - area);

  #ifndef GPU
  }
  #endif
}

// given box proposals, compute overlap between all box pairs
// (overlap = intersection area / union area)
// and then set mask-bit to 1 if a pair is significantly overlapped
//   num_boxes: number of box proposals given
//   boxes: "num_boxes x 5" array (x1, y1, x2, y2, score)
//   nms_thresh: threshold for determining "significant overlap"
//               if "intersection area / union area > nms_thresh",
//               two boxes are thought of as significantly overlapped
// the all-pair computation (num_boxes x num_boxes) is done by
// divide-and-conquer:
//   each GPU block (bj, bi) computes for "64 x 64" box pairs (j, i),
//     j = bj * 64 + { 0, 1, ..., 63 }
//     i = bi * 64 + { 0, 1, ..., 63 },
//   and each "1 x 64" results is saved into a 64-bit mask
//     mask: "num_boxes x num_blocks" array
//     for mask[j][bi], "di-th bit = 1" means:
//       box j is significantly overlapped with box i,
//       where i = bi * 64 + di
typedef unsigned long long uint64;
#define NMS_BLOCK_SIZE 64
#ifdef GPU
__global__
void nms_mask_gpu(const real boxes[], uint64 mask[],
                  const int num_boxes, const real nms_thresh)
{
  // block region
  //   j = j_start + { 0, ..., dj_end - 1 }
  //   i = i_start + { 0, ..., di_end - 1 }
  const int i_start = blockIdx.x * NMS_BLOCK_SIZE;
  const int di_end = MIN(num_boxes - i_start,  NMS_BLOCK_SIZE);
  const int j_start = blockIdx.y * NMS_BLOCK_SIZE;
  const int dj_end = MIN(num_boxes - j_start,  NMS_BLOCK_SIZE);

  // copy all i-th boxes to GPU cache
  //   i = i_start + { 0, ..., di_end - 1 }
  __shared__ real boxes_i[NMS_BLOCK_SIZE * 4];
  {
    const int di = threadIdx.x;
    if (di < di_end) {
      boxes_i[di * 4 + 0] = boxes[(i_start + di) * 5 + 0];
      boxes_i[di * 4 + 1] = boxes[(i_start + di) * 5 + 1];
      boxes_i[di * 4 + 2] = boxes[(i_start + di) * 5 + 2];
      boxes_i[di * 4 + 3] = boxes[(i_start + di) * 5 + 3];
    }
  }
  __syncthreads();

  // given j = j_start + dj,
  //   check whether box i is significantly overlapped with box j
  //   (i.e., IoU(box j, box i) > threshold)
  //   for all i = i_start + { 0, ..., di_end - 1 } except for i == j
  {
    const int dj = threadIdx.x;
    if (dj < dj_end) {
      // box j
      const real* const box_j = boxes + (j_start + dj) * 5;

      // mask for significant overlap
      //   if IoU(box j, box i) > threshold,  di-th bit = 1
      uint64 mask_j = 0;

      // check for all i = i_start + { 0, ..., di_end - 1 }
      // except for i == j
      const int di_start = (i_start == j_start) ? (dj + 1) : 0;
      for (int di = di_start; di < di_end; ++di) {
        // box i
        const real* const box_i = boxes_i + di * 4;

        // if IoU(box j, box i) > threshold,  di-th bit = 1
        if (iou(box_j, box_i) > nms_thresh) {
          mask_j |= 1ULL << di;
        }
      }

      // mask: "num_boxes x num_blocks" array
      //   for mask[j][bi], "di-th bit = 1" means:
      //     box j is significantly overlapped with box i = i_start + di,
      //     where i_start = bi * block_size
      {
        const int num_blocks = DIV_THEN_CEIL(num_boxes,  NMS_BLOCK_SIZE);
        const int bi = blockIdx.x;
        mask[(j_start + dj) * num_blocks + bi] = mask_j;
      }
    } // endif dj < dj_end
  }
}

__global__
void nms_sub(const int num_boxes, const real boxes[],
             const real nms_thresh, unsigned char is_dead[],
             int i)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index > i && index < num_boxes && !is_dead[index]) {
    const real iou_val = iou(&boxes[i * 5], &boxes[index * 5]);
    if (iou_val > nms_thresh) {
      is_dead[index] = 1;
    }
  }
}

void nms_(const int num_boxes, const real boxes[],
         int* const num_out, int keep_out[], const int base_index,
         const real nms_thresh, const int max_num_out,
         const int bbox_vote)
{
  unsigned char *is_dead_dev;
  unsigned char is_dead;
  real* boxes_dev;
  int num_to_keep = 0;

  const int num_threads = num_boxes;
  const int threads_per_block = 512;
  const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);

  hipMalloc(&is_dead_dev, num_boxes * sizeof(unsigned char));
  hipMemset(is_dead_dev, 0, num_boxes * sizeof(unsigned char));
  hipMalloc(&boxes_dev, num_boxes * 5 * sizeof(real));
  hipMemcpyAsync(boxes_dev, boxes, num_boxes * 5 * sizeof(real),
                  hipMemcpyHostToDevice);

  for (int i = 0; i < num_boxes; ++i) {
    hipMemcpy(&is_dead, is_dead_dev + i, sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (is_dead) {
      continue;
    }

    keep_out[num_to_keep++] = base_index + i;
    if (num_to_keep == max_num_out) {
      break;
    }

    nms_sub<<<num_blocks, threads_per_block>>>(
      num_boxes,  boxes_dev,  nms_thresh,  is_dead_dev,  i);
  }

  *num_out = num_to_keep;

  hipFree(is_dead_dev);
  hipFree(boxes_dev);
}

#else
void nms(const int num_boxes, real boxes[],
         int* const num_out, int keep_out[], const int base_index,
         const real nms_thresh, const int max_num_out,
         const int bbox_vote, const real vote_thresh)
{
  unsigned char* const is_dead =
      (unsigned char*)calloc(num_boxes, sizeof(unsigned char));
  int num_to_keep = 0;
  for (int i = 0; i < num_boxes; ++i) {
    if (is_dead[i]) {
      continue;
    }

    keep_out[num_to_keep++] = base_index + i;

    if (bbox_vote) {
      real sum_score = boxes[i * 5 + 4];
      real sum_box[4] = {
          sum_score * boxes[i * 5 + 0], sum_score * boxes[i * 5 + 1],
          sum_score * boxes[i * 5 + 2], sum_score * boxes[i * 5 + 3]
      };

      for (int j = 0; j < i; ++j) {
        if (is_dead[j] && iou(&boxes[i * 5], &boxes[j * 5]) > vote_thresh) {
          real score = boxes[j * 5 + 4];
          sum_box[0] += score * boxes[j * 5 + 0];
          sum_box[1] += score * boxes[j * 5 + 1];
          sum_box[2] += score * boxes[j * 5 + 2];
          sum_box[3] += score * boxes[j * 5 + 3];
          sum_score += score;
        }
      }
      for (int j = i + 1; j < num_boxes; ++j) {
        real iou_val = iou(&boxes[i * 5], &boxes[j * 5]);
        if (!is_dead[j] && iou_val > nms_thresh) {
          is_dead[j] = 1;
        }
        if (iou_val > vote_thresh) {
          real score = boxes[j * 5 + 4];
          sum_box[0] += score * boxes[j * 5 + 0];
          sum_box[1] += score * boxes[j * 5 + 1];
          sum_box[2] += score * boxes[j * 5 + 2];
          sum_box[3] += score * boxes[j * 5 + 3];
          sum_score += score;
        }
      }

      boxes[i * 5 + 0] = sum_box[0] / sum_score;
      boxes[i * 5 + 1] = sum_box[1] / sum_score;
      boxes[i * 5 + 2] = sum_box[2] / sum_score;
      boxes[i * 5 + 3] = sum_box[3] / sum_score;
    }

    else {
      for (int j = i + 1; j < num_boxes; ++j) {
        if (!is_dead[j] && iou(&boxes[i * 5], &boxes[j * 5]) > nms_thresh) {
          is_dead[j] = 1;
        }
      }
    }

    if (num_to_keep == max_num_out) {
      break;
    }
  }

  *num_out = num_to_keep;

  free(is_dead);
}

void nms_mask_cpu(const real boxes[], uint64 mask[],
                  const int num_boxes, const real nms_thresh)
{
  // number of blocks along each dimension
  const int num_blocks = DIV_THEN_CEIL(num_boxes,  NMS_BLOCK_SIZE);

  // the whole 2-dim computations "num_boxes x num_boxes" is done by
  // sweeping all "64 x 64"-sized blocks
  for (int j_start = 0; j_start < num_boxes; j_start += NMS_BLOCK_SIZE) {
    for (int i_start = 0; i_start < num_boxes; i_start += NMS_BLOCK_SIZE) {
      // block region
      //   j = j_start + { 0, ..., dj_end - 1 }
      //   i = i_start + { 0, ..., di_end - 1 }
      const int di_end = MIN(num_boxes - i_start,  NMS_BLOCK_SIZE);
      const int dj_end = MIN(num_boxes - j_start,  NMS_BLOCK_SIZE);

      // check whether box i is significantly overlapped with box j
      // for all j = j_start + { 0, ..., dj_end - 1 },
      //         i = i_start + { 0, ..., di_end - 1 },
      // except for i == j
      for (int dj = 0; dj < dj_end; ++dj) {
        // box j & overlap mask
        const real* const box_j = boxes + (j_start + dj) * 5;
        uint64 mask_j = 0;

        // check for all i = i_start + { 0, ..., di_end - 1 }
        // except for i == j
        const int di_start = (i_start == j_start) ? (dj + 1) : 0;
        for (int di = di_start; di < di_end; ++di) {
          // box i
          const real* const box_i = boxes + (i_start + di) * 5;

          // if IoU(box j, box i) > threshold,  di-th bit = 1
          if (iou(box_j, box_i) > nms_thresh) {
            mask_j |= 1ULL << di;
          }
        }

        // mask: "num_boxes x num_blocks" array
        //   for mask[j][bi], "di-th bit = 1" means:
        //     box j is significantly overlapped with box i = i_start + di,
        //     where i_start = bi * block_size
        {
          const int bi = i_start / NMS_BLOCK_SIZE;
          mask[(j_start + dj) * num_blocks + bi] = mask_j;
        }
      } // endfor dj
    } // endfor j_start
  } // endfor i_start
}
#endif



#ifdef GPU
// --------------------------------------------------------------------------
// operator code
//   nms: given a set of boxes, discard significantly-overlapped boxes
// --------------------------------------------------------------------------

// given box proposals (sorted in descending order of their scores),
// discard a box if it is significantly overlapped with
// one or more previous (= scored higher) boxes
//   num_boxes: number of box proposals given
//   boxes: "num_boxes x 5" array (x1, y1, x2, y2, score)
//          sorted in descending order of scores
//   num_out: number of remaining boxes
//   keep_out: "num_out x 1" array
//             indices of remaining boxes
//   base_index: a constant added to keep_out,  usually 0
//               keep_out[i] = base_index + actual index in boxes
//   nms_thresh: threshold for determining "significant overlap"
//               if "intersection area / union area > nms_thresh",
//               two boxes are thought of as significantly overlapped
//   bbox_vote: whether bounding-box voting is used (= 1) or not (= 0)
//   vote_thresh: threshold for selecting overlapped boxes
//                which are participated in bounding-box voting
void nms(const int num_boxes, real boxes[],
         int* const num_out, int keep_out[], const int base_index,
         const real nms_thresh, const int max_num_out,
         const int bbox_vote, const real vote_thresh)
{
  const int num_blocks = DIV_THEN_CEIL(num_boxes,  NMS_BLOCK_SIZE);
  uint64* const mask
      = (uint64*)malloc(num_boxes * num_blocks * sizeof(uint64));

  #ifdef GPU
  {
    real* boxes_dev;
    uint64* mask_dev;
    const dim3 blocks(num_blocks, num_blocks);

    // GPU memory allocation & copy box data
    hipMalloc(&boxes_dev, num_boxes * 5 * sizeof(real));
    hipMemcpyAsync(boxes_dev, boxes, num_boxes * 5 * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMalloc(&mask_dev, num_boxes * num_blocks * sizeof(uint64));

    // find all significantly-overlapped pairs of boxes
    nms_mask_gpu<<<blocks, NMS_BLOCK_SIZE>>>(
        boxes_dev,  mask_dev,  num_boxes,  nms_thresh);

    // copy mask data to main memory
    hipMemcpyAsync(mask, mask_dev, sizeof(uint64) * num_boxes * num_blocks,
                    hipMemcpyDeviceToHost);

    // GPU memory deallocation
    hipFree(boxes_dev);
    hipFree(mask_dev);
  }
  #else
  {
    // find all significantly-overlapped pairs of boxes
    nms_mask_cpu(boxes,  mask,  num_boxes,  nms_thresh);
  }
  #endif

  // discard i-th box if it is significantly overlapped with
  // one or more previous (= scored higher) boxes
  {
    int num_to_keep = 0;
    uint64* const remv = (uint64*)calloc(num_blocks, sizeof(uint64));

    for (int i = 0; i < num_boxes; ++i) {
      const int nblock = i / NMS_BLOCK_SIZE;
      const int inblock = i % NMS_BLOCK_SIZE;

      if (!(remv[nblock] & (1ULL << inblock))) {
        keep_out[num_to_keep++] = base_index + i;
        uint64* p = mask + i * num_blocks;
        for (int j = nblock; j < num_blocks; ++j) {
          remv[j] |= p[j];
        }

        if (num_to_keep == max_num_out) {
          break;
        }
      }
    }
    *num_out = num_to_keep;

    free(remv);
  }

  free(mask);
}
#endif
