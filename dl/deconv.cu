#include "hip/hip_runtime.h"
#include "layer.h"

// --------------------------------------------------------------------------
// kernel code
//   convert_top_{gpu, cpu}
// --------------------------------------------------------------------------

// convert top5d (C x kernel_h x kernel_w x H5 x W5)
//         -> top3d (C x H x W)
//   TODO: detailed description
#ifdef GPU
__global__
void convert_top_gpu(const real* const top5d,
                     real* const top3d,
                     const int C, const int H, const int W,
                     const int H5, const int W5,
                     const int kernel_h, const int kernel_w,
                     const int pad_h, const int pad_w,
                     const int stride_h, const int stride_w)
{
  // thread index: (c, h, w) = c*H*W + h*W + w
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  {
    // parse thread index -> (c, h, w)
    const int c = index / (H * W);
    const int h = (index / W) % H + pad_h;
    const int w = index % W + pad_w;

    // range of summation
    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    //   0 <= h5 <= 0
    //   0 <= w5 <= 0
    //   TODO: optimization & description
    const int h5_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int h5_end = min(h / stride_h + 1, H5);
    const int w5_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int w5_end = min(w / stride_w + 1, W5);
    const real* p_top5d = top5d +
                  (c * kernel_h * kernel_w + h * kernel_w + w) * H5 * W5;
    const int h5_coef = (1 - stride_h * kernel_w * H5) * W5;
    const int w5_coef = 1 - stride_w * H5 * W5;

    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    real val = 0;
    for (int h5 = h5_start; h5 < h5_end; ++h5) {
      for (int w5 = w5_start; w5 < w5_end; ++w5) {
        val += p_top5d[h5 * h5_coef + w5 * w5_coef];
      }
    }
    top3d[index] = val;
  }
}
#else
void convert_top_cpu(const real* const top5d,
                     real* const top3d,
                     const int C, const int H, const int W,
                     const int H5, const int W5,
                     const int kernel_h, const int kernel_w,
                     const int pad_h, const int pad_w,
                     const int stride_h, const int stride_w)
{
  // thread index: (c, h, w) = c*H*W + h*W + w
  for (int index = 0; index < C * H * W; ++index) {
    // parse thread index -> (c, h, w)
    const int c = index / (H * W);
    const int h = (index / W) % H + pad_h;
    const int w = index % W + pad_w;

    // range of summation
    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    //   0 <= h5 <= 0
    //   0 <= w5 <= 0
    //   TODO: optimization & description
    const int h5_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int h5_end = MIN(h / stride_h + 1, H5);
    const int w5_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int w5_end = MIN(w / stride_w + 1, W5);
    const real* p_top5d = top5d +
                  (c * kernel_h * kernel_w + h * kernel_w + w) * H5 * W5;
    const int h5_coef = (1 - stride_h * kernel_w * H5) * W5;
    const int w5_coef = 1 - stride_w * H5 * W5;

    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    real val = 0;
    for (int h5 = h5_start; h5 < h5_end; ++h5) {
      for (int w5 = w5_start; w5 < w5_end; ++w5) {
        val += p_top5d[h5 * h5_coef + w5 * w5_coef];
      }
    }
    top3d[index] = val;
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   deconv_forward
// --------------------------------------------------------------------------

// deconvolution: bottom -> top
//   G: number of groups
//   bottom: (G * C') x H' x W'
//   top: (G * C) x H x W
//   weight: G x C' x C x kernel_h x kernel_w
//   bias: (G * C) x 1
//   temp: (G * C * kernel_h * kernel_w) x (H' * W') array
//   const: 1 x (H * W) array,  const[i] = 1 for all i
void deconv_forward(const Tensor* const bottom3d,
                    Tensor* const top3d,
                    const Tensor* const weight5d,
                    const Tensor* const bias1d,
                    real* const temp_data,
                    const real* const const_data,
                    const ConvOption* const option)
{
  // weight shape: G x C' x C x kernel_h x kernel_w
  const int num_groups = weight5d->shape[0][0]; // G
  const int bottom_C = weight5d->shape[0][1];  // C'
  const int top_C = weight5d->shape[0][2];  // C
  const int kernel_h = weight5d->shape[0][3];
  const int kernel_w = weight5d->shape[0][4];

  // padding size & stride size
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom3d->data;
  real* p_top_item = top3d->data;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: (G * C') x H' x W'
    const int bottom_H = bottom3d->shape[n][1];  // H'
    const int bottom_W = bottom3d->shape[n][2];  // W'

    // set top shape: (G * C) x H x W
    //   H' = 1 + (H + 2 * pad_h - kernel_h) / stride_h
    //   -> H = stride_h * (H' - 1) - 2 * pad_h + kernel_h
    const int top_H = stride_h * (bottom_H - 1) - 2 * pad_h + kernel_h;
    const int top_W = stride_w * (bottom_W - 1) - 2 * pad_w + kernel_w;
    top3d->shape[n][0] = num_groups * top_C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // compute top[g] = dot(weight[g].transpose(), bottom[g])
    //   weight[g]: C' x (C * kernel_h * kernel_w)
    //   bottom[g]: C' x (H' * W')
    //   top[g]: (C * kernel_h * kernel_w) x (H' * W')
    for (int g = 0; g < num_groups; ++g) {
      const int kernel_size = top_C * kernel_h * kernel_w;
      const int bottom_area = bottom_H * bottom_W;
      const real* const p_bottom_g = p_bottom_item +
                                     g * bottom_C * bottom_area;
      const real* const p_weight_g = weight5d->data +
                                     g * bottom_C * kernel_size;
      real* const p_temp_g = temp_data + g * kernel_size * bottom_area;

      // compute Z = alpha * dot(X.transpose(), Y) + beta * Z
      //   X (= weight): p x m,  Y (= bottom): p x n,  Z (= top): m x n
      //   X, Y, Z: row-major order (e.g., Z[i][j] = Z[i * n + j])
    #ifdef GPU
      // input arguments:
      //   cublas handle,
      //   do_transpose_Y (= false),  do_transpose_X (= true),
      //   n (= H' * W'),  m (= C * kernel_h * kernel_w),  p (= C'),
      //   &alpha (= 1),
      //   &Y,  number of columns in Y (= n),
      //   &X,  number of columns in X (= m),
      //   &beta (= 0),
      //   &Z,  number of columns in Z (= n)
      const real one = 1.0f, zero = 0.0f;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_T,
                  bottom_area,  kernel_size,  bottom_C,
                  &one,
                  p_bottom_g,  bottom_area,
                  p_weight_g,  kernel_size,
                  &zero,
                  p_temp_g,  bottom_area);
    #else
      // input arguments:
      //   is_row_major_order (= true),
      //   do_transpose_X (= true),  do_transpose_Y (= false),
      //   m (= C * kernel_h * kernel_w),  n (= H' * W'),  p (= C'),
      //   alpha (= 1),
      //   &X,  number of columns in X (= m),
      //   &Y,  number of columns in Y (= n),
      //   beta (= 0),
      //   &Z,  number of columns in Z (= n)
      cblas_sgemm(CblasRowMajor,
                  CblasTrans,  CblasNoTrans,
                  kernel_size,  bottom_area,  bottom_C,
                  1.0f,
                  p_weight_g,  kernel_size,
                  p_bottom_g,  bottom_area,
                  0.0f,
                  p_temp_g,  bottom_area);
    #endif
    }

    // convert top shape
    //   (G * C * kernel_h * kernel_w) x (H' * W') -> (G * C) x (H * W)
    {
    #ifdef GPU
      // one thread computes one entry in top
      const int num_threads = num_groups * top_C * top_H * top_W;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads, threads_per_block);
      convert_top_gpu<<<num_blocks, threads_per_block>>>(
          temp_data,  p_top_item,
          num_groups * top_C,  top_H,  top_W,
          bottom_H,  bottom_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    #else
      convert_top_cpu(
          temp_data,  p_top_item,
          num_groups * top_C,  top_H,  top_W,
          bottom_H,  bottom_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    #endif
    }

    // compute top[i][j] = top[i][j] + bias[i]
    //   top: (G * C) x (H * W)
    //   bias: (G * C) x 1
    if (option->bias) {
      const int top_channels = num_groups * top_C;
      const int top_area = top_H * top_W;

      // the computation is equivalent to...
      //   top = top + dot(bias, constant)
      //   constant: 1 x (H * W),  constant[i] = 1 for all i
    #ifdef GPU
      // thus, input arguments:
      //   do_transpose_Y (= false),  do_transpose_X (= false),
      //   n = H * W,  m = G * C,  p = 1
      //   alpha = 1,  beta = 1
      const real one = 1.0;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_N,
                  top_area,  top_channels,  1,
                  &one,
                  const_data,  top_area,
                  bias1d->data,  1,
                  &one,
                  p_top_item,  top_area);
    #else
      // input arguments:
      //   do_transpose_X (= false),  do_transpose_Y (= false),
      //   m = G * C,  n = H * W,  p = 1
      //   alpha = 1,  beta = 1
      cblas_sgemm(CblasRowMajor,
                  CblasNoTrans,  CblasNoTrans,
                  top_channels,  top_area,  1,
                  1.0f,
                  bias1d->data,  1,
                  const_data,  top_area,
                  1.0f,
                  p_top_item,  top_area);
    #endif
    }

    // locate next item
    {
      const int bottom_size = num_groups * bottom_C * bottom_H * bottom_W;
      const int top_size = num_groups * top_C * top_H * top_W;
      p_bottom_item += bottom_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;
}



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST
#include <stdio.h>
#include <stdlib.h>

#define DATA_SIZE 512*36*46
#define WEIGHT_SIZE 512*1*1*4*4
#define BIAS_SIZE 512
#define CONST_SIZE 36*46

int main(int argc, char *argv[])
{
  // variable declaration & memory allocation
  Tensor X, Y, W, b;
  real* const X_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* const Y_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* const Y_true_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* const W_data = (real*)malloc(WEIGHT_SIZE * sizeof(real));
  real* const b_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* const const_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* p_temp_data;
  real* p_const_data;
  ConvOption option;

  // set option
  {
    option.num_groups = 512;
    option.out_channels = 512;
    option.kernel_h = 4;
    option.kernel_w = 4;
    option.pad_h = 1;
    option.pad_w = 1;
    option.stride_h = 2;
    option.stride_w = 2;
    option.bias = 0;
  }

  // set data shapes
  {
    X.ndim = 3;
    X.num_items = 1;
    for (int i = 0; i < X.num_items; ++i) {
      X.shape[i][0] = 512;
      X.shape[i][1] = 18;
      X.shape[i][2] = 23;
    }

    Y.ndim = X.ndim;
    Y.num_items = X.num_items;
    for (int i = 0; i < Y.num_items; ++i) {
      Y.shape[i][0] = option.out_channels;
      Y.shape[i][1] = option.stride_h * (X.shape[i][1] - 1)
                      - 2 * option.pad_h + option.kernel_h;
      Y.shape[i][2] = option.stride_w * (X.shape[i][2] - 1)
                      - 2 * option.pad_w + option.kernel_w;
    }

    W.ndim = 5; W.num_items = 1;
    W.shape[0][0] = option.num_groups;
    W.shape[0][1] = X.shape[0][0] / option.num_groups;
    W.shape[0][2] = option.out_channels / option.num_groups;
    W.shape[0][3] = option.kernel_h;
    W.shape[0][4] = option.kernel_w;

    b.ndim = 1; b.num_items = 1;
    b.shape[0][0] = option.out_channels;
  }

  // load data
  {
    FILE* fp;
    const int X_size = flatten_size(&X);
    const int Y_size = flatten_size(&Y);
    const int W_size = flatten_size(&W);
    const int b_size = flatten_size(&b);

    printf("data loading\n");

    fp = fopen("../data/temp/deconv_bottom0.bin", "rb");
    if ((int)fread(X_data, sizeof(real), X_size, fp) != X_size) {
      printf("Error while reading deconv_bottom0\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/deconv_param0.bin", "rb");
    if ((int)fread(W_data, sizeof(real), W_size, fp) != W_size) {
      printf("Error while reading deconv_param0\n");
    }
    fclose(fp);

    if (option.bias) {
      fp = fopen("../data/temp/deconv_param1.bin", "rb");
      if ((int)fread(b_data, sizeof(real), b_size, fp) != b_size) {
        printf("Error while reading deconv_param1\n");
      }
      fclose(fp);

      for (int i = 0; i < CONST_SIZE; ++i) {
        const_data[i] = 1;
      }
    }

    fp = fopen("../data/temp/deconv_top0.bin", "rb");
    if ((int)fread(Y_true_data, sizeof(real), Y_size, fp) != Y_size) {
      printf("Error while reading deconv_top0\n");
    }
    fclose(fp);
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    CUDA_CHECK(hipSetDevice(0));
    option.handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    if (hipblasCreate((hipblasHandle_t*)option.handle)
          != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const int X_size = flatten_size(&X);
    const int Y_size = flatten_size(&Y);
    const int W_size = flatten_size(&W);
    const int b_size = flatten_size(&b);
    const int temp_size = option.kernel_h * option.kernel_w *
                          Y.shape[0][0] * X.shape[0][1] * X.shape[0][2];

    printf("gpu malloc\n");
    CUDA_CHECK(hipMalloc(&X.data, X_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&Y.data, Y_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&W.data, W_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&b.data, b_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&p_temp_data, temp_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&p_const_data, CONST_SIZE * sizeof(real)));

    printf("memcpy: cpu -> gpu\n");
    CUDA_CHECK(hipMemcpyAsync(X.data, X_data, X_size * sizeof(real),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(W.data, W_data, W_size * sizeof(real),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(b.data, b_data, b_size * sizeof(real),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(p_const_data, const_data,
                          CONST_SIZE * sizeof(real),
                          hipMemcpyHostToDevice));
  }
  #else
  {
    const int temp_size = option.kernel_h * option.kernel_w *
                          Y.shape[0][0] * X.shape[0][1] * X.shape[0][2];

    X.data = X_data;
    Y.data = Y_data;
    W.data = W_data;
    b.data = b_data;
    p_temp_data = (real*)malloc(temp_size * sizeof(real));
    p_const_data = const_data;
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    deconv_forward(&X, &Y, &W, &b, p_temp_data, p_const_data, &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const int Y_size = flatten_size(&Y);

    printf("memcpy: cpu <- gpu\n");
    CUDA_CHECK(hipMemcpyAsync(Y_data, Y.data, Y_size * sizeof(real),
                          hipMemcpyDeviceToHost));
  }
  #endif

  // verify results
  {
    int i = 0;

    printf("verification\n");

    for (int n = 0; n < Y.num_items; ++n) {
      for (int c = 0; c < Y.shape[n][0]; ++c) {
        for (int h = 0; h < Y.shape[n][1]; ++h) {
          for (int w = 0; w < Y.shape[n][2]; ++w) {
            real diff = ABS(Y_data[i] - Y_true_data[i]);
            diff /= 1e-10f + MIN(ABS(Y_data[i]), ABS(Y_true_data[i]));
          #ifdef GPU
            if (diff > 0) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
          #else
            if (diff > 1e-3f) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
          #endif
            ++i;
          } // endfor w
        } // endfor h
      } // endfor c
    } // endfor n
  }

  // memory deallocation
  {
    printf("free\n");
    free(X_data);
    free(Y_data);
    free(Y_true_data);
    free(W_data);
    free(b_data);
    free(const_data);
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    CUDA_CHECK(hipFree(X.data));
    CUDA_CHECK(hipFree(Y.data));
    CUDA_CHECK(hipFree(W.data));
    CUDA_CHECK(hipFree(b.data));
    CUDA_CHECK(hipFree(p_temp_data));
    CUDA_CHECK(hipFree(p_const_data));

    if (hipblasDestroy(*((hipblasHandle_t*)option.handle))
          != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
    free(option.handle);
  }
  #else
  {
    free(p_temp_data);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
