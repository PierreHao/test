#include "hip/hip_runtime.h"
#include "layer.h"

#ifdef GPU
#include "cuda_settings.h"
#else
#include <cblas.h>
#endif

// convert top5d (C x kernel_h x kernel_w x H5 x W5)
//         -> top3d (C x H x W)
//   TODO: detailed description
#ifdef GPU
__global__
void convert_top_gpu(const real* const top5d,
                     real* const top3d,
                     const int C, const int H, const int W,
                     const int H5, const int W5,
                     const int kernel_h, const int kernel_w,
                     const int pad_h, const int pad_w,
                     const int stride_h, const int stride_w)
{
  // thread index: (c, h, w) = c*H*W + h*W + w
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < C * H * W;
       index += blockDim.x) {
    // parse thread index -> (c, h, w)
    const int c = index / (H * W);
    const int h = (index / W) % H + pad_h;
    const int w = index % W + pad_w;

    // range of summation
    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    //   0 <= h5 <= 0
    //   0 <= w5 <= 0
    //   TODO: optimization & description
    const int h5_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int h5_end = min(h / stride_h + 1, H5);
    const int w5_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int w5_end = min(w / stride_w + 1, W5);
    const real* p_top5d = top5d + (c * kernel_h * kernel_w + h * kernel_w + w) * H5 * W5;
    const int h5_coef = (1 - stride_h * kernel_w * H5) * W5;
    const int w5_coef = 1 - stride_w * H5 * W5;

    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    real val = 0;
    for (int h5 = h5_start; h5 < h5_end; ++h5) {
      for (int w5 = w5_start; w5 < w5_end; ++w5) {
        val += p_top5d[h5 * h5_coef + w5 * w5_coef];
      }
    }
    top3d[index] = val;
  }
}
#else
void convert_top_cpu(const real* const top5d,
                     real* const top3d,
                     const int C, const int H, const int W,
                     const int H5, const int W5,
                     const int kernel_h, const int kernel_w,
                     const int pad_h, const int pad_w,
                     const int stride_h, const int stride_w)
{
  // thread index: (c, h, w) = c*H*W + h*W + w
  for (int index = 0; index < C * H * W; ++index) {
    // parse thread index -> (c, h, w)
    const int c = index / (H * W);
    const int h = (index / W) % H + pad_h;
    const int w = index % W + pad_w;

    // range of summation
    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    //   0 <= h5 <= 0
    //   0 <= w5 <= 0
    //   TODO: optimization & description
    const int h5_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int h5_end = MIN(h / stride_h + 1, H5);
    const int w5_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int w5_end = MIN(w / stride_w + 1, W5);
    const real* p_top5d = top5d + (c * kernel_h * kernel_w + h * kernel_w + w) * H5 * W5;
    const int h5_coef = (1 - stride_h * kernel_w * H5) * W5;
    const int w5_coef = 1 - stride_w * H5 * W5;

    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    real val = 0;
    for (int h5 = h5_start; h5 < h5_end; ++h5) {
      for (int w5 = w5_start; w5 < w5_end; ++w5) {
        val += p_top5d[h5 * h5_coef + w5 * w5_coef];
      }
    }
    top3d[index] = val;
  }
}
#endif

// deconvolution: bottom -> top
//   bottom: (G * C') x H' x W'
//   top: (G * C) x H x W
//   weight: G x C' x C x kernel_h x kernel_w
//   bias: (G * C) x 1
//   temp: G * C * kernel_h * kernel_w * H' * W'
//   const: H * W,  const[i] = 1 for all i
//   G: number of groups
void deconv_forward(const Tensor* const bottom3d,
                    Tensor* const top3d,
                    const Tensor* const weight5d,
                    const Tensor* const bias1d,
                    real* const temp_data,
                    const real* const const_data,
                    const ConvOption* const option)
{
  // weight shape: G x C' x C x kernel_h x kernel_w
  const int num_groups = weight5d->shape[0][0]; // G
  const int bottom_C = weight5d->shape[0][1];  // C'
  const int top_C = weight5d->shape[0][2];  // C
  const int kernel_h = weight5d->shape[0][3];
  const int kernel_w = weight5d->shape[0][4];

  // padding size & stride size
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom3d->data;
  real* p_top_item = top3d->data;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: (G * C') x H' x W'
    const int bottom_H = bottom3d->shape[n][1];  // H'
    const int bottom_W = bottom3d->shape[n][2];  // W'

    // set top shape: (G * C) x H x W
    //   H' = 1 + (H + 2 * pad_h - kernel_h) / stride_h
    //   -> H = stride_h * (H' - 1) - 2 * pad_h + kernel_h
    const int top_H = stride_h * (bottom_H - 1) - 2 * pad_h + kernel_h;
    const int top_W = stride_w * (bottom_W - 1) - 2 * pad_w + kernel_w;
    top3d->shape[n][0] = num_groups * top_C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // compute top[g] = dot(weight[g].transpose(), bottom[g])
    //   weight[g]: C' x (C * kernel_h * kernel_w)
    //   bottom[g]: C' x (H' * W')
    //   top[g]: (C * kernel_h * kernel_w) x (H' * W')
    for (int g = 0; g < num_groups; ++g) {
      const int kernel_size = top_C * kernel_h * kernel_w;
      const int bottom_area = bottom_H * bottom_W;
      const real* const p_bottom_g = p_bottom_item + g * bottom_C * bottom_area;
      const real* const p_weight_g = weight5d->data + g * bottom_C * kernel_size;
      real* const p_temp_g = temp_data + g * kernel_size * bottom_area;

      // compute Z = alpha * dot(X.transpose(), Y) + beta * Z
      //   X (= weight): p x m,  Y (= bottom): p x n,  Z (= top): m x n
      //   X, Y, Z: row-major order (e.g., Z[i][j] = Z[i * n + j])
#ifdef GPU
      // input arguments:
      //   cublas handle,
      //   do_transpose_Y (= false),  do_transpose_X (= true),
      //   n (= H' * W'),  m (= C * kernel_h * kernel_w),  p (= C'),
      //   &alpha (= 1),
      //   &Y,  number of columns in Y (= n),
      //   &X,  number of columns in X (= m),
      //   &beta (= 0),
      //   &Z,  number of columns in Z (= n)
      const real one = 1.0f, zero = 0.0f;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_T,
                  bottom_area,  kernel_size,  bottom_C,
                  &one,
                  p_bottom_g,  bottom_area,
                  p_weight_g,  kernel_size,
                  &zero,
                  p_temp_g,  bottom_area);
#else
      // input arguments:
      //   is_row_major_order (= true),
      //   do_transpose_X (= true),  do_transpose_Y (= false),
      //   m (= C * kernel_h * kernel_w),  n (= H' * W'),  p (= C'),
      //   alpha (= 1),
      //   &X,  number of columns in X (= m),
      //   &Y,  number of columns in Y (= n),
      //   beta (= 0),
      //   &Z,  number of columns in Z (= n)
      cblas_sgemm(CblasRowMajor,
                  CblasTrans,  CblasNoTrans,
                  kernel_size,  bottom_area,  bottom_C,
                  1.0f,
                  p_weight_g,  kernel_size,
                  p_bottom_g,  bottom_area,
                  0.0f,
                  p_temp_g,  bottom_area);
#endif
    }

    // convert top shape
    //   (G * C * kernel_h * kernel_w) x (H' * W') -> (G * C) x (H * W)
    {
#ifdef GPU
      // one thread computes one entry in top
      const int num_threads = num_groups * top_C * top_H * top_W;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads, threads_per_block);
      convert_top_gpu<<<num_blocks, threads_per_block>>>(
          temp_data,  p_top_item,
          num_groups * top_C,  top_H,  top_W,
          bottom_H,  bottom_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
#else
      convert_top_cpu(
          temp_data,  p_top_item,
          num_groups * top_C,  top_H,  top_W,
          bottom_H,  bottom_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
#endif
    }

    // compute top[i][j] = top[i][j] + bias[i]
    //   top: (G * C) x (H * W)
    //   bias: (G * C) x 1
    if (option->bias) {
      const int top_channels = num_groups * top_C;
      const int top_area = top_H * top_W;

      // the computation is equivalent to...
      //   top = top + dot(bias, constant)
      //   constant: 1 x (H * W), constant[i] = 1 for all i
#ifdef GPU
      // thus, input arguments:
      //   do_transpose_Y (= false),  do_transpose_X (= false),
      //   n = H * W,  m = G * C,  p = 1
      //   alpha = 1,  beta = 1
      const real one = 1.0;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_N,
                  top_area,  top_channels,  1,
                  &one,
                  const_data,  top_area,
                  bias1d->data,  1,
                  &one,
                  p_top_item,  top_area);
#else
      // input arguments:
      //   do_transpose_X (= false),  do_transpose_Y (= false),
      //   m = G * C,  n = H * W,  p = 1
      //   alpha = 1,  beta = 1
      cblas_sgemm(CblasRowMajor,
                  CblasNoTrans,  CblasNoTrans,
                  top_channels,  top_area,  1,
                  1.0f,
                  bias1d->data,  1,
                  const_data,  top_area,
                  1.0f,
                  p_top_item,  top_area);
#endif
    }

    // locate next item
    {
      const int bottom_size = num_groups * bottom_C * bottom_H * bottom_W;
      const int top_size = num_groups * top_C * top_H * top_W;
      p_bottom_item += bottom_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;
}

// TODO
void deconv_backward(Tensor *top_grad, Tensor *bottom_grad, Tensor *top_layer, Tensor *bottom_layer, ConvOption *option)
{
  return;
}

#include <stdio.h>
#include <stdlib.h>

#define DATA_SIZE 512*36*46
#define WEIGHT_SIZE 512*1*1*4*4
#define BIAS_SIZE 512
#define CONST_SIZE 36*46

int main(int argc, char **argv)
{
  Tensor X, Y, W, b;
  real* X_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* Y_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* Y_true_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* W_data = (real*)malloc(WEIGHT_SIZE * sizeof(real));
  real* b_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* const_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* p_temp_data;
  real* p_const_data;
  ConvOption option;

  {
    option.num_groups = 512;
    option.out_channels = 512;
    option.kernel_h = 4;
    option.kernel_w = 4;
    option.pad_h = 1;
    option.pad_w = 1;
    option.stride_h = 2;
    option.stride_w = 2;
    option.bias = 0;
  }

  {
    X.ndim = 3;
    X.num_items = 1;
    for (int i = 0; i < X.num_items; ++i) {
      X.shape[i][0] = 512;
      X.shape[i][1] = 18;
      X.shape[i][2] = 23;
    }

    Y.ndim = X.ndim;
    Y.num_items = X.num_items;
    for (int i = 0; i < Y.num_items; ++i) {
      Y.shape[i][0] = option.out_channels;
      Y.shape[i][1] = option.stride_h * (X.shape[i][1] - 1) - 2 * option.pad_h + option.kernel_h;
      Y.shape[i][2] = option.stride_w * (X.shape[i][2] - 1) - 2 * option.pad_w + option.kernel_w;
    }

    W.ndim = 5; W.num_items = 1;
    W.shape[0][0] = option.num_groups;
    W.shape[0][1] = X.shape[0][0] / option.num_groups;
    W.shape[0][2] = option.out_channels / option.num_groups;
    W.shape[0][3] = option.kernel_h;
    W.shape[0][4] = option.kernel_w;

    b.ndim = 1; b.num_items = 1;
    b.shape[0][0] = option.out_channels;
  }

  {
    FILE* fp;
    int X_size = flatten_size(&X);
    int Y_size = flatten_size(&Y);
    int W_size = flatten_size(&W);
    int b_size = flatten_size(&b);

    printf("data loading\n");

    fp = fopen("../data/temp/deconv_bottom0.bin", "rb");
    if ((int)fread(X_data, sizeof(real), X_size, fp) != X_size) {
      printf("Error while reading deconv_bottom0\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/deconv_param0.bin", "rb");
    if ((int)fread(W_data, sizeof(real), W_size, fp) != W_size) {
      printf("Error while reading deconv_param0\n");
    }
    fclose(fp);

    if (option.bias) {
      fp = fopen("../data/temp/deconv_param1.bin", "rb");
      if ((int)fread(b_data, sizeof(real), b_size, fp) != b_size) {
        printf("Error while reading deconv_param1\n");
      }
      fclose(fp);
      for (int i = 0; i < CONST_SIZE; ++i) {
        const_data[i] = 1;
      }
    }

    fp = fopen("../data/temp/deconv_top0.bin", "rb");
    if ((int)fread(Y_true_data, sizeof(real), Y_size, fp) != Y_size) {
      printf("Error while reading deconv_top0\n");
    }
    fclose(fp);
  }

#ifdef GPU
  {
    printf("set device\n");
    CUDA_CHECK(hipSetDevice(0));
    //printf("get device\n");
    //CUDA_CHECK(hipGetDevice(0));
    printf("cublas initialization\n");
    option.handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    if (hipblasCreate((hipblasHandle_t*)option.handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
#endif

#ifdef GPU
  {
    int X_size = flatten_size(&X);
    int Y_size = flatten_size(&Y);
    int W_size = flatten_size(&W);
    int b_size = flatten_size(&b);
    int temp_size = option.kernel_h * option.kernel_w * Y.shape[0][0] * X.shape[0][1] * X.shape[0][2];

    printf("cuda malloc\n");
    CUDA_CHECK(hipMalloc(&X.data, X_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&Y.data, Y_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&W.data, W_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&b.data, b_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&p_temp_data, temp_size * sizeof(real)));
    CUDA_CHECK(hipMalloc(&p_const_data, CONST_SIZE * sizeof(real)));

    printf("memcopy\n");
    CUDA_CHECK(hipMemcpy(X.data, X_data, X_size * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(W.data, W_data, W_size * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(b.data, b_data, b_size * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(p_const_data, const_data, CONST_SIZE * sizeof(real), hipMemcpyHostToDevice));
  }
#else
  {
    int temp_size = option.kernel_h * option.kernel_w * Y.shape[0][0] * X.shape[0][1] * X.shape[0][2];

    X.data = X_data;
    Y.data = Y_data;
    W.data = W_data;
    b.data = b_data;
    p_temp_data = (real*)malloc(temp_size * sizeof(real));
    p_const_data = const_data;
  }
#endif

  {
    printf("do forward\n");
    deconv_forward(&X, &Y, &W, &b, p_temp_data, p_const_data, &option);
  }

#ifdef GPU
  {
    int Y_size = flatten_size(&Y);
    printf("memcpy\n");
    CUDA_CHECK(hipMemcpy(Y_data, Y.data, Y_size * sizeof(real), hipMemcpyDeviceToHost));
  }
#endif

  {
    int i = 0;
    for (int n = 0; n < Y.num_items; ++n) {
      for (int c = 0; c < Y.shape[n][0]; ++c) {
        for (int h = 0; h < Y.shape[n][1]; ++h) {
          for (int w = 0; w < Y.shape[n][2]; ++w) {
            real diff = ABS(Y_data[i] - Y_true_data[i]);
            diff /= 1e-10f + MIN(ABS(Y_data[i]), ABS(Y_true_data[i]));
#ifdef GPU
            if (diff > 0) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
#else
            if (diff > 1e-3f) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
#endif
            ++i;
          }
        }
      }
    }
  }

  {
    printf("free\n");
    free(X_data);
    free(Y_data);
    free(Y_true_data);
    free(W_data);
    free(b_data);
    free(const_data);
  }
#ifdef GPU
  {
    printf("cuda free\n");
    CUDA_CHECK(hipFree(X.data));
    CUDA_CHECK(hipFree(Y.data));
    CUDA_CHECK(hipFree(W.data));
    CUDA_CHECK(hipFree(b.data));
    CUDA_CHECK(hipFree(p_temp_data));
    CUDA_CHECK(hipFree(p_const_data));

    printf("cublas finalization\n");
    if (hipblasDestroy(*((hipblasHandle_t*)option.handle)) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
    free(option.handle);
  }
#else
  {
    free(p_temp_data);
  }
#endif

  return 0;
}
