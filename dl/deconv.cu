#include "hip/hip_runtime.h"
#include "layer.h"
#include "cuda_settings.h"

// convert top5d (C x kernel_h x kernel_w x H5 x W5)
//         -> top3d (C x H x W)
//   TODO: detailed description
__global__ void convert_top(const real* top5d, real* const top3d,
                            const int C, const int H, const int W,
                            const int H5, const int W5,
                            const int kernel_h, const int kernel_w,
                            const int pad_h, const int pad_w,
                            const int stride_h, const int stride_w)
{
  const int top_HW = H * W;
  const int top_CHW = top_HW * C;

  // thread index: (c, h, w) = c*H*W + h*W + w
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < top_CHW;
       index += blockDim.x) {
    // parse thread index -> (c, h, w)
    const int c = index / top_HW;
    const int h = (index / W) % H + pad_h;
    const int w = index % W + pad_w;

    // range of summation
    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    //   0 <= h5 <= 0
    //   0 <= w5 <= 0
    //   TODO: optimization & description
    const int h5_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int h5_end = min(h / stride_h + 1, H5);
    const int w5_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int w5_end = min(w / stride_w + 1, W5);
    const real* p_top5d = top5d + (c * kernel_h * kernel_w + h * kernel_w + w) * H5 * W5;
    const int h5_coef = (1 - stride_h * kernel_w * H5) * W5;
    const int w5_coef = 1 - stride_w * H5 * W5;

    // top3d[c][h][w] = sum_{h5,w5} top5d[]
    real val = 0;
    for (int h5 = h5_start; h5 < h5_end; ++h5) {
      for (int w5 = w5_start; w5 < w5_end; ++w5) {
        val += p_top5d[h5 * h5_coef + w5 * w5_coef];
      }
    }
    top3d[index] = val;
  }
}

void forward(const Tensor* bottom4d, Tensor* const top4d,
             const Tensor* weight5d, const Tensor* bias2d,
             real* const temp_data, const real* const_data,
             const ConvOption* option)
{
  // weight shape: G x C' x C x kernel_h x kernel_w
  //   G: number of groups
  const int num_groups = weight5d->shape[0][0]; // G
  const int bottom_C = weight5d->shape[0][1];  // C'
  const int top_C = weight5d->shape[0][2];  // C
  const int kernel_h = weight5d->shape[0][3];
  const int kernel_w = weight5d->shape[0][4];
  const int kernel_size = top_C * kernel_h * kernel_w;

  // padding size & stride size
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  const hipblasHandle_t* cublas_handle = (hipblasHandle_t*)option->handle;
  const real one = 1.0, zero = 0.0;

  // do forward-pass for each item in the batch
  const real* p_bottom_data = bottom4d->data;
  real* p_top_data = top4d->data;
  const int num_items = bottom4d->num_items;
  for (int n = 0; n < num_items; ++n) {
    // bottom shape: G x C' x H' x W'
    const int bottom_H = bottom4d->shape[n][2];  // H'
    const int bottom_W = bottom4d->shape[n][3];  // W'
    const int bottom_area = bottom_H * bottom_W;

    // set top shape: G x C x H x W
    //   H' = 1 + (H + 2 * pad_h - kernel_h) / stride_h
    //   -> H = stride_h * (H' - 1) - 2 * pad_h + kernel_h
    const int top_H = stride_h * (bottom_H - 1) - 2 * pad_h + kernel_h;
    const int top_W = stride_w * (bottom_W - 1) - 2 * pad_w + kernel_w;
    const int top_size = num_groups * top_C * top_H * top_W;
    top4d->shape[n][0] = num_groups;
    top4d->shape[n][1] = top_C;
    top4d->shape[n][2] = top_H;
    top4d->shape[n][3] = top_W;

   { // do matrix computation
    // top[g] = dot(weight[g].trans(), bottom[g])
    //   weight[g].trans(): (C * kernel_h * kernel_w) x C'
    //   bottom[g]: C' x (H' * W')
    //   top[g]: (C * kernel_h * kernel_w) x (H' * W')
    for (int g = 0; g < num_groups; ++g) {
      hipblasSgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                  bottom_area, kernel_size, bottom_C,
                  &one, p_bottom_data + g * bottom_C * bottom_area, bottom_area,
                  weight5d->data + g * bottom_C * kernel_size, kernel_size,
                  &zero, temp_data + g * kernel_size * bottom_area, bottom_area);
    }
   } // end matrix computation

   { // convert top shape: (C * kernel_h * kernel_w) x (H' * W') -> C x H x W
    const int num_threads = 1024;
    const int num_blocks = (num_threads - 1 + top_size) / num_threads;
    convert_top<<<num_blocks, num_threads>>>(temp_data, p_top_data,
                                             num_groups * top_C, top_H, top_W,
                                             bottom_H, bottom_W,
                                             kernel_h, kernel_w,
                                             pad_h, pad_w,
                                             stride_h, stride_w);
   } // end convert top shape

    // top = top + bias
    if (option->bias) {
      hipblasSgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                  top_H * top_W, num_groups * top_C, 1,
                  &one, const_data, top_H * top_W,
                  bias2d->data, 1,
                  &one, p_top_data, top_H * top_W);
    }

    // locate next data
    p_bottom_data += num_groups * bottom_C * bottom_area;
    p_top_data += top_size;
  } // endfor batch

  top4d->ndim = 4;
  top4d->num_items = num_items;
}

// TODO
void backward(Tensor *top_grad, Tensor *bottom_grad, Tensor *top_layer, Tensor *bottom_layer, ConvOption *option)
{
  return;
}

#define DATA_SIZE 512*18*23*4
#define WEIGHT_SIZE 512*1*1*4*4
#define BIAS_SIZE 512

int main(int argc, char **argv)
{
  Tensor X, Y, W, b;
  real* X_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* Y_data = (real*)malloc(DATA_SIZE * sizeof(real));
  real* W_data = (real*)malloc(WEIGHT_SIZE * sizeof(real));
  real* b_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* const_data = (real*)malloc(BIAS_SIZE * sizeof(real));
  real* p_temp_data;
  real* p_const_data;
  ConvOption option;
  hipblasHandle_t cublas_handle;
 {
  X.ndim = 4; X.num_items = 1;
  for (int i = 0; i < X.num_items; ++i) {
    X.shape[i][0] = 512;
    X.shape[i][1] = 1;
    X.shape[i][2] = 18;
    X.shape[i][3] = 23;
  }
  W.ndim = 5; W.num_items = 1;
  W.shape[0][0] = 512; W.shape[0][1] = 1; W.shape[0][2] = 1; W.shape[0][3] = 4; W.shape[0][4] = 4;
  b.ndim = 2; b.num_items = 1; b.shape[0][0] = 512; b.shape[0][1] = 1;
  option.kernel_h = 4;
  option.kernel_w = 4;
  option.pad_h = 1;
  option.pad_w = 1;
  option.stride_h = 2;
  option.stride_w = 2;
  option.bias = 0;
 }
 {
  FILE* fp;
  int X_size = flatten_size(&X);
  int W_size = flatten_size(&W);
  int b_size = flatten_size(&b);
  printf("data loading\n");
  fp = fopen("../data/temp/deconv_bottom0.txt", "r");
  for (int i = 0; i < X_size; ++i)
    fscanf(fp, "%f", &X_data[i]);
  fclose(fp);
  fp = fopen("../data/temp/deconv_param0.txt", "r");
  for (int i = 0; i < W_size; ++i)
    fscanf(fp, "%f", &W_data[i]);
  fclose(fp);
  if (option.bias) {
    fp = fopen("../data/temp/deconv_param1.txt", "r");
    for (int i = 0; i < b_size; ++i)
      fscanf(fp, "%f", &b_data[i]);
    fclose(fp);
    for (int i = 0; i < b_size; ++i) {
      const_data[i] = 1;
    }
  }

  printf("set device\n");
  CUDA_CHECK(hipSetDevice(1));
  //printf("get device\n");
  //CUDA_CHECK(hipGetDevice(0));
  printf("cublas initialization\n");
  if (hipblasCreate(&cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
    printf("cublas creation failed\n");
  }
  option.handle = &cublas_handle;

  printf("cuda malloc\n");
  CUDA_CHECK(hipMalloc(&X.data, DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&Y.data, DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&W.data, WEIGHT_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&b.data, BIAS_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&p_temp_data, 4*4*DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&p_const_data, BIAS_SIZE*sizeof(real)));

  printf("memcopy\n");
  CUDA_CHECK(hipMemcpy(X.data, X_data, X_size*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(W.data, W_data, W_size*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(b.data, b_data, b_size*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(p_const_data, const_data, b_size*sizeof(real), hipMemcpyHostToDevice));
 }
 {
  printf("do forward\n");
  forward(&X, &Y, &W, &b, p_temp_data, p_const_data, &option);
 }
 {
  int Y_size = flatten_size(&Y);
  printf("memcpy\n");
  CUDA_CHECK(hipMemcpy(Y_data, Y.data, Y_size*sizeof(real), hipMemcpyDeviceToHost));
 }
 {
  real* p_Y_data = &Y_data[0];
  for (int n = 0; n < Y.num_items; ++n) {
    for (int g = 0; g < Y.shape[n][0]; ++g) {
      for (int c = 0; c < Y.shape[n][1]; ++c) {
        for (int h = 0; h < Y.shape[n][2]; ++h) {
          for (int w = 0; w < Y.shape[n][3]; ++w) {
            printf("%.4f\n", *(p_Y_data++));
          }
        }
      }
    }
  }
 }
 {
  printf("cuda free\n");
  CUDA_CHECK(hipFree(X.data));
  CUDA_CHECK(hipFree(Y.data));
  CUDA_CHECK(hipFree(W.data));
  CUDA_CHECK(hipFree(b.data));
  CUDA_CHECK(hipFree(p_temp_data));
  CUDA_CHECK(hipFree(p_const_data));
  printf("cublas finalization\n");
  if (hipblasDestroy(cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
    printf("cublas destruction failed\n");
  }
  printf("free\n");
  free(X_data);
  free(Y_data);
  free(W_data);
  free(b_data);
  free(const_data);
  return 0;
 }
}
