#include "layer.h"
#include <stdio.h>
#include <string.h>

typedef struct PVANET_
{
  Tensor input;
  Tensor conv1_1, conv1_2;
  Tensor weight1_1, bias1_1, weight1_2, bias1_2;
  Tensor conv2_1, conv2_2;
  Tensor weight2_1, bias2_1, weight2_2, bias2_2;
  Tensor conv3_1, conv3_2, conv3_3;
  Tensor weight3_1, bias3_1, weight3_2, bias3_2, weight3_3, bias3_3;
  Tensor conv4_1, conv4_2, conv4_3;
  Tensor weight4_1, bias4_1, weight4_2, bias4_2, weight4_3, bias4_3;
  Tensor conv5_1, conv5_2, conv5_3;
  Tensor weight5_1, bias5_1, weight5_2, bias5_2, weight5_3, bias5_3;
  Tensor downsample, upsample, concat;
  Tensor weight_up, bias_up;
  Tensor convf;
  Tensor weightf, biasf;
} PVANET;

typedef struct SRPN_
{
  Tensor conv1, conv3, conv5;
  Tensor score1, score3, score5;
  Tensor bbox1, bbox3, bbox5;
  Tensor score, bbox;
  Tensor roi;
} SRPN;

typedef struct RCNN_
{
  Tensor roipool;
  Tensor fc6;
  Tensor fc7;
  Tensor score, bbox;
} RCNN;

PVANET pvanet;
SRPN srpn;
RCNN rcnn;
ConvOption conv_option1;
ConvOption conv_option2;
ConvOption conv1x1_option;
ConvOption deconv_option;
PoolOption pool_option;
ReluOption relu_option;

const Tensor* const concat_bottoms[3]
    = { &pvanet.downsample, &pvanet.conv4_3, &pvanet.upsample };

int max_layer_size = 0, max_param_size = 0;
int max_temp_size = 0, max_const_size = 0, max_tempint_size = 0;

real* layer1_data = NULL;
real* layer2_data = NULL;
real* backup1_data = NULL;
real* backup2_data = NULL;
real* backup3_data = NULL;
real* temp_data = NULL;
real* const_data = NULL;
int* tempint_data = NULL;

real* true_data = NULL;
real* input_data = NULL;
real* output_data = NULL;
real* param_data = NULL;

void load_tensor(const char* filename, Tensor* const tensor)
{
  int ndim;
  int shape[g_max_ndim];

  {
  #ifdef GPU
    int data_size = 1;
    load_data(filename, &ndim, shape, param_data);
    for (int i = 0; i < ndim; ++i) {
      data_size *= shape[i];
    }
    if (data_size != flatten_size(tensor)) {
      printf("[ERROR] Size mismatch: %s (%d) != tensor (%d)\n",
             filename, data_size, flatten_size(tensor));
    }
    hipMemcpyAsync(tensor->data, param_data, data_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    load_data(filename, &ndim, shape, tensor->data);
  #endif
  }
}

int malloc_tensor(Tensor* const tensor)
{
  const int data_size = flatten_size(tensor);

  #ifdef GPU
  hipMalloc(&tensor->data, data_size * sizeof(real));
  #else
  tensor->data = (real*)malloc(data_size * sizeof(real));
  #endif

  return data_size;
}

void print_tensor_info(const char* name, const Tensor* const tensor)
{
  printf("%s: ", name);
  if (tensor->num_items > 1) {
    printf("%d x ", tensor->num_items);
  }
  for (int i = 0; i < tensor->ndim - 1; ++i) {
    printf("%d x ", tensor->shape[0][i]);
  }
  printf("%d\n", tensor->shape[0][tensor->ndim - 1]);
}

void forward_frcnn_7_1_1(void)
{
  {
    // 1_1
    pvanet.input.data = layer1_data;
    pvanet.conv1_1.data = layer2_data;
    conv_option1.out_channels = 32;
    conv_forward(&pvanet.input, &pvanet.conv1_1,
                &pvanet.weight1_1, &pvanet.bias1_1,
                temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv1_1, &relu_option);
    print_tensor_info("input", &pvanet.input);
    print_tensor_info("conv1_1", &pvanet.conv1_1);

    // 1_2
    pvanet.conv1_2.data = layer1_data;
    conv_option2.out_channels = 32;
    conv_forward(&pvanet.conv1_1, &pvanet.conv1_2,
                 &pvanet.weight1_2, &pvanet.bias1_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv1_2, &relu_option);
    print_tensor_info("conv1_2", &pvanet.conv1_2);

    // 2_1
    pvanet.conv2_1.data = layer2_data;
    conv_option1.out_channels = 64;
    conv_forward(&pvanet.conv1_2, &pvanet.conv2_1,
                 &pvanet.weight2_1, &pvanet.bias2_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv2_1, &relu_option);
    print_tensor_info("conv2_1", &pvanet.conv2_1);

    // 2_2
    pvanet.conv2_2.data = layer1_data;
    conv_option2.out_channels = 64;
    conv_forward(&pvanet.conv2_1, &pvanet.conv2_2,
                 &pvanet.weight2_2, &pvanet.bias2_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv2_2, &relu_option);
    print_tensor_info("conv2_2", &pvanet.conv2_2);

    // 3_1
    pvanet.conv3_1.data = layer2_data;
    conv_option1.out_channels = 96;
    conv_forward(&pvanet.conv2_2, &pvanet.conv3_1,
                 &pvanet.weight3_1, &pvanet.bias3_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv3_1, &relu_option);
    print_tensor_info("conv3_1", &pvanet.conv3_1);

    // 3_2
    pvanet.conv3_2.data = layer1_data;
    conv_option2.out_channels = 64;
    conv_forward(&pvanet.conv3_1, &pvanet.conv3_2,
                 &pvanet.weight3_2, &pvanet.bias3_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv3_2, &relu_option);
    print_tensor_info("conv3_2", &pvanet.conv3_2);

    // 3_3
    pvanet.conv3_3.data = layer2_data;
    conv_option2.out_channels = 128;
    conv_forward(&pvanet.conv3_2, &pvanet.conv3_3,
                 &pvanet.weight3_3, &pvanet.bias3_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv3_3, &relu_option);
    print_tensor_info("conv3_3", &pvanet.conv3_3);

    // downsample
    pvanet.downsample.data = backup1_data;
    pool_forward(&pvanet.conv3_3, &pvanet.downsample,
                 tempint_data, &pool_option);
    print_tensor_info("downsample", &pvanet.downsample);

    // 4_1
    pvanet.conv4_1.data = layer1_data;
    conv_option1.out_channels = 192;
    conv_forward(&pvanet.conv3_3, &pvanet.conv4_1,
                 &pvanet.weight4_1, &pvanet.bias4_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv4_1, &relu_option);
    print_tensor_info("conv4_1", &pvanet.conv4_1);

    // 4_2
    pvanet.conv4_2.data = layer2_data;
    conv_option2.out_channels = 128;
    conv_forward(&pvanet.conv4_1, &pvanet.conv4_2,
                 &pvanet.weight4_2, &pvanet.bias4_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv4_2, &relu_option);
    print_tensor_info("conv4_2", &pvanet.conv4_2);

    // 4_3
    pvanet.conv4_3.data = backup2_data;
    conv_option2.out_channels = 256;
    conv_forward(&pvanet.conv4_2, &pvanet.conv4_3,
                 &pvanet.weight4_3, &pvanet.bias4_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv4_3, &relu_option);
    print_tensor_info("conv4_3", &pvanet.conv4_3);

    // 5_1
    pvanet.conv5_1.data = layer1_data;
    conv_option1.out_channels = 384;
    conv_forward(&pvanet.conv4_3, &pvanet.conv5_1,
                 &pvanet.weight5_1, &pvanet.bias5_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv5_1, &relu_option);
    print_tensor_info("conv5_1", &pvanet.conv5_1);

    // 5_2
    pvanet.conv5_2.data = layer2_data;
    conv_option2.out_channels = 256;
    conv_forward(&pvanet.conv5_1, &pvanet.conv5_2,
                 &pvanet.weight5_2, &pvanet.bias5_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv5_2, &relu_option);
    print_tensor_info("conv5_2", &pvanet.conv5_2);

    // 5_3
    pvanet.conv5_3.data = layer1_data;
    conv_option2.out_channels = 512;
    conv_forward(&pvanet.conv5_2, &pvanet.conv5_3,
                 &pvanet.weight5_3, &pvanet.bias5_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv5_3, &relu_option);
    print_tensor_info("conv5_3", &pvanet.conv5_3);

    // upsample
    pvanet.upsample.data = layer2_data;
    deconv_forward(&pvanet.conv5_3, &pvanet.upsample,
                   &pvanet.weight_up, &pvanet.bias_up,
                   temp_data, const_data, &deconv_option);
    print_tensor_info("upsample", &pvanet.upsample);

    // concat
    pvanet.concat.data = layer1_data;
    concat_forward(concat_bottoms, &pvanet.concat, 3);
    print_tensor_info("concat", &pvanet.concat);

    // convf
    pvanet.convf.data = backup1_data;
    conv1x1_option.out_channels = 512;
    conv_forward(&pvanet.concat, &pvanet.convf,
                 &pvanet.weightf, &pvanet.biasf,
                 temp_data, const_data, &conv1x1_option);
    //relu_forward_inplace(&pvanet.convf, &relu_option);
    print_tensor_info("convf", &pvanet.convf);
  }
}

void shape_frcnn_7_1_1(void)
{
  {
    int temp_size, const_size, tempint_size;

    // 1_1
    conv_option1.out_channels = 32;
    conv_shape(&pvanet.input, &pvanet.conv1_1,
               &pvanet.weight1_1, &pvanet.bias1_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.input));
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv1_1));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight1_1));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("input", &pvanet.input);
    print_tensor_info("conv1_1", &pvanet.conv1_1);

    // 1_2
    conv_option2.out_channels = 32;
    conv_shape(&pvanet.conv1_1, &pvanet.conv1_2,
               &pvanet.weight1_2, &pvanet.bias1_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv1_2));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight1_2));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv1_2", &pvanet.conv1_2);

    // 2_1
    conv_option1.out_channels = 64;
    conv_shape(&pvanet.conv1_2, &pvanet.conv2_1,
               &pvanet.weight2_1, &pvanet.bias2_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv2_1));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight2_1));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv2_1", &pvanet.conv2_1);

    // 2_2
    conv_option2.out_channels = 64;
    conv_shape(&pvanet.conv2_1, &pvanet.conv2_2,
               &pvanet.weight2_2, &pvanet.bias2_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv2_2));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight2_2));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv2_2", &pvanet.conv2_2);

    // 3_1
    conv_option1.out_channels = 96;
    conv_shape(&pvanet.conv2_2, &pvanet.conv3_1,
               &pvanet.weight3_1, &pvanet.bias3_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv3_1));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight3_1));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv3_1", &pvanet.conv3_1);

    // 3_2
    conv_option2.out_channels = 64;
    conv_shape(&pvanet.conv3_1, &pvanet.conv3_2,
               &pvanet.weight3_2, &pvanet.bias3_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv3_2));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight3_2));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv3_2", &pvanet.conv3_2);

    // 3_3
    conv_option2.out_channels = 128;
    conv_shape(&pvanet.conv3_2, &pvanet.conv3_3,
               &pvanet.weight3_3, &pvanet.bias3_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv3_3));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight3_3));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv3_3", &pvanet.conv3_3);

    // 4_1
    conv_option1.out_channels = 192;
    conv_shape(&pvanet.conv3_3, &pvanet.conv4_1,
               &pvanet.weight4_1, &pvanet.bias4_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv4_1));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight4_1));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv4_1", &pvanet.conv4_1);

    // 4_2
    conv_option2.out_channels = 128;
    conv_shape(&pvanet.conv4_1, &pvanet.conv4_2,
               &pvanet.weight4_2, &pvanet.bias4_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv4_2));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight4_2));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv4_2", &pvanet.conv4_2);

    // 4_3
    conv_option2.out_channels = 256;
    conv_shape(&pvanet.conv4_2, &pvanet.conv4_3,
               &pvanet.weight4_3, &pvanet.bias4_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv4_3));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight4_3));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv4_3", &pvanet.conv4_3);

    // 5_1
    conv_option1.out_channels = 384;
    conv_shape(&pvanet.conv4_3, &pvanet.conv5_1,
               &pvanet.weight5_1, &pvanet.bias5_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv5_1));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight5_1));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv5_1", &pvanet.conv5_1);

    // 5_2
    conv_option2.out_channels = 256;
    conv_shape(&pvanet.conv5_1, &pvanet.conv5_2,
               &pvanet.weight5_2, &pvanet.bias5_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv5_2));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight5_2));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv5_2", &pvanet.conv5_2);

    // 5_3
    conv_option2.out_channels = 512;
    conv_shape(&pvanet.conv5_2, &pvanet.conv5_3,
               &pvanet.weight5_3, &pvanet.bias5_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.conv5_3));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight5_3));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("conv5_3", &pvanet.conv5_3);

    // downsample
    pool_shape(&pvanet.conv3_3, &pvanet.downsample,
               &tempint_size, &pool_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.downsample));
    max_tempint_size = MAX(max_tempint_size, tempint_size);
    print_tensor_info("downsample", &pvanet.downsample);

    // upsample
    deconv_shape(&pvanet.conv5_3, &pvanet.upsample,
                 &pvanet.weight_up, &pvanet.bias_up,
                 &temp_size, &const_size, &deconv_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.upsample));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weight_up));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("upsample", &pvanet.upsample);

    // concat
    concat_shape(concat_bottoms, &pvanet.concat, 3);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.concat));
    print_tensor_info("concat", &pvanet.concat);

    // convf
    conv1x1_option.out_channels = 512;
    conv_shape(&pvanet.concat, &pvanet.convf,
               &pvanet.weightf, &pvanet.biasf,
               &temp_size, &const_size, &conv1x1_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&pvanet.convf));
    max_param_size = MAX(max_param_size, flatten_size(&pvanet.weightf));
    max_temp_size = MAX(max_temp_size, temp_size);
    max_const_size = MAX(max_const_size, const_size);
    print_tensor_info("convf", &pvanet.convf);
  }
}

void init_frcnn_7_1_1(void)
{
  // set fixed options
  {
    conv_option1.num_groups = 1;
    conv_option1.kernel_h = 3;
    conv_option1.kernel_w = 3;
    conv_option1.pad_h = 1;
    conv_option1.pad_w = 1;
    conv_option1.bias = 1;
    conv_option1.stride_h = 2;
    conv_option1.stride_w = 2;

    conv_option2 = conv_option1;
    conv_option2.stride_h = 1;
    conv_option2.stride_w = 1;

    conv1x1_option = conv_option2;
    conv1x1_option.kernel_h = 1;
    conv1x1_option.kernel_w = 1;
    conv1x1_option.pad_h = 0;
    conv1x1_option.pad_w = 0;

    deconv_option = conv_option1;
    deconv_option.num_groups = 512;
    deconv_option.out_channels = 512;
    deconv_option.kernel_h = 4;
    deconv_option.kernel_w = 4;
    deconv_option.bias = 0;

    pool_option.kernel_h = 3;
    pool_option.kernel_w = 3;
    pool_option.stride_h = 2;
    pool_option.stride_w = 2;
    pool_option.pad_h = 0;
    pool_option.pad_w = 0;

    relu_option.negative_slope = 0;
  }

  // calculate maximum size
  pvanet.input.num_items = 1;
  pvanet.input.ndim = 3;
  pvanet.input.shape[0][0] = 3;
  pvanet.input.shape[0][1] = 640;
  pvanet.input.shape[0][2] = 1024;
  shape_frcnn_7_1_1();

  // memory allocation
  {
    long int space = 0;

    // space for data loading
    {
      input_data = (real*)malloc(flatten_size(&pvanet.input) * sizeof(real));
      output_data = (real*)malloc(max_layer_size * sizeof(real));
      true_data = (real*)malloc(max_layer_size * sizeof(real));
      param_data = (real*)malloc(max_param_size * sizeof(real));
      space += flatten_size(&pvanet.input) + max_layer_size * 2
               + max_param_size;
    }

    // space required for forward-pass
    {
    #ifdef GPU
      hipMalloc(&layer1_data, max_layer_size * sizeof(real));
      hipMalloc(&layer2_data, max_layer_size * sizeof(real));
      hipMalloc(&temp_data, max_temp_size * sizeof(real));
      hipMalloc(&const_data, max_const_size * sizeof(real));
      hipMalloc(&tempint_data, max_tempint_size * sizeof(int));
      hipMalloc(&backup1_data, max_layer_size * sizeof(real));
      hipMalloc(&backup2_data, max_layer_size * sizeof(real));
      hipMalloc(&backup3_data, max_layer_size * sizeof(real));
    #else
      layer1_data = (real*)malloc(max_layer_size * sizeof(real));
      layer2_data = (real*)malloc(max_layer_size * sizeof(real));
      temp_data = (real*)malloc(max_temp_size * sizeof(real));
      const_data = (real*)malloc(max_const_size * sizeof(real));
      tempint_data = (int*)malloc(max_tempint_size * sizeof(int));
      backup1_data = (real*)malloc(max_layer_size * sizeof(real));
      backup2_data = (real*)malloc(max_layer_size * sizeof(real));
      backup3_data = (real*)malloc(max_layer_size * sizeof(real));
    #endif
      space += max_layer_size * 4 + max_temp_size + max_const_size
               + max_tempint_size;
    }

    // parameters
    {
      space += malloc_tensor(&pvanet.weight1_1);
      space += malloc_tensor(&pvanet.bias1_1);
      space += malloc_tensor(&pvanet.weight1_2);
      space += malloc_tensor(&pvanet.bias1_2);
      space += malloc_tensor(&pvanet.weight2_1);
      space += malloc_tensor(&pvanet.bias2_1);
      space += malloc_tensor(&pvanet.weight2_2);
      space += malloc_tensor(&pvanet.bias2_2);
      space += malloc_tensor(&pvanet.weight3_1);
      space += malloc_tensor(&pvanet.bias3_1);
      space += malloc_tensor(&pvanet.weight3_2);
      space += malloc_tensor(&pvanet.bias3_2);
      space += malloc_tensor(&pvanet.weight3_3);
      space += malloc_tensor(&pvanet.bias3_3);
      space += malloc_tensor(&pvanet.weight4_1);
      space += malloc_tensor(&pvanet.bias4_1);
      space += malloc_tensor(&pvanet.weight4_2);
      space += malloc_tensor(&pvanet.bias4_2);
      space += malloc_tensor(&pvanet.weight4_3);
      space += malloc_tensor(&pvanet.bias4_3);
      space += malloc_tensor(&pvanet.weight5_1);
      space += malloc_tensor(&pvanet.bias5_1);
      space += malloc_tensor(&pvanet.weight5_2);
      space += malloc_tensor(&pvanet.bias5_2);
      space += malloc_tensor(&pvanet.weight5_3);
      space += malloc_tensor(&pvanet.bias5_3);
      space += malloc_tensor(&pvanet.weight_up);
      space += malloc_tensor(&pvanet.bias_up);
      space += malloc_tensor(&pvanet.weightf);
      space += malloc_tensor(&pvanet.biasf);
    }

    space = DIV_THEN_CEIL(space * sizeof(real),  1000000);
    printf("%ldMB memory allocated\n", space);
  }

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < max_const_size; ++i) {
      output_data[i] = 1;
    }
    hipMemcpy(const_data, output_data, max_const_size * sizeof(real),
               hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < max_const_size; ++i) {
      const_data[i] = 1;
    }
  #endif
  }

  // parameter loading
  {
    load_tensor("../data/temp/conv1_1_param0.bin", &pvanet.weight1_1);
    load_tensor("../data/temp/conv1_1_param1.bin", &pvanet.bias1_1);
    load_tensor("../data/temp/conv1_2_param0.bin", &pvanet.weight1_2);
    load_tensor("../data/temp/conv1_2_param1.bin", &pvanet.bias1_2);
    load_tensor("../data/temp/conv2_1_param0.bin", &pvanet.weight2_1);
    load_tensor("../data/temp/conv2_1_param1.bin", &pvanet.bias2_1);
    load_tensor("../data/temp/conv2_2_param0.bin", &pvanet.weight2_2);
    load_tensor("../data/temp/conv2_2_param1.bin", &pvanet.bias2_2);
    load_tensor("../data/temp/conv3_1_param0.bin", &pvanet.weight3_1);
    load_tensor("../data/temp/conv3_1_param1.bin", &pvanet.bias3_1);
    load_tensor("../data/temp/conv3_2_param0.bin", &pvanet.weight3_2);
    load_tensor("../data/temp/conv3_2_param1.bin", &pvanet.bias3_2);
    load_tensor("../data/temp/conv3_3_param0.bin", &pvanet.weight3_3);
    load_tensor("../data/temp/conv3_3_param1.bin", &pvanet.bias3_3);
    load_tensor("../data/temp/conv4_1_param0.bin", &pvanet.weight4_1);
    load_tensor("../data/temp/conv4_1_param1.bin", &pvanet.bias4_1);
    load_tensor("../data/temp/conv4_2_param0.bin", &pvanet.weight4_2);
    load_tensor("../data/temp/conv4_2_param1.bin", &pvanet.bias4_2);
    load_tensor("../data/temp/conv4_3_param0.bin", &pvanet.weight4_3);
    load_tensor("../data/temp/conv4_3_param1.bin", &pvanet.bias4_3);
    load_tensor("../data/temp/conv5_1_param0.bin", &pvanet.weight5_1);
    load_tensor("../data/temp/conv5_1_param1.bin", &pvanet.bias5_1);
    load_tensor("../data/temp/conv5_2_param0.bin", &pvanet.weight5_2);
    load_tensor("../data/temp/conv5_2_param1.bin", &pvanet.bias5_2);
    load_tensor("../data/temp/conv5_3_param0.bin", &pvanet.weight5_3);
    load_tensor("../data/temp/conv5_3_param1.bin", &pvanet.bias5_3);
    load_tensor("../data/temp/upsample_param0.bin", &pvanet.weight_up);
    load_tensor("../data/temp/convf_param0.bin", &pvanet.weightf);
    load_tensor("../data/temp/convf_param1.bin", &pvanet.biasf);
  }
}

int main(int argc, char* argv[])
{
  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
    conv_option1.handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    if (hipblasCreate((hipblasHandle_t*)conv_option1.handle)
          != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
    conv_option2.handle = conv_option1.handle;
  }
  #endif

  // PVANET initialization
  init_frcnn_7_1_1();

  // input data loading
  {
    int ndim;
    int shape[g_max_ndim];
    int input_size;

    load_data("../data/temp/conv1_1_bottom0.bin", &ndim, shape, input_data);
    pvanet.input.num_items = shape[0];
    pvanet.input.ndim = ndim - 1;
    input_size = 0;
    for (int n = 0; n < pvanet.input.num_items; ++n) {
      int size_n = 1;
      for (int i = 0; i < pvanet.input.ndim; ++i) {
        pvanet.input.shape[n][i] = shape[i + 1];
        size_n *= shape[i + 1];
      }
      pvanet.input.start[n] = input_size;
      input_size += size_n;
    }

  #ifdef GPU
    hipMemcpyAsync(layer1_data, input_data, input_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    memcpy(layer1_data, input_data, input_size * sizeof(real));
  #endif

    print_tensor_info("input data loaded", &pvanet.input);
  }

  // network reshape
  shape_frcnn_7_1_1();

  // forward-pass
  printf("forward-pass start\n");
  forward_frcnn_7_1_1();
  printf("forward-pass end\n");

  // retrieve output
  {
    const int output_size = flatten_size(&pvanet.convf);

  #ifdef GPU
    hipMemcpyAsync(output_data, pvanet.convf.data,
                    output_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  #else
    memcpy(output_data, pvanet.convf.data, output_size * sizeof(real));
  #endif
  }

  // verify results
  {
    const int output_size = flatten_size(&pvanet.convf);

    int ndim;
    int shape[g_max_ndim];
    load_data("../data/temp/convf_top0.bin", &ndim, shape, true_data);

    for (int i = 0; i < output_size; ++i) {
      real diff = ABS(true_data[i] - output_data[i]);
      diff /= 1e-10f + MIN(ABS(true_data[i]),  ABS(output_data[i]));
      #ifdef GPU
      if (diff > 0) {
        printf("%d: %.6f %.6f\n", i, true_data[i], output_data[i]);
      }
      #else
      if (diff > 1e-3f) {
        printf("%d: %.6f %.6f\n", i, true_data[i], output_data[i]);
      }
      #endif
    }
  }

  // memory deallocation
  {
    if (true_data) free(true_data);
    if (input_data) free(input_data);
    if (output_data) free(output_data);
    if (param_data) free(param_data);
  }
  #ifdef GPU
  {
    if (layer1_data) hipFree(layer1_data);
    if (layer2_data) hipFree(layer2_data);
    if (backup1_data) hipFree(backup1_data);
    if (backup2_data) hipFree(backup2_data);
    if (backup3_data) hipFree(backup3_data);
    if (temp_data) hipFree(temp_data);
    if (tempint_data) hipFree(tempint_data);
    if (const_data) hipFree(const_data);

    if (hipblasDestroy(*((hipblasHandle_t*)conv_option1.handle))
        != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }

    hipFree(pvanet.weight1_1.data);
    hipFree(pvanet.bias1_1.data);
    hipFree(pvanet.weight1_2.data);
    hipFree(pvanet.bias1_2.data);
    hipFree(pvanet.weight2_1.data);
    hipFree(pvanet.bias2_1.data);
    hipFree(pvanet.weight2_2.data);
    hipFree(pvanet.bias2_2.data);
    hipFree(pvanet.weight3_1.data);
    hipFree(pvanet.bias3_1.data);
    hipFree(pvanet.weight3_2.data);
    hipFree(pvanet.bias3_2.data);
    hipFree(pvanet.weight3_3.data);
    hipFree(pvanet.bias3_3.data);
    hipFree(pvanet.weight4_1.data);
    hipFree(pvanet.bias4_1.data);
    hipFree(pvanet.weight4_2.data);
    hipFree(pvanet.bias4_2.data);
    hipFree(pvanet.weight4_3.data);
    hipFree(pvanet.bias4_3.data);
    hipFree(pvanet.weight5_1.data);
    hipFree(pvanet.bias5_1.data);
    hipFree(pvanet.weight5_2.data);
    hipFree(pvanet.bias5_2.data);
    hipFree(pvanet.weight5_3.data);
    hipFree(pvanet.bias5_3.data);
    hipFree(pvanet.weight_up.data);
    hipFree(pvanet.weightf.data);
    hipFree(pvanet.biasf.data);
  }
  #else
  {
    if (layer1_data) free(layer1_data);
    if (layer2_data) free(layer2_data);
    if (backup1_data) free(backup1_data);
    if (backup2_data) free(backup2_data);
    if (backup3_data) free(backup3_data);
    if (temp_data) free(temp_data);
    if (tempint_data) free(tempint_data);
    if (const_data) free(const_data);

    free(pvanet.weight1_1.data);
    free(pvanet.bias1_1.data);
    free(pvanet.weight1_2.data);
    free(pvanet.bias1_2.data);
    free(pvanet.weight2_1.data);
    free(pvanet.bias2_1.data);
    free(pvanet.weight2_2.data);
    free(pvanet.bias2_2.data);
    free(pvanet.weight3_1.data);
    free(pvanet.bias3_1.data);
    free(pvanet.weight3_2.data);
    free(pvanet.bias3_2.data);
    free(pvanet.weight3_3.data);
    free(pvanet.bias3_3.data);
    free(pvanet.weight4_1.data);
    free(pvanet.bias4_1.data);
    free(pvanet.weight4_2.data);
    free(pvanet.bias4_2.data);
    free(pvanet.weight4_3.data);
    free(pvanet.bias4_3.data);
    free(pvanet.weight5_1.data);
    free(pvanet.bias5_1.data);
    free(pvanet.weight5_2.data);
    free(pvanet.bias5_2.data);
    free(pvanet.weight5_3.data);
    free(pvanet.bias5_3.data);
    free(pvanet.weight_up.data);
    free(pvanet.weightf.data);
    free(pvanet.biasf.data);
  }
  #endif

  return 0;
}
