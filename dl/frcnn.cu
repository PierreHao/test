#include "hip/hip_runtime.h"
#include "layer.h"
#include <stdio.h>
#include <string.h>

typedef struct PVANET_
{
  Tensor input;
  Tensor conv1_1, conv1_2;
  Tensor weight1_1, bias1_1, weight1_2, bias1_2;
  Tensor conv2_1, conv2_2;
  Tensor weight2_1, bias2_1, weight2_2, bias2_2;
  Tensor conv3_1, conv3_2, conv3_3;
  Tensor weight3_1, bias3_1, weight3_2, bias3_2, weight3_3, bias3_3;
  Tensor conv4_1, conv4_2, conv4_3;
  Tensor weight4_1, bias4_1, weight4_2, bias4_2, weight4_3, bias4_3;
  Tensor conv5_1, conv5_2, conv5_3;
  Tensor weight5_1, bias5_1, weight5_2, bias5_2, weight5_3, bias5_3;
  Tensor downsample, upsample, concat;
  Tensor weight_up, bias_up;
  Tensor convf;
  Tensor weightf, biasf;
} PVANET;

typedef struct SRPN_
{
  Tensor conv1, conv3, conv5;
  Tensor weight_c1, bias_c1, weight_c3, bias_c3, weight_c5, bias_c5;
  Tensor score1, score3, score5;
  Tensor weight_s1, bias_s1, weight_s3, bias_s3, weight_s5, bias_s5;
  Tensor bbox1, bbox3, bbox5;
  Tensor weight_b1, bias_b1, weight_b3, bias_b3, weight_b5, bias_b5;
  Tensor score, pred, bbox;
  Tensor img_info;
  Tensor roi;
} SRPN;

typedef struct RCNN_
{
  Tensor roipool, roipool_flat;
  Tensor fc6, fc7;
  Tensor weight6, bias6, weight7, bias7;
  Tensor score, bbox, pred;
  Tensor weight_s, bias_s, weight_b, bias_b;
} RCNN;

PVANET pvanet;
SRPN srpn;
RCNN rcnn;

ConvOption conv_option1;
ConvOption conv_option2;
ConvOption conv1x1_option;
ConvOption conv5x5_option;
ConvOption deconv_option;
PoolOption pool_option;
ReluOption relu_option;
ProposalOption proposal_option;
ROIPoolOption roipool_option;
FCOption fc_option;

const Tensor* const concat_bottoms[3]
    = { &pvanet.downsample, &pvanet.conv4_3, &pvanet.upsample };
const Tensor* const score_bottoms[3]
    = { &srpn.score1, &srpn.score3, &srpn.score5 };
const Tensor* const bbox_bottoms[3]
    = { &srpn.bbox1, &srpn.bbox3, &srpn.bbox5 };

int max_layer_size = 0, max_param_size = 0;
int max_temp_size = 0, max_const_size = 0, max_tempint_size = 0;

real* layer1_data = NULL;
real* layer2_data = NULL;
real* layer3_data = NULL;
real* backup1_data = NULL;
real* backup2_data = NULL;
real* temp_data = NULL;
real* const_data = NULL;
int* tempint_data = NULL;

real* true_data = NULL;
real* input_data = NULL;
real* output_data = NULL;
real* param_data = NULL;

real* anchors = NULL;
real anchor_scales[5] = { 3.0f, 6.0f, 9.0f, 16.0f, 32.0f };
real anchor_ratios[5] = { 0.5f, 0.666f, 1.0f, 1.5f, 2.0f };
real* proposal_temp = NULL;
int* proposal_tempint = NULL;

void load_tensor(const char* filename, Tensor* const tensor)
{
  int ndim;
  int shape[g_max_ndim];

  {
  #ifdef GPU
    int data_size = 1;
    load_data(filename, &ndim, shape, param_data);
    for (int i = 0; i < ndim; ++i) {
      data_size *= shape[i];
    }
    if (data_size != flatten_size(tensor)) {
      printf("[ERROR] Size mismatch: %s (%d) != tensor (%d)\n",
             filename, data_size, flatten_size(tensor));
    }
    hipMemcpyAsync(tensor->data, param_data, data_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    load_data(filename, &ndim, shape, tensor->data);
  #endif
  }
}

int malloc_tensor(Tensor* const tensor)
{
  const int data_size = flatten_size(tensor);

  #ifdef GPU
  hipMalloc(&tensor->data, data_size * sizeof(real));
  #else
  tensor->data = (real*)malloc(data_size * sizeof(real));
  #endif

  return data_size * sizeof(real);
}

void print_tensor_info(const char* name, const Tensor* const tensor)
{
  printf("%s: ", name);
  if (tensor->num_items > 1) {
    printf("%d x ", tensor->num_items);
  }
  for (int i = 0; i < tensor->ndim - 1; ++i) {
    printf("%d x ", tensor->shape[0][i]);
  }
  printf("%d\n", tensor->shape[0][tensor->ndim - 1]);
}

void forward_frcnn_7_1_1(void)
{
  // PVANET
  {
    // 1_1
    pvanet.input.data = layer1_data;
    pvanet.conv1_1.data = layer2_data;
    conv_option1.out_channels = 32;
    conv_forward(&pvanet.input, &pvanet.conv1_1,
                &pvanet.weight1_1, &pvanet.bias1_1,
                temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv1_1, &relu_option);

    // 1_2
    pvanet.conv1_2.data = layer1_data;
    conv_option2.out_channels = 32;
    conv_forward(&pvanet.conv1_1, &pvanet.conv1_2,
                 &pvanet.weight1_2, &pvanet.bias1_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv1_2, &relu_option);

    // 2_1
    pvanet.conv2_1.data = layer2_data;
    conv_option1.out_channels = 64;
    conv_forward(&pvanet.conv1_2, &pvanet.conv2_1,
                 &pvanet.weight2_1, &pvanet.bias2_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv2_1, &relu_option);

    // 2_2
    pvanet.conv2_2.data = layer1_data;
    conv_option2.out_channels = 64;
    conv_forward(&pvanet.conv2_1, &pvanet.conv2_2,
                 &pvanet.weight2_2, &pvanet.bias2_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv2_2, &relu_option);

    // 3_1
    pvanet.conv3_1.data = layer2_data;
    conv_option1.out_channels = 96;
    conv_forward(&pvanet.conv2_2, &pvanet.conv3_1,
                 &pvanet.weight3_1, &pvanet.bias3_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv3_1, &relu_option);

    // 3_2
    pvanet.conv3_2.data = layer1_data;
    conv_option2.out_channels = 64;
    conv_forward(&pvanet.conv3_1, &pvanet.conv3_2,
                 &pvanet.weight3_2, &pvanet.bias3_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv3_2, &relu_option);

    // 3_3
    pvanet.conv3_3.data = layer2_data;
    conv_option2.out_channels = 128;
    conv_forward(&pvanet.conv3_2, &pvanet.conv3_3,
                 &pvanet.weight3_3, &pvanet.bias3_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv3_3, &relu_option);

    // downsample
    pvanet.downsample.data = backup1_data;
    pool_forward(&pvanet.conv3_3, &pvanet.downsample,
                 tempint_data, &pool_option);

    // 4_1
    pvanet.conv4_1.data = layer1_data;
    conv_option1.out_channels = 192;
    conv_forward(&pvanet.conv3_3, &pvanet.conv4_1,
                 &pvanet.weight4_1, &pvanet.bias4_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv4_1, &relu_option);

    // 4_2
    pvanet.conv4_2.data = layer2_data;
    conv_option2.out_channels = 128;
    conv_forward(&pvanet.conv4_1, &pvanet.conv4_2,
                 &pvanet.weight4_2, &pvanet.bias4_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv4_2, &relu_option);

    // 4_3
    pvanet.conv4_3.data = backup2_data;
    conv_option2.out_channels = 256;
    conv_forward(&pvanet.conv4_2, &pvanet.conv4_3,
                 &pvanet.weight4_3, &pvanet.bias4_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv4_3, &relu_option);

    // 5_1
    pvanet.conv5_1.data = layer1_data;
    conv_option1.out_channels = 384;
    conv_forward(&pvanet.conv4_3, &pvanet.conv5_1,
                 &pvanet.weight5_1, &pvanet.bias5_1,
                 temp_data, const_data, &conv_option1);
    relu_forward_inplace(&pvanet.conv5_1, &relu_option);

    // 5_2
    pvanet.conv5_2.data = layer2_data;
    conv_option2.out_channels = 256;
    conv_forward(&pvanet.conv5_1, &pvanet.conv5_2,
                 &pvanet.weight5_2, &pvanet.bias5_2,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv5_2, &relu_option);

    // 5_3
    pvanet.conv5_3.data = layer1_data;
    conv_option2.out_channels = 512;
    conv_forward(&pvanet.conv5_2, &pvanet.conv5_3,
                 &pvanet.weight5_3, &pvanet.bias5_3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&pvanet.conv5_3, &relu_option);

    // upsample
    pvanet.upsample.data = layer2_data;
    deconv_forward(&pvanet.conv5_3, &pvanet.upsample,
                   &pvanet.weight_up, &pvanet.bias_up,
                   temp_data, const_data, &deconv_option);

    // concat
    pvanet.concat.data = layer1_data;
    concat_forward(concat_bottoms, &pvanet.concat, 3);

    // convf
    pvanet.convf.data = backup1_data;
    conv1x1_option.out_channels = 512;
    conv_forward(&pvanet.concat, &pvanet.convf,
                 &pvanet.weightf, &pvanet.biasf,
                 temp_data, const_data, &conv1x1_option);
    relu_forward_inplace(&pvanet.convf, &relu_option);
  }

  // SRPN
  {
    // conv1
    srpn.conv1.data = layer1_data;
    conv1x1_option.out_channels = 128;
    conv_forward(&pvanet.convf, &srpn.conv1,
                 &srpn.weight_c1, &srpn.bias_c1,
                 temp_data, const_data, &conv1x1_option);
    relu_forward_inplace(&srpn.conv1, &relu_option);

    // conv3
    srpn.conv3.data = layer2_data;
    conv_option2.out_channels = 256;
    conv_forward(&pvanet.convf, &srpn.conv3,
                 &srpn.weight_c3, &srpn.bias_c3,
                 temp_data, const_data, &conv_option2);
    relu_forward_inplace(&srpn.conv3, &relu_option);

    // conv5
    srpn.conv5.data = layer3_data;
    conv5x5_option.out_channels = 128;
    conv_forward(&pvanet.convf, &srpn.conv5,
                 &srpn.weight_c5, &srpn.bias_c5,
                 temp_data, const_data, &conv5x5_option);
    relu_forward_inplace(&srpn.conv5, &relu_option);

    // score1
    conv1x1_option.out_channels = 50;
    conv_forward(&srpn.conv1, &srpn.score1,
                 &srpn.weight_s1, &srpn.bias_s1,
                 temp_data, const_data, &conv1x1_option);

    // score3
    conv1x1_option.out_channels = 50;
    conv_forward(&srpn.conv3, &srpn.score3,
                 &srpn.weight_s3, &srpn.bias_s3,
                 temp_data, const_data, &conv1x1_option);

    // score5
    conv1x1_option.out_channels = 50;
    conv_forward(&srpn.conv5, &srpn.score5,
                 &srpn.weight_s5, &srpn.bias_s5,
                 temp_data, const_data, &conv1x1_option);

    // bbox1
    conv1x1_option.out_channels = 100;
    conv_forward(&srpn.conv1, &srpn.bbox1,
                 &srpn.weight_b1, &srpn.bias_b1,
                 temp_data, const_data, &conv1x1_option);

    // bbox3
    conv1x1_option.out_channels = 100;
    conv_forward(&srpn.conv3, &srpn.bbox3,
                 &srpn.weight_b3, &srpn.bias_b3,
                 temp_data, const_data, &conv1x1_option);

    // bbox5
    conv1x1_option.out_channels = 100;
    conv_forward(&srpn.conv5, &srpn.bbox5,
                 &srpn.weight_b5, &srpn.bias_b5,
                 temp_data, const_data, &conv1x1_option);

    // score
    srpn.score.data = layer1_data;
    concat_forward(score_bottoms, &srpn.score, 3);

    // pred
    srpn.pred.ndim = 3;
    srpn.pred.num_items = srpn.score.num_items;
    srpn.pred.shape[0][0] = 2;
    srpn.pred.shape[0][1]
        = srpn.score.shape[0][0] / 2 * srpn.score.shape[0][1];
    srpn.pred.shape[0][2] = srpn.score.shape[0][2];
    srpn.pred.data = srpn.score.data;
    softmax_inplace_forward(&srpn.pred, temp_data);

    // pred reshape
    srpn.pred.ndim = 4;
    srpn.pred.num_items = srpn.score.num_items;
    for (int n = 0; n < srpn.score.num_items; ++n) {
      srpn.pred.shape[n][0] = 2;
      srpn.pred.shape[n][1] = srpn.score.shape[n][0] / 2;
      srpn.pred.shape[n][2] = srpn.score.shape[n][1];
      srpn.pred.shape[n][3] = srpn.score.shape[n][2];
    }

    // bbox
    srpn.bbox.data = layer2_data;
    concat_forward(bbox_bottoms, &srpn.bbox, 3);
    // bbox reshape
    srpn.bbox.ndim = 4;
    for (int n = 0; n < srpn.bbox.num_items; ++n) {
      const int C = srpn.bbox.shape[n][0];
      const int H = srpn.bbox.shape[n][1];
      const int W = srpn.bbox.shape[n][2];
      srpn.bbox.shape[n][0] = C / 4;
      srpn.bbox.shape[n][1] = 4;
      srpn.bbox.shape[n][2] = H;
      srpn.bbox.shape[n][3] = W;
    }

    // proposal
    proposal_forward(&srpn.pred, &srpn.bbox, &srpn.img_info,
                     &srpn.roi, anchors,
                     proposal_temp, proposal_tempint,
                     temp_data, tempint_data,
                     &proposal_option);
  }

  // R-CNN
  {
    // roipool
    rcnn.roipool.data = layer1_data;
    roipool_forward(&pvanet.convf, &srpn.roi, &rcnn.roipool,
                    tempint_data, &roipool_option);

    // roipool reshape
    {
      // calculate total number of RoI-pooled data
      int total_num_rois = 0;
      for (int n = 0; n < rcnn.roipool.num_items; ++n) {
        total_num_rois += rcnn.roipool.shape[n][0];
      }

      // reshape to 2d tensor: total_num_rois x (C * H * W)
      rcnn.roipool_flat.ndim = 2;
      rcnn.roipool_flat.num_items = 1;
      rcnn.roipool_flat.shape[0][0] = total_num_rois;
      rcnn.roipool_flat.shape[0][1] = rcnn.roipool.shape[0][1]
                                      * rcnn.roipool.shape[0][2]
                                      * rcnn.roipool.shape[0][3];
    }

    // fc6
    rcnn.roipool_flat.data = rcnn.roipool.data;
    rcnn.fc6.data = layer2_data;
    fc_option.out_channels = 4096;
    fc_forward(&rcnn.roipool_flat, &rcnn.fc6, &rcnn.weight6, &rcnn.bias6,
               const_data, &fc_option);

    // fc7
    rcnn.fc7.data = layer1_data;
    fc_option.out_channels = 4096;
    fc_forward(&rcnn.fc6, &rcnn.fc7, &rcnn.weight7, &rcnn.bias7,
               const_data, &fc_option);

    // bbox
    rcnn.bbox.data = backup2_data;
    fc_option.out_channels = 84;
    fc_forward(&rcnn.fc7, &rcnn.bbox, &rcnn.weight_b, &rcnn.bias_b,
               const_data, &fc_option);
    // bbox reshape
    rcnn.bbox.ndim = 3;
    rcnn.bbox.num_items = rcnn.roipool.num_items;
    for (int n = 0; n < rcnn.roipool.num_items; ++n) {
      rcnn.bbox.shape[n][0] = rcnn.roipool.shape[n][0];
      rcnn.bbox.shape[n][1] = 21;
      rcnn.bbox.shape[n][2] = 4;
    }

    // score
    rcnn.score.data = backup1_data;
    fc_option.out_channels = 21;
    fc_forward(&rcnn.fc7, &rcnn.score, &rcnn.weight_s, &rcnn.bias_s,
               const_data, &fc_option);

    // pred
    rcnn.pred.ndim = 3;
    rcnn.pred.num_items = rcnn.roipool_flat.shape[0][0];
    for (int n = 0; n < rcnn.pred.num_items; ++n) {
      rcnn.pred.shape[n][0] = 21;
      rcnn.pred.shape[n][1] = 1;
      rcnn.pred.shape[n][2] = 1;
    }
    rcnn.pred.data = rcnn.score.data;
    softmax_inplace_forward(&rcnn.pred, temp_data);

    // pred reshape
    rcnn.pred.ndim = 2;
    rcnn.pred.num_items = rcnn.roipool.num_items;
    for (int n = 0; n < rcnn.pred.num_items; ++n) {
      rcnn.pred.shape[n][0] = rcnn.roipool.shape[n][0];
      rcnn.pred.shape[n][1] = 21;
    }
  }
}

void shape_frcnn_7_1_1(const int print_network_info)
{
  int temp_size, const_size, tempint_size;

  // PVANET
  {
    // 1_1
    conv_option1.out_channels = 32;
    conv_shape(&pvanet.input, &pvanet.conv1_1,
               &pvanet.weight1_1, &pvanet.bias1_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.input));
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv1_1));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight1_1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 1_2
    conv_option2.out_channels = 32;
    conv_shape(&pvanet.conv1_1, &pvanet.conv1_2,
               &pvanet.weight1_2, &pvanet.bias1_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv1_2));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight1_2));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 2_1
    conv_option1.out_channels = 64;
    conv_shape(&pvanet.conv1_2, &pvanet.conv2_1,
               &pvanet.weight2_1, &pvanet.bias2_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv2_1));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight2_1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 2_2
    conv_option2.out_channels = 64;
    conv_shape(&pvanet.conv2_1, &pvanet.conv2_2,
               &pvanet.weight2_2, &pvanet.bias2_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv2_2));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight2_2));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 3_1
    conv_option1.out_channels = 96;
    conv_shape(&pvanet.conv2_2, &pvanet.conv3_1,
               &pvanet.weight3_1, &pvanet.bias3_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv3_1));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight3_1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 3_2
    conv_option2.out_channels = 64;
    conv_shape(&pvanet.conv3_1, &pvanet.conv3_2,
               &pvanet.weight3_2, &pvanet.bias3_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv3_2));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight3_2));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 3_3
    conv_option2.out_channels = 128;
    conv_shape(&pvanet.conv3_2, &pvanet.conv3_3,
               &pvanet.weight3_3, &pvanet.bias3_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv3_3));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight3_3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 4_1
    conv_option1.out_channels = 192;
    conv_shape(&pvanet.conv3_3, &pvanet.conv4_1,
               &pvanet.weight4_1, &pvanet.bias4_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv4_1));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight4_1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 4_2
    conv_option2.out_channels = 128;
    conv_shape(&pvanet.conv4_1, &pvanet.conv4_2,
               &pvanet.weight4_2, &pvanet.bias4_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv4_2));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight4_2));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 4_3
    conv_option2.out_channels = 256;
    conv_shape(&pvanet.conv4_2, &pvanet.conv4_3,
               &pvanet.weight4_3, &pvanet.bias4_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv4_3));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight4_3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 5_1
    conv_option1.out_channels = 384;
    conv_shape(&pvanet.conv4_3, &pvanet.conv5_1,
               &pvanet.weight5_1, &pvanet.bias5_1,
               &temp_size, &const_size, &conv_option1);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv5_1));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight5_1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 5_2
    conv_option2.out_channels = 256;
    conv_shape(&pvanet.conv5_1, &pvanet.conv5_2,
               &pvanet.weight5_2, &pvanet.bias5_2,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv5_2));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight5_2));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // 5_3
    conv_option2.out_channels = 512;
    conv_shape(&pvanet.conv5_2, &pvanet.conv5_3,
               &pvanet.weight5_3, &pvanet.bias5_3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.conv5_3));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight5_3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // downsample
    pool_shape(&pvanet.conv3_3, &pvanet.downsample,
               &tempint_size, &pool_option);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.downsample));
    max_tempint_size = MAX(max_tempint_size,  tempint_size);

    // upsample
    deconv_shape(&pvanet.conv5_3, &pvanet.upsample,
                 &pvanet.weight_up, &pvanet.bias_up,
                 &temp_size, &const_size, &deconv_option);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.upsample));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weight_up));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // concat
    concat_shape(concat_bottoms, &pvanet.concat, 3);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.concat));

    // convf
    conv1x1_option.out_channels = 512;
    conv_shape(&pvanet.concat, &pvanet.convf,
               &pvanet.weightf, &pvanet.biasf,
               &temp_size, &const_size, &conv1x1_option);
    max_layer_size = MAX(max_layer_size,  flatten_size(&pvanet.convf));
    max_param_size = MAX(max_param_size,  flatten_size(&pvanet.weightf));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);
  }

  if (print_network_info) {
    print_tensor_info("input", &pvanet.input);
    print_tensor_info("conv1_1", &pvanet.conv1_1);
    print_tensor_info("conv1_2", &pvanet.conv1_2);
    print_tensor_info("conv2_1", &pvanet.conv2_1);
    print_tensor_info("conv2_2", &pvanet.conv2_2);
    print_tensor_info("conv3_1", &pvanet.conv3_1);
    print_tensor_info("conv3_2", &pvanet.conv3_2);
    print_tensor_info("conv3_3", &pvanet.conv3_3);
    print_tensor_info("conv4_1", &pvanet.conv4_1);
    print_tensor_info("conv4_2", &pvanet.conv4_2);
    print_tensor_info("conv4_3", &pvanet.conv4_3);
    print_tensor_info("conv5_1", &pvanet.conv5_1);
    print_tensor_info("conv5_2", &pvanet.conv5_2);
    print_tensor_info("conv5_3", &pvanet.conv5_3);
    print_tensor_info("downsample", &pvanet.downsample);
    print_tensor_info("upsample", &pvanet.upsample);
    print_tensor_info("concat", &pvanet.concat);
    print_tensor_info("convf", &pvanet.convf);
  }

  // SRPN
  {
    // conv1
    conv1x1_option.out_channels = 128;
    conv_shape(&pvanet.convf, &srpn.conv1,
               &srpn.weight_c1, &srpn.bias_c1,
               &temp_size, &const_size, &conv1x1_option);
    max_layer_size = MAX(max_layer_size,  flatten_size(&srpn.conv1));
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_c1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // conv3
    conv_option2.out_channels = 256;
    conv_shape(&pvanet.convf, &srpn.conv3,
               &srpn.weight_c3, &srpn.bias_c3,
               &temp_size, &const_size, &conv_option2);
    max_layer_size = MAX(max_layer_size,  flatten_size(&srpn.conv3));
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_c3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // conv5
    conv5x5_option.out_channels = 128;
    conv_shape(&pvanet.convf, &srpn.conv5,
               &srpn.weight_c5, &srpn.bias_c5,
               &temp_size, &const_size, &conv5x5_option);
    max_layer_size = MAX(max_layer_size,  flatten_size(&srpn.conv5));
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_c5));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // score1
    conv1x1_option.out_channels = 50;
    conv_shape(&srpn.conv1, &srpn.score1,
               &srpn.weight_s1, &srpn.bias_s1,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_s1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // score3
    conv1x1_option.out_channels = 50;
    conv_shape(&srpn.conv3, &srpn.score3,
               &srpn.weight_s3, &srpn.bias_s3,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_s3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // score5
    conv1x1_option.out_channels = 50;
    conv_shape(&srpn.conv5, &srpn.score5,
               &srpn.weight_s5, &srpn.bias_s5,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_s5));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // bbox1
    conv1x1_option.out_channels = 100;
    conv_shape(&srpn.conv1, &srpn.bbox1,
               &srpn.weight_b1, &srpn.bias_b1,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_b1));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // bbox3
    conv1x1_option.out_channels = 100;
    conv_shape(&srpn.conv3, &srpn.bbox3,
               &srpn.weight_b3, &srpn.bias_b3,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_b3));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // bbox5
    conv1x1_option.out_channels = 100;
    conv_shape(&srpn.conv5, &srpn.bbox5,
               &srpn.weight_b5, &srpn.bias_b5,
               &temp_size, &const_size, &conv1x1_option);
    max_param_size = MAX(max_param_size,  flatten_size(&srpn.weight_b5));
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_const_size = MAX(max_const_size,  const_size);

    // score
    concat_shape(score_bottoms, &srpn.score, 3);
    max_layer_size = MAX(max_layer_size,  flatten_size(&srpn.score));

    // pred
    srpn.pred.ndim = 3;
    srpn.pred.num_items = srpn.score.num_items;
    srpn.pred.shape[0][0] = 2;
    srpn.pred.shape[0][1]
        = srpn.score.shape[0][0] / 2 * srpn.score.shape[0][1];
    srpn.pred.shape[0][2] = srpn.score.shape[0][2];

    // pred reshape
    srpn.pred.ndim = 4;
    srpn.pred.num_items = srpn.score.num_items;
    for (int n = 0; n < srpn.score.num_items; ++n) {
      srpn.pred.shape[n][0] = 2;
      srpn.pred.shape[n][1] = srpn.score.shape[n][0] / 2;
      srpn.pred.shape[n][2] = srpn.score.shape[n][1];
      srpn.pred.shape[n][3] = srpn.score.shape[n][2];
    }

    // bbox
    concat_shape(bbox_bottoms, &srpn.bbox, 3);
    max_layer_size = MAX(max_layer_size,  flatten_size(&srpn.bbox));
    // bbox reshape
    srpn.bbox.ndim = 4;
    for (int n = 0; n < srpn.bbox.num_items; ++n) {
      const int C = srpn.bbox.shape[n][0];
      const int H = srpn.bbox.shape[n][1];
      const int W = srpn.bbox.shape[n][2];
      srpn.bbox.shape[n][0] = C / 4;
      srpn.bbox.shape[n][1] = 4;
      srpn.bbox.shape[n][2] = H;
      srpn.bbox.shape[n][3] = W;
    }

    // img_info
    srpn.img_info.ndim = 1;
    srpn.img_info.num_items = srpn.bbox.num_items;
    for (int n = 0; n < srpn.img_info.num_items; ++n) {
      srpn.img_info.shape[n][0] = 4;
    }

    // proposal
    proposal_shape(&srpn.pred, &srpn.roi,
                   &temp_size, &tempint_size, &proposal_option);
    max_temp_size = MAX(max_temp_size,  temp_size);
    max_tempint_size = MAX(max_tempint_size,  tempint_size);
  }

  if (print_network_info) {
    print_tensor_info("rpn_conv1", &srpn.conv1);
    print_tensor_info("rpn_conv3", &srpn.conv3);
    print_tensor_info("rpn_conv5", &srpn.conv5);
    print_tensor_info("rpn_score1", &srpn.score1);
    print_tensor_info("rpn_score3", &srpn.score3);
    print_tensor_info("rpn_score5", &srpn.score5);
    print_tensor_info("rpn_bbox1", &srpn.bbox1);
    print_tensor_info("rpn_bbox3", &srpn.bbox3);
    print_tensor_info("rpn_bbox5", &srpn.bbox5);
    print_tensor_info("rpn_score", &srpn.score);
    print_tensor_info("rpn_pred", &srpn.pred);
    print_tensor_info("rpn_pred_reshape", &srpn.pred);
    print_tensor_info("rpn_bbox", &srpn.bbox);
    print_tensor_info("rpn_bbox_reshape", &srpn.bbox);
    print_tensor_info("img_info", &srpn.img_info);
    print_tensor_info("roi", &srpn.roi);
  }

  // R-CNN
  {
    // roipool
    roipool_shape(&pvanet.convf, &srpn.roi, &rcnn.roipool,
                  &tempint_size, &roipool_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&rcnn.roipool));
    max_tempint_size = MAX(max_tempint_size, tempint_size);

    // roipool reshape
    {
      // calculate total number of RoI-pooled data
      int total_num_rois = 0;
      for (int n = 0; n < rcnn.roipool.num_items; ++n) {
        total_num_rois += rcnn.roipool.shape[n][0];
      }

      // reshape to 2d tensor: total_num_rois x (C * H * W)
      rcnn.roipool_flat.ndim = 2;
      rcnn.roipool_flat.num_items = 1;
      rcnn.roipool_flat.shape[0][0] = total_num_rois;
      rcnn.roipool_flat.shape[0][1] = rcnn.roipool.shape[0][1]
                                      * rcnn.roipool.shape[0][2]
                                      * rcnn.roipool.shape[0][3]; 
    }

    // fc6
    fc_option.out_channels = 4096;
    fc_shape(&rcnn.roipool_flat, &rcnn.fc6, &rcnn.weight6, &rcnn.bias6,
             &const_size, &fc_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&rcnn.fc6));
    max_param_size = MAX(max_param_size,  flatten_size(&rcnn.weight6));
    max_const_size = MAX(max_const_size, const_size);

    // fc7
    fc_option.out_channels = 4096;
    fc_shape(&rcnn.fc6, &rcnn.fc7, &rcnn.weight7, &rcnn.bias7,
             &const_size, &fc_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&rcnn.fc7));
    max_param_size = MAX(max_param_size,  flatten_size(&rcnn.weight7));
    max_const_size = MAX(max_const_size, const_size);

    // bbox
    fc_option.out_channels = 84;
    fc_shape(&rcnn.fc7, &rcnn.bbox, &rcnn.weight_b, &rcnn.bias_b,
             &const_size, &fc_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&rcnn.bbox));
    max_param_size = MAX(max_param_size,  flatten_size(&rcnn.weight_b));
    max_const_size = MAX(max_const_size, const_size);
    // bbox reshape
    rcnn.bbox.ndim = 3;
    rcnn.bbox.num_items = rcnn.roipool.num_items;
    for (int n = 0; n < rcnn.roipool.num_items; ++n) {
      rcnn.bbox.shape[n][0] = rcnn.roipool.shape[n][0];
      rcnn.bbox.shape[n][1] = 21;
      rcnn.bbox.shape[n][2] = 4;
    }

    // score
    fc_option.out_channels = 21;
    fc_shape(&rcnn.fc7, &rcnn.score, &rcnn.weight_s, &rcnn.bias_s,
             &const_size, &fc_option);
    max_layer_size = MAX(max_layer_size, flatten_size(&rcnn.score));
    max_param_size = MAX(max_param_size,  flatten_size(&rcnn.weight_s));
    max_const_size = MAX(max_const_size, const_size);

    // pred
    rcnn.pred.ndim = 2;
    rcnn.pred.num_items = rcnn.roipool.num_items;
    for (int n = 0; n < rcnn.roipool.num_items; ++n) {
      rcnn.pred.shape[n][0] = rcnn.roipool.shape[n][0];
      rcnn.pred.shape[n][1] = 21;
    }
  }

  if (print_network_info) {
    print_tensor_info("roipool", &rcnn.roipool);
    print_tensor_info("roipool_flat", &rcnn.roipool_flat);
    print_tensor_info("fc6", &rcnn.fc6);
    print_tensor_info("fc7", &rcnn.fc7);
    print_tensor_info("bbox", &rcnn.bbox);
    print_tensor_info("score", &rcnn.score);
    print_tensor_info("pred", &rcnn.pred);
  }
}

void init_frcnn_7_1_1(void)
{
  // set fixed options
  {
    conv_option1.num_groups = 1;
    conv_option1.kernel_h = 3;
    conv_option1.kernel_w = 3;
    conv_option1.pad_h = 1;
    conv_option1.pad_w = 1;
    conv_option1.bias = 1;
    conv_option1.stride_h = 2;
    conv_option1.stride_w = 2;

    conv_option2 = conv_option1;
    conv_option2.stride_h = 1;
    conv_option2.stride_w = 1;

    conv1x1_option = conv_option2;
    conv1x1_option.kernel_h = 1;
    conv1x1_option.kernel_w = 1;
    conv1x1_option.pad_h = 0;
    conv1x1_option.pad_w = 0;

    conv5x5_option = conv_option2;
    conv5x5_option.kernel_h = 5;
    conv5x5_option.kernel_w = 5;
    conv5x5_option.pad_h = 2;
    conv5x5_option.pad_w = 2;

    deconv_option = conv_option1;
    deconv_option.num_groups = 512;
    deconv_option.out_channels = 512;
    deconv_option.kernel_h = 4;
    deconv_option.kernel_w = 4;
    deconv_option.bias = 0;

    pool_option.kernel_h = 3;
    pool_option.kernel_w = 3;
    pool_option.stride_h = 2;
    pool_option.stride_w = 2;
    pool_option.pad_h = 0;
    pool_option.pad_w = 0;

    relu_option.negative_slope = 0;

    proposal_option.scales = &anchor_scales[0];
    proposal_option.ratios = &anchor_ratios[0];
    proposal_option.num_scales = 5;
    proposal_option.num_ratios = 5;
    proposal_option.num_concats = 3;
    proposal_option.base_size = 16;
    proposal_option.feat_stride = 16;
    proposal_option.min_size = 16;
    proposal_option.pre_nms_topn = 6000;
    proposal_option.post_nms_topn = 300;
    proposal_option.nms_thresh = 0.7f;

    roipool_option.pooled_height = 6;
    roipool_option.pooled_width = 6;
    roipool_option.spatial_scale = 0.0625;

    fc_option.bias = 1;
  }

  // calculate maximum size
  pvanet.input.num_items = 1;
  pvanet.input.ndim = 3;
  pvanet.input.shape[0][0] = 3;
  pvanet.input.shape[0][1] = 640;
  pvanet.input.shape[0][2] = 1024;
  shape_frcnn_7_1_1(1);

  // memory allocation
  {
    // total memory size (in byte) always allocated in main memory
    long int space_cpu = 0;
    // allocated size in GPU memory (GPU mode) or main memory (CPU mode)
    long int space = 0;

    // space for data loading
    {
      input_data = (real*)malloc(flatten_size(&pvanet.input) * sizeof(real));
      output_data = (real*)malloc(max_layer_size * sizeof(real));
      true_data = (real*)malloc(max_layer_size * sizeof(real));
      param_data = (real*)malloc(max_param_size * sizeof(real));
      space_cpu += sizeof(real) * (flatten_size(&pvanet.input)
                                   + max_layer_size * 2 + max_param_size);
    }

    // space required for forward-pass
    {
      const int num_anchors = proposal_option.num_scales
                              * proposal_option.num_ratios
                              * proposal_option.num_concats;
    #ifdef GPU
      hipMalloc(&layer1_data, max_layer_size * sizeof(real));
      hipMalloc(&layer2_data, max_layer_size * sizeof(real));
      hipMalloc(&layer3_data, max_layer_size * sizeof(real));
      hipMalloc(&temp_data, max_temp_size * sizeof(real));
      hipMalloc(&const_data, max_const_size * sizeof(real));
      hipMalloc(&tempint_data, max_tempint_size * sizeof(int));
      hipMalloc(&backup1_data, max_layer_size * sizeof(real));
      hipMalloc(&backup2_data, max_layer_size * sizeof(real));
      hipMalloc(&anchors, num_anchors * 4 * sizeof(real));
    #else
      layer1_data = (real*)malloc(max_layer_size * sizeof(real));
      layer2_data = (real*)malloc(max_layer_size * sizeof(real));
      layer3_data = (real*)malloc(max_layer_size * sizeof(real));
      temp_data = (real*)malloc(max_temp_size * sizeof(real));
      const_data = (real*)malloc(max_const_size * sizeof(real));
      tempint_data = (int*)malloc(max_tempint_size * sizeof(int));
      backup1_data = (real*)malloc(max_layer_size * sizeof(real));
      backup2_data = (real*)malloc(max_layer_size * sizeof(real));
      anchors = (real*)malloc(num_anchors * 4 * sizeof(real));
    #endif
      space += sizeof(real) * (max_layer_size * 5 + max_temp_size
                               + max_const_size + num_anchors * 4)
             + sizeof(int) * (max_tempint_size);

      proposal_temp = (real*)malloc(max_temp_size * sizeof(real));
      proposal_tempint = (int*)malloc(max_tempint_size * sizeof(int));
      space_cpu += sizeof(real) * (max_temp_size)
                 + sizeof(int) * (max_tempint_size);
    }

    // PVANET parameters
    {
      space += malloc_tensor(&pvanet.weight1_1);
      space += malloc_tensor(&pvanet.bias1_1);
      space += malloc_tensor(&pvanet.weight1_2);
      space += malloc_tensor(&pvanet.bias1_2);
      space += malloc_tensor(&pvanet.weight2_1);
      space += malloc_tensor(&pvanet.bias2_1);
      space += malloc_tensor(&pvanet.weight2_2);
      space += malloc_tensor(&pvanet.bias2_2);
      space += malloc_tensor(&pvanet.weight3_1);
      space += malloc_tensor(&pvanet.bias3_1);
      space += malloc_tensor(&pvanet.weight3_2);
      space += malloc_tensor(&pvanet.bias3_2);
      space += malloc_tensor(&pvanet.weight3_3);
      space += malloc_tensor(&pvanet.bias3_3);
      space += malloc_tensor(&pvanet.weight4_1);
      space += malloc_tensor(&pvanet.bias4_1);
      space += malloc_tensor(&pvanet.weight4_2);
      space += malloc_tensor(&pvanet.bias4_2);
      space += malloc_tensor(&pvanet.weight4_3);
      space += malloc_tensor(&pvanet.bias4_3);
      space += malloc_tensor(&pvanet.weight5_1);
      space += malloc_tensor(&pvanet.bias5_1);
      space += malloc_tensor(&pvanet.weight5_2);
      space += malloc_tensor(&pvanet.bias5_2);
      space += malloc_tensor(&pvanet.weight5_3);
      space += malloc_tensor(&pvanet.bias5_3);
      space += malloc_tensor(&pvanet.weight_up);
      space += malloc_tensor(&pvanet.bias_up);
      space += malloc_tensor(&pvanet.weightf);
      space += malloc_tensor(&pvanet.biasf);
    }

    // SRPN parameters & layers
    {
      space += malloc_tensor(&srpn.weight_c1);
      space += malloc_tensor(&srpn.bias_c1);
      space += malloc_tensor(&srpn.weight_c3);
      space += malloc_tensor(&srpn.bias_c3);
      space += malloc_tensor(&srpn.weight_c5);
      space += malloc_tensor(&srpn.bias_c5);
      space += malloc_tensor(&srpn.weight_s1);
      space += malloc_tensor(&srpn.bias_s1);
      space += malloc_tensor(&srpn.weight_s3);
      space += malloc_tensor(&srpn.bias_s3);
      space += malloc_tensor(&srpn.weight_s5);
      space += malloc_tensor(&srpn.bias_s5);
      space += malloc_tensor(&srpn.weight_b1);
      space += malloc_tensor(&srpn.bias_b1);
      space += malloc_tensor(&srpn.weight_b3);
      space += malloc_tensor(&srpn.bias_b3);
      space += malloc_tensor(&srpn.weight_b5);
      space += malloc_tensor(&srpn.bias_b5);

      space += malloc_tensor(&srpn.score1);
      space += malloc_tensor(&srpn.score3);
      space += malloc_tensor(&srpn.score5);
      space += malloc_tensor(&srpn.bbox1);
      space += malloc_tensor(&srpn.bbox3);
      space += malloc_tensor(&srpn.bbox5);
      space += malloc_tensor(&srpn.img_info);
      space += malloc_tensor(&srpn.roi);

      // for convenience, img_info.data is always allocated in main memory
      srpn.img_info.data
          = (real*)malloc(flatten_size(&srpn.img_info) * sizeof(real));
      space_cpu += sizeof(real) * flatten_size(&srpn.img_info);
    }

    // RCNN parameters
    {
      space += malloc_tensor(&rcnn.weight6);
      space += malloc_tensor(&rcnn.bias6);
      space += malloc_tensor(&rcnn.weight7);
      space += malloc_tensor(&rcnn.bias7);
      space += malloc_tensor(&rcnn.weight_s);
      space += malloc_tensor(&rcnn.bias_s);
      space += malloc_tensor(&rcnn.weight_b);
      space += malloc_tensor(&rcnn.bias_b);
    }

    // print total memory size required
    {
    #ifdef GPU
      printf("%ldMB of main memory allocated\n",
             DIV_THEN_CEIL(space_cpu,  1000000));
      printf("%ldMB of GPU memory allocated\n",
             DIV_THEN_CEIL(space,  1000000));
    #else
      printf("%ldMB of main memory allocated\n",
             DIV_THEN_CEIL(space_cpu + space,  1000000));
    #endif
    }
  }

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < max_const_size; ++i) {
      output_data[i] = 1;
    }
    hipMemcpy(const_data, output_data, max_const_size * sizeof(real),
               hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < max_const_size; ++i) {
      const_data[i] = 1;
    }
  #endif
  }

  // anchor generation for proposal layer
  {
  #ifdef GPU
    const int num_anchors = proposal_option.num_scales
                            * proposal_option.num_ratios
                            * proposal_option.num_concats;
    generate_anchors(param_data, &proposal_option);
    hipMemcpy(anchors, param_data, num_anchors * 4 * sizeof(real),
               hipMemcpyHostToDevice);
  #else
    generate_anchors(anchors, &proposal_option);
  #endif
  }

  // PVANET parameter loading
  {
    load_tensor("../data/temp/conv1_1_param0.bin", &pvanet.weight1_1);
    load_tensor("../data/temp/conv1_1_param1.bin", &pvanet.bias1_1);
    load_tensor("../data/temp/conv1_2_param0.bin", &pvanet.weight1_2);
    load_tensor("../data/temp/conv1_2_param1.bin", &pvanet.bias1_2);
    load_tensor("../data/temp/conv2_1_param0.bin", &pvanet.weight2_1);
    load_tensor("../data/temp/conv2_1_param1.bin", &pvanet.bias2_1);
    load_tensor("../data/temp/conv2_2_param0.bin", &pvanet.weight2_2);
    load_tensor("../data/temp/conv2_2_param1.bin", &pvanet.bias2_2);
    load_tensor("../data/temp/conv3_1_param0.bin", &pvanet.weight3_1);
    load_tensor("../data/temp/conv3_1_param1.bin", &pvanet.bias3_1);
    load_tensor("../data/temp/conv3_2_param0.bin", &pvanet.weight3_2);
    load_tensor("../data/temp/conv3_2_param1.bin", &pvanet.bias3_2);
    load_tensor("../data/temp/conv3_3_param0.bin", &pvanet.weight3_3);
    load_tensor("../data/temp/conv3_3_param1.bin", &pvanet.bias3_3);
    load_tensor("../data/temp/conv4_1_param0.bin", &pvanet.weight4_1);
    load_tensor("../data/temp/conv4_1_param1.bin", &pvanet.bias4_1);
    load_tensor("../data/temp/conv4_2_param0.bin", &pvanet.weight4_2);
    load_tensor("../data/temp/conv4_2_param1.bin", &pvanet.bias4_2);
    load_tensor("../data/temp/conv4_3_param0.bin", &pvanet.weight4_3);
    load_tensor("../data/temp/conv4_3_param1.bin", &pvanet.bias4_3);
    load_tensor("../data/temp/conv5_1_param0.bin", &pvanet.weight5_1);
    load_tensor("../data/temp/conv5_1_param1.bin", &pvanet.bias5_1);
    load_tensor("../data/temp/conv5_2_param0.bin", &pvanet.weight5_2);
    load_tensor("../data/temp/conv5_2_param1.bin", &pvanet.bias5_2);
    load_tensor("../data/temp/conv5_3_param0.bin", &pvanet.weight5_3);
    load_tensor("../data/temp/conv5_3_param1.bin", &pvanet.bias5_3);
    load_tensor("../data/temp/upsample_param0.bin", &pvanet.weight_up);
    load_tensor("../data/temp/convf_param0.bin", &pvanet.weightf);
    load_tensor("../data/temp/convf_param1.bin", &pvanet.biasf);
  }

  // SRPN parameter loading
  {
    load_tensor("../data/temp/rpn_conv1_param0.bin", &srpn.weight_c1);
    load_tensor("../data/temp/rpn_conv1_param1.bin", &srpn.bias_c1);
    load_tensor("../data/temp/rpn_conv3_param0.bin", &srpn.weight_c3);
    load_tensor("../data/temp/rpn_conv3_param1.bin", &srpn.bias_c3);
    load_tensor("../data/temp/rpn_conv5_param0.bin", &srpn.weight_c5);
    load_tensor("../data/temp/rpn_conv5_param1.bin", &srpn.bias_c5);
    load_tensor("../data/temp/rpn_cls_score1_param0.bin", &srpn.weight_s1);
    load_tensor("../data/temp/rpn_cls_score1_param1.bin", &srpn.bias_s1);
    load_tensor("../data/temp/rpn_cls_score3_param0.bin", &srpn.weight_s3);
    load_tensor("../data/temp/rpn_cls_score3_param1.bin", &srpn.bias_s3);
    load_tensor("../data/temp/rpn_cls_score5_param0.bin", &srpn.weight_s5);
    load_tensor("../data/temp/rpn_cls_score5_param1.bin", &srpn.bias_s5);
    load_tensor("../data/temp/rpn_bbox_pred1_param0.bin", &srpn.weight_b1);
    load_tensor("../data/temp/rpn_bbox_pred1_param1.bin", &srpn.bias_b1);
    load_tensor("../data/temp/rpn_bbox_pred3_param0.bin", &srpn.weight_b3);
    load_tensor("../data/temp/rpn_bbox_pred3_param1.bin", &srpn.bias_b3);
    load_tensor("../data/temp/rpn_bbox_pred5_param0.bin", &srpn.weight_b5);
    load_tensor("../data/temp/rpn_bbox_pred5_param1.bin", &srpn.bias_b5);
  }

  // RCNN parameter loading
  {
    load_tensor("../data/temp/fc6_param0.bin", &rcnn.weight6);
    load_tensor("../data/temp/fc6_param1.bin", &rcnn.bias6);
    load_tensor("../data/temp/fc7_param0.bin", &rcnn.weight7);
    load_tensor("../data/temp/fc7_param1.bin", &rcnn.bias7);
    load_tensor("../data/temp/cls_score_param0.bin", &rcnn.weight_s);
    load_tensor("../data/temp/cls_score_param1.bin", &rcnn.bias_s);
    load_tensor("../data/temp/bbox_pred_param0.bin", &rcnn.weight_b);
    load_tensor("../data/temp/bbox_pred_param1.bin", &rcnn.bias_b);
  }
}

int main(int argc, char* argv[])
{
  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
    conv_option1.handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    if (hipblasCreate((hipblasHandle_t*)conv_option1.handle)
          != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
    conv_option2.handle = conv_option1.handle;
    conv1x1_option.handle = conv_option1.handle;
    conv5x5_option.handle = conv_option1.handle;
    fc_option.handle = conv_option1.handle;
  }
  #endif

  // PVANET initialization
  init_frcnn_7_1_1();

  // input data loading
  {
    int ndim;
    int shape[g_max_ndim];
    int input_size;

    // input image
    load_data("../data/temp/conv1_1_bottom0.bin",
              &ndim, shape, input_data);
    pvanet.input.num_items = shape[0];
    pvanet.input.ndim = ndim - 1;
    input_size = 0;
    for (int n = 0; n < pvanet.input.num_items; ++n) {
      int size_n = 1;
      for (int i = 0; i < pvanet.input.ndim; ++i) {
        pvanet.input.shape[n][i] = shape[i + 1];
        size_n *= shape[i + 1];
      }
      pvanet.input.start[n] = input_size;
      input_size += size_n;
    }

    // image info
    load_data("../data/temp/proposal_bottom2.bin",
              &ndim, shape, srpn.img_info.data);

  #ifdef GPU
    hipMemcpyAsync(layer1_data, input_data, input_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    memcpy(layer1_data, input_data, input_size * sizeof(real));
  #endif

    print_tensor_info("input data loaded", &pvanet.input);
  }

  // network reshape
  shape_frcnn_7_1_1(0);

  // forward-pass
  printf("forward-pass start\n");
  forward_frcnn_7_1_1();
  printf("forward-pass end\n");

  // retrieve output
  {
    const int output1_size = flatten_size(&rcnn.pred);
    const int output2_size = flatten_size(&rcnn.bbox);

  #ifdef GPU
    hipMemcpyAsync(output_data, rcnn.pred.data,
                    output1_size * sizeof(real),
                    hipMemcpyDeviceToHost);
    hipMemcpyAsync(output_data + output1_size, rcnn.bbox.data,
                    output2_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  #else
    memcpy(output_data, rcnn.pred.data, output1_size * sizeof(real));
    memcpy(output_data + output1_size, rcnn.bbox.data,
           output2_size * sizeof(real));
  #endif
  }

  {
    const int output1_size = flatten_size(&rcnn.pred);
    const int output2_size = flatten_size(&rcnn.bbox);
    const real* const p_pred = output_data;
    const real* const p_bbox = output_data + output1_size;
    int i = 0, j = 0;
    for (int n = 0; n < rcnn.pred.num_items; ++n) {
      for (int r = 0; r < rcnn.pred.shape[n][0]; ++r) {
        real maxpred = p_pred[i];
        int maxclass = 0;
        for (int c = 0; c < rcnn.pred.shape[n][1]; ++c) {
          if (p_pred[i] > maxpred) {
            maxclass = c;
            maxpred = p_pred[i];
          }
          ++i;
        }

        if (maxclass > 0 && maxpred >= 0.8f) {
          const int x1 = (int)ROUND(p_bbox[j + maxclass * 4 + 0]);
          const int x2 = (int)ROUND(p_bbox[j + maxclass * 4 + 1]);
          const int y1 = (int)ROUND(p_bbox[j + maxclass * 4 + 2]);
          const int y2 = (int)ROUND(p_bbox[j + maxclass * 4 + 3]);
          printf("[Image %d] Box (%d, %d, %d, %d): class %d, score = %.2f\n",
                 n, x1, x2, y1, y2, maxclass, maxpred);
        }
        j += 21 * 4;
      } // endfor r
    } // endfor n
  }

  // verify results
  #ifdef PASS
  {
    const int output_size = flatten_size(&rcnn.roipool);

    int ndim;
    int shape[g_max_ndim];
    load_data("../data/temp/roi_pool_conv5_top0.bin",
              &ndim, shape, true_data);

    for (int i = 0; i < output_size; ++i) {
      real diff = ABS(true_data[i] - output_data[i]);
      diff /= 1e-10f + MIN(ABS(true_data[i]),  ABS(output_data[i]));
      #ifdef GPU
      if (diff > 1e-5f) {
        printf("%d: %.6f %.6f\n", i, true_data[i], output_data[i]);
      }
      #else
      if (diff > 1e-3f) {
        printf("%d: %.6f %.6f\n", i, true_data[i], output_data[i]);
      }
      #endif
    }
  }
  #endif

  // memory deallocation
  {
    if (true_data) free(true_data);
    if (input_data) free(input_data);
    if (output_data) free(output_data);
    if (param_data) free(param_data);

    if (proposal_temp) free(proposal_temp);
    if (proposal_tempint) free(proposal_tempint);
  }
  #ifdef GPU
  {
    if (layer1_data) hipFree(layer1_data);
    if (layer2_data) hipFree(layer2_data);
    if (layer3_data) hipFree(layer3_data);
    if (backup1_data) hipFree(backup1_data);
    if (backup2_data) hipFree(backup2_data);
    if (temp_data) hipFree(temp_data);
    if (tempint_data) hipFree(tempint_data);
    if (const_data) hipFree(const_data);
    if (anchors) hipFree(anchors);

    if (hipblasDestroy(*((hipblasHandle_t*)conv_option1.handle))
        != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }

    hipFree(pvanet.weight1_1.data);
    hipFree(pvanet.bias1_1.data);
    hipFree(pvanet.weight1_2.data);
    hipFree(pvanet.bias1_2.data);
    hipFree(pvanet.weight2_1.data);
    hipFree(pvanet.bias2_1.data);
    hipFree(pvanet.weight2_2.data);
    hipFree(pvanet.bias2_2.data);
    hipFree(pvanet.weight3_1.data);
    hipFree(pvanet.bias3_1.data);
    hipFree(pvanet.weight3_2.data);
    hipFree(pvanet.bias3_2.data);
    hipFree(pvanet.weight3_3.data);
    hipFree(pvanet.bias3_3.data);
    hipFree(pvanet.weight4_1.data);
    hipFree(pvanet.bias4_1.data);
    hipFree(pvanet.weight4_2.data);
    hipFree(pvanet.bias4_2.data);
    hipFree(pvanet.weight4_3.data);
    hipFree(pvanet.bias4_3.data);
    hipFree(pvanet.weight5_1.data);
    hipFree(pvanet.bias5_1.data);
    hipFree(pvanet.weight5_2.data);
    hipFree(pvanet.bias5_2.data);
    hipFree(pvanet.weight5_3.data);
    hipFree(pvanet.bias5_3.data);
    hipFree(pvanet.weight_up.data);
    hipFree(pvanet.weightf.data);
    hipFree(pvanet.biasf.data);

    hipFree(srpn.weight_c1.data);
    hipFree(srpn.bias_c1.data);
    hipFree(srpn.weight_c3.data);
    hipFree(srpn.bias_c3.data);
    hipFree(srpn.weight_c5.data);
    hipFree(srpn.bias_c5.data);
    hipFree(srpn.weight_s1.data);
    hipFree(srpn.bias_s1.data);
    hipFree(srpn.weight_s3.data);
    hipFree(srpn.bias_s3.data);
    hipFree(srpn.weight_s5.data);
    hipFree(srpn.bias_s5.data);
    hipFree(srpn.weight_b1.data);
    hipFree(srpn.bias_b1.data);
    hipFree(srpn.weight_b3.data);
    hipFree(srpn.bias_b3.data);
    hipFree(srpn.weight_b5.data);
    hipFree(srpn.bias_b5.data);
    
    hipFree(srpn.score1.data);
    hipFree(srpn.score3.data);
    hipFree(srpn.score5.data);
    hipFree(srpn.bbox1.data);
    hipFree(srpn.bbox3.data);
    hipFree(srpn.bbox5.data);
    hipFree(srpn.img_info.data);
    hipFree(srpn.roi.data);

    hipFree(rcnn.weight6.data);
    hipFree(rcnn.bias6.data);
    hipFree(rcnn.weight7.data);
    hipFree(rcnn.bias7.data);
    hipFree(rcnn.weight_s.data);
    hipFree(rcnn.bias_s.data);
    hipFree(rcnn.weight_b.data);
    hipFree(rcnn.bias_b.data);
  }
  #else
  {
    if (layer1_data) free(layer1_data);
    if (layer2_data) free(layer2_data);
    if (layer3_data) free(layer3_data);
    if (backup1_data) free(backup1_data);
    if (backup2_data) free(backup2_data);
    if (temp_data) free(temp_data);
    if (tempint_data) free(tempint_data);
    if (const_data) free(const_data);
    if (anchors) free(anchors);

    free(pvanet.weight1_1.data);
    free(pvanet.bias1_1.data);
    free(pvanet.weight1_2.data);
    free(pvanet.bias1_2.data);
    free(pvanet.weight2_1.data);
    free(pvanet.bias2_1.data);
    free(pvanet.weight2_2.data);
    free(pvanet.bias2_2.data);
    free(pvanet.weight3_1.data);
    free(pvanet.bias3_1.data);
    free(pvanet.weight3_2.data);
    free(pvanet.bias3_2.data);
    free(pvanet.weight3_3.data);
    free(pvanet.bias3_3.data);
    free(pvanet.weight4_1.data);
    free(pvanet.bias4_1.data);
    free(pvanet.weight4_2.data);
    free(pvanet.bias4_2.data);
    free(pvanet.weight4_3.data);
    free(pvanet.bias4_3.data);
    free(pvanet.weight5_1.data);
    free(pvanet.bias5_1.data);
    free(pvanet.weight5_2.data);
    free(pvanet.bias5_2.data);
    free(pvanet.weight5_3.data);
    free(pvanet.bias5_3.data);
    free(pvanet.weight_up.data);
    free(pvanet.weightf.data);
    free(pvanet.biasf.data);

    free(srpn.weight_c1.data);
    free(srpn.bias_c1.data);
    free(srpn.weight_c3.data);
    free(srpn.bias_c3.data);
    free(srpn.weight_c5.data);
    free(srpn.bias_c5.data);
    free(srpn.weight_s1.data);
    free(srpn.bias_s1.data);
    free(srpn.weight_s3.data);
    free(srpn.bias_s3.data);
    free(srpn.weight_s5.data);
    free(srpn.bias_s5.data);
    free(srpn.weight_b1.data);
    free(srpn.bias_b1.data);
    free(srpn.weight_b3.data);
    free(srpn.bias_b3.data);
    free(srpn.weight_b5.data);
    free(srpn.bias_b5.data);

    free(srpn.score1.data);
    free(srpn.score3.data);
    free(srpn.score5.data);
    free(srpn.bbox1.data);
    free(srpn.bbox3.data);
    free(srpn.bbox5.data);
    free(srpn.img_info.data);
    free(srpn.roi.data);

    free(rcnn.weight6.data);
    free(rcnn.bias6.data);
    free(rcnn.weight7.data);
    free(rcnn.bias7.data);
    free(rcnn.weight_s.data);
    free(rcnn.bias_s.data);
    free(rcnn.weight_b.data);
    free(rcnn.bias_b.data);
  }
  #endif

  return 0;
}
