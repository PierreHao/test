#include "hip/hip_runtime.h"
#include "core/net.h"
#include <string.h>

// --------------------------------------------------------------------------
// kernel code
//   scale_const_{gpu, cpu}
// --------------------------------------------------------------------------

// element-wise addition
//   top[i] = top[i] + bottom[i]
#ifdef GPU
__global__
static
void eltwise_add_gpu(const real bottom[], real top[],
                     const long int data_size)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    top[index] += bottom[index];
  }
}
#else
static
void eltwise_add_cpu(const real bottom[], real top[],
                     const long int data_size)
{
  for (long int index = 0; index < data_size; ++index) {
    top[index] += bottom[index];
  }
}
#endif


// --------------------------------------------------------------------------
// layer operator code
//   eltwise_sum_forward
// --------------------------------------------------------------------------

// element-wise sum: top = bottoms[0] + ... + bottoms[num_bottoms-1]
static
void eltwise_sum_forward(const Tensor* const bottoms[],
                         Tensor* const top,
                         const int num_bottoms)
{
  if (num_bottoms > 0) {
    const int data_size = get_data_size(bottoms[0]);

    #ifdef GPU
    {
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(data_size,  threads_per_block);
      hipMemcpyAsync(top->data, bottoms[0]->data, data_size * sizeof(real),
                      hipMemcpyDeviceToDevice);
      for (int m = 1; m < num_bottoms; ++m) {
        eltwise_add_gpu<<<num_blocks, threads_per_block>>>(
            bottoms[m]->data, top->data, data_size);
      }
    }
    #else
    {
      memcpy(top->data, bottoms[0]->data, data_size * sizeof(real));
      for (int m = 1; m < num_bottoms; ++m) {
        eltwise_add_cpu(bottoms[m]->data, top->data, data_size);
      }
    }
    #endif
  }
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------

static
void eltwise_shape(const Tensor* const bottoms[],
                   Tensor* const top,
                   const int num_bottoms)
{
  const Tensor* const p_bottom = (num_bottoms > 0) ? bottoms[0] : NULL;

  // top shape = bottom shape
  if (p_bottom) {
    top->ndim = p_bottom->ndim;
    top->num_items = p_bottom->num_items;
    for (int n = 0; n < p_bottom->num_items; ++n) {
      for (int i = 0; i < p_bottom->ndim; ++i) {
        top->shape[n][i] = p_bottom->shape[n][i];
      }
    }
    for (int n = 0; n < p_bottom->num_items; ++n) {
      top->start[n] = p_bottom->start[n];
    }
  }
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void forward_eltwise_sum_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;

  eltwise_sum_forward(layer->p_bottoms, get_top(layer, 0),
                      layer->num_bottoms);
}

void shape_eltwise_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;

  eltwise_shape(layer->p_bottoms, get_top(layer, 0),
                layer->num_bottoms);
}
