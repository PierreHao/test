#include "hip/hip_runtime.h"
#include "layers/operator.h"
#include <string.h>

// --------------------------------------------------------------------------
// kernel code
//   minus_{gpu, cpu}
// --------------------------------------------------------------------------

// in-place negative transform bottom -> bottom
//   bottom[i] = -bottom[i]
#ifdef GPU
__global__
static
void minus_inplace_gpu(real bottom[], const int item_size)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < item_size) {
    bottom[index] = -bottom[index];
  }
}
#else
static
void minus_inplace_cpu(real bottom[], const int item_size)
{
  for (int index = 0; index < item_size; ++index) {
    bottom[index] = -bottom[index];
  }
}
#endif



// --------------------------------------------------------------------------
// layer-wise operator code
// --------------------------------------------------------------------------

static
void crelu_forward(const Tensor* const bottom,
                   Tensor* const top,
                   const LayerOption* const option)
{
  for (int n = bottom->num_items - 1; n >= 0; --n) {
    int item_size = 1;
    for (int i = 0; i < bottom->ndim; ++i) {
      item_size *= bottom->shape[n][i];
    }

    #ifdef GPU
    {
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(item_size,  threads_per_block);
      hipMemcpyAsync(top->data + top->start[n] + item_size,
                      bottom->data + bottom->start[n],
                      item_size * sizeof(real),
                      hipMemcpyDeviceToDevice);
      hipMemcpyAsync(top->data + top->start[n],
                      bottom->data + bottom->start[n],
                      item_size * sizeof(real),
                      hipMemcpyDeviceToDevice);
      minus_inplace_gpu<<<num_blocks, threads_per_block>>>(
          top->data + top->start[n] + item_size,
          item_size);
    }
    #else
    {
      memcpy(top->data + top->start[n] + item_size,
             bottom->data + bottom->start[n],
             item_size * sizeof(real));
      memcpy(top->data + top->start[n],
             bottom->data + bottom->start[n],
             item_size * sizeof(real));
      minus_inplace_cpu(top->data + top->start[n] + item_size,
                        item_size);
    }
    #endif
  }
}



// --------------------------------------------------------------------------
// output shape calculator code
// --------------------------------------------------------------------------

static
void crelu_shape(const Tensor* const bottom,
                 Tensor* const top)
{
  top->ndim = bottom->ndim;
  top->num_items = bottom->num_items;
  for (int n = 0; n < bottom->num_items; ++n) {
    top->shape[n][0] = bottom->shape[n][0] * 2; // 2x channels
    for (int i = 1; i < bottom->ndim; ++i) {
      top->shape[n][i] = bottom->shape[n][i];
    }
  }
  for (int n = 0; n < bottom->num_items; ++n) {
    top->start[n] = bottom->start[n] * 2;
  }
}



// --------------------------------------------------------------------------
// functions for layer instance
// --------------------------------------------------------------------------

void forward_crelu_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;
  crelu_forward(get_bottom(layer, 0), get_top(layer, 0),
                &layer->option);
}

void shape_crelu_layer(void* const net_, void* const layer_)
{
  Layer* const layer = (Layer*)layer_;
  crelu_shape(get_bottom(layer, 0), get_top(layer, 0));
}

void init_crelu_layer(void* const net_, void* const layer_)
{
  return;
}

void free_crelu_layer(void* const net_, void* const layer_)
{
  return;
}
