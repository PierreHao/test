#include "hip/hip_runtime.h"
#include "layer.h"
#include "limits.h"

// --------------------------------------------------------------------------
// kernel code
//   dropout_{gpu, cpu}
//   dropout_scaled_{gpu, cpu}
//   dropout_test_{gpu, cpu}
//   dropout_inplace_{gpu, cpu}
//   dropout_scaled_inplace_{gpu, cpu}
//   dropout_test_inplace_{gpu, cpu}
// --------------------------------------------------------------------------

// dropout transform bottom -> top
//   uint_thresh = UINT_MAX * threshold
//   top[i] = bottom[i] if mask[i] > uint_thresh, otherwise 0
#ifdef GPU
__global__
void dropout_gpu(const real* const bottom,
                 const unsigned int* const mask,
                 real* const top,
                 const long int data_size,
                 const unsigned int uint_thresh)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    top[index] = (mask[index] > uint_thresh) * bottom[index];
  }
}
#else
void dropout_cpu(const real* const bottom,
                 const unsigned int* const mask,
                 real* const top,
                 const long int data_size,
                 const unsigned int uint_thresh)
{
  for (long int index = 0; index < data_size; ++index) {
    top[index] = (mask[index] > uint_thresh) * bottom[index];
  }
}
#endif

// scaled dropout transform
//   uint_thresh = UINT_MAX * threshold
//   inv_scale = 1 / (1 - threshold)
//   top[i] = inv_scale * bottom[i] if mask[i] > uint_thresh
#ifdef GPU
__global__
void dropout_scaled_gpu(const real* const bottom,
                        const unsigned int* const mask,
                        real* const top,
                        const long int data_size,
                        const unsigned int uint_thresh,
                        const real inv_scale)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    top[index] = (mask[index] > uint_thresh) * inv_scale
                 * bottom[index];
  }
}
#else
void dropout_scaled_cpu(const real* const bottom,
                        const unsigned int* const mask,
                        real* const top,
                        const long int data_size,
                        const unsigned int uint_thresh,
                        const real inv_scale)
{
  for (long int index = 0; index < data_size; ++index) {
    top[index] = (mask[index] > uint_thresh) * inv_scale
                 * bottom[index];
  }
}
#endif

// testing-time dropout transform
//   scale = 1 - threshold
//   top[i] = scale * bottom[i]
#ifdef GPU
__global__
void dropout_test_gpu(const real* const bottom,
                      real* const top,
                      const long int data_size,
                      const real scale)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    top[index] = scale * bottom[index];
  }
}
#else
void dropout_test_cpu(const real* const bottom,
                      real* const top,
                      const long int data_size,
                      const real scale)
{
  for (long int index = 0; index < data_size; ++index) {
    top[index] = scale * bottom[index];
  }
}
#endif

// in-place dropout transform
#ifdef GPU
__global__
void dropout_inplace_gpu(real* const bottom,
                         const unsigned int* const mask,
                         const long int data_size,
                         const unsigned int uint_thresh)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    bottom[index] *= (mask[index] > uint_thresh);
  }
}
#else
void dropout_inplace_cpu(real* const bottom,
                         const unsigned int* const mask,
                         const long int data_size,
                         const unsigned int uint_thresh)
{
  for (long int index = 0; index < data_size; ++index) {
    bottom[index] *= (mask[index] > uint_thresh);
  }
}
#endif

// in-place scaled dropout transform
#ifdef GPU
__global__
void dropout_scaled_inplace_gpu(real* const bottom,
                                const unsigned int* const mask,
                                const long int data_size,
                                const unsigned int uint_thresh,
                                const real inv_scale)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    bottom[index] *= (mask[index] > uint_thresh) * inv_scale;
  }
}
#else
void dropout_scaled_inplace_cpu(real* const bottom,
                                const unsigned int* const mask,
                                const long int data_size,
                                const unsigned int uint_thresh,
                                const real inv_scale)
{
  for (long int index = 0; index < data_size; ++index) {
    bottom[index] *= (mask[index] > uint_thresh) * inv_scale;
  }
}
#endif

// testing-time in-place dropout transform
#ifdef GPU
__global__
void dropout_test_inplace_gpu(real* const bottom,
                              const long int data_size,
                              const real scale)
{
  const long int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < data_size) {
    bottom[index] *= scale;
  }
}
#else
void dropout_test_inplace_cpu(real* const bottom,
                              const long int data_size,
                              const real scale)
{
  for (long int index = 0; index < data_size; ++index) {
    bottom[index] *= scale;
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   dropout_forward
//   dropout_forward_inplace
// --------------------------------------------------------------------------

// dropout transform: bottom -> top
//   if option->scaled = 1, perform scaled dropout
//   if option->test = 1, perform testing-time dropout
//   if both = 1, perform testing-time scaled dropout,
//                which is actually do nothing:  top[i] = bottom[i]
//   if both = 0, perform dropout
//   data size: total number of nodes (N * C * H * W or something)
//   mask: data_size x 1 temporary array
void dropout_forward(const Tensor* const bottom,
                     unsigned int* const mask,
                     Tensor* const top,
                     const LayerOption* const option)
{
  const long int data_size = flatten_size(bottom);

  // perform dropout transform
  #ifdef GPU
  {
    const int threads_per_block = 512;
    const int num_blocks = DIV_THEN_CEIL(data_size,  threads_per_block);
    if (option->test) {
      if (option->scaled) {
        // testing-time scaled dropout  (= no operation)
        hipMemcpyAsync(top->data, bottom->data, data_size * sizeof(real),
                        hipMemcpyDeviceToDevice);
      }
      else {
        // testing-time dropout
        dropout_test_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  top->data,  data_size,  1.0f - option->threshold);
      }
    }
    else {
      // TODO: random number generation

      unsigned int uint_thresh = (unsigned int)option->threshold * UINT_MAX;
      if (option->scaled) {
        // scaled dropout
        dropout_scaled_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  mask,  top->data,  data_size,  uint_thresh,
            1.0f / (1.0f - option->threshold));
      }
      else {
        // dropout
        dropout_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  mask,  top->data,  data_size,  uint_thresh);
      }
    }
  }
  #else
  {
    if (option->test) {
      if (option->scaled) {
        // testing-time scaled dropout  (= no operation)
        for (int i = 0; i < data_size; ++i) {
          top->data[i] = bottom->data[i];
        }
      }
      else {
        // testing-time dropout
        dropout_test_cpu(
            bottom->data,  top->data,  data_size,  1.0f - option->threshold);
      }
    }
    else {
      // TODO: random number generation

      unsigned int uint_thresh = (unsigned int)option->threshold * UINT_MAX;
      if (option->scaled) {
        // scaled dropout
        dropout_scaled_cpu(
            bottom->data,  mask,  top->data,  data_size,  uint_thresh,
            1.0f / (1.0f - option->threshold));
      }
      else {
        // dropout
        dropout_cpu(
            bottom->data,  mask,  top->data,  data_size,  uint_thresh);
      }
    }
  }
  #endif

  // set top shape (= bottom shape)
  {
    top->ndim = bottom->ndim;
    top->num_items = bottom->num_items;
    for (int n = 0; n < bottom->num_items; ++n) {
      for (int i = 0; i < bottom->ndim; ++i) {
        top->shape[n][i] = bottom->shape[n][i];
      }
    }
  }
}

// in-place dropout transform: bottom -> bottom
void dropout_forward_inplace(Tensor* const bottom,
                             unsigned int* const mask,
                             const LayerOption* const option)
{
  const long int data_size = flatten_size(bottom);

  // perform dropout transform
  #ifdef GPU
  {
    const int threads_per_block = 512;
    const int num_blocks = DIV_THEN_CEIL(data_size,  threads_per_block);
    if (option->test) {
      if (option->scaled) {
        // testing-time scaled dropout  (= no operation)
        return;
      }
      else {
        // testing-time dropout
        dropout_test_inplace_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  data_size,  option->threshold);
      }
    }
    else {
      // TODO: random number generation

      if (option->scaled) {
        // scaled dropout
        dropout_scaled_inplace_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  mask,  data_size,  option->threshold,
            1.0f / (1.0f - option->threshold));
      }
      else {
        // dropout
        dropout_inplace_gpu<<<num_blocks, threads_per_block>>>(
            bottom->data,  mask,  data_size,  option->threshold);
      }
    }
  }
  #else
  {
    if (option->test) {
      if (option->scaled) {
        // testing-time scaled dropout  (= no operation)
        return;
      }
      else {
        // testing-time dropout
        dropout_test_inplace_cpu(
            bottom->data,  data_size,  option->threshold);
      }
    }
    else {
      // TODO: random number generation

      if (option->scaled) {
        // scaled dropout
        dropout_scaled_inplace_cpu(
            bottom->data,  mask,  data_size,  option->threshold,
            1.0f / (1.0f - option->threshold));
      }
      else {
        // dropout
        dropout_inplace_cpu(
            bottom->data, mask, data_size, option->threshold);
      }
    }
  }
  #endif
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------

void dropout_shape(const Tensor* const bottom,
                   Tensor* const top)
{
  // top shape = bottom shape
  top->ndim = bottom->ndim;
  top->num_items = bottom->num_items;
  for (int n = 0; n < bottom->num_items; ++n) {
    for (int i = 0; i < bottom->ndim; ++i) {
      top->shape[n][i] = bottom->shape[n][i];
    }
  }
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void forward_dropout_layer(Net* const net, Layer* const layer)
{
  dropout_forward(layer->p_bottoms[0], (unsigned int*)net->tempint_data,
                  &layer->tops[0], &layer->option);
  print_tensor_info(layer->name, &layer->tops[0]);
}

void forward_inplace_dropout_layer(Net* const net, Layer* const layer)
{
  dropout_forward_inplace(&layer->tops[0], (unsigned int*)net->tempint_data,
                          &layer->option);
  print_tensor_info(layer->name, &layer->tops[0]);
}

void shape_dropout_layer(Net* const net, Layer* const layer)
{
  dropout_shape(layer->p_bottoms[0], &layer->tops[0]);
}



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST
#include <stdio.h>

int main(int argc, char* argv[])
{
  // variable declaration & memory allocation
  Tensor X, Y;
  real *X_data = NULL, *Y_data = NULL;
  unsigned int *mask_data = NULL;
  LayerOption option;

  // set option
  {
    option.scaled = 0;
    option.test = 1;
    option.threshold = 0.7f;
  }

  // load data
  {
    int ndim;
    int shape[g_max_ndim];
    int total_size;

    X_data = load_data("../data/temp/conv_bottom0.bin",
                       &ndim, shape, NULL);
    X.num_items = shape[0];
    X.ndim = ndim - 1;
    total_size = 0;
    for (int n = 0; n < X.num_items; ++n) {
      int size_n = 1;
      for (int i = 0; i < X.ndim; ++i) {
        X.shape[n][i] = shape[i + 1];
        size_n *= shape[i + 1];
      }
      X.start[n] = total_size;
      total_size += size_n;
    }

    dropout_shape(&X, &Y);

    Y_data = (real*)malloc(flatten_size(&Y) * sizeof(real));
  }
 
  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const long int X_size = flatten_size(&X);
    const long int Y_size = flatten_size(&Y);

    printf("gpu malloc\n");
    hipMalloc(&X.data, X_size * sizeof(real));
    hipMalloc(&mask_data, X_size * sizeof(unsigned int));
    hipMalloc(&Y.data, Y_size * sizeof(real));

    printf("memcpy: cpu -> gpu\n");
    hipMemcpyAsync(X.data, X_data, X_size * sizeof(real),
                    hipMemcpyHostToDevice);
  }
  #else
  {
    const long int X_size = flatten_size(&X);

    X.data = X_data;
    Y.data = Y_data;
    mask_data = (unsigned int*)malloc(X_size * sizeof(unsigned int));
  }
  #endif

  // do forward operation
  {
    printf("do forward (testing-time dropout)\n");
    dropout_forward(&X, mask_data, &Y, &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const long int Y_size = flatten_size(&Y);

    printf("memcpy: cpu <- gpu\n");
    hipMemcpyAsync(Y_data, Y.data, Y_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  }
  #endif

  // verify results
  {
    const long int Y_size = flatten_size(&Y);

    printf("verification\n");

    for (int i = 0; i < Y_size; ++i) {
      if (ABS(Y_data[i] / (1.0f - option.threshold) - X_data[i]) > 1e-3) {
        printf("top[%d] = %.6f, bottom[%d] = %.6f\n",
               i, Y_data[i], i, X_data[i]);
      }
    }
  }

  // memory deallocation
  {
    printf("free\n");
    free(X_data);
    free(Y_data);
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    hipFree(X.data);
    hipFree(Y.data);
    hipFree(mask_data);
  }
  #else
  {
    free(mask_data);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
