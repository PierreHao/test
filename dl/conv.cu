#include "hip/hip_runtime.h"
#include "layer.h"
#include <string.h>
#include <omp.h>

#include "boost/date_time/posix_time/posix_time.hpp"

static float a_time[8] = { 0, };
static clock_t tick0, tick1, tick00, tick01;

#ifndef GPU
void conv_k3s1p1(const real* const bottom3d,
                 const real* const weight4d,
                 real* const temp_data,
                 real* const top3d,
                 const int top_C, const int bottom_C,
                 const int H, const int W)
{
  const int H2 = DIV_THEN_CEIL(H,  2);
  const int W2 = DIV_THEN_CEIL(W,  2);
  real* const p_weight4x4 = temp_data;
  real* const p_bottom4x4 = temp_data + top_C * bottom_C * 4 * 4;
  real* const p_temp4x4 = p_bottom4x4 + bottom_C * H2 * W2 * 4 * 4;
  real d[4][4];
  real uv[16];

  tick0 = clock();
  tick01 = tick0;
  {
    const int stride = top_C * bottom_C;
    for (int k = 0; k < top_C; ++k) {
      for (int c = 0; c < bottom_C; ++c) {
        const real* const g = weight4d + (k * bottom_C + c) * 3 * 3;
        real* const u = p_weight4x4 + k * bottom_C + c;
        const real g_sum = (g[0] + g[1] + g[2] +
                            g[3] + g[4] + g[5] +
                            g[6] + g[7] + g[8]) / 4;

        u[0 * stride] = g[0];
        u[1 * stride] = (g[0] + g[1] + g[2]) / 2;
        u[2 * stride] = (g[0] - g[1] + g[2]) / 2;
        u[3 * stride] = g[2];
        u[4 * stride] = (g[0] + g[3] + g[6]) / 2;
        u[5 * stride] = g_sum;
        u[6 * stride] = g_sum - (g[1] + g[4] + g[7]) / 2;
        u[7 * stride] = (g[2] + g[5] + g[8]) / 2;
        u[8 * stride] = (g[0] - g[3] + g[6]) / 2;
        u[9 * stride] = g_sum - (g[3] + g[4] + g[5]) / 2;
        u[10 * stride] = g_sum - (g[1] + g[3] + g[5] + g[7]) / 2;
        u[11 * stride] = (g[2] - g[5] + g[8]) / 2;
        u[12 * stride] = g[6];
        u[13 * stride] = (g[6] + g[7] + g[8]) / 2;
        u[14 * stride] = (g[6] - g[7] + g[8]) / 2;
        u[15 * stride] = g[8];
      } // endfor c
    } // endfor k
  }
  tick1 = clock();
  a_time[0] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

  tick0 = clock();
  {
    const int stride = bottom_C * H2 * W2;
    for (int c = 0; c < bottom_C; ++c) {
    for (int h = 0; h < H; h += 2) {
    for (int w = 0; w < W; w += 2) {
      const real* const p_patch = bottom3d + (c * H + h - 1) * W + w - 1;
      //real* const v = p_bottom4x4 + (h / 2 * W2 + w / 2) * bottom_C + c;
      real* const v = p_bottom4x4 + (c * H2 + h / 2) * W2 + w / 2;

      for (int j = 0; j < 4; ++j) {
        for (int i = 0; i < 4; ++i) {
          const int hh = h - 1 + j;
          const int ww = w - 1 + i;
          d[j][i] = (hh >= 0 && hh < H && ww >= 0 && ww < W) ?
                    p_patch[j * W + i] : 0;
        }
      }

      v[0 * stride] = d[0][0] - d[0][2] - d[2][0] + d[2][2];
      v[1 * stride] = d[0][1] + d[0][2] - d[2][1] - d[2][2];
      v[2 * stride] = -d[0][1] + d[0][2] + d[2][1] - d[2][2];
      v[3 * stride] = d[0][1] - d[0][3] - d[2][1] + d[2][3];
      v[4 * stride] = d[1][0] - d[1][2] + d[2][0] - d[2][2];
      v[5 * stride] = d[1][1] + d[1][2] + d[2][1] + d[2][2];
      v[6 * stride] = -d[1][1] + d[1][2] - d[2][1] + d[2][2];
      v[7 * stride] = d[1][1] - d[1][3] + d[2][1] - d[2][3];
      v[8 * stride] = -d[1][0] + d[1][2] + d[2][0] - d[2][2];
      v[9 * stride] = -d[1][1] - d[1][2] + d[2][1] + d[2][2];
      v[10 * stride] = d[1][1] - d[1][2] - d[2][1] + d[2][2];
      v[11 * stride] = -d[1][1] + d[1][3] + d[2][1] - d[2][3];
      v[12 * stride] = d[1][0] - d[1][2] - d[3][0] + d[3][2];
      v[13 * stride] = d[1][1] + d[1][2] - d[3][1] - d[3][2];
      v[14 * stride] = -d[1][1] + d[1][2] + d[3][1] - d[3][2];
      v[15 * stride] = d[1][1] - d[1][3] - d[3][1] + d[3][3];
    }}} // endfor chw
  }
  tick1 = clock();
  a_time[1] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

  tick0 = clock();
  //#pragma omp parallel num_threads(2)
  {
    const int top_area = H2 * W2;
    //int i;
    //#pragma omp for nowait
    for (int i = 0; i < 16; ++i) {
      const real* const u = p_weight4x4 + i * top_C * bottom_C;
      const real* const v = p_bottom4x4 + i * bottom_C * top_area;
      real* const uv_ = p_temp4x4 + i * top_C * top_area;
      cblas_sgemm(CblasRowMajor,
                  CblasNoTrans,  CblasNoTrans,
                  top_C,  top_area,  bottom_C,
                  1,
                  u,  bottom_C,
                  v,  top_area,//bottom_C,
                  0,
                  uv_,  top_area);
    }
  }
  tick1 = clock();
  a_time[2] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

  tick0 = clock();
  {
    const int stride = top_C * H2 * W2;
    for (int k = 0; k < top_C; ++k) {
    for (int h = 0; h < H; h += 2) {
    for (int w = 0; w < W; w += 2) {
      const real* const uv_ = p_temp4x4 + k * H2 * W2 + h / 2 * W2 + w / 2;
      real* const y = top3d + (k * H + h) * W + w;

      for (int i = 0; i < 16; ++i) {
        uv[i] =  uv_[i * stride];
      }

      y[0] = uv[0] + uv[1] + uv[2] +
             uv[4] + uv[5] + uv[6] +
             uv[8] + uv[9] + uv[10];
      if (w + 1 < W) {
        y[1] = uv[1] - uv[2] - uv[3] +
               uv[5] - uv[6] - uv[7] +
               uv[9] - uv[10] - uv[11];
      }
      if (h + 1 < H) {
        y[W] = uv[4] + uv[5] + uv[6]
               - uv[8] - uv[9] - uv[10]
               - uv[12] - uv[13] - uv[14];
        if (w + 1 < W) {
          y[W + 1] = uv[5] - uv[6] - uv[7]
                     - uv[9] + uv[10] + uv[11]
                     - uv[13] + uv[14] + uv[15];
        }
      }
    }}} // endfor khw
  }
  tick1 = clock();
  a_time[3] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;
  a_time[4] = (float)(tick1 - tick01) / CLOCKS_PER_SEC;
}

void conv_str1(const real* const bottom3d,
               const real* const weight4d,
               real* const top3d,
               const int C, const int H, const int W,
               const int C5, const int H5, const int W5,
               const int pad_h, const int pad_w)
{
  clock_t tick0, tick1, tick00;

  real u_[1024][16];
  real d_[4][4];
  real v[16];
  real uv[16];

  if (H != H5 || W != W5) {
    printf("[ERROR] Size mismatch! bottom:(%d x %d) vs. top:(%d x %d)\n",
           H, W, H5, W5);
  }

  tick0 = clock();
  tick00 = clock();

  memset(top3d, 0, sizeof(real) * C5 * H5 * W5);

  tick1 = clock();
  a_time[0] += (float)(tick1 - tick0) / CLOCKS_PER_SEC;

  for (int c = 0; c < C; ++c) {
    tick0 = clock();
    {
      for (int k = 0; k < C5; ++k) {
        const real* const g = weight4d + (k * C + c) * 9;
        real* const u = u_[k];
        const real g_sum = (g[0] + g[1] + g[2] +
                            g[3] + g[4] + g[5] +
                            g[6] + g[7] + g[8]) / 4;
        u[0] = g[0];
        u[1] = (g[0] + g[1] + g[2]) / 2;
        u[2] = (g[0] - g[1] + g[2]) / 2;
        u[3] = g[2];
        u[4] = (g[0] + g[3] + g[6]) / 2;
        u[5] = g_sum;
        u[6] = g_sum - (g[1] + g[4] + g[7]) / 2;
        u[7] = (g[2] + g[5] + g[8]) / 2;
        u[8] = (g[0] - g[3] + g[6]) / 2;
        u[9] = g_sum - (g[3] + g[4] + g[5]) / 2;
        u[10] = g_sum - (g[1] + g[3] + g[5] + g[7]) / 2;
        u[11] = (g[2] - g[5] + g[8]) / 2;
        u[12] = g[6];
        u[13] = (g[6] + g[7] + g[8]) / 2;
        u[14] = (g[6] - g[7] + g[8]) / 2;
        u[15] = g[8];
      }
    }
    tick1 = clock();
    a_time[1] += (float)(tick1 - tick0) / CLOCKS_PER_SEC;

    tick0 = clock();
    {
      for (int h = 0; h < H; h += 2) {
      for (int w = 0; w < W; w += 2) {
        const real* const d
            = bottom3d + (c * H + h - pad_h) * W + w - pad_w;
        for (int j = 0; j < 4; ++j) {
          for (int i = 0; i < 4; ++i) {
            const int hh = h - pad_h + j;
            const int ww = w - pad_w + i;
            d_[j][i] = (hh >= 0 && hh < H && ww >= 0 && ww < W) ?
                       d[j * W + i] : 0;
          }
        }

        v[0] = d_[0][0] - d_[0][2] - d_[2][0] + d_[2][2];
        v[1] = d_[0][1] + d_[0][2] - d_[2][1] - d_[2][2];
        v[2] = -d_[0][1] + d_[0][2] + d_[2][1] - d_[2][2];
        v[3] = d_[0][1] - d_[0][3] - d_[2][1] + d_[2][3];
        v[4] = d_[1][0] - d_[1][2] + d_[2][0] - d_[2][2];
        v[5] = d_[1][1] + d_[1][2] + d_[2][1] + d_[2][2];
        v[6] = -d_[1][1] + d_[1][2] - d_[2][1] + d_[2][2];
        v[7] = d_[1][1] - d_[1][3] + d_[2][1] - d_[2][3];
        v[8] = -d_[1][0] + d_[1][2] + d_[2][0] - d_[2][2];
        v[9] = -d_[1][1] - d_[1][2] + d_[2][1] + d_[2][2];
        v[10] = d_[1][1] - d_[1][2] - d_[2][1] + d_[2][2];
        v[11] = -d_[1][1] + d_[1][3] + d_[2][1] - d_[2][3];
        v[12] = d_[1][0] - d_[1][2] - d_[3][0] + d_[3][2];
        v[13] = d_[1][1] + d_[1][2] - d_[3][1] - d_[3][2];
        v[14] = -d_[1][1] + d_[1][2] + d_[3][1] - d_[3][2];
        v[15] = d_[1][1] - d_[1][3] - d_[3][1] + d_[3][3];

        for (int k = 0; k < C5; ++k) {
          const real* const u = u_[k];
          for (int i = 0; i < 16; ++i) {
            uv[i] = u[i] * v[i];
          }

          real* const y = top3d + (k * H + h) * W + w;

          y[0] += uv[0] + uv[1] + uv[2] +
                  uv[4] + uv[5] + uv[6] +
                  uv[8] + uv[9] + uv[10];
          if (w + 1 < W) {
            y[1] += uv[1] - uv[2] - uv[3] +
                    uv[5] - uv[6] - uv[7] +
                    uv[9] - uv[10] - uv[11];
          }
          if (h + 1 < H) {
            y[W] += uv[4] + uv[5] + uv[6]
                    - uv[8] - uv[9] - uv[10]
                    - uv[12] - uv[13] - uv[14];
            if (w + 1 < W) {
              y[W + 1] += uv[5] - uv[6] - uv[7]
                          - uv[9] + uv[10] + uv[11]
                          - uv[13] + uv[14] + uv[15];
            }
          }
        }
      }}
    }
    tick1 = clock();
    a_time[2] += (float)(tick1 - tick0) / CLOCKS_PER_SEC;
  }

  tick1 = clock();
  a_time[4] += (float)(tick1 - tick00) / CLOCKS_PER_SEC;
}
#endif

// --------------------------------------------------------------------------
// kernel code
//   convert_bottom_{gpu, cpu}
// --------------------------------------------------------------------------

// convert bottom3d (C x H x W)
//         -> bottom5d (C x kernel_h x kernel_w x H5 x W5)
//   given (c, h5, w5),
//     bottom5d[c][kh][kw][h5][w5] = bottom3d[c][h][w]
//       h = (-pad_h + stride_h * h5) + kh,  kh = { 0, 1, ..., kernel_h - 1 }
//       w = (-pad_w + stride_w * w5) + kw,  kw = { 0, 1, ..., kernel_w - 1 }
//       if !(0 <= h < H) or !(0 <= w < W), assign 0
#ifdef GPU
__global__
void convert_bottom_gpu(const real* const bottom3d,
                        real* const bottom5d,
                        const int C, const int H, const int W,
                        const int H5, const int W5,
                        const int kernel_h, const int kernel_w,
                        const int pad_h, const int pad_w,
                        const int stride_h, const int stride_w)
{
  // thread index: (c, h5, w5) = c*H5*W5 + h5*W5 + w5
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int H5W5 = H5 * W5;
  if (index < C * H5W5) {
    // parse thread index -> (c, h5, w5)
    const int c = index / H5W5;
    const int h5 = (index / W5) % H5;
    const int w5 = index % W5; 
    // p_bottom5d initially points to bottom5d[c][kh = 0][kw = 0][h5][w5]
    real* p_bottom5d = bottom5d + index +
                       (c * H5W5) * (kernel_h * kernel_w - 1);

    // (h_start, w_start): upper-left corner of bottom3d's kernel patch
    const int h_start = h5 * stride_h - pad_h;
    const int w_start = w5 * stride_w - pad_w;
    const real* p_bottom3d = bottom3d + (c * H + h_start) * W + w_start;

    // bottom5d[c][kh][kw][h5][w5] = bottom3d[c][h][w]
    //   h = h_start + kh,  kh = {0, 1, ..., kernel_h - 1}
    //   w = w_start + kw,  kw = {0, 1, ..., kernel_w - 1}
    //   if (h, w) is in a zero-padded region, assign 0
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        const int h = h_start + kh;
        const int w = w_start + kw;
        p_bottom5d[(kh * kernel_w + kw) * H5W5] =
          (h >= 0 && h < H && w >= 0 && w < W) ? p_bottom3d[kh * W + kw] : 0;
      }
    }
  }
}
#else
void convert_bottom_cpu(const real* const bottom3d,
                        real* const bottom5d,
                        const int C, const int H, const int W,
                        const int H5, const int W5,
                        const int kernel_h, const int kernel_w,
                        const int pad_h, const int pad_w,
                        const int stride_h, const int stride_w)
{
  for (int c = 0; c < C; ++c) {
   for (int kh = 0; kh < kernel_h; ++kh) {
    for (int kw = 0; kw < kernel_w; ++kw) {
      // pointer to bottom5d[c][kh][kw][h5 = 0][w5 = 0]
      real* const p_bottom5d = bottom5d +
                    ((c * kernel_h + kh) * kernel_w + kw) * H5 * W5;
      int h = -pad_h + kh;
      int h5 = 0;

      // for h < 0 (zero-padded region): bottom5d[c][kh][kw][h5][:] = 0
      for (; h < 0; h += stride_h, ++h5) {
        for (int w5 = 0; w5 < W5; ++w5) {
          p_bottom5d[h5 * W5 + w5] = 0;
        }
      }

      // for 0 <= h < H (data region)
      for (; h < H && h5 < H5; h += stride_h, ++h5) {
        // pointer to bottom3d[c][h][w = 0]
        int w = -pad_w + kw;
        int w5 = 0;

        // for w < 0 (zero-padded region): bottom5d[c][kh][kw][h5][w5] = 0
        for (; w < 0; w += stride_w, ++w5) {
          p_bottom5d[h5 * W5 + w5] = 0;
        }

        // for 0 <= w < W (data region):
        //   bottom5d[c][kh][kw][h5][w5] = bottom3d[c][h][w]
        for (; w < W && w5 < W5; w += stride_w, ++w5) {
          p_bottom5d[h5 * W5 + w5] = bottom3d[(c * H + h) * W + w];
        }

        // for w >= W (zero-padded region): bottom5d[c][kh][kw][h5][w5] = 0
        for (; w5 < W5; ++w5) {
          p_bottom5d[h5 * W5 + w5] = 0;
        }
      }

      // for h >= H (zero-padded region): bottom5d[c][kh][kw][h5][:] = 0
      for (; h5 < H5; ++h5) {
        for (int w5 = 0; w5 < W5; ++w5) {
          p_bottom5d[h5 * W5 + w5] = 0;
        }
      }
    } // endfor kw
   } // endfor kh
  } // endfor c
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   conv_forward
// --------------------------------------------------------------------------

// convolution: bottom -> top
//   G: number of groups
//   bottom: (G * C) x H x W
//   top: (G * C') x H' x W'
//   weight: G x C' x C x kernel_h x kernel_w
//   bias: (G * C') x 1
//   temp: (G * C * kernel_h * kernel_w) x (H' * W') array
//   const: 1 x (H' * W') array,  const[i] = 1 for all i
void conv_forward(const Tensor* const bottom3d,
                  Tensor* const top3d,
                  const Tensor* const weight5d,
                  const Tensor* const bias1d,
                  real* const temp_data,
                  const real* const const_data,
                  const LayerOption* const option)
{
  tick00 = clock();

  // weight shape: G x C' x C x kernel_h x kernel_w
  const int num_groups = weight5d->shape[0][0]; // G
  const int top_C = weight5d->shape[0][1];  // C'
  const int bottom_C = weight5d->shape[0][2];  // C
  const int kernel_h = weight5d->shape[0][3];
  const int kernel_w = weight5d->shape[0][4];

  // padding size & stride size
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom3d->data;
  real* p_top_item = top3d->data;
  real* p_temp_data = temp_data;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: (G * C) x H x W
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // set top shape: (G * C') x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H = 1 + (bottom_H + 2 * pad_h - kernel_h) / stride_h;
    const int top_W = 1 + (bottom_W + 2 * pad_w - kernel_w) / stride_w;
    top3d->shape[n][0] = num_groups * top_C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;
  #ifndef GPU
    if (top_C >= 64 &&
        kernel_h == 3 && kernel_w == 3 && stride_h == 1 && stride_w == 1) {
      //conv_str1(p_bottom_item, weight5d->data, p_top_item,
      //          bottom_C, bottom_H, bottom_W, top_C, top_H, top_W,
      //          pad_h, pad_w);
      conv_k3s1p1(p_bottom_item, weight5d->data, temp_data, p_top_item,
                  top_C, bottom_C, bottom_H, bottom_W);
    }
    else {
  #endif

    a_time[0] = a_time[1] = a_time[2] = a_time[3] = a_time[4] = 0;
    tick01 = clock();

    // convert bottom shape
    //   (G * C) x H x W -> (G * C * kernel_h * kernel_w) x (H' * W')
    {
    #ifdef GPU
      // one thread computes "kernel_h * kernel_w" entries in top
      const int num_threads = num_groups * bottom_C * top_H * top_W;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);
      convert_bottom_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item,  p_temp_data,
          num_groups * bottom_C,  bottom_H,  bottom_W,
          top_H,  top_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    #else
      convert_bottom_cpu(
          p_bottom_item,  p_temp_data,
          num_groups * bottom_C,  bottom_H,  bottom_W,
          top_H,  top_W,
          kernel_h,  kernel_w,  pad_h,  pad_w,  stride_h,  stride_w);
    #endif
    }

    tick1 = clock();
    a_time[1] = (float)(tick1 - tick01) / CLOCKS_PER_SEC;
    tick0 = clock();

    // compute top[g] = dot(weight[g], bottom[g])
    //   weight[g]: C' x (C * kernel_h * kernel_w)
    //   bottom[g]: (C * kernel_h * kernel_w) x (H' * W')
    //   top[g]: C' x H' x W'
    for (int g = 0; g < num_groups; ++g) {
      const int kernel_size = bottom_C * kernel_h * kernel_w;
      const int top_area = top_H * top_W;
      const real* const p_temp_g = p_temp_data +
                          g * kernel_size * top_area;
      const real* const p_weight_g = weight5d->data +
                          g * top_C * kernel_size;
      real* const p_top_g = p_top_item + g * top_C * top_area;

      // compute Z = alpha * dot(X, Y) + beta * Z
      //   X (= weight): m x p,  Y (= bottom): p x n,  Z (= top): m x n
      //   X, Y, Z: row-major order (e.g., Z[i][j] = Z[i * n + j])
    #ifdef GPU
      // input arguments:
      //   cublas handle,
      //   do_transpose_Y (= false),  do_transpose_X (= false),
      //   n (= H' * W'),  m (= C'),  p (= C * kernel_h * kernel_w),
      //   &alpha (= 1),
      //   &Y,  number of columns in Y (= n),
      //   &X,  number of columns in X (= p),
      //   &beta (= 0),
      //   &Z,  number of columns in Z (= n)
      const real one = 1.0f, zero = 0.0f;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_N,
                  top_area,  top_C,  kernel_size,
                  &one,
                  p_temp_g,  top_area,
                  p_weight_g,  kernel_size,
                  &zero,
                  p_top_g,  top_area);
    #else
      // input arguments:
      //   is_row_major_order (= true),
      //   do_transpose_X (= false),  do_transpose_Y (= false),
      //   m (= C'),  n (= H' * W'),  p (= C * kernel_h * kernel_w),
      //   alpha (= 1),
      //   &X,  number of columns in X (= p),
      //   &Y,  number of columns in Y (= n),
      //   beta (= 0),
      //   &Z,  number of columns in Z (= n)
      cblas_sgemm(CblasRowMajor,
                  CblasNoTrans,  CblasNoTrans,
                  top_C,  top_area,  kernel_size,
                  1,
                  p_weight_g,  kernel_size,
                  p_temp_g,  top_area,
                  0,
                  p_top_g,  top_area);
    #endif
    }

    tick1 = clock();
    a_time[2] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;
    a_time[4] = (float)(tick1 - tick01) / CLOCKS_PER_SEC;
    tick0 = clock();

  #ifndef GPU
    }
  #endif

    // compute top[i][j] = top[i][j] + bias[i]
    //   top: (G * C') x (H' * W')
    //   bias: (G * C') x 1
    if (option->bias) {
      const int top_channels = num_groups * top_C;
      const int top_area = top_H * top_W;

      // the computation is equivalent to...
      //   top = top + dot(bias, constant)
      //   constant: 1 x (H' * W'),  constant[i] = 1 for all i
    #ifdef GPU
      // thus, input arguments:
      //   do_transpose_Y (= false),  do_transpose_X (= false),
      //   n = H' * W',  m = G * C',  p = 1
      //   alpha = 1,  beta = 1
      const real one = 1.0f;
      hipblasSgemm(*((hipblasHandle_t*)option->handle),
                  HIPBLAS_OP_N,  HIPBLAS_OP_N,
                  top_area,  top_channels,  1,
                  &one,
                  const_data,  top_area,
                  bias1d->data,  1,
                  &one,
                  p_top_item,  top_area);
    #else
      // input arguments:
      //   do_transpose_X (= false),  do_transpose_Y (= false),
      //   m = G * C',  n = H' * W',  p = 1
      //   alpha = 1,  beta = 1
/*
      cblas_sgemm(CblasRowMajor,
                  CblasNoTrans,  CblasNoTrans,
                  top_channels,  top_area,  1,
                  1,
                  bias1d->data,  1,
                  const_data,  top_area,
                  1,
                  p_top_item,  top_area);
*/
      cblas_sger(CblasRowMajor,
                 top_channels,  top_area,
                 1,
                 bias1d->data,  1,
                 const_data,  1,
                 p_top_item,  top_area);
    #endif
    }

    tick1 = clock();
    a_time[5] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

    // locate next item
    {
      const int bottom_size = num_groups * bottom_C * bottom_H * bottom_W;
      const int top_size = num_groups * top_C * top_H * top_W;
      //const int temp_size =
      //    num_groups * bottom_C * kernel_h * kernel_w * top_H * top_W;
      p_bottom_item += bottom_size;
      p_top_item += top_size;
      //p_temp_data += temp_size;
    }
  } // endfor batch

  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;

  tick1 = clock();
  a_time[6] = (float)(tick1 - tick00) / CLOCKS_PER_SEC;
  a_time[7] += (float)(tick1 - tick00) / CLOCKS_PER_SEC;
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------

void conv_shape(const Tensor* const bottom3d,
                Tensor* const top3d,
                Tensor* const weight5d,
                Tensor* const bias1d,
                int* const temp_size,
                int* const const_size,
                const LayerOption* const option)
{
  const int num_groups = option->num_groups; // G
  const int top_C = option->out_channels / option->num_groups;  // C'
  const int bottom_C = bottom3d->shape[0][0] / option->num_groups;  // C
  const int kernel_h = option->kernel_h;
  const int kernel_w = option->kernel_w;
  const int pad_h = option->pad_h;
  const int pad_w = option->pad_w;
  const int stride_h = option->stride_h;
  const int stride_w = option->stride_w;

  // calculate shape for each item in the batch
  int total_size = 0, total_top_area = 0, max_top_area = 0;
  for (int n = 0; n < bottom3d->num_items; ++n) {
    // bottom shape: (G * C) x H x W
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // top shape: (G * C') x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H = 1 + (bottom_H + 2 * pad_h - kernel_h) / stride_h;
    const int top_W = 1 + (bottom_W + 2 * pad_w - kernel_w) / stride_w;
    const int top_area = top_H * top_W;
    top3d->shape[n][0] = num_groups * top_C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

    // start position for n-th item in top3d->data
    top3d->start[n] = total_size;
    total_size += num_groups * top_C * top_H * top_W;

    // sum(H' * W') & max(H' * W') in the batch
    total_top_area += top_area;
    max_top_area = MAX(max_top_area,  top_area);
  }
  top3d->ndim = 3;
  top3d->num_items = bottom3d->num_items;

  // weight shape: G x C' x C x kernel_h x kernel_w
  weight5d->num_items = 1;
  weight5d->ndim = 5;
  weight5d->shape[0][0] = num_groups;
  weight5d->shape[0][1] = top_C;
  weight5d->shape[0][2] = bottom_C;
  weight5d->shape[0][3] = kernel_h;
  weight5d->shape[0][4] = kernel_w;
  weight5d->start[0] = 0;

  // bias shape: (G * C') x 1
  if (option->bias) {
    bias1d->num_items = 1;
    bias1d->ndim = 1;
    bias1d->shape[0][0] = num_groups * top_C;
    bias1d->start[0] = 0;
  }
  else if (bias1d) {
    bias1d->num_items = 0;
    bias1d->ndim = 0;
    bias1d->shape[0][0] = 0;
    bias1d->start[0] = 0;
  }

  // temporary data size: G * C * kernel_h * kernel_w * sum(H' * W')
  *temp_size = num_groups * bottom_C * kernel_h * kernel_w * max_top_area
               + num_groups * top_C * max_top_area * 4
               + num_groups * top_C * bottom_C * 4 * 4;

  // constant data size: max(H' * W')
  *const_size = max_top_area;
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void forward_conv_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  Tensor* p_bias = (layer->option.bias) ? &layer->params[1] : NULL;

  conv_forward(layer->p_bottoms[0], &layer->tops[0],
               &layer->params[0], p_bias,
               net->temp_data, net->const_data, &layer->option);

  print_tensor_info(layer->name, &layer->tops[0]);
  #ifdef DEBUG
  {
    for (int i = 0; i < 8; ++i) {
      printf("%4.2f\t", a_time[i] * 1000);
    }
    printf("\n");
  }
  #endif
}

void shape_conv_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  int temp_size, const_size;
  Tensor* p_bias = (layer->option.bias) ? &layer->params[1] : NULL;

  conv_shape(layer->p_bottoms[0], &layer->tops[0],
             &layer->params[0], p_bias,
             &temp_size, &const_size, &layer->option);

  update_net_size(net, layer, temp_size, 0, const_size);
}

#ifdef PASS
void init_conv_layer(void* const net_, void* const layer_,
                     const void* const entry_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;
  LayerOption* const option = &layer->option;

  layer->num_params = 2;
  option->num_groups = 1;
  option->pad_h = 0;
  option->pad_w = 0;
  option->stride_h = 1;
  option->stride_w = 1;
  option->bias = 1;
  #ifdef GPU
  option->handle = (void*)&net->cublas_handle;
  #endif

  option->out_channels = 0;
  option->kernel_h = 0;
  option->kernel_w = 0;

  {
    const HashEntry* const p_entry =
        find_value_from_hash_entry((HashEntry*)entry_, "convolution_param");

    if (p_entry) {
      for (int n = 0; n < p_entry->num_values; ++n) {
        HashEntry* p_child = (HashEntry*)p_entry->p_values[n];
        if (strcmp(p_child->p_name, "num_output") == 0) {
          option->out_channels = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "kernel_size") == 0) {
          option->kernel_h = atoi((char*)p_child->p_values[0]);
          option->kernel_w = option->kernel_h;
        }
        else if (strcmp(p_child->p_name, "stride") == 0) {
          option->stride_h = atoi((char*)p_child->p_values[0]);
          option->stride_w = option->stride_h;
        }
        else if (strcmp(p_child->p_name, "pad") == 0) {
          option->pad_h = atoi((char*)p_child->p_values[0]);
          option->pad_w = option->pad_h;
        }
        else if (strcmp(p_child->p_name, "kernel_h") == 0) {
          option->kernel_h = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "kernel_w") == 0) {
          option->kernel_w = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "stride_h") == 0) {
          option->stride_h = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "stride_w") == 0) {
          option->stride_w = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "pad_h") == 0) {
          option->pad_h = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "pad_w") == 0) {
          option->pad_w = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "group") == 0) {
          option->num_groups = atoi((char*)p_child->p_values[0]);
        }
        else if (strcmp(p_child->p_name, "bias_term") == 0) {  
          if (strcmp((char*)p_child->p_values[0], "false") == 0) {
            option->bias = 0;
            layer->num_params = 1;
          }
        }
      }
    }
  }

  if (!option->out_channels || !option->kernel_h || !option->kernel_w) {
    printf("[ERROR] Essential parameters are not given for Layer %s\n",
           layer->name);
  }
}
#endif



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST

int main(int argc, char* argv[])
{
  // variable declaration & memory allocation
  Tensor X, Y, W, b;
  real *X_data = NULL, *Y_data = NULL, *Y_true_data = NULL;
  real *W_data = NULL, *b_data = NULL;
  real *p_temp_data = NULL, *const_data = NULL, *p_const_data = NULL;
  LayerOption option;
  int temp_size, const_size;

  // set option
  {
    option.num_groups = 1;
    option.out_channels = 512;
    option.kernel_h = 1;
    option.kernel_w = 1;
    option.pad_h = 0;
    option.pad_w = 0;
    option.stride_h = 1;
    option.stride_w = 1;
    option.bias = 1;
  }

  // load data
  {
    int ndim;
    int shape[g_max_ndim];
    int total_size;

    X_data = load_data("../data/temp/conv_bottom0.bin",
                       &ndim, shape, NULL);
    X.num_items = shape[0];
    X.ndim = ndim - 1;
    total_size = 0;
    for (int n = 0; n < X.num_items; ++n) {
      int size_n = 1;
      for (int i = 0; i < X.ndim; ++i) {
        X.shape[n][i] = shape[i + 1];
        size_n *= shape[i + 1];
      }
      X.start[n] = total_size;
      total_size += size_n;
    }
    conv_shape(&X, &Y, &W, &b, &temp_size, &const_size, &option);

    Y_true_data = load_data("../data/temp/conv_top0.bin",
                            &ndim, shape, NULL);
    Y_data = (real*)malloc(flatten_size(&Y) * sizeof(real));

    W_data = load_data("../data/temp/conv_param0.bin",
                       &ndim, shape, NULL);

    if (option.bias) {
      b_data = load_data("../data/temp/conv_param1.bin",
                         &ndim, shape, NULL);

      const_data = (real*)malloc(const_size * sizeof(real));
      for (int i = 0; i < const_size; ++i) {
        const_data[i] = 1;
      }
    }
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
    option.handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    if (hipblasCreate((hipblasHandle_t*)option.handle)
          != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const int X_size = flatten_size(&X);
    const int Y_size = flatten_size(&Y);
    const int W_size = flatten_size(&W);
    const int b_size = flatten_size(&b);

    printf("gpu malloc\n");
    hipMalloc(&X.data, X_size * sizeof(real));
    hipMalloc(&Y.data, Y_size * sizeof(real));
    hipMalloc(&W.data, W_size * sizeof(real));
    hipMalloc(&p_temp_data, temp_size * sizeof(real));
    if (option.bias) {
      hipMalloc(&b.data, b_size * sizeof(real));
      hipMalloc(&p_const_data, const_size * sizeof(real));
    }
    else {
      b.data = NULL;
    }

    printf("memcpy: cpu -> gpu\n");
    hipMemcpyAsync(X.data, X_data, X_size * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMemcpyAsync(W.data, W_data, W_size * sizeof(real),
                    hipMemcpyHostToDevice);
    if (option.bias) {
      hipMemcpyAsync(b.data, b_data, b_size * sizeof(real),
                      hipMemcpyHostToDevice);
      hipMemcpyAsync(p_const_data, const_data, const_size * sizeof(real),
                      hipMemcpyHostToDevice);
    }
  }
  #else
  {
    X.data = X_data;
    Y.data = Y_data;
    W.data = W_data;
    p_temp_data = (real*)malloc(temp_size * sizeof(real));
    if (option.bias) {
      b.data = b_data;
      p_const_data = const_data;
    }
    else {
      b.data = NULL;
    }
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    conv_forward(&X, &Y, &W, &b, p_temp_data, p_const_data, &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const int Y_size = flatten_size(&Y);

    printf("memcpy: cpu <- gpu\n");
    hipMemcpyAsync(Y_data, Y.data, Y_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  }
  #endif

  // verify results
  {
    int i = 0;

    printf("verification\n");

    for (int n = 0; n < Y.num_items; ++n) {
      for (int c = 0; c < Y.shape[n][0]; ++c) {
        for (int h = 0; h < Y.shape[n][1]; ++h) {
          for (int w = 0; w < Y.shape[n][2]; ++w) {
            real diff = ABS(Y_data[i] - Y_true_data[i]);
            diff /= 1e-10f + MIN(ABS(Y_data[i]),  ABS(Y_true_data[i]));
          #ifdef GPU
            if (diff > 0) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
          #else
            if (diff > 1e-3f) {
              printf("Y[%d,%d,%d,%d] = %.6f  Y_true[%d,%d,%d,%d] = %.6f\n",
                     n, c, h, w, Y_data[i], n, c, h, w, Y_true_data[i]);
            }
          #endif
            ++i;
          } // endfor w
        } // endfor h
      } // endfor c
    } // endfor n
  }

  // memory deallocation
  {
    printf("free\n");
    free(X_data);
    free(Y_data);
    free(Y_true_data);
    free(W_data);
    if (option.bias) {
      free(b_data);
      free(const_data);
    }
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    hipFree(X.data);
    hipFree(Y.data);
    hipFree(W.data);
    hipFree(p_temp_data);
    if (option.bias) {
      hipFree(b.data);
      hipFree(p_const_data);
    }

    if (hipblasDestroy(*((hipblasHandle_t*)option.handle))
        != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
    free(option.handle);
  }
  #else
  {
    free(p_temp_data);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
