#include "hip/hip_runtime.h"
#include "layer.h"
#include "cuda_settings.h"

#ifdef PASS
__global__ void convert_bottom_patch(const real* bottom3d_patch,
                                     real* const bottom5d_hyperpatch,
                                     const int kernel_h, const int kernel_w,
                                     const int h_min, const int h_max,
                                     const int w_min, const int w_max,
                                     const int W, const int stride_hyperpatch)
{
  real* p_bottom5d = bottom5d_hyperpatch;
  for (int kh = blockIdx.x * blockDim.x + threadIdx.x;
       kh < kernel_h;
       kh += blockDim.x) {
    for (int kw = blockIdx.y * blockDim.y + threadIdx.y;
         kw < kernel_w;
         kw += blockDim.y) {
      const int index = kh * W + kw;
      if (kh >= h_min && kh < h_max && kw >= w_min && kw < w_max) {
        p_bottom5d[index * stride_hyperpatch] = bottom3d_patch[index];
      }
      else {
        p_bottom5d[index * stride_hyperpatch] = 0;
      }
    }
  }
}
#endif

// convert bottom3d (C x H x W)
//         -> bottom5d (C x kernel_h x kernel_w x H5 x W5)
//   given (c, h5, w5), for kh: [0, ..., kernel_h) and kw: [0, ..., kernel_w),
//     bottom5d[c][kh][kw][h5][w5] = bottom3d[c][h][w]
//       h = (-pad_h + stride_h * h5) + kh
//       w = (-pad_w + stride_w * w5) + kw
//       if !(0 <= h < H) or !(0 <= w < W), assign 0
__global__ void convert_bottom(const real* bottom3d, real* const bottom5d,
                               const int C, const int H, const int W,
                               const int H5, const int W5,
                               const int kernel_h, const int kernel_w,
                               const int pad_h, const int pad_w,
                               const int stride_h, const int stride_w)
{
  const int top_HW = H5 * W5;
  const int top_CHW = top_HW * C;

  // thread index: (c, h5, w5) = c*H5*W5 + h5*W5 + w5
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < top_CHW;
       index += blockDim.x) {
    // parse thread index -> (c, h5, w5)
    const int c = index / top_HW;
    const int h5 = (index / W5) % H5;
    const int w5 = index % W5; 
    // p_bottom5d initially points to bottom5d[c][kh = 0][kw = 0][h5][w5]
    real* p_bottom5d = bottom5d + index + (c * top_HW) * (kernel_h * kernel_w - 1);

    // (h0, w0): upper-left corner location of bottom3d's kernel patch
    const int h0 = h5 * stride_h - pad_h;
    const int w0 = w5 * stride_w - pad_w;
    const real* p_bottom3d = bottom3d + (c * H + h0) * W + w0;

#ifdef PASS
    dim3 num_threads(3, 3);
    dim3 num_blocks((kernel_h + 3 - 1)/3, (kernel_w + 3 - 1)/3);
    convert_bottom_patch<<<num_blocks, num_threads>>>(p_bottom3d, p_bottom5d,
                                                      kernel_h, kernel_w,
                                                      -h0, H-h0, -w0, W-w0,
                                                      W, top_HW);
#else
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        if (h0 + kh >= 0 && h0 + kh < H && w0 + kw >= 0 && w0 + kw < W) {
          // bottom5d[c][kh][kw][h5][w5] = bottom3d[c][h0 + kh][w0 + kw]
          p_bottom5d[(kh * kernel_w + kw) * top_HW] = p_bottom3d[kh * W + kw];
        }
        else {
          // if [h0 + kh][w0 + kw] is in a zero-padded region, assign 0
          p_bottom5d[(kh * kernel_w + kw) * top_HW] = 0;
        }
      }
    }
#endif
  }
}

void forward(const Tensor* bottom3d, Tensor* const top3d,
             const Tensor* weight4d, const Tensor* bias1d,
             real* const temp_data, const real* const_data,
             const ConvOption* options)
{
  // weight shape: C' x C x kernel_h x kernel_w
  const int top_C = weight4d->shape[0][0];  // C'
  const int bottom_C = weight4d->shape[0][1];  // C
  const int kernel_h = weight4d->shape[0][2];
  const int kernel_w = weight4d->shape[0][3];

  // padding size & stride size
  const int pad_h = options->pad_h;
  const int pad_w = options->pad_w;
  const int stride_h = options->stride_h;
  const int stride_w = options->stride_w;

  // do forward-pass for each item in the batch
  const real* p_bottom_data = bottom3d->data;
  real* p_top_data = top3d->data;
  const int num_items = bottom3d->num_items;
  for (int n = 0; n < num_items; ++n) {
    // bottom shape: C x H x W
    const int bottom_H = bottom3d->shape[n][1];  // H
    const int bottom_W = bottom3d->shape[n][2];  // W

    // set top shape: C' x H' x W'
    //   H' = 1 + (H + 2*pad_h - kernel_h) / stride_h
    //   W' = 1 + (W + 2*pad_w - kernel_w) / stride_w
    const int top_H = 1 + (bottom_H + 2 * pad_h - kernel_h) / stride_h;
    const int top_W = 1 + (bottom_W + 2 * pad_w - kernel_w) / stride_w;
    top3d->ndim = 3;
    top3d->num_items = num_items;
    top3d->shape[n][0] = top_C;
    top3d->shape[n][1] = top_H;
    top3d->shape[n][2] = top_W;

   { // convert bottom shape: C x H x W -> (C * kernel_h * kernel_w) x (H' * W')
    const int num_threads = 1024;
    const int num_blocks = (num_threads - 1 + bottom_C * top_H * top_W) / num_threads;
    convert_bottom<<<num_blocks, num_threads>>>(p_bottom_data, temp_data,
                                                bottom_C, bottom_H, bottom_W,
                                                top_H, top_W,
                                                kernel_h, kernel_w,
                                                pad_h, pad_w,
                                                stride_h, stride_w);
   } // end convert bottom shape

   { // do matrix computation
    const hipblasHandle_t* cublas_handle = (hipblasHandle_t*)options->handle;
    const real one = 1.0, zero = 0.0;

    // top = dot(weight, bottom)
    //   weight: C' x (C * kernel_h * kernel_w)
    //   bottom: (C * kernel_h * kernel_w) x (H' * W')
    //   top: C' x H' x W'
    hipblasSgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                top_H * top_W, top_C, bottom_C * kernel_h * kernel_w,
                &one, temp_data, top_H * top_W,
                weight4d->data, bottom_C * kernel_h * kernel_w,
                &zero, p_top_data, top_H * top_W);

    // top = top + bias
    hipblasSgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                top_H * top_W, top_C, 1,
                &one, const_data, top_H * top_W,
                bias1d->data, 1,
                &one, p_top_data, top_H * top_W);
   } // end matrix computation

    // locate next data
    p_bottom_data += bottom_C * bottom_H * bottom_W;
    p_top_data += top_C * top_H * top_W;
  } // endfor batch
}

void backward(Tensor *top_grad, Tensor *bottom_grad, Tensor *top_layer, Tensor *bottom_layer, ConvOption *options)
{
  return;
}

#define DATA_SIZE 30000
#define WEIGHT_SIZE 100000
#define BIAS_SIZE 200

int main(int argc, char **argv)
{
  Tensor X, Y, W, b;
  real X_data[DATA_SIZE], Y_data[DATA_SIZE], W_data[WEIGHT_SIZE], b_data[BIAS_SIZE], temp_data[DATA_SIZE];
  ConvOption option;
  real* p_temp_data;
  real* p_const_data;
  hipblasHandle_t cublas_handle;
 {
  X.ndim = 3; X.num_items = 10;
  for (int i = 0; i < X.num_items; ++i) {
    X.shape[i][0] = 100;
    X.shape[i][1] = 5;
    X.shape[i][2] = 5;
  }
  W.ndim = 4; W.num_items = 1; W.shape[0][0] = 100; W.shape[0][1] = 100; W.shape[0][2] = 3; W.shape[0][3] = 3;
  b.ndim = 1; b.num_items = 1; b.shape[0][0] = 100;
  X.data = &X_data[0];
  Y.data = &Y_data[0];
  W.data = &W_data[0];
  b.data = &b_data[0];
  option.kernel_h = 3;
  option.kernel_w = 3;
  option.pad_h = 1;
  option.pad_w = 1;
  option.stride_h = 1;
  option.stride_w = 1;
 }
 {
  printf("set device\n");
  CUDA_CHECK(hipSetDevice(1));
  //printf("get device\n");
  //CUDA_CHECK(hipGetDevice(0));
  printf("cublas initialization\n");
  if (hipblasCreate(&cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
    printf("cublas creation failed\n");
  }
  option.handle = &cublas_handle;
 }
 {
  printf("cuda malloc\n");
  CUDA_CHECK(hipMalloc(&X.data, DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&Y.data, DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&W.data, WEIGHT_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&b.data, BIAS_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&p_temp_data, DATA_SIZE*sizeof(real)));
  CUDA_CHECK(hipMalloc(&p_const_data, DATA_SIZE*sizeof(real)));
 }
 {
  FILE* fp;
  int X_size = flatten_size(&X);
  int W_size = flatten_size(&W);
  int b_size = flatten_size(&b);
  printf("data loading\n");
  fp = fopen("X.txt", "r");
  for (int i = 0; i < X_size; ++i)
    fscanf(fp, "%f", &X_data[i]);
  fclose(fp);
  fp = fopen("W.txt", "r");
  for (int i = 0; i < W_size; ++i)
    fscanf(fp, "%f", &W_data[i]);
  fclose(fp);
  fp = fopen("b.txt", "r");
  for (int i = 0; i < b_size; ++i)
    fscanf(fp, "%f", &b_data[i]);
  fclose(fp);
  for (int i = 0; i < DATA_SIZE; ++i) {
    temp_data[i] = 1;
  }
 }
 {
  printf("memcopy\n");
  CUDA_CHECK(hipMemcpy(X.data, X_data, DATA_SIZE*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(W.data, W_data, WEIGHT_SIZE*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(b.data, b_data, BIAS_SIZE*sizeof(real), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(p_const_data, temp_data, DATA_SIZE*sizeof(real), hipMemcpyHostToDevice));
 }
 {
  real* p_Y_data = &Y_data[0];

  printf("do forward\n");
  for (int i = 0; i < 100; ++i) {
    forward(&X, &Y, &W, &b, p_temp_data, p_const_data, &option);
    forward(&Y, &X, &W, &b, p_temp_data, p_const_data, &option);
  }

  printf("memcpy\n");
  CUDA_CHECK(hipMemcpy(Y_data, X.data, DATA_SIZE*sizeof(real), hipMemcpyDeviceToHost));

  for (int n = 0; n < Y.num_items; ++n) {
    printf("Y[%d] (%d x %d x %d)\n", n, Y.shape[n][0], Y.shape[n][1], Y.shape[n][2]);
    for (int c = 0; c < Y.shape[n][0]; ++c) {
      for (int h = 0; h < Y.shape[n][1]; ++h) {
        for (int w = 0; w < Y.shape[n][2]; ++w) {
          printf("%03.5f ", p_Y_data[(c * Y.shape[n][1] + h) * Y.shape[n][2] + w]);
        }
        printf("\n");
      }
      printf("\n\n");
    }
    p_Y_data += Y.shape[n][0] * Y.shape[n][1] * Y.shape[n][2];
    printf("\n\n===============================\n\n");
  }
 }
 {
  printf("cuda free\n");
  CUDA_CHECK(hipFree(X.data));
  CUDA_CHECK(hipFree(Y.data));
  CUDA_CHECK(hipFree(W.data));
  CUDA_CHECK(hipFree(b.data));
  CUDA_CHECK(hipFree(p_temp_data));
  CUDA_CHECK(hipFree(p_const_data));
  printf("cublas finalization\n");
  if (hipblasDestroy(cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
    printf("cublas destruction failed\n");
  }
 }
  return 0;
}
