#include "hip/hip_runtime.h"
#include "layer.h"
#include <stdlib.h>
#include <math.h>

#ifdef GPU
#include "cuda_settings.h"
#endif

// data structure for a bounding-box in a given image
//   (x1, y1): upper-left corner location of the box in the image
//   (x2, y2): lower-right corner location of the box in the image
//   score: objectness score of the region bounded by the box
typedef struct BoundingBox_
{
  real x1, y1, x2, y2;
  real score;
} BoundingBox;


/*
 * functions for NMS operation:
 *   iou: compute overlap between two boxes
 *   nms_mask: given a set of boxes, compute overlap between all box pairs
 *   nms: given a set of boxes, discard significantly-overlapped boxes
 */

// "IoU = intersection area / union area" of two boxes A, B
//   A, B: 4-dim array (x1, y1, x2, y2)
#ifdef GPU
__device__
#endif
inline real iou(const real* const A, const real* const B)
{
  // overlapped region (= box)
  const real x1 = MAX(A[0], B[0]);
  const real y1 = MAX(A[1], B[1]);
  const real x2 = MIN(A[2], B[2]);
  const real y2 = MIN(A[3], B[3]);

  // intersection area
  const real width = MAX(0.0f,  x2 - x1 + 1.0f);
  const real height = MAX(0.0f,  y2 - y1 + 1.0f);
  const real area = width * height;

  // area of A, B
  const real A_area = (A[2] - A[0] + 1.0f) * (A[3] - A[1] + 1.0f);
  const real B_area = (B[2] - B[0] + 1.0f) * (B[3] - B[1] + 1.0f);

  // IoU
  return area / (A_area + B_area - area);
}

// given box proposals, compute overlap between all box pairs
// (overlap = intersection area / union area)
// and then set mask-bit to 1 if a pair is significantly overlapped
//   num_boxes: number of box proposals given
//   boxes: "num_boxes x 5" array (x1, y1, x2, y2, score)
//   nms_thresh: threshold for determining "significant overlap"
//               if "intersection area / union area > nms_thresh",
//               two boxes are thought of as significantly overlapped
// the all-pair computation (num_boxes x num_boxes) is done by
// divide-and-conquer:
//   each GPU block (bj, bi) computes for "64 x 64" box pairs (j, i),
//     j = bj * 64 + { 0, 1, ..., 63 }
//     i = bi * 64 + { 0, 1, ..., 63 },
//   and each "1 x 64" results is saved into a 64-bit mask
//     mask: "num_boxes x num_blocks" array
//     for mask[j][bi], "di-th bit = 1" means:
//       box j is significantly overlapped with box i,
//       where i = bi * 64 + di
typedef unsigned long long uint64;
#define NMS_BLOCK_SIZE 64
#ifdef GPU
__global__
void nms_mask_gpu(const real* const boxes,
                  uint64* const mask,
                  const int num_boxes, const real nms_thresh)
{
  // block region
  //   j = j_start + { 0, ..., dj_end - 1 }
  //   i = i_start + { 0, ..., di_end - 1 }
  const int i_start = blockIdx.x * NMS_BLOCK_SIZE;
  const int di_end = MIN(num_boxes - i_start,  NMS_BLOCK_SIZE);
  const int j_start = blockIdx.y * NMS_BLOCK_SIZE;
  const int dj_end = MIN(num_boxes - j_start,  NMS_BLOCK_SIZE);

  // copy all i-th boxes to GPU cache
  //   i = i_start + { 0, ..., di_end - 1 }
  __shared__ real boxes_i[NMS_BLOCK_SIZE * 5];
  {
    const int di = threadIdx.x;
    if (di < di_end) {
      boxes_i[di * 5 + 0] = boxes[(i_start + di) * 5 + 0];
      boxes_i[di * 5 + 1] = boxes[(i_start + di) * 5 + 1];
      boxes_i[di * 5 + 2] = boxes[(i_start + di) * 5 + 2];
      boxes_i[di * 5 + 3] = boxes[(i_start + di) * 5 + 3];
      boxes_i[di * 5 + 4] = boxes[(i_start + di) * 5 + 4];
    }
  }
  __syncthreads();

  // given j = j_start + dj,
  //   check whether box i is significantly overlapped with box j
  //   (i.e., IoU(box j, box i) > threshold)
  //   for all i = i_start + { 0, ..., di_end - 1 } except for i == j
  {
    const int dj = threadIdx.x;
    if (dj < dj_end) {
      // box j
      const real* const box_j = boxes + (j_start + dj) * 5;

      // mask for significant overlap
      //   if IoU(box j, box i) > threshold,  di-th bit = 1
      uint64 mask_j = 0;

      // check for all i = i_start + { 0, ..., di_end - 1 }
      // except for i == j
      const int di_start = (i_start == j_start) ? (dj + 1) : 0;
      for (int di = di_start; di < di_end; ++di) {
        // box i
        const real* const box_i = boxes_i + di * 5;

        // if IoU(box j, box i) > threshold,  di-th bit = 1
        if (iou(box_j, box_i) > nms_thresh) {
          mask_j |= 1ULL << di;
        }
      }

      // mask: "num_boxes x num_blocks" array
      //   for mask[j][bi], "di-th bit = 1" means:
      //     box j is significantly overlapped with box i = i_start + di,
      //     where i_start = bi * block_size
      {
        const int num_blocks = DIV_THEN_CEIL(num_boxes, NMS_BLOCK_SIZE);
        const int bi = blockIdx.x;
        mask[(j_start + dj) * num_blocks + bi] = mask_j;
      }
    } // endif dj < dj_end
  }
}
#else
void nms_mask_cpu(const real* const boxes,
                  uint64* const mask,
                  const int num_boxes, const real nms_thresh)
{
  // number of blocks along each dimension
  const int num_blocks = DIV_THEN_CEIL(num_boxes, NMS_BLOCK_SIZE);

  // the whole 2-dim computations "num_boxes x num_boxes" is done by
  // sweeping all "64 x 64"-sized blocks
  for (int j_start = 0; j_start < num_boxes; j_start += NMS_BLOCK_SIZE) {
    for (int i_start = 0; i_start < num_boxes; i_start += NMS_BLOCK_SIZE) {
      // block region
      //   j = j_start + { 0, ..., dj_end - 1 }
      //   i = i_start + { 0, ..., di_end - 1 }
      const int di_end = MIN(num_boxes - i_start,  NMS_BLOCK_SIZE);
      const int dj_end = MIN(num_boxes - j_start,  NMS_BLOCK_SIZE);

      // check whether box i is significantly overlapped with box j
      // for all j = j_start + { 0, ..., dj_end - 1 },
      //         i = i_start + { 0, ..., di_end - 1 },
      // except for i == j
      for (int dj = 0; dj < dj_end; ++dj) {
        // box j & overlap mask
        const real* const box_j = boxes + (j_start + dj) * 5;
        uint64 mask_j = 0;

        // check for all i = i_start + { 0, ..., di_end - 1 }
        // except for i == j
        const int di_start = (i_start == j_start) ? (dj + 1) : 0;
        for (int di = di_start; di < di_end; ++di) {
          // box i
          const real* const box_i = boxes + (i_start + di) * 5;

          // if IoU(box j, box i) > threshold,  di-th bit = 1
          if (iou(box_j, box_i) > nms_thresh) {
            mask_j |= 1ULL << di;
          }
        }

        // mask: "num_boxes x num_blocks" array
        //   for mask[j][bi], "di-th bit = 1" means:
        //     box j is significantly overlapped with box i = i_start + di,
        //     where i_start = bi * block_size
        {
          const int bi = i_start / NMS_BLOCK_SIZE;
          mask[(j_start + dj) * num_blocks + bi] = mask_j;
        }
      } // endfor dj
    } // endfor j_start
  } // endfor i_start
}
#endif

// given box proposals (sorted in descending order of their scores),
// discard a box if it is significantly overlapped with
// one or more previous (= scored higher) boxes
//   num_boxes: number of box proposals given
//   boxes: "num_boxes x 5" array (x1, y1, x2, y2, score)
//          sorted in descending order of scores
//   num_out: number of remaining boxes
//   keep_out: "num_out x 1" array
//             indices of remaining boxes
//   nms_thresh: threshold for determining "significant overlap"
//               if "intersection area / union area > nms_thresh",
//               two boxes are thought of as significantly overlapped
void nms(const int num_boxes, const real* const boxes,
         int* const num_out, int* const keep_out,
         const real nms_thresh)
{
  const int num_blocks = DIV_THEN_CEIL(num_boxes, NMS_BLOCK_SIZE);
  uint64* const mask
      = (uint64*)malloc(num_boxes * num_blocks * sizeof(uint64));

  #ifdef GPU
  {
    uint64* mask_dev;
    real* boxes_dev;
    const dim3 blocks(num_blocks, num_blocks);

    // GPU memory allocation & copy box data
    CUDA_CHECK(hipMalloc(&boxes_dev, num_boxes * 5 * sizeof(real)));
    CUDA_CHECK(hipMemcpy(boxes_dev, boxes, num_boxes * 5 * sizeof(real),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&mask_dev,
                          num_boxes * num_blocks * sizeof(uint64)));

    // find all significantly-overlapped pairs of boxes
    nms_mask_gpu<<<blocks, NMS_BLOCK_SIZE>>>(
        boxes_dev, mask_dev, num_boxes, nms_thresh);

    // copy mask data to main memory
    CUDA_CHECK(hipMemcpy(mask, mask_dev,
                          sizeof(uint64) * num_boxes * num_blocks,
                          hipMemcpyDeviceToHost));

    // GPU memory deallocation
    CUDA_CHECK(hipFree(boxes_dev));
    CUDA_CHECK(hipFree(mask_dev));
  }
  #else
  {
    // find all significantly-overlapped pairs of boxes
    nms_mask_cpu(boxes, mask, num_boxes, nms_thresh);
  }
  #endif

  // discard i-th box if it is significantly overlapped with
  // one or more previous (= scored higher) boxes
  {
    int num_to_keep = 0;
    uint64* const remv = (uint64*)calloc(num_blocks, sizeof(uint64));

    for (int i = 0; i < num_boxes; ++i) {
      const int nblock = i / NMS_BLOCK_SIZE;
      const int inblock = i % NMS_BLOCK_SIZE;

      if (!(remv[nblock] & (1ULL << inblock))) {
        keep_out[num_to_keep++] = i;
        uint64* p = mask + i * num_blocks;
        for (int j = nblock; j < num_blocks; ++j) {
          remv[j] |= p[j];
        }
      }
    }
    *num_out = num_to_keep;

    free(remv);
  }

  free(mask);
}


/*
 * functions for box data structure
 *   transform_box: transform a box according to a given gradient
 *   generate_anchors: generate anchor boxes of varying sizes and ratios
 *   sort_box: sort a list of boxes in descending order of their scores
 */

// transform a box according to a given gradient
//   box: (x1, y1, x2, y2)
//   gradient: dx, dy, d(log w), d(log h)
int transform_box(BoundingBox* const box,
                  const real dx, const real dy,
                  const real d_log_w, const real d_log_h,
                  const real im_w, const real im_h,
                  const real min_w, const real min_h)
{
  // width & height of box
  const real w = box->x2 - box->x1 + 1.0f;
  const real h = box->y2 - box->y1 + 1.0f;
  // center location of box
  const real ctr_x = box->x1 + 0.5f * w;
  const real ctr_y = box->y1 + 0.5f * h;

  // new center location according to gradient (dx, dy)
  const real pred_ctr_x = dx * w + ctr_x;
  const real pred_ctr_y = dy * h + ctr_y;
  // new width & height according to gradient d(log w), d(log h)
  const real pred_w = exp(d_log_w) * w;
  const real pred_h = exp(d_log_h) * h;

  // update upper-left corner location
  box->x1 = pred_ctr_x - 0.5f * pred_w;
  box->y1 = pred_ctr_y - 0.5f * pred_h;
  // update lower-right corner location
  box->x2 = pred_ctr_x + 0.5f * pred_w;
  box->y2 = pred_ctr_y + 0.5f * pred_h;

  // adjust new corner locations to be within the image region,
  box->x1 = MAX(0.0f,  MIN(box->x1,  im_w - 1.0f));
  box->y1 = MAX(0.0f,  MIN(box->y1,  im_h - 1.0f));
  box->x2 = MAX(0.0f,  MIN(box->x2,  im_w - 1.0f));
  box->y2 = MAX(0.0f,  MIN(box->y2,  im_h - 1.0f));

  // recompute new width & height
  const real box_w = box->x2 - box->x1 + 1.0f;
  const real box_h = box->y2 - box->y1 + 1.0f;

  // check if new box's size >= threshold
  if (box_w >= min_w && box_h >= min_h) return 1;
  return 0;
}

// given a base box, enumerate transformed boxes of varying sizes and ratios
//   option->base_size: base box's width & height (i.e., base box is square)
//   option->scales: "option->num_scales x 1" array
//                   varying scale factor for base box
//   option->ratios: "option->num_ratios x 1" array
//                   varying height-width ratio
//   option->num_concats: repeat count of anchor set generation
//                        (required for separated RPN)
//   anchors: "num_boxes x 4" array,  (x1, y1, x2, y2) for each box
//     num_boxes = total number of transformations
//         = option->num_scales * option->num_ratios * option->num_concats
#define MAX_NUM_RATIO_SCALE 10
void generate_anchors(real* const anchors,
                      const ProposalOption* const option)
{
  // base box's width & height & center location
  const real base_area = option->base_size * option->base_size;
  const real ctr = 0.5f * (option->base_size - 1.0f);

  // transformed width & height for given ratios
  real wr[MAX_NUM_RATIO_SCALE];
  real hr[MAX_NUM_RATIO_SCALE];
  for (int i = 0; i < option->num_ratios; ++i) {
    wr[i] = ROUND(sqrt(base_area / option->ratios[i]));
    hr[i] = ROUND(wr[i] * option->ratios[i]);
  }

  // enumerate all transformed boxes
  {
    real* p_anchors = anchors;
    for (int c = 0; c < option->num_concats; ++c) {
      for (int i = 0; i < option->num_ratios; ++i) {
        for (int j = 0; j < option->num_scales; ++j) {
          // transformed width & height for given ratios & scales
          const real ws = 0.5f * (wr[i] * option->scales[j] - 1.0f);
          const real hs = 0.5f * (hr[i] * option->scales[j] - 1.0f);
          // (x1, y1, x2, y2) for transformed box
          p_anchors[0] = ctr - ws;
          p_anchors[1] = ctr - hs;
          p_anchors[2] = ctr + ws;
          p_anchors[3] = ctr + hs;
          p_anchors += 4;
        } // endfor j
      } // endfor i
    } // endfor c
  }
}

// quick-sort a list of boxes in descending order of their scores
//   if num_top <= end,  only top-k results are guaranteed to be sorted
//   (for efficient computation)
void sort_box(BoundingBox* const list, const int start, const int end,
              const int num_top)
{
  const real pivot_score = list[start].score;
  int left = start + 1, right = end;
  BoundingBox temp;
  while (left <= right) {
    while (left <= end && list[left].score >= pivot_score) ++left;
    while (right > start && list[right].score <= pivot_score) --right;
    if (left <= right) {
      temp = list[left];
      list[left] = list[right];
      list[right] = temp;
      ++left;
      --right;
    }
  }
  if (right > start) {
    temp = list[right];
    list[right] = list[start];
    list[start] = temp;
  }
  if (start < right - 1) {
    sort_box(list, start, right - 1, num_top);
  }
  if (right + 1 < num_top && right + 1 < end) {
    sort_box(list, right + 1, end, num_top);
  }
}


/*
 * finally, proposal operator
 */

// proposal: bottom -> top
//   bottom: 2 x num_anchors x H x W tensor
//     bottom[0, n, h, w] = foreground score of anchor n at node (h, w)
//     bottom[1, n, h, w] = background score of anchor n at node (h, w)
//   pred_box: num_anchors x 4 x H x W tensor
//     pred_box[n, :, h, w] = predicted box (d x1, d y1, d log w, d log h)
//                            of anchor n at pixel (h, w)
//   img_info: 4 x 1 tensor,  (w, h, min_w, min_h) of raw image
//     min_w: minimum box width in raw image
//     min_h: minimum box height in raw image
//   top: num_RoIs x 4 tensor,  (x1, y1, x2, y2) of each RoI
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) of each anchor
#define MAX_DATA_WIDTH 80
#define MAX_DATA_HEIGHT 80
#define MAX_NUM_PROPOSAL 6000
void proposal_forward(const Tensor* const bottom4d,
                      const Tensor* const pred_box4d,
                      const Tensor* const img_info1d,
                      Tensor* const top2d,
                      const real* const anchors,
                      const ProposalOption* const option)
{
  BoundingBox* const proposals
      = (BoundingBox*)malloc(MAX_NUM_RATIO_SCALE * MAX_NUM_RATIO_SCALE *
                             MAX_DATA_WIDTH * MAX_DATA_HEIGHT *
                             sizeof(BoundingBox));
  real* const sorted_dets
      = (real*)malloc(MAX_NUM_PROPOSAL * 5 * sizeof(real));
  int* const keep = (int*)malloc(MAX_NUM_PROPOSAL * sizeof(int));

  // bottom4d: N x 2 x num_anchors x H x W
  // pred_box4d: N x num_anchors x 4 x H x W
  // img_info1d: N x 4
  // top2d: N x num_rois x 4
  const real* p_bottom_item = bottom4d->data;
  const real* p_pred_box_item = pred_box4d->data;
  const real* p_img_info = img_info1d->data;
  real* p_top_item = top2d->data;
  const int num_anchors
      = option->num_concats * option->num_ratios * option->num_scales;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    const real im_w = p_img_info[1];
    const real im_h = p_img_info[0];
    const real min_w = option->min_size * p_img_info[2];
    const real min_h = option->min_size * p_img_info[3];

    // enumerate all proposals
    // TODO: GPU code
    int num_proposals = 0;
    for (int h = 0; h < bottom_H; ++h) {
      for (int w = 0; w < bottom_W; ++w) {
        const real x = w * option->feat_stride;
        const real y = h * option->feat_stride;
        const real* p_box = p_pred_box_item + h * bottom_W + w;
        const real* p_score
            = p_bottom_item + num_anchors * bottom_area + h * bottom_W + w;
        for (int k = 0; k < num_anchors; ++k) {
          const real dx = p_box[(k * 4 + 0) * bottom_area];
          const real dy = p_box[(k * 4 + 1) * bottom_area];
          const real dw = p_box[(k * 4 + 2) * bottom_area];
          const real dh = p_box[(k * 4 + 3) * bottom_area];
          proposals[num_proposals].x1 = x + anchors[k * 4 + 0];
          proposals[num_proposals].y1 = y + anchors[k * 4 + 1];
          proposals[num_proposals].x2 = x + anchors[k * 4 + 2];
          proposals[num_proposals].y2 = y + anchors[k * 4 + 3];
          proposals[num_proposals].score = p_score[k * bottom_area];
          {
            const int box_created = transform_box(&proposals[num_proposals],
                                                  dx, dy, dw, dh,
                                                  im_w, im_h, min_w, min_h);
            if (box_created) ++num_proposals;
          }
        } // endfor k
      } // endfor w
    } // endfor h

    // choose candidates according to scores
    // TODO: copy proposals to GPU memory directly
    sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
    if (num_proposals > option->pre_nms_topn) {
      num_proposals = option->pre_nms_topn;
    }
    for (int i = 0; i < num_proposals; ++i) {
      sorted_dets[i * 5 + 0] = proposals[i].x1;
      sorted_dets[i * 5 + 1] = proposals[i].y1;
      sorted_dets[i * 5 + 2] = proposals[i].x2;
      sorted_dets[i * 5 + 3] = proposals[i].y2;
      sorted_dets[i * 5 + 4] = proposals[i].score;
    }

    // NMS & RoI retrieval
    {
      int num_rois = 0;
      nms(num_proposals, sorted_dets, &num_rois, keep, option->nms_thresh);

      if (num_rois > option->post_nms_topn) {
        num_rois = option->post_nms_topn;
      }
      top2d->shape[n][0] = num_rois;
      top2d->shape[n][1] = 4;
      for (int i = 0; i < num_rois; ++i) {
        p_top_item[i * 4 + 0] = proposals[keep[i]].x1;
        p_top_item[i * 4 + 1] = proposals[keep[i]].y1;
        p_top_item[i * 4 + 2] = proposals[keep[i]].x2;
        p_top_item[i * 4 + 3] = proposals[keep[i]].y2;
      }
    }

    // locate next item
    p_top_item += 4 * top2d->shape[n][0];
    p_bottom_item += 2 * num_anchors * bottom_area;
    p_pred_box_item += 4 * num_anchors * bottom_area;
    p_img_info += 4;
  } // endfor num_items

  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;

  free(proposals);
  free(sorted_dets);
  free(keep);
}

// test code
#ifdef TEST
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char *argv[])
{
  // variable declaration & memory allocation
  Tensor score, bbox, im_info, roi;
  real score_data[150*36*46], bbox_data[300*36*46], im_info_data[4];
  real roi_data[300*4], roi_true_data[300*5];
  int num_rois_true;
  real anchors[3*5*5*4];
  real scales[5] = {3, 6, 9, 16, 32};
  real ratios[5] = {0.5, 0.666, 1.0, 1.5, 2.0};
  int num_anchors;
  ProposalOption option;

  // set option
  {
    option.num_concats = 3;
    option.base_size = 16;
    option.feat_stride = 16;
    option.pre_nms_topn = 6000;
    option.post_nms_topn = 300;
    option.nms_thresh = 0.7;
    option.min_size = 16;
    option.scales = &scales[0];
    option.ratios = &ratios[0];
    option.num_scales = 5;
    option.num_ratios = 5;
  }

  // generate anchors
  {
    generate_anchors(anchors, &option);
    num_anchors = option.num_concats * option.num_scales * option.num_ratios;
  }

  // set data shapes
  {
    score.ndim = 4; score.num_items = 1; score.data = &score_data[0];
    for (int i = 0; i < score.num_items; ++i) {
      score.shape[i][0] = 2;
      score.shape[i][1] = num_anchors;
      score.shape[i][2] = 36;
      score.shape[i][3] = 46;
    }

    bbox.ndim = 4; bbox.num_items = score.num_items; bbox.data = &bbox_data[0];
    for (int i = 0; i < bbox.num_items; ++i) {
      bbox.shape[i][0] = num_anchors;
      bbox.shape[i][1] = 4;
      bbox.shape[i][2] = score.shape[i][2];
      bbox.shape[i][3] = score.shape[i][3];
    }

    im_info.ndim = 1; im_info.num_items = score.num_items; im_info.data = &im_info_data[0];
    for (int i = 0; i < im_info.num_items; ++i) {
      im_info.shape[i][0] = 4;
    }

    roi.ndim = 2; roi.num_items = score.num_items; roi.data = &roi_data[0];
    for (int i = 0; i < roi.num_items; ++i) {
      roi.shape[i][0] = option.post_nms_topn;
      roi.shape[i][1] = 4;
    }
  }

  // load data
  {
    FILE* fp;
    const int score_size = flatten_size(&score);
    const int bbox_size = flatten_size(&bbox);
    const int im_info_size = flatten_size(&im_info);

    printf("data loading\n");

    fp = fopen("../data/temp/proposal_bottom0.bin", "rb");
    if ((int)fread(score_data, sizeof(real), score_size, fp) != score_size) {
      printf("Error occurred while reading proposal_bottom0\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/proposal_bottom1.bin", "rb");
    if ((int)fread(bbox_data, sizeof(real), bbox_size, fp) != bbox_size) {
      printf("Error occurred while reading proposal_bottom1\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/proposal_bottom2.bin", "rb");
    if ((int)fread(im_info_data, sizeof(real), im_info_size, fp)
        != im_info_size) {
      printf("Error occurred while reading proposal_bottom2\n");
    }
    fclose(fp);

    fp = fopen("../data/temp/proposal_top0.bin", "rb");
    if ((int)fread(&num_rois_true, sizeof(int), 1, fp) != 1) {
      printf("Error occurred while reading proposal_top0_size\n");
    }
    if ((int)fread(roi_true_data, sizeof(real), num_rois_true * 5, fp)
        != num_rois_true * 5) {
      printf("Error occurred while reading proposal_top0\n");
    }
    fclose(fp);
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    CUDA_CHECK(hipSetDevice(0));
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    proposal_forward(&score, &bbox, &im_info, &roi, anchors, &option);
  }

  // verify results
  {
    const int roi_size = flatten_size(&roi);
    const int roi_true_size = num_rois_true * 5;
    int i = 0, i_true = 1; // for true data, 0-th element = batch index
    for (; i < roi_size && i_true < roi_true_size; i += 4, i_true += 5) {
      real diff = 0.0f;
      for (int di = 0; di < 4; ++di) {
        diff += ABS(roi_data[i + di] - roi_true_data[i_true + di]) /
                (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + di]));
      }
      if (diff > 1e-3f) {
        real diff1 = 0.0f;
        for (int di = 0; i_true + 5 + di < roi_true_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + di] - roi_true_data[i_true + 5 + di]) /
            (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + 5 + di]));
        }
        if (diff1 < 1e-3f) {
          printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
                 i_true / 5,
                 roi_true_data[i_true + 0], roi_true_data[i_true + 1],
                 roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
          i_true += 5;
          continue;
        }
        real diff2 = 0.0f;
        for (int di = 0; i + 4 + di < roi_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + 4 + di] - roi_true_data[i_true + di]) /
            (1e-10f + MIN(roi_data[i + 4 + di], roi_true_data[i_true + di]));
        }
        if (diff2 < 1e-3f) {
          printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
                 i / 4, roi_data[i + 0], roi_data[i + 1],
                 roi_data[i + 2], roi_data[i + 3]);
          i += 4;
          continue;
        }
        printf("RoI[%d]: %.2f %.2f %.2f %.2f  ",
               i / 4, roi_data[i + 0], roi_data[i + 1],
               roi_data[i + 2], roi_data[i + 3]);
        printf("RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
               i_true / 5,
               roi_true_data[i_true + 0], roi_true_data[i_true + 1],
               roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
      }
    }
    for (; i < roi_size; i += 4) {
      printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
             i / 4, roi_data[i + 0], roi_data[i + 1],
             roi_data[i + 2], roi_data[i + 3]);
    }
    for (; i_true < roi_true_size; i_true += 5) {
      printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
             i_true / 5,
             roi_true_data[i_true + 0], roi_true_data[i_true + 1],
             roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
    }
  }

  return 0;
}
#endif // endifdef TEST
