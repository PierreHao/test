#include "hip/hip_runtime.h"
/*
  Original version (25.8ms)
    1. [1ms] memcpy, D->H
      1-1. scores (75*2*36*46*float = 993.6KB)
      1-2. bbox (75*4*36*46*float = 1987.2KB)
    2. [15ms] all candidate enumeration & sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [3.3ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing (bitwise calculations)
    7. [4ms] roi -> top

  Improved version (6.3ms)
    1. [0ms] no memcpy required
    2. [2.6ms] all candidate enumeration & sort
      2-1. [0.3ms] all candidate enumeration
      2-2. [0.6ms] memcpy, D->H, all candidates (75*36*46*5*float = 2484KB)
      2-3. [1.6ms] partial quick-sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [1.1ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing
    7. [0.1ms] roi -> top

  TODO
    - GPU sort (improve 2-2, 2-3) - speedup
    - GPU nms post processing (remove 5)
*/

#include "layer.h"
#include <math.h>

// --------------------------------------------------------------------------
// kernel code
//   generate_anchors: generate anchor boxes of varying sizes and ratios
//   transform_box: transform a box according to a given gradient
//   sort_box: sort a list of boxes in descending order of their scores
//   enumerate_proposals: generate all candidate boxes with their scores
//   retrieve_rois: retrieve boxes that are determined to be kept by NMS
// --------------------------------------------------------------------------

// given a base box, enumerate transformed boxes of varying sizes and ratios
//   option->base_size: base box's width & height (i.e., base box is square)
//   option->scales: "option->num_scales x 1" array
//                   varying scale factor for base box
//   option->ratios: "option->num_ratios x 1" array
//                   varying height-width ratio
//   option->num_concats: repeat count of anchor set generation
//                        (required for separated RPN)
//   anchors: "num_boxes x 4" array,  (x1, y1, x2, y2) for each box
//     num_boxes = total number of transformations
//         = option->num_scales * option->num_ratios * option->num_concats
#define MAX_NUM_RATIO_SCALE 10
void generate_anchors(real* const anchors,
                      const LayerOption* const option)
{
  // base box's width & height & center location
  const real base_area = option->base_size * option->base_size;
  const real ctr = 0.5f * (option->base_size - 1.0f);

  // transformed width & height for given ratios
  real wr[MAX_NUM_RATIO_SCALE];
  real hr[MAX_NUM_RATIO_SCALE];
  for (int i = 0; i < option->num_ratios; ++i) {
    wr[i] = ROUND(sqrt(base_area / option->ratios[i]));
    hr[i] = ROUND(wr[i] * option->ratios[i]);
  }

  // enumerate all transformed boxes
  {
    real* p_anchors = anchors;
    for (int c = 0; c < option->num_concats; ++c) {
      for (int i = 0; i < option->num_ratios; ++i) {
        for (int j = 0; j < option->num_scales; ++j) {
          // transformed width & height for given ratios & scales
          const real ws = 0.5f * (wr[i] * option->scales[j] - 1.0f);
          const real hs = 0.5f * (hr[i] * option->scales[j] - 1.0f);
          // (x1, y1, x2, y2) for transformed box
          p_anchors[0] = ctr - ws;
          p_anchors[1] = ctr - hs;
          p_anchors[2] = ctr + ws;
          p_anchors[3] = ctr + hs;
          p_anchors += 4;
        } // endfor j
      } // endfor i
    } // endfor c
  }
}

// transform a box according to a given gradient
//   box: (x1, y1, x2, y2)
//   gradient: dx, dy, d(log w), d(log h)
#ifdef GPU
__device__
#endif
static
int transform_box(real* const box,
                  const real dx, const real dy,
                  const real d_log_w, const real d_log_h,
                  const real img_W, const real img_H,
                  const real min_box_W, const real min_box_H)
{
  // width & height of box
  const real w = box[2] - box[0] + 1.0f;
  const real h = box[3] - box[1] + 1.0f;
  // center location of box
  const real ctr_x = box[0] + 0.5f * w;
  const real ctr_y = box[1] + 0.5f * h;

  // new center location according to gradient (dx, dy)
  const real pred_ctr_x = dx * w + ctr_x;
  const real pred_ctr_y = dy * h + ctr_y;
  // new width & height according to gradient d(log w), d(log h)
  const real pred_w = exp(d_log_w) * w;
  const real pred_h = exp(d_log_h) * h;

  // update upper-left corner location
  box[0] = pred_ctr_x - 0.5f * pred_w;
  box[1] = pred_ctr_y - 0.5f * pred_h;
  // update lower-right corner location
  box[2] = pred_ctr_x + 0.5f * pred_w;
  box[3] = pred_ctr_y + 0.5f * pred_h;

  // adjust new corner locations to be within the image region,
  box[0] = MAX(0.0f,  MIN(box[0],  img_W - 1.0f));
  box[1] = MAX(0.0f,  MIN(box[1],  img_H - 1.0f));
  box[2] = MAX(0.0f,  MIN(box[2],  img_W - 1.0f));
  box[3] = MAX(0.0f,  MIN(box[3],  img_H - 1.0f));

  // recompute new width & height
  const real box_w = box[2] - box[0] + 1.0f;
  const real box_h = box[3] - box[1] + 1.0f;

  // check if new box's size >= threshold
  return (box_w >= min_box_W) * (box_h >= min_box_H);
}

// bitonic sort a list of boxes in descending order of their scores (GPU)
//   list: num_boxes x 5 array,  (x1, y1, x2, y2, score) for each box
//     in bitoninc sort, total space allocated for list should be
//     a power of 2 >= num_boxes,
//     and scores of virtually-padded boxes { num_boxes, ..., 2^n - 1 }
//     should be set smaller than mininum score of actual boxes
#ifdef GPU
__global__
void bitonic_sort_step(real* list, const int idx_major, const int idx_minor)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int index_xor = index ^ idx_minor;
  real temp[5];

  // the threads with the lowest ids sort the array
  if (index_xor > index) {
    if (index & idx_major) {
      // sort ascending
      if (list[index * 5 + 4] > list[index_xor * 5 + 4]) {
        for (int i = 0; i < 5; ++i) {
          temp[i] = list[index * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index * 5 + i] = list[index_xor * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index_xor * 5 + i] = temp[i];
        }
      }
    }
    else {
      // sort descending
      if (list[index * 5 + 4] < list[index_xor * 5 + 4]) {
        for (int i = 0; i < 5; ++i) {
          temp[i] = list[index * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index * 5 + i] = list[index_xor * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index_xor * 5 + i] = temp[i];
        }
      }
    }
  }
}
void bitonic_sort_box(real* const list, const int num_boxes)
{
  int num_power_of_2 = 1;
  while (num_power_of_2 < num_boxes) num_power_of_2 *= 2;
  const int num_threads = num_power_of_2;
  const int threads_per_block = 512;
  const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);

  // major step
  for (int idx_major = 2; idx_major <= num_threads; idx_major *= 2) {
    // minor step
    for (int idx_minor = idx_major / 2; idx_minor > 0; idx_minor /= 2) {
      bitonic_sort_step<<<num_blocks, threads_per_block>>>(
          list, idx_major, idx_minor);
    }
  }
}
#endif

// quick-sort a list of boxes in descending order of their scores (CPU)
//   list: num_boxes x 5 array,  (x1, y1, x2, y2, score) for each box
//   if num_top <= end,  only top-k results are guaranteed to be sorted
//   (for efficient computation)
static
void sort_box(real* const list, const int start, const int end,
              const int num_top)
{
  const real pivot_score = list[start * 5 + 4];
  int left = start + 1, right = end;
  real temp[5];
  while (left <= right) {
    while (left <= end && list[left * 5 + 4] >= pivot_score) ++left;
    while (right > start && list[right * 5 + 4] <= pivot_score) --right;
    if (left <= right) {
      for (int i = 0; i < 5; ++i) {
        temp[i] = list[left * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[left * 5 + i] = list[right * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[right * 5 + i] = temp[i];
      }
      ++left;
      --right;
    }
  }

  if (right > start) {
    for (int i = 0; i < 5; ++i) {
      temp[i] = list[start * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[start * 5 + i] = list[right * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[right * 5 + i] = temp[i];
    }
  }

  if (start < right - 1) {
    sort_box(list, start, right - 1, num_top);
  }
  if (right + 1 < num_top && right + 1 < end) {
    sort_box(list, right + 1, end, num_top);
  }
}

// generate all candidate boxes with their scores
//   bottom: 1 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   num_anchors: number of anchors  (= # concats * # scales * # ratios)
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
//   img_H, img_W: scaled image height & width
//   min_box_H, min_box_W: minimum box height & width
//   feat_stride: scaled image height (width) / bottom height (width)
//   proposals: num_proposals * 5 array
//     num_proposals = num_anchors * H * W
//     (x1, y1, x2, y2, score) for each proposal
#ifdef GPU
__global__
void enumerate_proposals_gpu(const real* const bottom4d,
                             const real* const d_anchor4d,
                             const real* const anchors,
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real* const proposals)
{
  const int bottom_area = bottom_H * bottom_W;
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_anchors * bottom_area) {
    const int h = index / num_anchors / bottom_W;
    const int w = (index / num_anchors) % bottom_W;
    const int k = index % num_anchors;
    const real x = w * feat_stride;
    const real y = h * feat_stride;
    const real* p_box = d_anchor4d + h * bottom_W + w;
    const real* p_score = bottom4d + h * bottom_W + w;

    const real dx = p_box[(k * 4 + 0) * bottom_area];
    const real dy = p_box[(k * 4 + 1) * bottom_area];
    const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
    const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

    proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
    proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
    proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
    proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

    proposals[index * 5 + 4]
        = transform_box(&proposals[index * 5],
                        dx, dy, d_log_w, d_log_h,
                        img_W, img_H, min_box_W, min_box_H)
          * p_score[k * bottom_area];
  }
  else {
    // in GPU mode, total space allocated for proposals should be
    // a power of 2 >= actual number of proposals,
    // thus, scores of virtually-padded boxes should be set smaller than
    // mininum score of actual boxes
    // (in RPN, 0 is the smallest possible score)
    proposals[index * 5 + 0] = 0;
    proposals[index * 5 + 1] = 0;
    proposals[index * 5 + 2] = 0;
    proposals[index * 5 + 3] = 0;
    proposals[index * 5 + 4] = 0;
  }
}
#else
void enumerate_proposals_cpu(const real* const bottom4d,
                             const real* const d_anchor4d,
                             const real* const anchors,
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real* const proposals)
{
  const int bottom_area = bottom_H * bottom_W;
  for (int h = 0; h < bottom_H; ++h) {
    for (int w = 0; w < bottom_W; ++w) {
      const real x = w * feat_stride;
      const real y = h * feat_stride;
      const real* p_box = d_anchor4d + h * bottom_W + w;
      const real* p_score = bottom4d + h * bottom_W + w;
      for (int k = 0; k < num_anchors; ++k) {
        const real dx = p_box[(k * 4 + 0) * bottom_area];
        const real dy = p_box[(k * 4 + 1) * bottom_area];
        const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
        const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

        const int index = (h * bottom_W + w) * num_anchors + k;
        proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
        proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
        proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
        proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

        proposals[index * 5 + 4]
            = transform_box(&proposals[index * 5],
                            dx, dy, d_log_w, d_log_h,
                            img_W, img_H, min_box_W, min_box_H)
              * p_score[k * bottom_area];
      } // endfor k
    } // endfor w
  } // endfor h
}
#endif

// retrieve proposals that are determined to be kept as RoIs by NMS
//   proposals : "num_boxes x 5" array,  (x1, y1, x2, y2, score) for each box
//   num_rois: number of RoIs to be retrieved
//   keep: "num_rois x 1" array
//     keep[i]: index of i-th RoI in proposals
//   rois: "num_rois x 4" array,  (x1, y1, x2, y2) for each RoI
#ifdef GPU
__global__
void retrieve_rois_gpu(const real* const proposals,
                       const int* const keep,
                       real* const rois,
                       const int num_rois)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_rois) {
    const real* const proposals_index = proposals + keep[index] * 5;
    rois[index * 4 + 0] = proposals_index[0];
    rois[index * 4 + 1] = proposals_index[1];
    rois[index * 4 + 2] = proposals_index[2];
    rois[index * 4 + 3] = proposals_index[3];
  }
}
#else
void retrieve_rois_cpu(const real* const proposals,
                       const int* const keep,
                       real* const rois,
                       const int num_rois)
{
  for (int i = 0; i < num_rois; ++i) {
    const real* const proposals_index = proposals + keep[i] * 5;
    rois[i * 4 + 0] = proposals_index[0];
    rois[i * 4 + 1] = proposals_index[1];
    rois[i * 4 + 2] = proposals_index[2];
    rois[i * 4 + 3] = proposals_index[3];
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   proposal_forward
// --------------------------------------------------------------------------

// proposal: bottom -> top
//   bottom: 2 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = background score of anchor k at node (h, w)
//     bottom[1, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   img_info: 6 x 1 tensor,  (img_H, img_W, scale_H, scale_W, raw_H, raw_W)
//     img_H, img_W: scaled image height & width
//     scale_H: height scale factor
//              img_H = raw image height * scale_H
//     scale_W: width scale factor
//              img_W = raw image width * scale_W
//     raw_H, raw_W: raw image height & width
//   top: num_RoIs x 4 tensor,  (x1, y1, x2, y2) of each RoI
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
//   4 temporary arrays
//     proposals: all box proposals with their scores
//       "num_boxes x 5" array,  (x1, y1, x2, y2, score) for each box
//       in GPU mode, if proposals = NULL, use bitonic sort in GPU
//       if proposals != NULL & allocated in main memory, quicksort in CPU
//     keep: indices of proposals to be retrieved as RoIs
//       "num_rois x 1" array,  keep[i]: index of i-th RoI in proposals
//       TODO: always stored in main memory due to implementation issue
//     proposals_dev: GPU memory space, required in GPU mode
//       in GPU mode, total space allocated for proposals should be
//       a power of 2 >= num_boxes
//     keep_dev: GPU memory space, required in GPU mode
void proposal_forward(const Tensor* const bottom4d,
                      const Tensor* const d_anchor4d,
                      const Tensor* const img_info1d,
                      Tensor* const top2d,
                      const real* const anchors,
                      real* const proposals,
                      int* const keep,
                      real* const proposals_dev,
                      int* const keep_dev,
                      const LayerOption* const option)
{
  // number of anchors  (= number of concats * scales * ratios)
  const int num_anchors
      = option->num_concats * option->num_ratios * option->num_scales;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom4d->data;
  const real* p_d_anchor_item = d_anchor4d->data;
  const real* p_img_info = img_info1d->data;
  real* p_top_item = top2d->data;
  int total_top_size = 0;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    // bottom shape: 2 x num_anchors x H x W
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    // input image height & width
    const real img_H = p_img_info[0];
    const real img_W = p_img_info[1];
    // scale factor for height & width
    const real scale_H = p_img_info[2];
    const real scale_W = p_img_info[3];
    // minimum box width & height
    const real min_box_H = option->min_size * scale_H;
    const real min_box_W = option->min_size * scale_W;

    // enumerate all proposals
    //   num_proposals = num_anchors * H * W
    //   (x1, y1, x2, y2, score) for each proposal
    // NOTE: for bottom, only foreground scores are passed
    #ifdef GPU
    {
      // in GPU mode, total space allocated for proposals is
      // a power of 2 >= num_proposals (due to bitonic sort algorithm)
      // thus, scores of virtually-padded boxes should be set smaller than
      // mininum score of actual boxes
      const int num_proposals = num_anchors * bottom_area;
      int num_power_of_2 = 1;
      while (num_power_of_2 < num_proposals) num_power_of_2 *= 2;
      const int num_threads = num_power_of_2;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);
      enumerate_proposals_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals_dev);
    }
    #else
    {
      enumerate_proposals_cpu(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals);
    }
    #endif

    // choose candidates according to scores
    #ifdef GPU
    {
      const int num_proposals = num_anchors * bottom_area;
      if (!proposals) {
        // in GPU mode, if proposals = NULL, use bitonic sort in GPU
        bitonic_sort_box(proposals_dev, num_proposals);
      }
      else {
        // if proposals != NULL & allocated in main memory, quicksort in CPU
        hipMemcpyAsync(proposals, proposals_dev,
                        num_proposals * 5 * sizeof(real),
                        hipMemcpyDeviceToHost);
        sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
        hipMemcpyAsync(proposals_dev, proposals,
                        num_proposals * 5 * sizeof(real),
                        hipMemcpyHostToDevice);
      }
    }
    #else
    {
      const int num_proposals = num_anchors * bottom_area;
      sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
    }
    #endif

    // NMS & RoI retrieval
    {
      // NMS
      const int num_proposals
          = MIN(num_anchors * bottom_area,  option->pre_nms_topn);
      int num_rois = 0;
      nms(num_proposals,  proposals,  &num_rois,  keep,  0,
          option->nms_thresh,  option->post_nms_topn);

      // RoI retrieval
      #ifdef GPU
      {
        const int num_threads = num_rois;
        const int threads_per_block = 128;
        const int num_blocks
            = DIV_THEN_CEIL(num_threads,  threads_per_block);

        hipMemcpyAsync(keep_dev, keep, num_rois * sizeof(int),
                        hipMemcpyHostToDevice);

        retrieve_rois_gpu<<<num_blocks, threads_per_block>>>(
            proposals_dev,  keep_dev,  p_top_item,  num_rois);
      }
      #else
      {
        retrieve_rois_cpu(
            proposals,  keep,  p_top_item,  num_rois);
      }
      #endif

      // set top shape: num_rois x 4,  (x1, y1, x2, y2) for each RoI
      top2d->shape[n][0] = num_rois;
      top2d->shape[n][1] = 4;
      top2d->start[n] = total_top_size;
      total_top_size += num_rois * 4;
    }

    // locate next item
    {
      const int bottom_size = 2 * num_anchors * bottom_area;
      const int d_anchor_size = 4 * num_anchors * bottom_area;
      const int img_info_size = 6;
      const int top_size = 4 * top2d->shape[n][0];
      p_bottom_item += bottom_size;
      p_d_anchor_item += d_anchor_size;
      p_img_info += img_info_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------
void proposal_shape(const Tensor* const bottom4d,
                    Tensor* const top2d,
                    int* const proposals_size,
                    int* const keep_size,
                    const LayerOption* const option)
{
  int max_area = 0;

  // calculate shape for each item in the batch
  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    // calculate maximum area size for determining temporary space size
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    max_area = MAX(max_area,  bottom_area);

    // top shape <= post_nms_topn x 4
    //   exact row size will be determined after forward-pass
    top2d->shape[n][0] = option->post_nms_topn;
    top2d->shape[n][1] = 4;
    top2d->start[n] = top2d->shape[n][0] * top2d->shape[n][1];
  }

  // temporary space size
  //   in GPU mode, total space allocated for proposals should be
  //   a power of 2 >= actual number of proposals
  {
    const int num_anchors 
        = option->num_concats * option->num_ratios * option->num_scales;
    const int num_proposals = num_anchors * max_area;
    int num_power_of_2 = 1;
    while (num_power_of_2 < num_proposals) num_power_of_2 *= 2;
    *proposals_size = num_power_of_2 * 5;
    *keep_size = option->post_nms_topn;
  }
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void init_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  const int num_anchors = layer->option.num_scales
                          * layer->option.num_ratios
                          * layer->option.num_concats;

  #ifdef GPU
  {
    hipMalloc(&layer->p_aux_data[0], num_anchors * 4 * sizeof(real));
    generate_anchors(net->param_cpu_data, &layer->option);
    hipMemcpyAsync(layer->p_aux_data[0], net->param_cpu_data,
                    num_anchors * 4 * sizeof(real),
                    hipMemcpyHostToDevice);
  }
  #else
  {
    layer->p_aux_data[0] = (real*)malloc(num_anchors * 4 * sizeof(real));
    generate_anchors(layer->p_aux_data[0], &layer->option);
  }
  #endif

  net->space += num_anchors * 4 * sizeof(real);
}

void forward_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  proposal_forward(layer->p_bottoms[0], layer->p_bottoms[1],
                   layer->p_bottoms[2],
                   &layer->tops[0], layer->p_aux_data[0],
                   net->temp_cpu_data, net->tempint_cpu_data,
                   net->temp_data, net->tempint_data,
                   &layer->option);

  print_tensor_info(layer->name, &layer->tops[0]);
}

void shape_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  int temp_size, tempint_size;

  proposal_shape(layer->p_bottoms[0], &layer->tops[0],
                 &temp_size, &tempint_size, &layer->option);

  update_net_size(net, layer, temp_size, tempint_size, 0);
}



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST
#include <stdio.h>

int main(int argc, char* argv[])
{
  // variable declaration & memory allocation
  Tensor score, d_anchor, img_info, roi, roi_true;
  real *score_data = NULL, *d_anchor_data = NULL, *img_info_data = NULL;
  real *roi_data = NULL, *roi_true_data = NULL;
  real scales[5] = {3, 6, 9, 16, 32};
  real ratios[5] = {0.5, 0.666, 1.0, 1.5, 2.0};
  real *anchors = NULL, *p_anchors = NULL;
  real *proposals = NULL, *proposals_dev = NULL;
  int *keep = NULL, *keep_dev = NULL;
  int num_anchors;
  LayerOption option;

  // set option
  {
    option.scales = &scales[0];
    option.ratios = &ratios[0];
    option.num_scales = 5;
    option.num_ratios = 5;
    option.num_concats = 3;
    option.base_size = 16;
    option.feat_stride = 16;
    option.min_size = 16;
    option.pre_nms_topn = 6000;
    option.post_nms_topn = 300;
    option.nms_thresh = 0.7;
  }

  // generate anchors
  {
    num_anchors = option.num_scales * option.num_ratios * option.num_concats;
    // 4 real variables for each anchor: (x1, y1, x2, y2)
    anchors = (real*)malloc(num_anchors * 4 * sizeof(real));
    generate_anchors(anchors, &option);
  }

  // load data
  {
    int ndim;
    int shape[g_max_ndim];
    int total_size;

    // score: 2 x num_anchors x H x W tensor
    score_data = load_data("../data/temp/proposal_bottom0.bin",
                           &ndim, shape, NULL);
    score.num_items = shape[0];
    score.ndim = 4;
    total_size = 0;
    for (int n = 0; n < score.num_items; ++n) {
      score.shape[n][0] = 2;
      score.shape[n][1] = num_anchors;
      score.shape[n][2] = shape[2];
      score.shape[n][3] = shape[3];
      score.start[n] = total_size;
      total_size += 2 * num_anchors * shape[2] * shape[3];
    }

    // d_anchor: num_anchors x 4 x H x W tensor
    d_anchor_data = load_data("../data/temp/proposal_bottom1.bin",
                              &ndim, shape, NULL);
    d_anchor.num_items = shape[0];
    d_anchor.ndim = 4;
    total_size = 0;
    for (int n = 0; n < d_anchor.num_items; ++n) {
      d_anchor.shape[n][0] = num_anchors;
      d_anchor.shape[n][1] = 4;
      d_anchor.shape[n][2] = shape[2];
      d_anchor.shape[n][3] = shape[3];
      d_anchor.start[n] = total_size;
      total_size += num_anchors * 4 * shape[2] * shape[3];
    }

    // img_info: 6 x 1 tensor
    img_info_data = load_data("../data/temp/proposal_bottom2.bin",
                              &ndim, shape, NULL);
    img_info.num_items = 1;
    img_info.ndim = 1;
    img_info.shape[0][0] = shape[0];

    // roi_true: num_rois x 4 tensor
    roi_true_data = load_data("../data/temp/proposal_top0.bin",
                              &ndim, shape, NULL);
    {
      const int num_rois = shape[0];
      int num_items = 0;
      for (int i = 0; i < num_rois; ++i) {
        const int n = (int)ROUND(roi_true_data[i * 5 + 0]);
        const real x1 = roi_true_data[i * 5 + 1];
        const real y1 = roi_true_data[i * 5 + 2];
        const real x2 = roi_true_data[i * 5 + 3];
        const real y2 = roi_true_data[i * 5 + 4];
        ++roi_true.shape[n][0];
        roi_true_data[i * 4 + 0] = x1;
        roi_true_data[i * 4 + 1] = y1;
        roi_true_data[i * 4 + 2] = x2;
        roi_true_data[i * 4 + 3] = y2;
        num_items = MAX(num_items,  n);
      }
      roi_true.num_items = num_items + 1;
    }
    roi_true.ndim = 2;
    for (int n = 0; n < roi_true.num_items; ++n) {
      roi_true.shape[n][1] = 4;
    }

    // memory allocation for output & temporary data
    {
      int proposals_size, keep_size;
      proposal_shape(&score, &roi, &proposals_size, &keep_size, &option);

      // temporary space for proposal_forward operation
      proposals = (real*)malloc(proposals_size * sizeof(real));
      keep = (int*)malloc(keep_size * sizeof(int));
      #ifdef GPU
      hipMalloc(&proposals_dev, proposals_size * sizeof(real));
      hipMalloc(&keep_dev, keep_size * sizeof(int));
      #endif

      // output data
      roi_data = (real*)malloc(flatten_size(&roi) * sizeof(real));
    }
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const long int score_size = flatten_size(&score);
    const long int d_anchor_size = flatten_size(&d_anchor);
    const long int roi_size = flatten_size(&roi);

    printf("gpu malloc\n");
    hipMalloc(&score.data, score_size * sizeof(real));
    hipMalloc(&d_anchor.data, d_anchor_size * sizeof(real));
    hipMalloc(&p_anchors, num_anchors * 4 * sizeof(real));
    hipMalloc(&roi.data, roi_size * sizeof(real));

    printf("memcpy: cpu -> gpu\n");
    hipMemcpyAsync(score.data, score_data,
                    score_size * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_anchor.data, d_anchor_data,
                    d_anchor_size * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMemcpyAsync(p_anchors, anchors,
                    num_anchors * 4 * sizeof(real),
                    hipMemcpyHostToDevice);

    img_info.data = img_info_data;
  }
  #else
  {
    score.data = score_data;
    d_anchor.data = d_anchor_data;
    p_anchors = anchors;
    img_info.data = img_info_data;
    roi.data = roi_data;
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    proposal_forward(&score, &d_anchor, &img_info, &roi, p_anchors,
                     proposals, keep, proposals_dev, keep_dev,
                     &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const long int roi_size = flatten_size(&roi);

    printf("memcpy: cpu <- gpu\n");
    hipMemcpyAsync(roi_data, roi.data, roi_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  }
  #endif

  // verify results
  {
    const long int roi_size = flatten_size(&roi);
    const long int roi_true_size = flatten_size(&roi_true);
    int i = 0, i_true = 0;

    printf("verification\n");

    for (; i < roi_size && i_true < roi_true_size; i += 4, i_true += 4) {
      real diff = 0.0f;
      for (int di = 0; di < 4; ++di) {
        diff += ABS(roi_data[i + di] - roi_true_data[i_true + di]) /
                (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + di]));
      }
      if (diff > 1e-3f) {
        real diff1 = 0.0f;
        for (int di = 0; i_true + 4 + di < roi_true_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + di] - roi_true_data[i_true + 4 + di]) /
            (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + 4 + di]));
        }
        if (diff1 < 1e-3f) {
          printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
                 i_true / 4,
                 roi_true_data[i_true + 0], roi_true_data[i_true + 1],
                 roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
          i_true += 4;
          continue;
        }
        real diff2 = 0.0f;
        for (int di = 0; i + 4 + di < roi_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + 4 + di] - roi_true_data[i_true + di]) /
            (1e-10f + MIN(roi_data[i + 4 + di], roi_true_data[i_true + di]));
        }
        if (diff2 < 1e-3f) {
          printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
                 i / 4, roi_data[i + 0], roi_data[i + 1],
                 roi_data[i + 2], roi_data[i + 3]);
          i += 4;
          continue;
        }
        printf("RoI[%d]: %.2f %.2f %.2f %.2f  ",
               i / 4, roi_data[i + 0], roi_data[i + 1],
               roi_data[i + 2], roi_data[i + 3]);
        printf("RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
               i_true / 4,
               roi_true_data[i_true + 0], roi_true_data[i_true + 1],
               roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
      }
    }
    for (; i < roi_size; i += 4) {
      printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
             i / 4, roi_data[i + 0], roi_data[i + 1],
             roi_data[i + 2], roi_data[i + 3]);
    }
    for (; i_true < roi_true_size; i_true += 4) {
      printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
             i_true / 4,
             roi_true_data[i_true + 0], roi_true_data[i_true + 1],
             roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
    }
  }

  // memory deallocation
  {
    free(score_data);
    free(d_anchor_data);
    free(img_info_data);
    free(roi_data);
    free(roi_true_data);
    free(anchors);
    free(proposals);
    free(keep);
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    hipFree(score.data);
    hipFree(d_anchor.data);
    hipFree(roi.data);
    hipFree(p_anchors);
    hipFree(proposals_dev);
    hipFree(keep_dev);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
