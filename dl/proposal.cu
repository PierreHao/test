#include "hip/hip_runtime.h"
/*
  Original version (25.8ms)
    1. [1ms] memcpy, D->H
      1-1. scores (75*2*36*46*float = 993.6KB)
      1-2. bbox (75*4*36*46*float = 1987.2KB)
    2. [15ms] all candidate enumeration & sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [3.3ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing (bitwise calculations)
    7. [4ms] roi -> top

  Improved version (6.3ms)
    1. [0ms] no memcpy required
    2. [2.6ms] all candidate enumeration & sort
      2-1. [0.3ms] all candidate enumeration
      2-2. [0.6ms] memcpy, D->H, all candidates (75*36*46*5*float = 2484KB)
      2-3. [1.6ms] partial quick-sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [1.1ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing
    7. [0.1ms] roi -> top

  TODO
    - GPU sort (improve 2-2, 2-3)
    - GPU nms post processing (remove 5)
*/

#include "layer.h"
#include <math.h>

// --------------------------------------------------------------------------
// kernel code
//   transform_box: transform a box according to a given gradient
//   generate_anchors: generate anchor boxes of varying sizes and ratios
//   sort_box: sort a list of boxes in descending order of their scores
//   enumerate_proposals: generate all candidate boxes with their scores
//   retrieve_rois: retrieve boxes that are determined to be kept by NMS
// --------------------------------------------------------------------------

// transform a box according to a given gradient
//   box: (x1, y1, x2, y2)
//   gradient: dx, dy, d(log w), d(log h)
#ifdef GPU
__device__
#endif
int transform_box(real* const box,
                  const real dx, const real dy,
                  const real d_log_w, const real d_log_h,
                  const real img_W, const real img_H,
                  const real min_box_W, const real min_box_H)
{
  // width & height of box
  const real w = box[2] - box[0] + 1.0f;
  const real h = box[3] - box[1] + 1.0f;
  // center location of box
  const real ctr_x = box[0] + 0.5f * w;
  const real ctr_y = box[1] + 0.5f * h;

  // new center location according to gradient (dx, dy)
  const real pred_ctr_x = dx * w + ctr_x;
  const real pred_ctr_y = dy * h + ctr_y;
  // new width & height according to gradient d(log w), d(log h)
  const real pred_w = exp(d_log_w) * w;
  const real pred_h = exp(d_log_h) * h;

  // update upper-left corner location
  box[0] = pred_ctr_x - 0.5f * pred_w;
  box[1] = pred_ctr_y - 0.5f * pred_h;
  // update lower-right corner location
  box[2] = pred_ctr_x + 0.5f * pred_w;
  box[3] = pred_ctr_y + 0.5f * pred_h;

  // adjust new corner locations to be within the image region,
  box[0] = MAX(0.0f,  MIN(box[0],  img_W - 1.0f));
  box[1] = MAX(0.0f,  MIN(box[1],  img_H - 1.0f));
  box[2] = MAX(0.0f,  MIN(box[2],  img_W - 1.0f));
  box[3] = MAX(0.0f,  MIN(box[3],  img_H - 1.0f));

  // recompute new width & height
  const real box_w = box[2] - box[0] + 1.0f;
  const real box_h = box[3] - box[1] + 1.0f;

  // check if new box's size >= threshold
  return (box_w >= min_box_W) * (box_h >= min_box_H);
}

// given a base box, enumerate transformed boxes of varying sizes and ratios
//   option->base_size: base box's width & height (i.e., base box is square)
//   option->scales: "option->num_scales x 1" array
//                   varying scale factor for base box
//   option->ratios: "option->num_ratios x 1" array
//                   varying height-width ratio
//   option->num_concats: repeat count of anchor set generation
//                        (required for separated RPN)
//   anchors: "num_boxes x 4" array,  (x1, y1, x2, y2) for each box
//     num_boxes = total number of transformations
//         = option->num_scales * option->num_ratios * option->num_concats
#define MAX_NUM_RATIO_SCALE 10
void generate_anchors(real* const anchors,
                      const ProposalOption* const option)
{
  // base box's width & height & center location
  const real base_area = option->base_size * option->base_size;
  const real ctr = 0.5f * (option->base_size - 1.0f);

  // transformed width & height for given ratios
  real wr[MAX_NUM_RATIO_SCALE];
  real hr[MAX_NUM_RATIO_SCALE];
  for (int i = 0; i < option->num_ratios; ++i) {
    wr[i] = ROUND(sqrt(base_area / option->ratios[i]));
    hr[i] = ROUND(wr[i] * option->ratios[i]);
  }

  // enumerate all transformed boxes
  {
    real* p_anchors = anchors;
    for (int c = 0; c < option->num_concats; ++c) {
      for (int i = 0; i < option->num_ratios; ++i) {
        for (int j = 0; j < option->num_scales; ++j) {
          // transformed width & height for given ratios & scales
          const real ws = 0.5f * (wr[i] * option->scales[j] - 1.0f);
          const real hs = 0.5f * (hr[i] * option->scales[j] - 1.0f);
          // (x1, y1, x2, y2) for transformed box
          p_anchors[0] = ctr - ws;
          p_anchors[1] = ctr - hs;
          p_anchors[2] = ctr + ws;
          p_anchors[3] = ctr + hs;
          p_anchors += 4;
        } // endfor j
      } // endfor i
    } // endfor c
  }
}

// quick-sort a list of boxes in descending order of their scores
//   if num_top <= end,  only top-k results are guaranteed to be sorted
//   (for efficient computation)
void sort_box(real* const list, const int start, const int end,
              const int num_top)
{
  const real pivot_score = list[start * 5 + 4];
  int left = start + 1, right = end;
  real temp[5];
  while (left <= right) {
    while (left <= end && list[left * 5 + 4] >= pivot_score) ++left;
    while (right > start && list[right * 5 + 4] <= pivot_score) --right;
    if (left <= right) {
      for (int i = 0; i < 5; ++i) {
        temp[i] = list[left * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[left * 5 + i] = list[right * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[right * 5 + i] = temp[i];
      }
      ++left;
      --right;
    }
  }

  if (right > start) {
    for (int i = 0; i < 5; ++i) {
      temp[i] = list[start * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[start * 5 + i] = list[right * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[right * 5 + i] = temp[i];
    }
  }

  if (start < right - 1) {
    sort_box(list, start, right - 1, num_top);
  }
  if (right + 1 < num_top && right + 1 < end) {
    sort_box(list, right + 1, end, num_top);
  }
}

// generate all candidate boxes with their scores
//   bottom: 1 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   num_anchors: number of anchors  (= # concats * # scales * # ratios)
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
//   img_H, img_W: scaled image height & width
//   min_box_H, min_box_W: minimum box height & width
//   feat_stride: scaled image height (width) / bottom height (width)
//   proposals: num_proposals * 5 array
//     num_proposals = num_anchors * H * W
//     (x1, y1, x2, y2, score) for each proposal
#ifdef GPU
__global__
void enumerate_proposals_gpu(const real* const bottom4d,
                             const real* const d_anchor4d,
                             const real* const anchors,
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real* const proposals)
{
  const int bottom_area = bottom_H * bottom_W;
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_anchors * bottom_area) {
    const int h = index / num_anchors / bottom_W;
    const int w = (index / num_anchors) % bottom_W;
    const int k = index % num_anchors;
    const real x = w * feat_stride;
    const real y = h * feat_stride;
    const real* p_box = d_anchor4d + h * bottom_W + w;
    const real* p_score = bottom4d + h * bottom_W + w;

    const real dx = p_box[(k * 4 + 0) * bottom_area];
    const real dy = p_box[(k * 4 + 1) * bottom_area];
    const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
    const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

    proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
    proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
    proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
    proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

    proposals[index * 5 + 4]
        = transform_box(&proposals[index * 5],
                        dx, dy, d_log_w, d_log_h,
                        img_W, img_H, min_box_W, min_box_H)
          * p_score[k * bottom_area];
  }
}
#else
void enumerate_proposals_cpu(const real* const bottom4d,
                             const real* const d_anchor4d,
                             const real* const anchors,
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real* const proposals)
{
  const int bottom_area = bottom_H * bottom_W;
  for (int h = 0; h < bottom_H; ++h) {
    for (int w = 0; w < bottom_W; ++w) {
      const real x = w * feat_stride;
      const real y = h * feat_stride;
      const real* p_box = d_anchor4d + h * bottom_W + w;
      const real* p_score = bottom4d + h * bottom_W + w;
      for (int k = 0; k < num_anchors; ++k) {
        const real dx = p_box[(k * 4 + 0) * bottom_area];
        const real dy = p_box[(k * 4 + 1) * bottom_area];
        const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
        const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

        const int index = (h * bottom_W + w) * num_anchors + k;
        proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
        proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
        proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
        proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

        proposals[index * 5 + 4]
            = transform_box(&proposals[index * 5],
                            dx, dy, d_log_w, d_log_h,
                            img_W, img_H, min_box_W, min_box_H)
              * p_score[k * bottom_area];
      } // endfor k
    } // endfor w
  } // endfor h
}
#endif

// retrieve proposals that are determined to be kept as RoIs by NMS
//   proposals : "num_boxes x 5" array,  (x1, y1, x2, y2, score) for each box
//   num_rois: number of RoIs to be retrieved
//   keep: "num_rois x 1" array
//     keep[i]: index of i-th RoI in proposals
//   rois: "num_rois x 4" array,  (x1, y1, x2, y2) for each RoI
#ifdef GPU
__global__
void retrieve_rois_gpu(const real* const proposals,
                       const int* const keep,
                       real* const rois,
                       const int num_rois)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_rois) {
    const real* const proposals_index = proposals + keep[index] * 5;
    rois[index * 4 + 0] = proposals_index[0];
    rois[index * 4 + 1] = proposals_index[1];
    rois[index * 4 + 2] = proposals_index[2];
    rois[index * 4 + 3] = proposals_index[3];
  }
}
#else
void retrieve_rois_cpu(const real* const proposals,
                       const int* const keep,
                       real* const rois,
                       const int num_rois)
{
  for (int i = 0; i < num_rois; ++i) {
    const real* const proposals_index = proposals + keep[i] * 5;
    rois[i * 4 + 0] = proposals_index[0];
    rois[i * 4 + 1] = proposals_index[1];
    rois[i * 4 + 2] = proposals_index[2];
    rois[i * 4 + 3] = proposals_index[3];
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   proposal_forward
// --------------------------------------------------------------------------

// proposal: bottom -> top
//   bottom: 2 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = background score of anchor k at node (h, w)
//     bottom[1, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   img_info: 4 x 1 tensor,  (img_H, img_W, min_box_W, min_box_H)
//     img_H, img_W: scaled image height & width
//     min_box_W: minimum box width in raw image
//     min_box_H: minimum box height in raw image
//   top: num_RoIs x 4 tensor,  (x1, y1, x2, y2) of each RoI
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
void proposal_forward(const Tensor* const bottom4d,
                      const Tensor* const d_anchor4d,
                      const Tensor* const img_info1d,
                      Tensor* const top2d,
                      const real* const anchors,
                      const ProposalOption* const option)
{

  const int num_anchors
      = option->num_concats * option->num_ratios * option->num_scales;

  real* const proposals = (real*)malloc(num_anchors * 80*80 * 5 * sizeof(real));
  int* const keep = (int*)malloc(option->post_nms_topn * sizeof(int));

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom4d->data;
  const real* p_d_anchor_item = d_anchor4d->data;
  const real* p_img_info = img_info1d->data;
  real* p_top_item = top2d->data;

  #ifdef GPU
  real* proposals_dev;
  int* keep_dev;
  hipMalloc(&proposals_dev, num_anchors * 80*80 * 5 * sizeof(real));
  hipMalloc(&keep_dev, option->post_nms_topn * sizeof(int));
  #endif

  for (int n = 0; n < bottom4d->num_items; ++n) {
    // bottom shape: 2 x num_anchors x H x W
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    // input image height & width
    const real img_H = p_img_info[0];
    const real img_W = p_img_info[1];
    // minimum box width & height
    const real min_box_W = option->min_size * p_img_info[2];
    const real min_box_H = option->min_size * p_img_info[3];

    // enumerate all proposals
    //   num_proposals = num_anchors * H * W
    //   (x1, y1, x2, y2, score) for each proposal
    // NOTE: for bottom, only foreground scores are passed
    #ifdef GPU
    {
      const int num_threads = num_anchors * bottom_area;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);
      enumerate_proposals_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals_dev);
      hipMemcpyAsync(proposals, proposals_dev,
                      num_threads * 5 * sizeof(real),
                      hipMemcpyDeviceToHost);
    }
    #else
    {
      enumerate_proposals_cpu(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals);
    }
    #endif

    // choose candidates according to scores
    // TODO: copy proposals to GPU memory directly
    {
      const int num_proposals = num_anchors * bottom_area;
      sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
    }

    // NMS & RoI retrieval
    {
      // NMS
      const int num_proposals
          = MIN(num_anchors * bottom_area,  option->pre_nms_topn);
      int num_rois = 0;
      nms(num_proposals,  proposals,  &num_rois,  keep,
          option->nms_thresh,  option->post_nms_topn);

      // RoI retrieval
      #ifdef GPU
      {
        const int num_threads = num_rois;
        const int threads_per_block = 128;
        const int num_blocks
            = DIV_THEN_CEIL(num_threads,  threads_per_block);

        hipMemcpyAsync(keep_dev, keep, num_rois * sizeof(int),
                        hipMemcpyHostToDevice);
        hipMemcpyAsync(proposals_dev, proposals,
                        num_proposals * 5 * sizeof(int),
                        hipMemcpyHostToDevice);

        retrieve_rois_gpu<<<num_blocks, threads_per_block>>>(
            proposals_dev,  keep_dev,  p_top_item,  num_rois);
      }
      #else
      {
        retrieve_rois_cpu(
            proposals,  keep,  p_top_item,  num_rois);
      }
      #endif

      // set top shape: num_rois x 4,  (x1, y1, x2, y2) for each RoI
      top2d->shape[n][0] = num_rois;
      top2d->shape[n][1] = 4;
    }

    // locate next item
    {
      const int bottom_size = 2 * num_anchors * bottom_area;
      const int d_anchor_size = 4 * num_anchors * bottom_area;
      const int img_info_size = 4;
      const int top_size = 4 * top2d->shape[n][0];
      p_bottom_item += bottom_size;
      p_d_anchor_item += d_anchor_size;
      p_img_info += img_info_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;

  // memory deallocation
  {
    free(proposals);
    free(keep);

  #ifdef GPU
    hipFree(proposals_dev);
    hipFree(keep_dev);
  #endif
  }
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------
void proposal_shape(const Tensor* const bottom4d,
                    Tensor* const top2d,
                    const ProposalOption* const option)
{
  // calculate shape for each item in the batch
  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    // top shape <= post_nms_topn x 4
    //   exact row size will be determined after forward-pass
    top2d->shape[n][0] = option->post_nms_topn;
    top2d->shape[n][1] = 4;
  }
}



// --------------------------------------------------------------------------
// test code
// --------------------------------------------------------------------------

#ifdef TEST
#include <stdio.h>

int main(int argc, char* argv[])
{
  // variable declaration & memory allocation
  Tensor score, d_anchor, img_info, roi, roi_true;
  real *score_data = NULL, *d_anchor_data = NULL, *img_info_data = NULL;
  real *roi_data, *roi_true_data;
  real scales[5] = {3, 6, 9, 16, 32};
  real ratios[5] = {0.5, 0.666, 1.0, 1.5, 2.0};
  real *anchors, *p_anchors;
  int num_anchors;
  ProposalOption option;

  // set option
  {
    option.scales = &scales[0];
    option.ratios = &ratios[0];
    option.num_scales = 5;
    option.num_ratios = 5;
    option.num_concats = 3;
    option.base_size = 16;
    option.feat_stride = 16;
    option.min_size = 16;
    option.pre_nms_topn = 6000;
    option.post_nms_topn = 300;
    option.nms_thresh = 0.7;
  }

  // generate anchors
  {
    num_anchors = option.num_scales * option.num_ratios * option.num_concats;
    // 4 real variables for each anchor: (x1, y1, x2, y2)
    anchors = (real*)malloc(num_anchors * 4 * sizeof(real));
    generate_anchors(anchors, &option);
  }

  // load data
  {
    int ndim;
    int shape[g_max_ndim];
    int total_size;

    // score: 2 x num_anchors x H x W tensor
    score_data
        = load_data("../data/temp/proposal_bottom0.bin", &ndim, shape);
    score.num_items = shape[0];
    score.ndim = 4;
    total_size = 0;
    for (int n = 0; n < score.num_items; ++n) {
      score.shape[n][0] = 2;
      score.shape[n][1] = num_anchors;
      score.shape[n][2] = shape[2];
      score.shape[n][3] = shape[3];
      score.start[n] = total_size;
      total_size += 2 * num_anchors * shape[2] * shape[3];
    }

    // d_anchor: num_anchors x 4 x H x W tensor
    d_anchor_data
        = load_data("../data/temp/proposal_bottom1.bin", &ndim, shape);
    d_anchor.num_items = shape[0];
    d_anchor.ndim = 4;
    total_size = 0;
    for (int n = 0; n < d_anchor.num_items; ++n) {
      d_anchor.shape[n][0] = num_anchors;
      d_anchor.shape[n][1] = 4;
      d_anchor.shape[n][2] = shape[2];
      d_anchor.shape[n][3] = shape[3];
      d_anchor.start[n] = total_size;
      total_size += num_anchors * 4 * shape[2] * shape[3];
    }

    // img_info: 4 x 1 tensor
    img_info_data
        = load_data("../data/temp/proposal_bottom2.bin", &ndim, shape);
    img_info.num_items = 1;
    img_info.ndim = 1;
    img_info.shape[0][0] = shape[0];

    // roi_true: num_rois x 4 tensor
    roi_true_data
        = load_data("../data/temp/proposal_top0.bin", &ndim, shape);
    {
      const int num_rois = shape[0];
      int num_items = 0;
      for (int i = 0; i < num_rois; ++i) {
        const int n = (int)ROUND(roi_true_data[i * 5 + 0]);
        const real x1 = roi_true_data[i * 5 + 1];
        const real y1 = roi_true_data[i * 5 + 2];
        const real x2 = roi_true_data[i * 5 + 3];
        const real y2 = roi_true_data[i * 5 + 4];
        ++roi_true.shape[n][0];
        roi_true_data[i * 4 + 0] = x1;
        roi_true_data[i * 4 + 1] = y1;
        roi_true_data[i * 4 + 2] = x2;
        roi_true_data[i * 4 + 3] = y2;
        num_items = MAX(num_items,  n);
      }
      roi_true.num_items = num_items + 1;
    }
    roi_true.ndim = 2;
    for (int n = 0; n < roi_true.num_items; ++n) {
      roi_true.shape[n][1] = 4;
    }

    proposal_shape(&score, &roi, &option);
    roi_data = (real*)malloc(flatten_size(&roi) * sizeof(real));
  }

  // CUDA initialization
  #ifdef GPU
  {
    printf("set device\n");
    hipSetDevice(0);
  }
  #endif

  // bind loaded data to corresponding tensors
  #ifdef GPU
  {
    const int score_size = flatten_size(&score);
    const int d_anchor_size = flatten_size(&d_anchor);
    const int roi_size = flatten_size(&roi);

    printf("gpu malloc\n");
    hipMalloc(&score.data, score_size * sizeof(real));
    hipMalloc(&d_anchor.data, d_anchor_size * sizeof(real));
    hipMalloc(&p_anchors, num_anchors * 4 * sizeof(real));
    hipMalloc(&roi.data, roi_size * sizeof(real));

    printf("memcpy: cpu -> gpu\n");
    hipMemcpyAsync(score.data, score_data,
                    score_size * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_anchor.data, d_anchor_data,
                    d_anchor_size * sizeof(real),
                    hipMemcpyHostToDevice);
    hipMemcpyAsync(p_anchors, anchors,
                    num_anchors * 4 * sizeof(real),
                    hipMemcpyHostToDevice);

    img_info.data = img_info_data;
  }
  #else
  {
    score.data = score_data;
    d_anchor.data = d_anchor_data;
    p_anchors = anchors;
    img_info.data = img_info_data;
    roi.data = roi_data;
  }
  #endif

  // do forward operation
  {
    printf("do forward\n");
    proposal_forward(&score, &d_anchor, &img_info, &roi, p_anchors, &option);
  }

  // copy GPU data to main memory
  #ifdef GPU
  {
    const int roi_size = flatten_size(&roi);

    printf("memcpy: cpu <- gpu\n");
    hipMemcpyAsync(roi_data, roi.data, roi_size * sizeof(real),
                    hipMemcpyDeviceToHost);
  }
  #endif

  // verify results
  {
    const int roi_size = flatten_size(&roi);
    const int roi_true_size = flatten_size(&roi_true);
    int i = 0, i_true = 0;

    printf("verification\n");

    for (; i < roi_size && i_true < roi_true_size; i += 4, i_true += 4) {
      real diff = 0.0f;
      for (int di = 0; di < 4; ++di) {
        diff += ABS(roi_data[i + di] - roi_true_data[i_true + di]) /
                (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + di]));
      }
      if (diff > 1e-3f) {
        real diff1 = 0.0f;
        for (int di = 0; i_true + 4 + di < roi_true_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + di] - roi_true_data[i_true + 4 + di]) /
            (1e-10f + MIN(roi_data[i + di], roi_true_data[i_true + 4 + di]));
        }
        if (diff1 < 1e-3f) {
          printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
                 i_true / 4,
                 roi_true_data[i_true + 0], roi_true_data[i_true + 1],
                 roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
          i_true += 4;
          continue;
        }
        real diff2 = 0.0f;
        for (int di = 0; i + 4 + di < roi_size && di < 4; ++di) {
          diff1 += ABS(roi_data[i + 4 + di] - roi_true_data[i_true + di]) /
            (1e-10f + MIN(roi_data[i + 4 + di], roi_true_data[i_true + di]));
        }
        if (diff2 < 1e-3f) {
          printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
                 i / 4, roi_data[i + 0], roi_data[i + 1],
                 roi_data[i + 2], roi_data[i + 3]);
          i += 4;
          continue;
        }
        printf("RoI[%d]: %.2f %.2f %.2f %.2f  ",
               i / 4, roi_data[i + 0], roi_data[i + 1],
               roi_data[i + 2], roi_data[i + 3]);
        printf("RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
               i_true / 4,
               roi_true_data[i_true + 0], roi_true_data[i_true + 1],
               roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
      }
    }
    for (; i < roi_size; i += 4) {
      printf("[False Positive] RoI[%d]: %.2f %.2f %.2f %.2f\n",
             i / 4, roi_data[i + 0], roi_data[i + 1],
             roi_data[i + 2], roi_data[i + 3]);
    }
    for (; i_true < roi_true_size; i_true += 4) {
      printf("[False Negative] RoI_true[%d]: %.2f %.2f %.2f %.2f\n",
             i_true / 4,
             roi_true_data[i_true + 0], roi_true_data[i_true + 1],
             roi_true_data[i_true + 2], roi_true_data[i_true + 3]);
    }
  }

  // memory deallocation
  {
    free(anchors);
    free(score_data);
    free(d_anchor_data);
    free(img_info_data);
    free(roi_data);
    free(roi_true_data);
  }
  #ifdef GPU
  {
    printf("gpu free\n");
    hipFree(score.data);
    hipFree(d_anchor.data);
    hipFree(p_anchors);
    hipFree(roi.data);
  }
  #endif

  return 0;
}
#endif // endifdef TEST
