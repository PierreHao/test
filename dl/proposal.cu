#include "hip/hip_runtime.h"
#include "layer.h"
#include "cuda_settings.h"

static const int num_threads = sizeof(unsigned long long) * 8;

__device__ inline real iou_kernel(const real* const a, const real* const b)
{
  const real left = max(a[0], b[0]);
  const real right = min(a[2], b[2]);
  const real top = max(a[1], b[1]);
  const real bottom = min(a[3], b[3]);
  const real width = max(right - left + 1, 0.0f);
  const real height = max(bottom - top + 1, 0.0f);
  const real interS = width * height;
  const real Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  const real Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const real nms_thresh,
                           const real* dev_boxes, unsigned long long* dev_mask)
{
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  const int row_size =
        min(n_boxes - row_start * num_threads, num_threads);
  const int col_size =
        min(n_boxes - col_start * num_threads, num_threads);

  __shared__ real block_boxes[num_threads * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(num_threads * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(num_threads * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(num_threads * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(num_threads * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(num_threads * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = num_threads * row_start + threadIdx.x;
    const real* cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (iou_kernel(cur_box, block_boxes + i * 5) > nms_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = (n_boxes + num_threads - 1) / num_threads;
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _nms_gpu(int* keep_out, int* num_out, const real* boxes_host,
              const int boxes_num, const int boxes_dim, const real nms_thresh)
{
  real* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = (boxes_num + num_threads - 1) / num_threads;

  CUDA_CHECK(hipMalloc(&boxes_dev, boxes_num * boxes_dim * sizeof(real)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(real),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev, boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(num_threads);
  nms_kernel<<<blocks, threads>>>(boxes_num, nms_thresh, boxes_dev, mask_dev);

  unsigned long long* mask_host
      = (unsigned long long*)malloc(boxes_num * col_blocks * sizeof(unsigned long long));
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));

  unsigned long long* remv
      = (unsigned long long*)malloc(col_blocks * sizeof(unsigned long long));
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / num_threads;
    int inblock = i % num_threads;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  free(mask_host);
  free(remv);
}

typedef struct BoundingBox_
{
  real x1, y1, x2, y2;
  real score;
} BoundingBox;

bool transform_box(BoundingBox* box,
                   real dx, real dy, real dw, real dh,
                   real im_w, real im_h, real min_w, real min_h)
{
  real w = box->x2 - box->x1 + 1.0f;
  real h = box->y2 - box->y1 + 1.0f;
  real ctr_x = box->x1 + 0.5f * w;
  real ctr_y = box->y1 + 0.5f * h;

  real pred_ctr_x = dx * w + ctr_x;
  real pred_ctr_y = dy * h + ctr_y;
  real pred_w = exp(dw) * w;
  real pred_h = exp(dh) * h;

  box->x1 = pred_ctr_x - 0.5f * pred_w;
  box->y1 = pred_ctr_y - 0.5f * pred_h;
  box->x2 = pred_ctr_x + 0.5f * pred_w;
  box->y2 = pred_ctr_y + 0.5f * pred_h;

  box->x1 = max(min(box->x1, im_w - 1.0f), 0.0f);
  box->y1 = max(min(box->y1, im_h - 1.0f), 0.0f);
  box->x2 = max(min(box->x2, im_w - 1.0f), 0.0f);
  box->y2 = max(min(box->y2, im_h - 1.0f), 0.0f);

  w = box->x2 - box->x1 + 1.0f;
  h = box->y2 - box->y1 + 1.0f;

  if (w >= min_w && h >= min_h) return true;
  return false;
}

typedef struct ProposalOption_
{
  int num_concats;
  real* ratios;
  int num_ratios;
  real* scales;
  int num_scales;
  int base_size;
  int feat_stride;
  int min_size;
  int pre_nms_topn;
  int post_nms_topn;
  real nms_thresh;
} ProposalOption;

#define MAX_NUM_RATIO_SCALE 10
#define MAX_DATA_WIDTH 80
#define MAX_DATA_HEIGHT 80
#define MAX_NUM_PROPOSAL 6000

void generate_anchors(real* const anchors, const ProposalOption* option)
{
  real base_area = option->base_size * option->base_size;
  real ctr = 0.5f * (option->base_size - 1.0f);
  real wr[MAX_NUM_RATIO_SCALE];
  real hr[MAX_NUM_RATIO_SCALE];
  for (int i = 0; i < option->num_ratios; ++i) {
    wr[i] = round(sqrt(base_area / option->ratios[i]));
    hr[i] = round(wr[i] * option->ratios[i]);
  }
 { // anchor generation
  real* p_anchors = &anchors[0];
  for (int c = 0; c < option->num_concats; ++c) {
    for (int i = 0; i < option->num_ratios; ++i) {
      for (int j = 0; j < option->num_scales; ++j) {
        const real ws = 0.5f * (wr[i] * option->scales[j] - 1.0f);
        const real hs = 0.5f * (hr[i] * option->scales[j] - 1.0f);
        p_anchors[0] = ctr - ws;
        p_anchors[1] = ctr - hs;
        p_anchors[2] = ctr + ws;
        p_anchors[3] = ctr + hs;
        p_anchors += 4;
      }
    }
  }
 } // end anchor generation
}

void sort_box(BoundingBox *list, int start, int end, const int num_top)
{
  int left = start + 1, right = end;
  real pivot_score = list[start].score;
  BoundingBox temp;
  while (left <= right) {
    while (left <= end && list[left].score >= pivot_score) ++left;
    while (right > start && list[right].score <= pivot_score) --right;
    if (left <= right) {
      temp = list[left];
      list[left] = list[right];
      list[right] = temp;
      ++left;
      --right;
    }
  }
  if (right > start) {
    temp = list[right];
    list[right] = list[start];
    list[start] = temp;
  }
  if (start < right - 1) {
    sort_box(list, start, right - 1, num_top);
  }
  if (right + 1 < num_top && right + 1 < end) {
    sort_box(list, right + 1, end, num_top);
  }
}

void forward(const Tensor* bottom4d, const Tensor* pred_box4d, const Tensor* img_info1d,
             Tensor* const top2d, const real* anchors, const ProposalOption* option)
{
  BoundingBox* proposals
      = (BoundingBox*)malloc(MAX_NUM_RATIO_SCALE * MAX_NUM_RATIO_SCALE *
                             MAX_DATA_WIDTH * MAX_DATA_HEIGHT * sizeof(BoundingBox));
  real* sorted_dets = (real*)malloc(MAX_NUM_PROPOSAL * 5 * sizeof(real));
  int* keep = (int*)malloc(MAX_NUM_PROPOSAL * sizeof(int));

  // bottom4d: N x 2 x num_anchors x H x W
  // pred_box4d: N x num_anchors x 4 x H x W
  // img_info1d: N x 4
  // top2d: N x num_rois x 4
  real* p_bottom_data = bottom4d->data;
  real* p_pred_box_data = pred_box4d->data;
  real* p_img_info = img_info1d->data;
  real* p_top_data = top2d->data;
  const int num_anchors = option->num_concats * option->num_ratios * option->num_scales;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    const int H = bottom4d->shape[n][2];
    const int W = bottom4d->shape[n][3];
    const int HW = H * W;
    const real im_w = p_img_info[1];
    const real im_h = p_img_info[0];
    const real min_w = option->min_size * p_img_info[2];
    const real min_h = option->min_size * p_img_info[3];

    // enumerate all proposals
    int num_proposals = 0;
    for (int h = 0; h < H; ++h) {
      for (int w = 0; w < W; ++w) {
        const real x = w * option->feat_stride;
        const real y = h * option->feat_stride;
        const real* p_box = &p_pred_box_data[h * W + w];
        const real* p_score = &p_bottom_data[num_anchors * HW + h * W + w];
        for (int k = 0; k < num_anchors; ++k) {
          const real dx = p_box[(k * 4 + 0) * HW];
          const real dy = p_box[(k * 4 + 1) * HW];
          const real dw = p_box[(k * 4 + 2) * HW];
          const real dh = p_box[(k * 4 + 3) * HW];
          proposals[num_proposals].x1 = x + anchors[k * 4 + 0];
          proposals[num_proposals].y1 = y + anchors[k * 4 + 1];
          proposals[num_proposals].x2 = x + anchors[k * 4 + 2];
          proposals[num_proposals].y2 = y + anchors[k * 4 + 3];
          proposals[num_proposals].score = p_score[k * HW];
          const bool box_created = transform_box(&proposals[num_proposals],
                                                 dx, dy, dw, dh,
                                                 im_w, im_h, min_w, min_h);
          if (box_created) ++num_proposals;
        }
      }
    }

    // choose candidates according to scores
    sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
    if (num_proposals > option->pre_nms_topn)
      num_proposals = option->pre_nms_topn;
    for (int i = 0; i < num_proposals; ++i) {
      sorted_dets[i * 5 + 0] = proposals[i].x1;
      sorted_dets[i * 5 + 1] = proposals[i].y1;
      sorted_dets[i * 5 + 2] = proposals[i].x2;
      sorted_dets[i * 5 + 3] = proposals[i].y2;
      sorted_dets[i * 5 + 4] = proposals[i].score;
    }

   { // roi retrieval
    int num_rois = 0;
    _nms_gpu(keep, &num_rois, sorted_dets, num_proposals, 5, option->nms_thresh);

    if (num_rois > option->post_nms_topn)
      num_rois = option->post_nms_topn;
    top2d->shape[n][0] = num_rois;
    top2d->shape[n][1] = 4;
    for (int i = 0; i < num_rois; ++i) {
      p_top_data[i * 4 + 0] = proposals[keep[i]].x1;
      p_top_data[i * 4 + 1] = proposals[keep[i]].y1;
      p_top_data[i * 4 + 2] = proposals[keep[i]].x2;
      p_top_data[i * 4 + 3] = proposals[keep[i]].y2;
    }
   } // end roi retrieval

    // locate next item
    p_top_data += 4 * top2d->shape[n][0];
    p_bottom_data += 2 * num_anchors * HW;
    p_pred_box_data += 4 * num_anchors * HW;
    p_img_info += 4;
  } // endfor num_items

  top2d->ndim = 2;

  free(proposals);
  free(sorted_dets);
  free(keep);
}

#include <stdlib.h>

int main(void)
{
  real anchors[100];
  real scales[5] = {3, 6, 9, 16, 32};
  real ratios[5] = {0.5, 0.666, 1.0, 1.5, 2.0};
  ProposalOption option;
  option.num_concats = 3;
  option.base_size = 16;
  option.feat_stride = 16;
  option.pre_nms_topn = 6000;
  option.post_nms_topn = 300;
  option.nms_thresh = 0.7;
  option.min_size = 16;
  option.scales = &scales[0];
  option.ratios = &ratios[0];
  option.num_scales = 5;
  option.num_ratios = 5;
  generate_anchors(anchors, &option);
  int num_anchors = option.num_concats * option.num_scales * option.num_ratios;

#ifdef PASS
  BoundingBox box[5000];
  for (int i = 0; i < 5000; ++i) {
    box[i].score = rand() % 1000;
  }
  int num_top = 200;
  sort_box(box, 0, 5000 - 1, num_top);
  for (int i = 1; i < num_top; ++i) {
    if (box[i-1].score < box[i].score)
    printf("%d:%.2f > %d:%.2f\n", i-1, box[i-1].score, i, box[i].score);
  }
  for (int i = num_top; i < 5000; ++i) {
    if (box[i].score > box[num_top-1].score)
      printf("%d:%.2f > %d:%.2f\n", i, box[i].score, num_top-1, box[num_top-1].score);
  }
#endif

  Tensor score, bbox, im_info, roi;
  real score_data[150*36*46], bbox_data[300*36*46], im_info_data[4], roi_data[300*4];
  score.ndim = 4; score.num_items = 1; score.data = &score_data[0];
  for (int i = 0; i < score.num_items; ++i) {
    score.shape[i][0] = 2;
    score.shape[i][1] = num_anchors;
    score.shape[i][2] = 36;
    score.shape[i][3] = 46;
  }
  bbox.ndim = 4; bbox.num_items = score.num_items; bbox.data = &bbox_data[0];
  for (int i = 0; i < bbox.num_items; ++i) {
    bbox.shape[i][0] = num_anchors;
    bbox.shape[i][1] = 4;
    bbox.shape[i][2] = 36;
    bbox.shape[i][3] = 46;
  }
  im_info.ndim = 1; im_info.num_items = score.num_items; im_info.data = &im_info_data[0];
  for (int i = 0; i < im_info.num_items; ++i) {
    im_info.shape[i][0] = 4;
  }
  roi.ndim = 2; roi.num_items = score.num_items; roi.data = &roi_data[0];

  FILE* fp = fopen("bottom.txt", "r");
  for (int i = 0; i < flatten_size(&score); ++i)
    fscanf(fp, "%f", &score_data[i]);
  fclose(fp);
  fp = fopen("bbox.txt", "r");
  for (int i = 0; i < flatten_size(&bbox); ++i)
    fscanf(fp, "%f", &bbox_data[i]);
  fclose(fp);
  fp = fopen("im_info.txt", "r");
  for (int i = 0; i < flatten_size(&im_info); ++i)
    fscanf(fp, "%f", &im_info_data[i]);
  fclose(fp);

  forward(&score, &bbox, &im_info, &roi, anchors, &option);

  real* p_roi_data = roi.data;
  for (int n = 0; n < roi.num_items; ++n) {
    printf("batch %d: %d x %d\n", n, roi.shape[n][0], roi.shape[n][1]);
    for (int i = 0; i < roi.shape[n][0]; ++i) {
      for (int j = 0; j < roi.shape[n][1]; ++j) {
        printf("%.2f ", *(p_roi_data++));
      }
      printf("\n");
    }
  }

  return 0;
}
