#include "hip/hip_runtime.h"
/*
  Original version (25.8ms)
    1. [1ms] memcpy, D->H
      1-1. scores (75*2*36*46*float = 993.6KB)
      1-2. bbox (75*4*36*46*float = 1987.2KB)
    2. [15ms] all candidate enumeration & sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [3.3ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing (bitwise calculations)
    7. [4ms] roi -> top

  Improved version (6.3ms)
    1. [0ms] no memcpy required
    2. [2.6ms] all candidate enumeration & sort
      2-1. [0.3ms] all candidate enumeration
      2-2. [0.6ms] memcpy, D->H, all candidates (75*36*46*5*float = 2484KB)
      2-3. [1.6ms] partial quick-sort
    3. [0ms] memcpy, H->D, 6000*5*float = 120KB
    4. [1.1ms] nms kernel
    5. [1.8ms] memcpy, D->H, 6000*94*uint64 = 4512KB
    6. [0.7ms] nms post processing
    7. [0.1ms] roi -> top

  TODO
    - GPU sort (improve 2-2, 2-3) - speedup
    - GPU nms post processing (remove 5)
*/

#include "layer.h"

#include <time.h>

static float a_time[8] = { 0, };
static clock_t tick0, tick1, tick00;

// --------------------------------------------------------------------------
// kernel code
//   generate_anchors: generate anchor boxes of varying sizes and ratios
//   transform_box: transform a box according to a given gradient
//   sort_box: sort a list of boxes in descending order of their scores
//   enumerate_proposals: generate all candidate boxes with their scores
//   retrieve_rois: retrieve boxes that are determined to be kept by NMS
// --------------------------------------------------------------------------

// given a base box, enumerate transformed boxes of varying sizes and ratios
//   option->base_size: base box's width & height (i.e., base box is square)
//   option->scales: "option->num_scales x 1" array
//                   varying scale factor for base box
//   option->ratios: "option->num_ratios x 1" array
//                   varying height-width ratio
//   anchors: "num_anchors x 4" array,  (x1, y1, x2, y2) for each box
//   num_anchors: total number of transformations
//                = option->num_scales * option->num_ratios
#define MAX_NUM_RATIO_SCALE 10
void generate_anchors(real anchors[],
                      const LayerOption* const option)
{
  // base box's width & height & center location
  const real base_area = (real)(option->base_size * option->base_size);
  const real ctr = 0.5f * (option->base_size - 1.0f);

  // transformed width & height for given ratios
  real wr[MAX_NUM_RATIO_SCALE];
  real hr[MAX_NUM_RATIO_SCALE];
  for (int i = 0; i < option->num_ratios; ++i) {
    wr[i] = (real)ROUND(sqrt(base_area / option->ratios[i]));
    hr[i] = (real)ROUND(wr[i] * option->ratios[i]);
  }

  // enumerate all transformed boxes
  {
    real* p_anchors = anchors;
    for (int j0 = 0; j0 < option->num_scales; j0 += option->num_ratios) {
      for (int i = 0; i < option->num_ratios; ++i) {
        for (int j = 0; j < option->num_ratios; ++j) {
          // transformed width & height for given ratios & scales
          const real ws = 0.5f * (wr[i] * option->scales[j0 + j] - 1.0f);
          const real hs = 0.5f * (hr[i] * option->scales[j0 + j] - 1.0f);
          // (x1, y1, x2, y2) for transformed box
          p_anchors[0] = ctr - ws;
          p_anchors[1] = ctr - hs;
          p_anchors[2] = ctr + ws;
          p_anchors[3] = ctr + hs;
          p_anchors += 4;
        } // endfor j
      } // endfor i
    } // endfor j0
  }
}

// transform a box according to a given gradient
//   box: (x1, y1, x2, y2)
//   gradient: dx, dy, d(log w), d(log h)
#ifdef GPU
__device__
#endif
static
int transform_box(real box[],
                  const real dx, const real dy,
                  const real d_log_w, const real d_log_h,
                  const real img_W, const real img_H,
                  const real min_box_W, const real min_box_H)
{
  // width & height of box
  const real w = box[2] - box[0] + 1.0f;
  const real h = box[3] - box[1] + 1.0f;
  // center location of box
  const real ctr_x = box[0] + 0.5f * w;
  const real ctr_y = box[1] + 0.5f * h;

  // new center location according to gradient (dx, dy)
  const real pred_ctr_x = dx * w + ctr_x;
  const real pred_ctr_y = dy * h + ctr_y;
  // new width & height according to gradient d(log w), d(log h)
  const real pred_w = exp(d_log_w) * w;
  const real pred_h = exp(d_log_h) * h;

  // update upper-left corner location
  box[0] = pred_ctr_x - 0.5f * pred_w;
  box[1] = pred_ctr_y - 0.5f * pred_h;
  // update lower-right corner location
  box[2] = pred_ctr_x + 0.5f * pred_w;
  box[3] = pred_ctr_y + 0.5f * pred_h;

  // adjust new corner locations to be within the image region,
  box[0] = MAX(0.0f,  MIN(box[0],  img_W - 1.0f));
  box[1] = MAX(0.0f,  MIN(box[1],  img_H - 1.0f));
  box[2] = MAX(0.0f,  MIN(box[2],  img_W - 1.0f));
  box[3] = MAX(0.0f,  MIN(box[3],  img_H - 1.0f));

  // recompute new width & height
  const real box_w = box[2] - box[0] + 1.0f;
  const real box_h = box[3] - box[1] + 1.0f;

  // check if new box's size >= threshold
  return (box_w >= min_box_W) * (box_h >= min_box_H);
}

// bitonic sort a list of boxes in descending order of their scores (GPU)
//   list: num_boxes x 5 array,  (x1, y1, x2, y2, score) for each box
//     in bitoninc sort, total space allocated for list should be
//     a power of 2 >= num_boxes,
//     and scores of virtually-padded boxes { num_boxes, ..., 2^n - 1 }
//     should be set smaller than mininum score of actual boxes
#ifdef GPU
__global__
void bitonic_sort_step(real list[], const int idx_major, const int idx_minor)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int index_xor = index ^ idx_minor;
  real temp[5];

  // the threads with the lowest ids sort the array
  if (index_xor > index) {
    if (index & idx_major) {
      // sort ascending
      if (list[index * 5 + 4] > list[index_xor * 5 + 4]) {
        for (int i = 0; i < 5; ++i) {
          temp[i] = list[index * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index * 5 + i] = list[index_xor * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index_xor * 5 + i] = temp[i];
        }
      }
    }
    else {
      // sort descending
      if (list[index * 5 + 4] < list[index_xor * 5 + 4]) {
        for (int i = 0; i < 5; ++i) {
          temp[i] = list[index * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index * 5 + i] = list[index_xor * 5 + i];
        }
        for (int i = 0; i < 5; ++i) {
          list[index_xor * 5 + i] = temp[i];
        }
      }
    }
  }
}
void bitonic_sort_box(real list[], const int num_boxes)
{
  int num_power_of_2 = 1;
  while (num_power_of_2 < num_boxes) num_power_of_2 *= 2;
  const int num_threads = num_power_of_2;
  const int threads_per_block = 512;
  const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);

  // major step
  for (int idx_major = 2; idx_major <= num_threads; idx_major *= 2) {
    // minor step
    for (int idx_minor = idx_major / 2; idx_minor > 0; idx_minor /= 2) {
      bitonic_sort_step<<<num_blocks, threads_per_block>>>(
          list, idx_major, idx_minor);
    }
  }
}
#endif

// quick-sort a list of boxes in descending order of their scores (CPU)
//   list: num_boxes x 5 array,  (x1, y1, x2, y2, score) for each box
//   if num_top <= end,  only top-k results are guaranteed to be sorted
//   (for efficient computation)
static
void sort_box(real list[], const int start, const int end,
              const int num_top)
{
  const real pivot_score = list[start * 5 + 4];
  int left = start + 1, right = end;
  real temp[5];
  while (left <= right) {
    while (left <= end && list[left * 5 + 4] >= pivot_score) ++left;
    while (right > start && list[right * 5 + 4] <= pivot_score) --right;
    if (left <= right) {
      for (int i = 0; i < 5; ++i) {
        temp[i] = list[left * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[left * 5 + i] = list[right * 5 + i];
      }
      for (int i = 0; i < 5; ++i) {
        list[right * 5 + i] = temp[i];
      }
      ++left;
      --right;
    }
  }

  if (right > start) {
    for (int i = 0; i < 5; ++i) {
      temp[i] = list[start * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[start * 5 + i] = list[right * 5 + i];
    }
    for (int i = 0; i < 5; ++i) {
      list[right * 5 + i] = temp[i];
    }
  }

  if (start < right - 1) {
    sort_box(list, start, right - 1, num_top);
  }
  if (right + 1 < num_top && right + 1 < end) {
    sort_box(list, right + 1, end, num_top);
  }
}

// generate all candidate boxes with their scores
//   bottom: 1 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   num_anchors: number of anchors  (= # scales * # ratios)
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
//   img_H, img_W: scaled image height & width
//   min_box_H, min_box_W: minimum box height & width
//   feat_stride: scaled image height (width) / bottom height (width)
//   proposals: num_proposals * 5 array
//     num_proposals = num_anchors * H * W
//     (x1, y1, x2, y2, score) for each proposal
#ifdef GPU
__global__
void enumerate_proposals_gpu(const real bottom4d[],
                             const real d_anchor4d[],
                             const real anchors[],
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real proposals[])
{
  const int bottom_area = bottom_H * bottom_W;
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_anchors * bottom_area) {
    const int h = index / num_anchors / bottom_W;
    const int w = (index / num_anchors) % bottom_W;
    const int k = index % num_anchors;
    const real x = w * feat_stride;
    const real y = h * feat_stride;
    const real* p_box = d_anchor4d + h * bottom_W + w;
    const real* p_score = bottom4d + h * bottom_W + w;

    const real dx = p_box[(k * 4 + 0) * bottom_area];
    const real dy = p_box[(k * 4 + 1) * bottom_area];
    const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
    const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

    proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
    proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
    proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
    proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

    proposals[index * 5 + 4]
        = transform_box(&proposals[index * 5],
                        dx, dy, d_log_w, d_log_h,
                        img_W, img_H, min_box_W, min_box_H)
          * p_score[k * bottom_area];
  }
  else {
    // in GPU mode, total space allocated for proposals should be
    // a power of 2 >= actual number of proposals,
    // thus, scores of virtually-padded boxes should be set smaller than
    // mininum score of actual boxes
    // (in RPN, 0 is the smallest possible score)
    proposals[index * 5 + 0] = 0;
    proposals[index * 5 + 1] = 0;
    proposals[index * 5 + 2] = 0;
    proposals[index * 5 + 3] = 0;
    proposals[index * 5 + 4] = 0;
  }
}
#else
void enumerate_proposals_cpu(const real bottom4d[],
                             const real d_anchor4d[],
                             const real anchors[],
                             const int num_anchors,
                             const int bottom_H, const int bottom_W,
                             const real img_H, const real img_W,
                             const real min_box_H, const real min_box_W,
                             const int feat_stride,
                             real proposals[])
{
  const int bottom_area = bottom_H * bottom_W;
  for (int h = 0; h < bottom_H; ++h) {
    for (int w = 0; w < bottom_W; ++w) {
      const real x = w * feat_stride;
      const real y = h * feat_stride;
      const real* p_box = d_anchor4d + h * bottom_W + w;
      const real* p_score = bottom4d + h * bottom_W + w;
      for (int k = 0; k < num_anchors; ++k) {
        const real dx = p_box[(k * 4 + 0) * bottom_area];
        const real dy = p_box[(k * 4 + 1) * bottom_area];
        const real d_log_w = p_box[(k * 4 + 2) * bottom_area];
        const real d_log_h = p_box[(k * 4 + 3) * bottom_area];

        const int index = (h * bottom_W + w) * num_anchors + k;
        proposals[index * 5 + 0] = x + anchors[k * 4 + 0];
        proposals[index * 5 + 1] = y + anchors[k * 4 + 1];
        proposals[index * 5 + 2] = x + anchors[k * 4 + 2];
        proposals[index * 5 + 3] = y + anchors[k * 4 + 3];

        proposals[index * 5 + 4]
            = transform_box(&proposals[index * 5],
                            dx, dy, d_log_w, d_log_h,
                            img_W, img_H, min_box_W, min_box_H)
              * p_score[k * bottom_area];
      } // endfor k
    } // endfor w
  } // endfor h
}
#endif

// retrieve proposals that are determined to be kept as RoIs by NMS
//   proposals : "num_boxes x 5" array,  (x1, y1, x2, y2, score) for each box
//   num_rois: number of RoIs to be retrieved
//   keep: "num_rois x 1" array
//     keep[i]: index of i-th RoI in proposals
//   rois: "num_rois x 5" array,  (x1, y1, x2, y2, score) for each RoI
#ifdef GPU
__global__
void retrieve_rois_gpu(const real proposals[],
                       const int keep[],
                       real rois[],
                       const int num_rois)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_rois) {
    const real* const proposals_index = proposals + keep[index] * 5;
    rois[index * 5 + 0] = proposals_index[0];
    rois[index * 5 + 1] = proposals_index[1];
    rois[index * 5 + 2] = proposals_index[2];
    rois[index * 5 + 3] = proposals_index[3];
    rois[index * 5 + 4] = proposals_index[4];
  }
}
#else
void retrieve_rois_cpu(const real proposals[],
                       const int keep[],
                       real rois[],
                       const int num_rois)
{
  for (int i = 0; i < num_rois; ++i) {
    const real* const proposals_index = proposals + keep[i] * 5;
    rois[i * 5 + 0] = proposals_index[0];
    rois[i * 5 + 1] = proposals_index[1];
    rois[i * 5 + 2] = proposals_index[2];
    rois[i * 5 + 3] = proposals_index[3];
    rois[i * 5 + 4] = proposals_index[4];
  }
}
#endif



// --------------------------------------------------------------------------
// layer operator code
//   proposal_forward
// --------------------------------------------------------------------------

// proposal: bottom -> top
//   bottom: 2 x num_anchors x H x W tensor
//     bottom[0, k, h, w] = background score of anchor k at node (h, w)
//     bottom[1, k, h, w] = foreground score of anchor k at node (h, w)
//   d_anchor: num_anchors x 4 x H x W tensor
//     d_anchor[k, :, h, w] = gradient (dx, dy, d(log w), d(log h))
//                            of anchor k at center location (h, w)
//   img_info: 6 x 1 tensor,  (img_H, img_W, scale_H, scale_W, raw_H, raw_W)
//     img_H, img_W: scaled image height & width
//     scale_H: height scale factor
//              img_H = raw image height * scale_H
//     scale_W: width scale factor
//              img_W = raw image width * scale_W
//     raw_H, raw_W: raw image height & width
//   top: num_RoIs x 5 tensor,  (x1, y1, x2, y2, score) of each RoI
//   anchors: num_anchors * 4 array,  (x1, y1, x2, y2) for each anchor
//   4 temporary arrays
//     proposals: all box proposals with their scores
//       "num_boxes x 5" array,  (x1, y1, x2, y2, score) for each box
//       in GPU mode, if proposals = NULL, use bitonic sort in GPU
//       if proposals != NULL & allocated in main memory, quicksort in CPU
//     keep: indices of proposals to be retrieved as RoIs
//       "num_rois x 1" array,  keep[i]: index of i-th RoI in proposals
//       TODO: always stored in main memory due to implementation issue
//     proposals_dev: GPU memory space, required in GPU mode
//       in GPU mode, total space allocated for proposals should be
//       a power of 2 >= num_boxes
//     keep_dev: GPU memory space, required in GPU mode
void proposal_forward(const Tensor* const bottom4d,
                      const Tensor* const d_anchor4d,
                      const Tensor* const img_info1d,
                      Tensor* const top2d,
                      const real anchors[],
                      real proposals[],
                      int keep[],
                      real proposals_dev[],
                      int keep_dev[],
                      const LayerOption* const option)
{
  // number of anchors  (= number of scales * ratios)
  const int num_anchors = option->num_ratios * option->num_scales;

  // do forward-pass for each item in the batch
  const real* p_bottom_item = bottom4d->data;
  const real* p_d_anchor_item = d_anchor4d->data;
  const real* p_img_info = img_info1d->data;
  real* p_top_item = top2d->data;
  int total_top_size = 0;

  tick00 = clock();

  for (int n = 0; n < bottom4d->num_items; ++n) {
    // bottom shape: 2 x num_anchors x H x W
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    // input image height & width
    const real img_H = p_img_info[0];
    const real img_W = p_img_info[1];
    // scale factor for height & width
    const real scale_H = p_img_info[2];
    const real scale_W = p_img_info[3];
    // minimum box width & height
    const real min_box_H = option->min_size * scale_H;
    const real min_box_W = option->min_size * scale_W;

    tick0 = clock();
    // enumerate all proposals
    //   num_proposals = num_anchors * H * W
    //   (x1, y1, x2, y2, score) for each proposal
    // NOTE: for bottom, only foreground scores are passed
    #ifdef GPU
    {
      // in GPU mode, total space allocated for proposals is
      // a power of 2 >= num_proposals (due to bitonic sort algorithm)
      // thus, scores of virtually-padded boxes should be set smaller than
      // mininum score of actual boxes
      const int num_proposals = num_anchors * bottom_area;
      int num_power_of_2 = 1;
      while (num_power_of_2 < num_proposals) num_power_of_2 *= 2;
      const int num_threads = num_power_of_2;
      const int threads_per_block = 512;
      const int num_blocks = DIV_THEN_CEIL(num_threads,  threads_per_block);
      enumerate_proposals_gpu<<<num_blocks, threads_per_block>>>(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals_dev);
    }
    #else
    {
      enumerate_proposals_cpu(
          p_bottom_item + num_anchors * bottom_area,
          p_d_anchor_item,  anchors,  num_anchors,
          bottom_H,  bottom_W,  img_H,  img_W,  min_box_H,  min_box_W,
          option->feat_stride,
          proposals);
    }
    #endif
    tick1 = clock();
    a_time[0] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

    tick0 = clock();
    // choose candidates according to scores
    #ifdef GPU
    {
      const int num_proposals = num_anchors * bottom_area;
      if (!proposals) {
        // in GPU mode, if proposals = NULL, use bitonic sort in GPU
        bitonic_sort_box(proposals_dev, num_proposals);
      }
      else {
        // if proposals != NULL & allocated in main memory, quicksort in CPU
        hipMemcpyAsync(proposals, proposals_dev,
                        num_proposals * 5 * sizeof(real),
                        hipMemcpyDeviceToHost);
        sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
        hipMemcpyAsync(proposals_dev, proposals,
                        num_proposals * 5 * sizeof(real),
                        hipMemcpyHostToDevice);
      }
    }
    #else
    {
      const int num_proposals = num_anchors * bottom_area;
      sort_box(proposals, 0, num_proposals - 1, option->pre_nms_topn);
    }
    #endif
    tick1 = clock();
    a_time[1] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

    tick0 = clock();
    // NMS & RoI retrieval
    {
      // NMS
      const int num_proposals
          = MIN(num_anchors * bottom_area,  option->pre_nms_topn);
      int num_rois = 0;
      nms(num_proposals,  proposals,  &num_rois,  keep,  0,
          option->nms_thresh,  option->post_nms_topn,
          option->bbox_vote,  option->vote_thresh);

      // RoI retrieval
      #ifdef GPU
      {
        const int num_threads = num_rois;
        const int threads_per_block = 128;
        const int num_blocks
            = DIV_THEN_CEIL(num_threads,  threads_per_block);

        hipMemcpyAsync(keep_dev, keep, num_rois * sizeof(int),
                        hipMemcpyHostToDevice);

        retrieve_rois_gpu<<<num_blocks, threads_per_block>>>(
            proposals_dev,  keep_dev,  p_top_item,  num_rois);
      }
      #else
      {
        retrieve_rois_cpu(
            proposals,  keep,  p_top_item,  num_rois);
      }
      #endif

      // set top shape: num_rois x 5,  (x1, y1, x2, y2, score) for each RoI
      top2d->shape[n][0] = num_rois;
      top2d->shape[n][1] = 5;
      top2d->start[n] = total_top_size;
      total_top_size += num_rois * 5;
    }
    tick1 = clock();
    a_time[2] = (float)(tick1 - tick0) / CLOCKS_PER_SEC;

    // locate next item
    {
      const int bottom_size = 2 * num_anchors * bottom_area;
      const int d_anchor_size = 4 * num_anchors * bottom_area;
      const int img_info_size = 6;
      const int top_size = 5 * top2d->shape[n][0];
      p_bottom_item += bottom_size;
      p_d_anchor_item += d_anchor_size;
      p_img_info += img_info_size;
      p_top_item += top_size;
    }
  } // endfor batch

  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;

  tick1 = clock();
  a_time[3] = (float)(tick1 - tick00) / CLOCKS_PER_SEC;
  a_time[7] += (float)(tick1 - tick00) / CLOCKS_PER_SEC;
}



// --------------------------------------------------------------------------
// layer shape calculator code
// --------------------------------------------------------------------------
void proposal_shape(const Tensor* const bottom4d,
                    Tensor* const top2d,
                    int* const proposals_size,
                    int* const keep_size,
                    const LayerOption* const option)
{
  int max_area = 0;

  // calculate shape for each item in the batch
  top2d->ndim = 2;
  top2d->num_items = bottom4d->num_items;
  for (int n = 0; n < bottom4d->num_items; ++n) {
    // calculate maximum area size for determining temporary space size
    const int bottom_H = bottom4d->shape[n][2];
    const int bottom_W = bottom4d->shape[n][3];
    const int bottom_area = bottom_H * bottom_W;
    max_area = MAX(max_area,  bottom_area);

    // top shape <= post_nms_topn x 5
    //   exact row size will be determined after forward-pass
    top2d->shape[n][0] = option->post_nms_topn;
    top2d->shape[n][1] = 5;
    top2d->start[n] = top2d->shape[n][0] * top2d->shape[n][1];
  }

  // temporary space size
  //   in GPU mode, total space allocated for proposals should be
  //   a power of 2 >= actual number of proposals
  {
    const int num_anchors = option->num_ratios * option->num_scales;
    const int num_proposals = num_anchors * max_area;
    int num_power_of_2 = 1;
    while (num_power_of_2 < num_proposals) num_power_of_2 *= 2;
    *proposals_size = num_power_of_2 * 5;
    *keep_size = option->post_nms_topn;
  }
}



// --------------------------------------------------------------------------
// API code
// --------------------------------------------------------------------------

void init_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  const int num_anchors
      = layer->option.num_scales * layer->option.num_ratios;

  #ifdef GPU
  {
    if (layer->p_aux_data[0]) {
      hipFree(layer->p_aux_data[0]);
    }
    hipMalloc(&layer->p_aux_data[0], num_anchors * 4 * sizeof(real));
    generate_anchors(net->param_cpu_data, &layer->option);
    hipMemcpyAsync(layer->p_aux_data[0], net->param_cpu_data,
                    num_anchors * 4 * sizeof(real),
                    hipMemcpyHostToDevice);
  }
  #else
  {
    if (layer->p_aux_data[0]) {
      free(layer->p_aux_data[0]);
    }
    layer->p_aux_data[0] = (real*)malloc(num_anchors * 4 * sizeof(real));
    generate_anchors(layer->p_aux_data[0], &layer->option);
  }
  #endif

  net->space += num_anchors * 4 * sizeof(real);
}

void forward_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  proposal_forward(layer->p_bottoms[0], layer->p_bottoms[1],
                   layer->p_bottoms[2],
                   layer->p_tops[0], layer->p_aux_data[0],
                   net->temp_cpu_data, net->tempint_cpu_data,
                   net->temp_data, net->tempint_data,
                   &layer->option);

  #ifdef DEBUG
  {
    printf("%s:  ", layer->name);
    for (int i = 0; i < 8; ++i) {
      printf("%4.2f\t", a_time[i] * 1000);
    }
    printf("\n");
  }
  #endif
}

void shape_proposal_layer(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  int temp_size, tempint_size;

  proposal_shape(layer->p_bottoms[0], layer->p_tops[0],
                 &temp_size, &tempint_size, &layer->option);

  update_net_size(net, layer, temp_size, tempint_size, 0);
}
