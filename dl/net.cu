#include "layer.h"
#include <string.h>

void init_layer(Layer* const layer)
{
  memset(layer, 0, sizeof(Layer));
}

long int malloc_layer(Layer* const layer)
{
  long int space_cpu = 0;

  layer->p_bottoms = (layer->num_bottoms <= 0) ? NULL :
                     (Tensor**)calloc(layer->num_bottoms, sizeof(Tensor*));
  space_cpu += layer->num_bottoms * sizeof(Tensor*);

  layer->tops = (layer->num_tops <= 0) ? NULL :
                (Tensor*)calloc(layer->num_tops, sizeof(Tensor));
  layer->allocate_top_data = (layer->num_tops <= 0) ? NULL:
                             (int*)calloc(layer->num_tops, sizeof(int));
  layer->p_top_data_backup = (layer->num_tops <= 0) ? NULL:
                             (real**)calloc(layer->num_tops, sizeof(real*));
  space_cpu += layer->num_tops * (sizeof(Tensor) + sizeof(int));

  layer->params = (layer->num_params <= 0) ? NULL : 
                  (Tensor*)calloc(layer->num_params, sizeof(Tensor));
  space_cpu += layer->num_params * sizeof(Tensor);

  layer->p_aux_data = (layer->num_aux_data <= 0) ? NULL : 
                      (real**)calloc(layer->num_aux_data, sizeof(real*));
  space_cpu += layer->num_aux_data * sizeof(real*);

  return space_cpu;
}

long int malloc_load_layer_data(Layer* const layer,
                                const char* const param_path,
                                const char* const name,
                                real* const temp_cpu_space)
{
  long int space = 0;

  #ifdef DEBUG
  printf("%s %d %d\n", name, layer->num_tops, layer->num_params);
  #endif

  for (int i = 0; i < layer->num_tops; ++i) {
    layer->tops[i].max_data_size = flatten_size(&layer->tops[i]);
    if (layer->allocate_top_data[i]) {
      space += malloc_tensor_data(&layer->tops[i]);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    char path[1024];
    layer->params[i].max_data_size = flatten_size(&layer->params[i]);
    space += malloc_tensor_data(&layer->params[i]);
    sprintf(path, "%s/%s_param%d.bin", param_path, name, i);
    load_tensor(path, &layer->params[i], temp_cpu_space);
  }

  return space;
}

long int malloc_top_data(Layer* const layer,
                         const int top_id)
{
  long int space = 0;

  if (!layer->allocate_top_data[top_id] &&
      !layer->p_top_data_backup[top_id])
  {
    layer->p_top_data_backup[top_id] = layer->tops[top_id].data;
    space = malloc_tensor_data(&layer->tops[top_id]);
    layer->allocate_top_data[top_id] = 1;
    printf("[Layer %s] malloc for top[%d], +%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

long int free_top_data(Layer* const layer,
                       const int top_id)
{
  long int space = 0;

  if (layer->allocate_top_data[top_id] &&
      layer->p_top_data_backup[top_id])
  {
    space = free_tensor_data(&layer->tops[top_id]);
    layer->tops[top_id].data = layer->p_top_data_backup[top_id];
    layer->p_top_data_backup[top_id] = NULL;
    layer->allocate_top_data[top_id] = 0;
    printf("[Layer %s] dealloc for top[%d], -%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

void free_layer(Layer* const layer)
{
  if (layer->p_bottoms) {
    free(layer->p_bottoms);
  }

  if (layer->tops) {
    for (int i = 0; i < layer->num_tops; ++i) {
      if (layer->allocate_top_data[i]) {
        free_tensor_data(&layer->tops[i]);
      }
    }
    free(layer->tops);
    free(layer->allocate_top_data);
    free(layer->p_top_data_backup);
  }

  if (layer->params) {
    for (int i = 0; i < layer->num_params; ++i) {
      free_tensor_data(&layer->params[i]);
    }
    free(layer->params);
  }

  if (layer->p_aux_data) {
    for (int i = 0; i < layer->num_aux_data; ++i) {
      #ifdef GPU
      hipFree(layer->p_aux_data[i]);
      #else
      free(layer->p_aux_data[i]);
      #endif
      layer->p_aux_data[i] = NULL;
    }
    free(layer->p_aux_data);
  }

  memset(layer, 0, sizeof(Layer));
  free(layer);
}

void init_net(Net* const net)
{
  memset(net, 0, sizeof(Net));
}

void malloc_net(Net* const net)
{
  long int space_cpu = 0;
  long int space = 0;

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipMalloc(&net->layer_data[i], net->layer_size * sizeof(real));
    #else
    net->layer_data[i] = (real*)malloc(net->layer_size * sizeof(real));
    printf("Allocate layer data %d: %x (%ld)\n", i, net->layer_data[i], net->layer_size);
    #endif
    net->reserved_layer_data[i] = 0;
  }
  space += net->num_layer_data * net->layer_size * sizeof(real);

  #ifdef GPU
  {
    hipMalloc(&net->temp_data, net->temp_size * sizeof(real));
    hipMalloc(&net->tempint_data, net->tempint_size * sizeof(int));
    hipMalloc(&net->const_data, net->const_size * sizeof(real));

    net->input_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->output_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));

  }
  #else
  {
    net->temp_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_data = (int*)malloc(net->tempint_size * sizeof(int));
    net->const_data = (real*)malloc(net->const_size * sizeof(real));

    net->input_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->output_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #endif
  space += sizeof(real) * (net->temp_size + net->const_size)
           + sizeof(int) * (net->tempint_size);
  space_cpu += sizeof(real) * (2 * net->layer_size + net->param_size
                               + net->temp_size)
               + sizeof(int) * (net->tempint_size);

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < net->const_size; ++i) {
      net->output_cpu_data[i] = 1;
    }
    hipMemcpyAsync(net->const_data, net->output_cpu_data,
                    net->const_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < net->const_size; ++i) {
      net->const_data[i] = 1;
    }
  #endif
  }

  // memory allocation for layers
  for (int i = 0; i < net->num_layers; ++i) {
    space += malloc_load_layer_data(net->layers[i], net->param_path,
                                    net->layers[i]->name,
                                    net->param_cpu_data);
  }

  {
    if (net->img_info) {
      const int img_info_size = net->layers[0]->tops[0].num_items * 6;
      net->img_info->data
          = (real*)malloc(img_info_size * sizeof(real));
      space_cpu += sizeof(real) * img_info_size;
    }
  }

  // acquire CuBLAS handle
  #ifdef GPU
  {
    if (hipblasCreate(&net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  net->space_cpu += space_cpu;
  net->space += space;

  net->initialized = 1;
}

void free_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    free_layer(net->layers[i]);
  }

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipFree(net->layer_data[i]);
    #else
    free(net->layer_data[i]);
    #endif
    net->layer_data[i] = NULL;
  }

  #ifdef GPU
  {
    hipFree(net->temp_data);
    hipFree(net->tempint_data);
    hipFree(net->const_data);

    free(net->input_cpu_data);
    free(net->output_cpu_data);
    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #else
  {
    free(net->temp_data);
    free(net->tempint_data);
    free(net->const_data);

    free(net->input_cpu_data);
    free(net->output_cpu_data);
    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #endif

  if (net->img_info) {
    free_tensor_data(net->img_info);
    free(net->img_info);
  }

  #ifdef GPU
  {
    if (hipblasDestroy(net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
  }
  #endif

  memset(net, 0, sizeof(Net));
}

void init_layers(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_init[j]) {
        (*layer->f_init[j])(net, layer);
      }
    }
  }
}

void forward_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_forward[j]) {
        (*layer->f_forward[j])(net, layer);
      }
    }
  }
}

void shape_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_shape[j]) {
        (*layer->f_shape[j])(net, layer);
        #ifdef DEBUG
        for (int k = 0; k < layer->num_tops; ++k) {
          print_tensor_info(layer->name, &layer->tops[k]);
        }
        #endif
      }
    }
  }
}

void update_net_size(Net* const net,
                     const Layer* const layer,
                     const int temp_size,
                     const int tempint_size,
                     const int const_size)
{
  if (!net->initialized) {
    long int top_size = 0, param_size = 0;
    for (int i = 0; i < layer->num_tops; ++i) {
      if (!layer->allocate_top_data[i]) {
        top_size = MAX(top_size,  flatten_size(&layer->tops[i]));
      }
    }
    for (int i = 0; i < layer->num_params; ++i) {
      param_size = MAX(param_size,  flatten_size(&layer->params[i]));
    }

    net->layer_size = MAX(net->layer_size,  top_size);
    net->param_size = MAX(net->param_size,  param_size);
    net->temp_size = MAX(net->temp_size,  (long)temp_size);
    net->tempint_size = MAX(net->tempint_size,  (long)tempint_size);
    net->const_size = MAX(net->const_size,  (long)const_size);
  }
}

real* get_layer_data(Net* const net)
{
  for (int i = 0; i < net->num_layer_data; ++i) {
    if (!net->reserved_layer_data[i]) {
      net->reserved_layer_data[i] = 1;
      return net->layer_data[i];
    }
  }

  printf("[ERROR] Not enough temporary space for storing layer output!\n");
  return NULL;
}

void save_layer_tops(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    char path[1024];
    sprintf(path, "%s/%s_top%d.rt.bin", net->param_path, layer->name, i);
    save_tensor_data(path, &layer->tops[i], net->output_cpu_data);
  }
}

void print_layer_tops(void* const net_, void* const layer_)
{
  const Net* const net = (Net*)net_;
  const Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    const long int size = flatten_size(&layer->tops[i]);
    const Tensor* const t = &layer->tops[i];
    int idx[MAX_NDIM + 1] = { 0, };

    #ifdef GPU
    hipMemcpyAsync(net->output_cpu_data, layer->tops[i].data,
                    size * sizeof(real),
                    hipMemcpyDeviceToHost);
    #else
    memcpy(net->output_cpu_data, layer->tops[i].data, size * sizeof(real));
    #endif

    for (int j = 0; j < size; ++j) {
      const int n = idx[0];

      printf("Layer %s / Top %d / Image %d [", layer->name, i, n);
      for (int d = 1; d < t->ndim; ++d) {
        printf("%d, ", idx[d]);
      }
      printf("%d]: %f\n", idx[t->ndim]++, net->output_cpu_data[j]);

      for (int d = t->ndim; d > 0; --d) {
        if (idx[d] == t->shape[n][d - 1]) {
          idx[d] = 0;
          ++idx[d - 1];
        }
      }
    }
  } // endfor i
}
