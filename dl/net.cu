#include "layer.h"
#include <string.h>
#include <stdio.h>

long int malloc_layer(Layer* const layer)
{
  long int space_cpu = 0;

  layer->p_bottoms = (layer->num_bottoms <= 0) ? NULL :
                     (Tensor**)malloc(layer->num_bottoms * sizeof(Tensor*));
  space_cpu += layer->num_bottoms * sizeof(Tensor*);

  layer->tops = (layer->num_tops <= 0) ? NULL :
                (Tensor*)malloc(layer->num_tops * sizeof(Tensor));
  layer->allocate_top_data = (layer->num_tops <= 0) ? NULL:
                             (int*)calloc(layer->num_tops, sizeof(int));
  space_cpu += layer->num_tops * (sizeof(Tensor) + sizeof(int));

  layer->params = (layer->num_params <= 0) ? NULL : 
                  (Tensor*)malloc(layer->num_params * sizeof(Tensor));
  space_cpu += layer->num_params * sizeof(Tensor);

  return space_cpu;
}

long int malloc_load_layer_data(Layer* const layer,
                                const char* const name,
                                real* const temp_cpu_space)
{
  long int space = 0;

  for (int i = 0; i < layer->num_tops; ++i) {
    if (layer->allocate_top_data[i]) {
      space += malloc_tensor(&layer->tops[i]);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    char path[1024];
    //printf("malloc param %d\n", i);
    space += malloc_tensor(&layer->params[i]);
    sprintf(path, "params/%s_param%d.bin", name, i);
    //printf("load param %s\n", path);
    load_tensor(path, &layer->params[i], temp_cpu_space);
  }

  return space;
}

void free_layer(Layer* const layer)
{
  if (layer->p_bottoms) {
    free(layer->p_bottoms);
    layer->p_bottoms = NULL;
  }

  if (layer->tops) {
    for (int i = 0; i < layer->num_tops; ++i) {
      if (layer->allocate_top_data[i]) {
        #ifdef GPU
        hipFree(layer->tops[i].data);
        layer->tops[i].data = NULL;
        #else
        free(layer->tops[i].data);
        layer->tops[i].data = NULL;
        #endif
      }
    }
    free(layer->tops);
    layer->tops = NULL;
    free(layer->allocate_top_data);
    layer->allocate_top_data = NULL;
  }

  if (layer->params) {
    for (int i = 0; i < layer->num_params; ++i) {
      #ifdef GPU
      hipFree(layer->params[i].data);
      layer->params[i].data = NULL;
      #else
      free(layer->params[i].data);
      layer->params[i].data = NULL;
      #endif
    }
    free(layer->params);
    layer->params = NULL;
  }

  free(layer);
}

void malloc_net(Net* const net)
{
  long int space_cpu = 0;
  long int space = 0;

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipMalloc(&net->layer_data[i], net->layer_size * sizeof(real));
    #else
    net->layer_data[i] = (real*)malloc(net->layer_size * sizeof(real));
    #endif
    net->reserved_layer_data[i] = 0;
  }
  space += net->num_layer_data * net->layer_size * sizeof(real);

  #ifdef GPU
  {
    hipMalloc(&net->temp_data, net->temp_size * sizeof(real));
    hipMalloc(&net->tempint_data, net->tempint_size * sizeof(int));
    hipMalloc(&net->const_data, net->const_size * sizeof(real));
/*
    hipHostMalloc(&net->input_cpu_data, net->layer_size * sizeof(real));
    hipHostMalloc(&net->output_cpu_data, net->layer_size * sizeof(real));
    hipHostMalloc(&net->param_cpu_data, net->param_size * sizeof(real));
    hipHostMalloc(&net->temp_cpu_data, net->temp_size * sizeof(real));
    hipHostMalloc(&net->tempint_cpu_data, net->tempint_size * sizeof(int));
*/
    net->input_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->output_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #else
  {
    net->temp_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_data = (int*)malloc(net->tempint_size * sizeof(int));
    net->const_data = (real*)malloc(net->const_size * sizeof(real));

    net->input_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->output_cpu_data = (real*)malloc(net->layer_size * sizeof(real));
    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #endif
  space += sizeof(real) * (net->temp_size + net->const_size)
           + sizeof(int) * (net->tempint_size);
  space_cpu += sizeof(real) * (2 * net->layer_size + net->param_size
                               + net->temp_size)
               + sizeof(int) * (net->tempint_size);

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < net->const_size; ++i) {
      net->output_cpu_data[i] = 1;
    }
    hipMemcpyAsync(net->const_data, net->output_cpu_data,
                    net->const_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < net->const_size; ++i) {
      net->const_data[i] = 1;
    }
  #endif
  }

  for (int i = 0; i < net->num_layers; ++i) {
    space += malloc_load_layer_data(net->layers[i], net->layers[i]->name,
                                    net->param_cpu_data);
  }

  net->img_info->data
      = (real*)malloc(flatten_size(net->img_info) * sizeof(real));
  space_cpu += sizeof(real) * flatten_size(net->img_info);

  // acquire CuBLAS handle
  #ifdef GPU
  {
    if (hipblasCreate(&net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  net->space_cpu = space_cpu;
  net->space = space;

  net->initialized = 1;
}

void free_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    free_layer(net->layers[i]);
  }

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipFree(net->layer_data[i]);
    #else
    free(net->layer_data[i]);
    #endif
    net->layer_data[i] = NULL;
  }

  #ifdef GPU
  {
    hipFree(net->temp_data);
    hipFree(net->tempint_data);
    hipFree(net->const_data);
    hipFree(net->anchors);

    hipHostFree(net->input_cpu_data);
    hipHostFree(net->output_cpu_data);
    hipHostFree(net->param_cpu_data);
    hipHostFree(net->temp_cpu_data);
    hipHostFree(net->tempint_cpu_data);
  }
  #else
  {
    free(net->temp_data);
    free(net->tempint_data);
    free(net->const_data);
    free(net->anchors);

    free(net->input_cpu_data);
    free(net->output_cpu_data);
    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #endif


  free(net->img_info->data);
  free(net->img_info);

  net->temp_data = NULL;
  net->tempint_data = NULL;
  net->const_data = NULL;
  net->input_cpu_data = NULL;
  net->output_cpu_data = NULL;
  net->param_cpu_data = NULL;
  net->temp_cpu_data = NULL;
  net->tempint_cpu_data = NULL;
  net->anchors = NULL;
  net->img_info = NULL;

  #ifdef GPU
  {
    if (hipblasDestroy(net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
  }
  #endif

  net->initialized = 0;
}

void update_net_size(Net* const net,
                     const Layer* const layer,
                     const int temp_size,
                     const int tempint_size,
                     const int const_size)
{
  if (!net->initialized) {
    long int top_size = 0, param_size = 0;
    for (int i = 0; i < layer->num_tops; ++i) {
      if (!layer->allocate_top_data[i]) {
        top_size = MAX(top_size,  flatten_size(&layer->tops[i]));
      }
    }
    for (int i = 0; i < layer->num_params; ++i) {
      param_size = MAX(param_size,  flatten_size(&layer->params[i]));
    }

    net->layer_size = MAX(net->layer_size,  top_size);
    net->param_size = MAX(net->param_size,  param_size);
    net->temp_size = MAX(net->temp_size,  (long)temp_size);
    net->tempint_size = MAX(net->tempint_size,  (long)tempint_size);
    net->const_size = MAX(net->const_size,  (long)const_size);
  }
}

real* get_layer_data(Net* const net)
{
  for (int i = 0; i < net->num_layer_data; ++i) {
    if (!net->reserved_layer_data[i]) {
      net->reserved_layer_data[i] = 1;
      return net->layer_data[i];
    }
  }

  printf("[ERROR] Not enough temporary space for storing layer output!\n");
  return NULL;
}

void print_layer_tops(const Net* const net,
                      const Layer* const layer)
{
  for (int i = 0; i < layer->num_tops; ++i) {
    const long int size = flatten_size(&layer->tops[i]);
    #ifdef GPU
    hipMemcpyAsync(net->output_cpu_data, layer->tops[i].data,
                    size * sizeof(real),
                    hipMemcpyDeviceToHost);
    #else
    memcpy(net->output_cpu_data, layer->tops[i].data, size * sizeof(real));
    #endif
    char path[1024];
    sprintf(path, "params/%s_top%d.txt", layer->name, i);
    FILE* fp = fopen(path, "w");
    const Tensor* const t = &layer->tops[0];
    int j = 0;
    for (int n = 0; n < t->num_items; ++n) {
      for (int c = 0; c < t->shape[n][0]; ++c)
        for (int h = 0; h < t->shape[n][1]; ++h)
          for (int w = 0; w < t->shape[n][2]; ++w)
            fprintf(fp, "%d %d %d %d %f\n",
                    n, c, h, w, net->output_cpu_data[j++]);
    }
    fclose(fp);
  }
}
