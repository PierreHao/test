#include "layer.h"
#include <string.h>

void init_layer(Layer* const layer)
{
  memset(layer, 0, sizeof(Layer));
}

void set_layer_name(Layer* const layer, const char* const name)
{
  strcpy(layer->name, name);
  for (int i = 0; i < layer->num_tops; ++i) {
    sprintf(layer->tops[i].name, "%s[%d]", name, i);
  }
  for (int i = 0; i < layer->num_params; ++i) {
    sprintf(layer->params[i].name, "%s_param%d", name, i);
  }
}

long int malloc_layer(Net* const net,
                      Layer* const layer)
{
  long int space = 0;

  #ifdef DEBUG
  printf("%s %d %d\n", layer->name, layer->num_tops, layer->num_params);
  #endif

  for (int i = 0; i < layer->num_tops; ++i) {
    Tensor* const tensor = &layer->tops[i];

    tensor->max_data_size = flatten_size(tensor);

    if (tensor->has_own_memory) {
      space += malloc_tensor_data(tensor);
    }
    else if (tensor->data_id > 0) {
      tensor->data = net->layer_data[tensor->data_id - 1];
    }
    else {
      printf("[ERROR] Wrong data id %d for layer %s[%d]\n",
             tensor->data_id, layer->name, i);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    char path[1024];
    Tensor* const tensor = &layer->params[i];
    tensor->max_data_size = flatten_size(tensor);
    space += malloc_tensor_data(tensor);
    sprintf(path, "%s/%s.bin", net->param_path, tensor->name);
    load_tensor(path, tensor, net->temp_cpu_data);
  }

  return space;
}

long int malloc_top_data(Net* const net,
                         Layer* const layer,
                         const int top_id)
{
  Tensor* const tensor = &layer->tops[top_id];
  long int space = 0;

  if (!tensor->has_own_memory && tensor->data_id > 0) {
    tensor->has_own_memory = 1;
    tensor->data = NULL;
    space = malloc_tensor_data(tensor);
    net->space += space;
    printf("[Layer %s] malloc for top[%d], +%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

long int free_top_data(Net* const net,
                       Layer* const layer,
                       const int top_id)
{
  Tensor* const tensor = &layer->tops[top_id];
  long int space = 0;

  if (tensor->has_own_memory && tensor->data_id > 0) {
    tensor->has_own_memory = 0;
    space = free_tensor_data(tensor);
    tensor->data = net->layer_data[tensor->data_id - 1];
    net->space -= space;
    printf("[Layer %s] dealloc for top[%d], -%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

void free_layer(Layer* const layer)
{
  for (int i = 0; i < layer->num_tops; ++i) {
    if (layer->tops[i].has_own_memory) {
      free_tensor_data(&layer->tops[i]);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    free_tensor_data(&layer->params[i]);
  }

  for (int i = 0; i < layer->num_aux_data; ++i) {
    #ifdef GPU
    hipFree(layer->p_aux_data[i]);
    #else
    free(layer->p_aux_data[i]);
    #endif
  }

  memset(layer, 0, sizeof(Layer));
}

void assign_layer_data(Net* const net)
{
  // compute lifetime for each tensor
  for (int layer_id = net->num_layers - 1; layer_id >= 0; --layer_id) {
    Layer* const layer = &net->layers[layer_id];
    for (int bottom_id = 0; bottom_id < layer->num_bottoms; ++bottom_id) {
      if (!layer->p_bottoms[bottom_id]->alive_until) {
        layer->p_bottoms[bottom_id]->alive_until = (void*)layer;
      }
    }
  }

  // lifetime for output tensors
  for (int layer_id = 0; layer_id < net->num_layers; ++layer_id) {
    Layer* const layer = &net->layers[layer_id];
    for (int top_id = 0; top_id < layer->num_tops; ++top_id) {
      Tensor* const tensor = &layer->tops[top_id];
      if (!tensor->alive_until) {
        tensor->alive_until = (void*)&net->layers[net->num_layers - 1];
      }
    }
  }

  // assign layer_data to each tensor according to its lifetime
  for (int layer_id = 0; layer_id < net->num_layers; ++layer_id) {
    Layer* const layer = &net->layers[layer_id];

    for (int top_id = 0; top_id < layer->num_tops; ++top_id) {
      Tensor* const tensor = &layer->tops[top_id];

      if (!tensor->has_own_memory) {
        for (int data_id = 0; data_id < net->num_layer_data; ++data_id) {
          if (!net->reserved_until[data_id]) {
            tensor->data_id = data_id + 1;
            net->reserved_until[data_id] = tensor->alive_until;
            printf("%s: assigned layer_data[%d], reserved until %s\n",
                   tensor->name, data_id,
                   ((Layer*)tensor->alive_until)->name);
            break;
          }
        }
        if (!tensor->data_id) {
          printf("[ERROR] Failed to assign layer_data for %s\n",
                 tensor->name);
        }
      }

      for (int data_id = 0; data_id < net->num_layer_data; ++data_id) {
        if (net->reserved_until[data_id] == (void*)layer) {
          net->reserved_until[data_id] = NULL;
        }
      }
    }
  }
}

void init_net(Net* const net)
{
  memset(net, 0, sizeof(Net));
  for (int i = 0; i < MAX_NUM_LAYERS; ++i) {
    init_layer(&net->layers[i]);
  }
}

void malloc_net(Net* const net)
{
  long int space_cpu = 0;
  long int space = 0;

  space_cpu += net->num_layers * sizeof(Layer);

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipMalloc(&net->layer_data[i], net->layer_size * sizeof(real));
    #else
    net->layer_data[i] = (real*)malloc(net->layer_size * sizeof(real));
    #endif
  }
  space += net->num_layer_data * net->layer_size * sizeof(real);

  #ifdef GPU
  {
    hipMalloc(&net->temp_data, net->temp_size * sizeof(real));
    hipMalloc(&net->tempint_data, net->tempint_size * sizeof(int));
    hipMalloc(&net->const_data, net->const_size * sizeof(real));

    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #else
  {
    net->temp_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_data = (int*)malloc(net->tempint_size * sizeof(int));
    net->const_data = (real*)malloc(net->const_size * sizeof(real));

    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #endif
  space += sizeof(real) * (net->temp_size + net->const_size)
           + sizeof(int) * (net->tempint_size);
  space_cpu += sizeof(real) * (2 * net->layer_size + net->param_size
                               + net->temp_size)
               + sizeof(int) * (net->tempint_size);

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < net->const_size; ++i) {
      net->temp_cpu_data[i] = 1;
    }
    hipMemcpyAsync(net->const_data, net->temp_cpu_data,
                    net->const_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < net->const_size; ++i) {
      net->const_data[i] = 1;
    }
  #endif
  }

  assign_layer_data(net);

  // memory allocation for layers
  for (int i = 0; i < net->num_layers; ++i) {
    space += malloc_layer(net, &net->layers[i]);
  }

  {
    Tensor* img_info = &net->img_info;
    const long int img_info_size = flatten_size(img_info);
    img_info->data = (real*)malloc(img_info_size * sizeof(real));
    space_cpu += sizeof(real) * img_info_size;
  }

  // acquire CuBLAS handle
  #ifdef GPU
  {
    if (hipblasCreate(&net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  net->space_cpu += space_cpu;
  net->space += space;

  net->initialized = 1;
}

void free_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    free_layer(&net->layers[i]);
  }

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipFree(net->layer_data[i]);
    #else
    free(net->layer_data[i]);
    #endif
  }

  #ifdef GPU
  {
    hipFree(net->temp_data);
    hipFree(net->tempint_data);
    hipFree(net->const_data);

    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #else
  {
    free(net->temp_data);
    free(net->tempint_data);
    free(net->const_data);

    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #endif

  free(net->img_info.data);

  #ifdef GPU
  {
    if (hipblasDestroy(net->cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
  }
  #endif

  memset(net, 0, sizeof(Net));
}

void init_layers(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = &net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_init[j]) {
        (*layer->f_init[j])(net, layer);
      }
    }
  }
}

void forward_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = &net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_forward[j]) {
        (*layer->f_forward[j])(net, layer);
      }
    }
  }
}

void shape_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = &net->layers[i];

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_shape[j]) {
        (*layer->f_shape[j])(net, layer);
        #ifdef DEBUG
        for (int k = 0; k < layer->num_tops; ++k) {
          print_tensor_info(layer->name, &layer->tops[k]);
        }
        #endif
      }
    }
  }
}

void _assign_layer_data(Net* const net, Tensor* const tensor)
{
  if (tensor->data_id) {
    printf("[WARNING] Reallocate layer_data for %s\n", tensor->name);
  }

  for (int i = 0; i < MAX_NUM_LAYER_DATA; ++i) {
    if (!net->reserved_until[i]) {
      tensor->data_id = i + 1;
      net->reserved_until[i] = (void*)1;
      #ifdef DEBUG
      printf("%s: assigned layer_data[%d]\n", tensor->name, i);
      #endif
      return;
    }
  }
  printf("[ERROR] Failed to assign layer_data for %s\n", tensor->name);
}

void _deallocate_layer_data(Net* const net, Tensor* const tensor)
{
  if (tensor->data_id) {
    net->reserved_until[tensor->data_id - 1] = 0;
    #ifdef DEBUG
    printf("%s: deallocated layer_data[%d]\n",
           tensor->name, tensor->data_id - 1);
    #endif
  }
}

void update_net_size(Net* const net,
                     const Layer* const layer,
                     const int temp_size,
                     const int tempint_size,
                     const int const_size)
{
  if (!net->initialized) {
    long int top_size = 0, param_size = 0;
    for (int i = 0; i < layer->num_tops; ++i) {
      if (!layer->tops[i].has_own_memory) {
        top_size = MAX(top_size,  flatten_size(&layer->tops[i]));
      }
    }
    for (int i = 0; i < layer->num_params; ++i) {
      param_size = MAX(param_size,  flatten_size(&layer->params[i]));
    }

    net->layer_size = MAX(net->layer_size,  top_size);
    net->param_size = MAX(net->param_size,  param_size);
    net->temp_size = MAX(net->temp_size,  (long)temp_size);
    net->tempint_size = MAX(net->tempint_size,  (long)tempint_size);
    net->const_size = MAX(net->const_size,  (long)const_size);
  }
}

void save_layer_tops(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    char path[1024];
    sprintf(path, "%s/%s_top%d.rt.bin", net->param_path, layer->name, i);
    save_tensor_data(path, &layer->tops[i], net->temp_cpu_data);
  }
}

void print_layer_tops(void* const net_, void* const layer_)
{
  const Net* const net = (Net*)net_;
  const Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    const long int size = flatten_size(&layer->tops[i]);
    const Tensor* const t = &layer->tops[i];
    int idx[MAX_NDIM + 1] = { 0, };

    #ifdef GPU
    hipMemcpyAsync(net->temp_cpu_data, layer->tops[i].data,
                    size * sizeof(real),
                    hipMemcpyDeviceToHost);
    #else
    memcpy(net->temp_cpu_data, layer->tops[i].data, size * sizeof(real));
    #endif

    for (int j = 0; j < size; ++j) {
      const int n = idx[0];

      printf("Layer %s / Top %d / Image %d [", layer->name, i, n);
      for (int d = 1; d < t->ndim; ++d) {
        printf("%d, ", idx[d]);
      }
      printf("%d]: %f\n", idx[t->ndim]++, net->temp_cpu_data[j]);

      for (int d = t->ndim; d > 0; --d) {
        if (idx[d] == t->shape[n][d - 1]) {
          idx[d] = 0;
          ++idx[d - 1];
        }
      }
    }
  } // endfor i
}
