#include "layer.h"
#include <string.h>

void init_layer(Layer* const layer)
{
  memset(layer, 0, sizeof(Layer));
}

void set_layer_name(Layer* const layer, const char* const name)
{
  strcpy(layer->name, name);
}

void set_bottom(Layer* const layer, const int bottom_id,
                Tensor* const tensor)
{
  if (bottom_id >= layer->num_bottoms) {
    printf("[ERROR] Layer %s: out-of-bound input index %d\n",
           layer->name, bottom_id);
    return;
  }
  layer->p_bottoms[bottom_id] = tensor;
}

void set_top(Layer* const layer, const int top_id,
             Tensor* const tensor)
{
  if (top_id >= layer->num_tops) {
    printf("[ERROR] Layer %s: out-of-bound output index %d\n",
           layer->name, top_id);
    return;
  }
  layer->p_tops[top_id] = tensor;
}

void set_param(Layer* const layer, const int param_id,
               Tensor* const tensor)
{
  if (param_id >= layer->num_params) {
    printf("[ERROR] Layer %s: out-of-bound parameter index %d\n",
           layer->name, param_id);
    return;
  }
  layer->p_params[param_id] = tensor;
}

void add_bottom(Layer* const layer, Tensor* const tensor)
{
  if (layer->num_bottoms == MAX_NUM_BOTTOMS) {
    printf("[ERROR] Layer %s: cannot add more input\n", layer->name);
    return;
  }
  ++layer->num_bottoms;
  set_bottom(layer, layer->num_bottoms - 1, tensor);
}

void add_top(Layer* const layer, Tensor* const tensor)
{
  if (layer->num_tops == MAX_NUM_TOPS) {
    printf("[ERROR] Layer %s: cannot add more output\n", layer->name);
    return;
  }
  ++layer->num_tops;
  set_top(layer, layer->num_tops - 1, tensor);
}

void add_param(Layer* const layer, Tensor* const tensor)
{
  if (layer->num_params == MAX_NUM_PARAMS) {
    printf("[ERROR] Layer %s: cannot add more parameter\n", layer->name);
    return;
  }
  ++layer->num_params;
  set_param(layer, layer->num_params - 1, tensor);
}

Tensor* get_bottom(const Layer* const layer, const int bottom_id)
{
  #ifdef DEBUG
  if (bottom_id >= layer->num_bottoms) {
    printf("[ERROR] Layer %s: out-of-bound input index %d\n",
           layer->name, bottom_id);
    return NULL;
  }
  #endif
  return layer->p_bottoms[bottom_id];
}

Tensor* get_top(const Layer* const layer, const int top_id)
{
  #ifdef DEBUG
  if (top_id >= layer->num_tops) {
    printf("[ERROR] Layer %s: out-of-bound output index %d\n",
           layer->name, top_id);
    return NULL;
  }
  #endif
  return layer->p_tops[top_id];
}

Tensor* get_param(const Layer* const layer, const int param_id)
{
  #ifdef DEBUG
  if (param_id >= layer->num_params) {
    printf("[ERROR] Layer %s: out-of-bound parameter index %d\n",
           layer->name, param_id);
    return NULL;
  }
  #endif
  return layer->p_params[param_id];
}

long int malloc_layer(Net* const net,
                      Layer* const layer)
{
  long int space = 0;

  #ifdef DEBUG
  printf("%s %d %d\n", layer->name, layer->num_tops, layer->num_params);
  #endif

  for (int i = 0; i < layer->num_tops; ++i) {
    Tensor* const tensor = get_top(layer, i);

    tensor->max_data_size = flatten_size(tensor);

    if (tensor->has_own_memory) {
      space += malloc_tensor_data(tensor);
    }
    else if (tensor->data_id > 0) {
      tensor->data = net->layer_data[tensor->data_id - 1];
    }
    else {
      printf("[ERROR] Wrong data id %d for layer %s[%d]\n",
             tensor->data_id, layer->name, i);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    char path[1024];
    Tensor* const tensor = get_param(layer, i);
    tensor->max_data_size = flatten_size(tensor);
    space += malloc_tensor_data(tensor);
    sprintf(path, "%s/%s.bin", net->param_path, tensor->name);
    load_tensor(path, tensor, net->temp_cpu_data);
  }

  return space;
}

long int malloc_top_data(Net* const net,
                         Layer* const layer,
                         const int top_id)
{
  Tensor* const tensor = get_top(layer, top_id);
  long int space = 0;

  if (!tensor->has_own_memory && tensor->data_id > 0) {
    tensor->has_own_memory = 1;
    tensor->data = NULL;
    space = malloc_tensor_data(tensor);
    net->space += space;
    printf("[Layer %s] malloc for top[%d], +%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

long int free_top_data(Net* const net,
                       Layer* const layer,
                       const int top_id)
{
  Tensor* const tensor = get_top(layer, top_id);
  long int space = 0;

  if (tensor->has_own_memory && tensor->data_id > 0) {
    tensor->has_own_memory = 0;
    space = free_tensor_data(tensor);
    tensor->data = net->layer_data[tensor->data_id - 1];
    net->space -= space;
    printf("[Layer %s] dealloc for top[%d], -%.2fKB\n",
           layer->name, top_id, (float)(space / 1000.0f));
  }

  return space;
}

void free_layer(Layer* const layer)
{
  for (int i = 0; i < layer->num_tops; ++i) {
    Tensor* const tensor = get_top(layer, i);
    if (tensor->has_own_memory) {
      free_tensor_data(tensor);
    }
  }

  for (int i = 0; i < layer->num_params; ++i) {
    Tensor* const tensor = get_param(layer, i);
    free_tensor_data(tensor);
  }

  for (int i = 0; i < layer->num_aux_data; ++i) {
    if (layer->p_aux_data[i]) {
      #ifdef GPU
      hipFree(layer->p_aux_data[i]);
      #else
      free(layer->p_aux_data[i]);
      #endif
    }
  }

  memset(layer, 0, sizeof(Layer));
}

Tensor* get_tensor(Net* const net, const int tensor_id)
{
  #ifdef DEBUG
  if (tensor_id >= net->num_tensors) {
    printf("[ERROR] Net: out-of-bound tensor index %d\n", tensor_id);
    return NULL;
  }
  #endif
  return &net->tensors[tensor_id];
} 

Layer* get_layer(Net* const net, const int layer_id)
{
  #ifdef DEBUG
  if (layer_id >= net->num_layers) {
    printf("[ERROR] Net: out-of-bound layer index %d\n", layer_id);
    return NULL;
  }
  #endif
  return &net->layers[layer_id];
} 

Tensor* find_tensor_by_name(Net* const net, const char* const name)
{
  for (int i = 0; i < net->num_tensors; ++i) {
    Tensor* const tensor = get_tensor(net, i);
    if (strcmp(tensor->name, name) == 0) {
      return tensor;
    }
  }
  return NULL;
}

Layer* find_layer_by_name(Net* const net, const char* const name)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);
    if (strcmp(layer->name, name) == 0) {
      return layer;
    }
  }
  return NULL;
}

Tensor* add_tensor(Net* const net, const char* const name)
{
  {
    Tensor* const tensor = find_tensor_by_name(net, name);
    if (tensor) {
      printf("[ERROR] Net: Tensor %s already exists\n", name);
      return tensor;
    }
  }

  if (net->num_tensors == MAX_NUM_TENSORS) {
    printf("[ERROR] Net: cannot add more tensor\n");
    return NULL;
  }
  ++net->num_tensors;

  {
    Tensor* const tensor = get_tensor(net, net->num_tensors - 1);
    set_tensor_name(tensor, name);
    return tensor;
  }
}

Layer* add_layer(Net* const net, const char* const name)
{
  {
    Layer* const layer = find_layer_by_name(net, name);
    if (layer) {
      printf("[ERROR] Net: Layer %s already exists\n", name);
      return layer;
    }
  }

  if (net->num_layers == MAX_NUM_LAYERS) {
    printf("[ERROR] Net: cannot add more layer\n");
    return NULL;
  }
  ++net->num_layers;

  {
    Layer* const layer = get_layer(net, net->num_layers - 1);
    set_layer_name(layer, name);
    return layer;
  }
}

Tensor* find_or_add_tensor(Net* const net, const char* const name)
{
  Tensor* tensor = find_tensor_by_name(net, name);
  if (!tensor) {
    tensor = add_tensor(net, name);
  }
  return tensor;
}

Layer* find_or_add_layer(Net* const net, const char* const name)
{
  Layer* layer = find_layer_by_name(net, name);
  if (!layer) {
    layer = add_layer(net, name);
  }
  return layer;
}

Tensor* get_tensor_by_name(Net* const net, const char* const name)
{
  Tensor* const tensor = find_tensor_by_name(net, name);
  if (!tensor) {
    printf("[ERROR] Cannot find tensor %s\n", name);
  }
  return tensor;
}

Layer* get_layer_by_name(Net* const net, const char* const name)
{
  Layer* const layer = find_layer_by_name(net, name);
  if (!layer) {
    printf("[ERROR] Cannot find layer %s\n", name);
  }
  return layer;
}

void assign_layer_data(Net* const net)
{
  // compute lifetime for each tensor
  for (int layer_id = net->num_layers - 1; layer_id >= 0; --layer_id) {
    Layer* const layer = get_layer(net, layer_id);
    for (int bottom_id = 0; bottom_id < layer->num_bottoms; ++bottom_id) {
      Tensor* const tensor = get_bottom(layer, bottom_id);
      if (!tensor->alive_until) {
        tensor->alive_until = (void*)layer;
      }
    }
  }

  // lifetime for output tensors
  for (int layer_id = 0; layer_id < net->num_layers; ++layer_id) {
    Layer* const layer = get_layer(net, layer_id);
    for (int top_id = 0; top_id < layer->num_tops; ++top_id) {
      Tensor* const tensor = get_top(layer, top_id);
      if (!tensor->alive_until) {
        const Layer* const last_layer = get_layer(net, net->num_layers - 1);
        tensor->alive_until = (void*)last_layer;
      }
    }
  }

  // assign layer_data to each tensor according to its lifetime
  for (int layer_id = 0; layer_id < net->num_layers; ++layer_id) {
    Layer* const layer = get_layer(net, layer_id);

    for (int top_id = 0; top_id < layer->num_tops; ++top_id) {
      Tensor* const tensor = get_top(layer, top_id);

      if (!tensor->has_own_memory) {
        for (int data_id = 0; data_id < net->num_layer_data; ++data_id) {
          if (!net->reserved_until[data_id]) {
            tensor->data_id = data_id + 1;
            net->reserved_until[data_id] = tensor->alive_until;
            printf("%s: assigned layer_data[%d], reserved until %s\n",
                   tensor->name, data_id,
                   ((Layer*)tensor->alive_until)->name);
            break;
          }
        }
        if (!tensor->data_id) {
          printf("[ERROR] Failed to assign layer_data for %s\n",
                 tensor->name);
        }
      }

      for (int data_id = 0; data_id < net->num_layer_data; ++data_id) {
        if (net->reserved_until[data_id] == (void*)layer) {
          net->reserved_until[data_id] = NULL;
        }
      }
    }
  }
}

void init_net(Net* const net)
{
  memset(net, 0, sizeof(Net));
}

void malloc_net(Net* const net)
{
  long int space_cpu = 0;
  long int space = 0;

  space_cpu += net->num_layers * sizeof(Layer);

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipMalloc(&net->layer_data[i], net->layer_size * sizeof(real));
    #else
    net->layer_data[i] = (real*)malloc(net->layer_size * sizeof(real));
    #endif
  }
  space += net->num_layer_data * net->layer_size * sizeof(real);

  #ifdef GPU
  {
    hipMalloc(&net->temp_data, net->temp_size * sizeof(real));
    hipMalloc(&net->tempint_data, net->tempint_size * sizeof(int));
    hipMalloc(&net->const_data, net->const_size * sizeof(real));

    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #else
  {
    net->temp_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_data = (int*)malloc(net->tempint_size * sizeof(int));
    net->const_data = (real*)malloc(net->const_size * sizeof(real));

    net->param_cpu_data = (real*)malloc(net->param_size * sizeof(real));
    net->temp_cpu_data = (real*)malloc(net->temp_size * sizeof(real));
    net->tempint_cpu_data = (int*)malloc(net->tempint_size * sizeof(int));
  }
  #endif
  space += sizeof(real) * (net->temp_size + net->const_size)
           + sizeof(int) * (net->tempint_size);
  space_cpu += sizeof(real) * (2 * net->layer_size + net->param_size
                               + net->temp_size)
               + sizeof(int) * (net->tempint_size);

  // data initialization
  {
  #ifdef GPU
    for (int i = 0; i < net->const_size; ++i) {
      net->temp_cpu_data[i] = 1;
    }
    hipMemcpyAsync(net->const_data, net->temp_cpu_data,
                    net->const_size * sizeof(real),
                    hipMemcpyHostToDevice);
  #else
    for (int i = 0; i < net->const_size; ++i) {
      net->const_data[i] = 1;
    }
  #endif
  }

  assign_layer_data(net);

  // memory allocation for layers
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);
    space += malloc_layer(net, layer);
  }

  {
    Tensor* img_info = &net->img_info;
    const long int img_info_size = flatten_size(img_info);
    img_info->data = (real*)malloc(img_info_size * sizeof(real));
    space_cpu += sizeof(real) * img_info_size;
  }

  // acquire CuBLAS handle
  #ifdef GPU
  {
    if (hipblasCreate(&net->blas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas creation failed\n");
    }
  }
  #endif

  net->space_cpu += space_cpu;
  net->space += space;

  net->initialized = 1;
}

void free_net(Net* const net)
{
  if (!net->initialized) {
    return;
  }

  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);
    free_layer(layer);
  }

  for (int i = 0; i < net->num_layer_data; ++i) {
    #ifdef GPU
    hipFree(net->layer_data[i]);
    #else
    free(net->layer_data[i]);
    #endif
  }

  #ifdef GPU
  {
    hipFree(net->temp_data);
    hipFree(net->tempint_data);
    hipFree(net->const_data);

    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #else
  {
    free(net->temp_data);
    free(net->tempint_data);
    free(net->const_data);

    free(net->param_cpu_data);
    free(net->temp_cpu_data);
    free(net->tempint_cpu_data);
  }
  #endif

  free(net->img_info.data);

  #ifdef GPU
  {
    if (hipblasDestroy(net->blas_handle) != HIPBLAS_STATUS_SUCCESS) {
      printf("cublas destruction failed\n");
    }
  }
  #endif

  memset(net, 0, sizeof(Net));
}

void init_layers(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_init[j]) {
        (*layer->f_init[j])(net, layer);
      }
    }
  }
}

void forward_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_forward[j]) {
        (*layer->f_forward[j])(net, layer);
      }
    }
  }
}

void shape_net(Net* const net)
{
  for (int i = 0; i < net->num_layers; ++i) {
    Layer* const layer = get_layer(net, i);

    for (int j = 0; j < MAX_NUM_OPS_PER_LAYER; ++j) {
      if (layer->f_shape[j]) {
        (*layer->f_shape[j])(net, layer);
        #ifdef DEBUG
        for (int top_id = 0; top_id < layer->num_tops; ++top_id) {
          const Tensor* const tensor = get_top(layer, top_id);
          print_tensor_info(layer->name, tensor);
        }
        #endif
      }
    }
  }
}

void _assign_layer_data(Net* const net, Tensor* const tensor)
{
  if (tensor->data_id) {
    printf("[WARNING] Reallocate layer_data for %s\n", tensor->name);
  }

  for (int i = 0; i < MAX_NUM_LAYER_DATA; ++i) {
    if (!net->reserved_until[i]) {
      tensor->data_id = i + 1;
      net->reserved_until[i] = (void*)1;
      #ifdef DEBUG
      printf("%s: assigned layer_data[%d]\n", tensor->name, i);
      #endif
      return;
    }
  }
  printf("[ERROR] Failed to assign layer_data for %s\n", tensor->name);
}

void _deallocate_layer_data(Net* const net, Tensor* const tensor)
{
  if (tensor->data_id) {
    net->reserved_until[tensor->data_id - 1] = 0;
    #ifdef DEBUG
    printf("%s: deallocated layer_data[%d]\n",
           tensor->name, tensor->data_id - 1);
    #endif
  }
}

void update_net_size(Net* const net,
                     const Layer* const layer,
                     const int temp_size,
                     const int tempint_size,
                     const int const_size)
{
  if (!net->initialized) {
    long int top_size = 0, param_size = 0;
    for (int i = 0; i < layer->num_tops; ++i) {
      const Tensor* const tensor = get_top(layer, i);
      if (!tensor->has_own_memory) {
        top_size = MAX(top_size,  flatten_size(tensor));
      }
    }
    for (int i = 0; i < layer->num_params; ++i) {
      const Tensor* const tensor = get_param(layer, i);
      param_size = MAX(param_size,  flatten_size(tensor));
    }

    net->layer_size = MAX(net->layer_size,  top_size);
    net->param_size = MAX(net->param_size,  param_size);
    net->temp_size = MAX(net->temp_size,  (long)temp_size);
    net->tempint_size = MAX(net->tempint_size,  (long)tempint_size);
    net->const_size = MAX(net->const_size,  (long)const_size);
  }
}

void save_layer_tops(void* const net_, void* const layer_)
{
  Net* const net = (Net*)net_;
  Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    char path[1024];
    const Tensor* const tensor = get_top(layer, i);
    sprintf(path, "%s/%s_top%d.rt.bin", net->param_path, layer->name, i);
    save_tensor_data(path, tensor, net->temp_cpu_data);
  }
}

void print_layer_tops(void* const net_, void* const layer_)
{
  const Net* const net = (Net*)net_;
  const Layer* const layer = (Layer*)layer_;

  for (int i = 0; i < layer->num_tops; ++i) {
    const Tensor* const tensor = get_top(layer, i);
    const long int size = flatten_size(tensor);
    int idx[MAX_NDIM + 1] = { 0, };

    #ifdef GPU
    hipMemcpyAsync(net->temp_cpu_data, tensor->data,
                    size * sizeof(real),
                    hipMemcpyDeviceToHost);
    #else
    memcpy(net->temp_cpu_data, tensor->data, size * sizeof(real));
    #endif

    for (int j = 0; j < size; ++j) {
      const int n = idx[0];

      printf("Layer %s / Top %d / Image %d [", layer->name, i, n);
      for (int d = 1; d < tensor->ndim; ++d) {
        printf("%d, ", idx[d]);
      }
      printf("%d]: %f\n", idx[tensor->ndim]++, net->temp_cpu_data[j]);

      for (int d = tensor->ndim; d > 0; --d) {
        if (idx[d] == tensor->shape[n][d - 1]) {
          idx[d] = 0;
          ++idx[d - 1];
        }
      }
    }
  } // endfor i
}
